#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "extern_var.h"

void SETUP_TFSF(void){

    //length of the incident field vectors
    //inc_Length=4*ceil(sqrt(NCELLX*NCELLX+NCELLY*NCELLY+NCELLZ*NCELLZ));
    //polarization and incident angles in degrees
    inc_phi=0;
    inc_theta=0;
    polar_psi=0;
    //convert to radians
    inc_phi*=pi/180.0;
    inc_theta*=pi/180.0;
    polar_psi*=pi/180.0 ;

    d_1D=dx;//*pow(pow(sin(inc_theta),4)*(pow(sin(inc_phi),4)+pow(cos(inc_phi),4))+pow(cos(inc_theta),4),0.5);
   // printf("%e\n",d_1D);
    /*printf("\n\n\n ex: %f\n",(cos(polar_psi)*sin(inc_phi)-sin(polar_psi)*cos(inc_theta)*cos(inc_phi)));
    printf("ey: %f\n",(-cos(polar_psi)*cos(inc_phi)-sin(polar_psi)*cos(inc_theta)*sin(inc_phi)));
    printf("ez: %f\n",(sin(polar_psi)*sin(inc_theta)));
    printf("hx: %f\n",(sin(polar_psi)*sin(inc_phi)+cos(polar_psi)*cos(inc_theta)*cos(inc_phi)));
    printf("hy: %f\n",(-sin(polar_psi)*cos(inc_phi)+cos(polar_psi)*cos(inc_theta)*sin(inc_phi)));
    printf("hz: %f\n\n\n",(-cos(polar_psi)*sin(inc_theta)));*/

    //where 1-D array meets 3-D array
    m0=floor(inc_Length/2);

    i_0=NtfsfX;
    j_0=NtfsfY;
    k_0=NtfsfZ;

    //incident pulse parameter;

    real BW=BandWidth;
    width_real = 6/(BW*2*pi);
    width=ceil((6/(BW*2*pi))/dt);
    delay=6*width;

    printf("delay = %f \t width = %f\n", delay, width);
  // f_0=k_rho*c0/(2*pi)+BW/1.3;
  // f_0;
  //f_0=k_rho*c0/(2*pi)+BW/2;

  printf("f_0 = %e\n",f_0);
	//f_0 = 6283200.000000*c0/(2*pi)+BW/2;
   // f_0=1e10;
    f_min=k_rho*c0/(2*pi);


    //first order Mur ABC buffers
    e1=e2=0;

}

// void TFSF_CORRECT(void){
//
//    UPDATE_h_inc();
//    // printf("HERE1");
//
//   if(PBC_CTW == 0){
//     if(Periodic_XY==1) {
//
//       CORRECT_Z();
//     }
//     else{
//       CORRECT_X();
//
//       CORRECT_Y();
//
//       CORRECT_Z();
//
//     //  printf("Here");
//
//     }
//
//   }
//
// else{
//   // SOURCE_IN_B();
//   // SOURCE_IN_E();
//   SOURCE_IN();
// }
//     UPDATE_e_inc();
//     //SOURCE_IN();
//
// }
//update 1-D incident fields
//
// void UPDATE_e_inc(void){
//     real pulse;
//     real factor = c0*dt;
//     int i,m;
//
//     //1st Order Mur ABC buffers
//     e1=e_inc[1];
//
//     e2=e_inc[inc_Length-2];
//
//     //#pragma omp parallel for private(i)// schedule(guided)
//     for(i=1;i<inc_Length-1;i++){
//         e_inc[i]=e_inc[i]-(dt/(z0*ep0*d_1D))*(h_inc[i]-h_inc[i-1]);
//     }
//   //  printf("%f\t%f\n",(dt/(z0*ep0*d_1D))*(h_inc[i]-h_inc[i-1]),((c0*dt-d_1D)/(c0*dt+d_1D)));
//     //1st order Mur ABC
//
//     e_inc[0]=e1+((factor-d_1D)/(factor+d_1D))*(e_inc[1]-e_inc[0]);
//
//     e_inc[inc_Length-1]=e2+((factor-d_1D)/(factor+d_1D))*(e_inc[inc_Length-2]-e_inc[inc_Length-1]);
//
//     //introduce source
//     #ifdef DOUBLECOMPLEX
//     pulse=exp(-pow((real)(t-delay)/(real)width,2)/2.0)*cexp(I*2*pi*f_0*(t)*dt);
//     #endif
//     #ifdef DOUBLEPRECISION
//     pulse=exp(-pow((real)(t-delay)/(real)width,2)/2.0)*sin(2*pi*f_0*(t)*dt);
//     #endif
//     for(m=0;m<NUM_freq;m++){
//       E_incident[m] += pulse*cexp(I*2*pi*t*dt*freq[m]);
//     }
//     e_inc[m0-50]+=pulse;
//
//   //  printf("%f\n",pulse);
// }
//
// void UPDATE_h_inc(void){
//     int i;
//
// //    //#pragma omp parallel for private(i)
//     for(i=0;i<inc_Length-1;i++){
//         h_inc[i]=h_inc[i]-(z0*dt/(mu0*d_1D))*(e_inc[i+1]-e_inc[i]);
//     }
// }
//
// //correcting
// void CORRECT_Y(void){
//  int i,j,k;
//  real d,d_prime,d_2_prime,e_inc_d,h_inc_d,e_x_inc,e_z_inc,h_z_inc,h_x_inc;
// ////#pragma omp parallel for collapse(2)
//  for(i=NtfsfX;i<=NCELLX-NtfsfX;i++){
//     for(k=NtfsfZ;k<=NCELLZ-NtfsfZ;k++){
//         j=NtfsfY;
//         d=(i-i_0)*sin(inc_theta)*cos(inc_phi)+(j-j_0)*sin(inc_theta)*sin(inc_phi)+(k+0.5-k_0)*cos(inc_theta);
//         d=(dx/d_1D)*d;
//         d_prime=d-(int)d;
//         e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
//
//         e_z_inc=e_inc_d*(sin(polar_psi)*sin(inc_theta));
//
//         if(k != NCELLZ-NtfsfZ){
//            hx[ThreeDMap(i,j-1,k,NCELLZ,NCELLY)]+=Chxe[ThreeDMap(i,j-1,k,NCELLZ,NCELLY)]*e_z_inc/dy;
//         }
//
//         j=NtfsfY;
//         d=(i-i_0+0.5)*sin(inc_theta)*cos(inc_phi)+(j-j_0)*sin(inc_theta)*sin(inc_phi)+(k-k_0)*cos(inc_theta);
//         d=(dx/d_1D)*d;
//         d_prime=d-(int)d;
//         e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
//
//         e_x_inc=e_inc_d*(cos(polar_psi)*sin(inc_phi)-sin(polar_psi)*cos(inc_theta)*cos(inc_phi));
//
//          if(i != NCELLX-NtfsfX ){
//            hz[ThreeDMap(i,j-1,k,NCELLZ,NCELLY)]-=Chze[ThreeDMap(i,j-1,k,NCELLZ,NCELLY)]*e_x_inc/dy;
//         }
//
//         j=NtfsfY;
//         d=(i+0.5-i_0)*sin(inc_theta)*cos(inc_phi)+(j-0.5-j_0)*sin(inc_theta)*sin(inc_phi)+(k-k_0)*cos(inc_theta);
//         d=(dx/d_1D)*d;
//         d_2_prime=d+0.5;
//         d_prime=d_2_prime-(int)(d_2_prime);
//         h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];
//
//         h_z_inc=h_inc_d*(-cos(polar_psi)*sin(inc_theta));
//
//         if(i != NCELLX-NtfsfX){
//             ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*h_z_inc/dy;
//         }
//
//
//         j=NtfsfY;
//         d=(i-i_0)*sin(inc_theta)*cos(inc_phi)+(j-0.5-j_0)*sin(inc_theta)*sin(inc_phi)+(k+0.5-k_0)*cos(inc_theta);
//         d=(dx/d_1D)*d;
//         d_2_prime=d+0.5;
//         d_prime=d_2_prime-(int)(d_2_prime);
//         h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];
//
//         h_x_inc=h_inc_d*(sin(polar_psi)*sin(inc_phi)+cos(polar_psi)*cos(inc_theta)*cos(inc_phi));
//
//         if(k != NCELLZ-NtfsfZ){
//             ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]+=Cezh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*h_x_inc/dy;
//         }
//
//
//         j=NCELLY-NtfsfY;
//         d=(i-i_0)*sin(inc_theta)*cos(inc_phi)+(j-j_0)*sin(inc_theta)*sin(inc_phi)+(k+0.5-k_0)*cos(inc_theta);
//         d=(dx/d_1D)*d;
//         d_prime=d-(int)d;
//         e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
//
//         e_z_inc=e_inc_d*(sin(polar_psi)*sin(inc_theta));
//
//         if(k != NCELLZ-NtfsfZ){
//            hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-=Chxe[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*e_z_inc/dy;
//         }
//
//
//         j=NCELLY-NtfsfY;
//         d=(i+0.5-i_0)*sin(inc_theta)*cos(inc_phi)+(j-j_0)*sin(inc_theta)*sin(inc_phi)+(k-k_0)*cos(inc_theta);
//         d=(dx/d_1D)*d;
//         d_prime=d-(int)d;
//         e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
//
//         e_x_inc=e_inc_d*(cos(polar_psi)*sin(inc_phi)-sin(polar_psi)*cos(inc_theta)*cos(inc_phi));
//
//         if(i != NCELLX-NtfsfX ){
//            hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]+=Chze[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*e_x_inc/dy;
//         }
//
//
//         j=NCELLY-NtfsfY;
//         d=(i+0.5-i_0)*sin(inc_theta)*cos(inc_phi)+(j+0.5-j_0)*sin(inc_theta)*sin(inc_phi)+(k-k_0)*cos(inc_theta);
//         d=(dx/d_1D)*d;
//         d_2_prime=d+0.5;
//         d_prime=d_2_prime-(int)(d_2_prime);
//         h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];
//
//         h_z_inc=h_inc_d*(-cos(polar_psi)*sin(inc_theta));
//
//         if(i != NCELLX-NtfsfX){
//             ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]+=Cexh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*h_z_inc/dy;
//         }
//
//         j=NCELLY-NtfsfY;
//         d=(i-i_0)*sin(inc_theta)*cos(inc_phi)+(j+0.5-j_0)*sin(inc_theta)*sin(inc_phi)+(k+0.5-k_0)*cos(inc_theta);
//         d=(dx/d_1D)*d;
//         d_2_prime=d+0.5;
//         d_prime=d_2_prime-(int)(d_2_prime);
//         h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];
//
//         h_x_inc=h_inc_d*(sin(polar_psi)*sin(inc_phi)+cos(polar_psi)*cos(inc_theta)*cos(inc_phi));
//
//         if(k != NCELLZ-NtfsfZ){
//             ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-=Cezh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*h_x_inc/dy;
//         }
//
//     }
//  }
// }
//
// void CORRECT_X(void){
//  int i,j,k;
//  real d,d_prime,d_2_prime,e_inc_d,h_inc_d,e_y_inc,e_z_inc,h_z_inc,h_y_inc;
//  ////#pragma omp parallel for collapse(2)
//  for(j=NtfsfY;j<=NCELLY-NtfsfY;j++){
//         for(k=NtfsfZ;k<=NCELLZ-NtfsfZ; k++){
//
//             i=NtfsfX;
//             d=(i-i_0)*sin(inc_theta)*cos(inc_phi)+(j-j_0)*sin(inc_theta)*sin(inc_phi)+(k+0.5-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_prime=d-(int)d;
//             e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
//
//             e_z_inc=e_inc_d*(sin(polar_psi)*sin(inc_theta));
//
//             if(k != NCELLZ-NtfsfZ){
//                 hy[ThreeDMap(i-1,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMap(i-1,j,k,NCELLZ,NCELLY)]*e_z_inc/dx;
//             }
//
//
//             i=NtfsfX;
//             d=(i-i_0)*sin(inc_theta)*cos(inc_phi)+(j+0.5-j_0)*sin(inc_theta)*sin(inc_phi)+(k-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_prime=d-(int)d;
//             e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
//
//             e_y_inc=e_inc_d*(-cos(polar_psi)*cos(inc_phi)-sin(polar_psi)*cos(inc_theta)*sin(inc_phi));
//
//             if(j != NCELLY-NtfsfY ){
//                 hz[ThreeDMap(i-1,j,k,NCELLZ,NCELLY)]+=Chze[ThreeDMap(i-1,j,k,NCELLZ,NCELLY)]*e_y_inc/dx;
//             }
//
//
//             i=NtfsfX;
//             d=(i-0.5-i_0)*sin(inc_theta)*cos(inc_phi)+(j+0.5-j_0)*sin(inc_theta)*sin(inc_phi)+(k-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_2_prime=d+0.5;
//             d_prime=d_2_prime-(int)(d_2_prime);
//             h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];
//
//             h_z_inc=h_inc_d*(-cos(polar_psi)*sin(inc_theta));
//
//             if(j != NCELLY-NtfsfY){
//                 ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]+=Cexh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*h_z_inc/dx;
//             }
//
//
//             i=NtfsfX;
//             d=(i-0.5-i_0)*sin(inc_theta)*cos(inc_phi)+(j-j_0)*sin(inc_theta)*sin(inc_phi)+(k+0.5-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_2_prime=d+0.5;
//             d_prime=d_2_prime-(int)(d_2_prime);
//             h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];
//
//             h_y_inc=h_inc_d*(-sin(polar_psi)*cos(inc_phi)+cos(polar_psi)*cos(inc_theta)*sin(inc_phi));
//
//             if(k != NCELLZ-NtfsfZ){
//                 ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-=Cezh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*h_y_inc/dx;
//             }
//
//
//
//             i=NCELLX-NtfsfX;
//             d=(i-i_0)*sin(inc_theta)*cos(inc_phi)+(j-j_0)*sin(inc_theta)*sin(inc_phi)+(k+0.5-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_prime=d-(int)d;
//             e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
//
//             e_z_inc=e_inc_d*(sin(polar_psi)*sin(inc_theta));
//
//             if(k != NCELLZ-NtfsfZ){
//                 hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]+=Chye[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*e_z_inc/dx;
//             }
//
//
//             i=NCELLX-NtfsfX;
//             d=(i-i_0)*sin(inc_theta)*cos(inc_phi)+(j+0.5-j_0)*sin(inc_theta)*sin(inc_phi)+(k-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_prime=d-(int)d;
//             e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
//
//             e_y_inc=e_inc_d*(-cos(polar_psi)*cos(inc_phi)-sin(polar_psi)*cos(inc_theta)*sin(inc_phi));
//
//             if(j != NCELLY-NtfsfY ){
//                 hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-=Chze[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*e_y_inc/dx;
//             }
//
//
//             i=NCELLX-NtfsfX;
//             d=(i+0.5-i_0)*sin(inc_theta)*cos(inc_phi)+(j+0.5-j_0)*sin(inc_theta)*sin(inc_phi)+(k-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_2_prime=d+0.5;
//             d_prime=d_2_prime-(int)(d_2_prime);
//             h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];
//
//             h_z_inc=h_inc_d*(-cos(polar_psi)*sin(inc_theta));
//
//             if(j != NCELLY-NtfsfY){
//                 ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*h_z_inc/dx;
//             }
//
//
//             i=NCELLX-NtfsfX;
//             d=(i+0.5-i_0)*sin(inc_theta)*cos(inc_phi)+(j-j_0)*sin(inc_theta)*sin(inc_phi)+(k+0.5-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_2_prime=d+0.5;
//             d_prime=d_2_prime-(int)(d_2_prime);
//             h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];
//
//             h_y_inc=h_inc_d*(-sin(polar_psi)*cos(inc_phi)+cos(polar_psi)*cos(inc_theta)*sin(inc_phi));
//
//             if(k != NCELLZ-NtfsfZ){
//                 ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]+=Cezh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*h_y_inc/dx;
//             }
//
//     }
//  }
// }
//
// void CORRECT_Z(void){
//  int i,j,k;
//  real d,d_prime,d_2_prime,e_inc_d,h_inc_d,e_y_inc,e_x_inc,h_x_inc,h_y_inc;
// // //#pragma omp parallel for collapse(2)
//  for(i=NtfsfX;i<=NCELLX-NtfsfX;i++){
//     for(j=NtfsfY;j<=NCELLY-NtfsfY;j++){
// //    if(t==2001)printf("%d\t%d\n",i,j);
//             k=NtfsfZ;
//             d=(i-i_0)*sin(inc_theta)*cos(inc_phi)+(j+0.5-j_0)*sin(inc_theta)*sin(inc_phi)+(k-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_prime=d-(int)d;
//           //  if(t==2001) printf("%f,%d",d,(int)d);
//
//             e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
//             e_y_inc=e_inc_d*(-cos(polar_psi)*cos(inc_phi)-sin(polar_psi)*cos(inc_theta)*sin(inc_phi));
//
//             if(j != NCELLY-NtfsfY){
//                 hx[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]-=Chxe[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]*e_y_inc/dz;
//             }
//
//
//             k=NtfsfZ;
//             d=(i+0.5-i_0)*sin(inc_theta)*cos(inc_phi)+(j-j_0)*sin(inc_theta)*sin(inc_phi)+(k-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_prime=d-(int)d;
//             e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
//
//             e_x_inc=e_inc_d*(cos(polar_psi)*sin(inc_phi)-sin(polar_psi)*cos(inc_theta)*cos(inc_phi));
//
//             if(i != NCELLX-NtfsfX ){
//                 hy[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]+=Chye[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]*e_x_inc/dz;
//             }
//
//
//
//             k=NtfsfZ;
//             d=(i-i_0)*sin(inc_theta)*cos(inc_phi)+(j+0.5-j_0)*sin(inc_theta)*sin(inc_phi)+(k-0.5-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_2_prime=d+0.5;
//             d_prime=d_2_prime-(int)(d_2_prime);
//             h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];
//
//             h_x_inc=h_inc_d*(sin(polar_psi)*sin(inc_phi)+cos(polar_psi)*cos(inc_theta)*cos(inc_phi));
//
//             if(j != NCELLY-NtfsfY){
//                 ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*h_x_inc/dz;
//             }
//
//
//             k=NtfsfZ;
//             d=(i+0.5-i_0)*sin(inc_theta)*cos(inc_phi)+(j-j_0)*sin(inc_theta)*sin(inc_phi)+(k-0.5-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_2_prime=d+0.5;
//             d_prime=d_2_prime-(int)(d_2_prime);
//             h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];
//
//             h_y_inc=h_inc_d*(-sin(polar_psi)*cos(inc_phi)+cos(polar_psi)*cos(inc_theta)*sin(inc_phi));
//
//             if(i != NCELLX-NtfsfX ){
//                 ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]+=Cexh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*h_y_inc/dz;
//             }
//
// if(Periodic_XY == 0){
//             k=NCELLZ-NtfsfZ;
//             d=(i-i_0)*sin(inc_theta)*cos(inc_phi)+(j+0.5-j_0)*sin(inc_theta)*sin(inc_phi)+(k-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_prime=d-(int)d;
//             e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
//
//             e_y_inc=e_inc_d*(-cos(polar_psi)*cos(inc_phi)-sin(polar_psi)*cos(inc_theta)*sin(inc_phi));
//
//             if(j != NCELLY-NtfsfY){
//                 hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]+=Chxe[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*e_y_inc/dz;
//             }
//             if(i != NCELLX-NtfsfX ){
//                 hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*e_x_inc/dz;
//             }
//
//
//             k=NCELLZ-NtfsfZ;
//             d=(i+0.5-i_0)*sin(inc_theta)*cos(inc_phi)+(j-j_0)*sin(inc_theta)*sin(inc_phi)+(k-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_prime=d-(int)d;
//             e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
//
//             e_x_inc=e_inc_d*(cos(polar_psi)*sin(inc_phi)-sin(polar_psi)*cos(inc_theta)*cos(inc_phi));
//
//             if(i != NCELLX-NtfsfX ){
//                 hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*e_x_inc/dz;
//             }
//
//
//             k=NCELLZ-NtfsfZ;
//             d=(i-i_0)*sin(inc_theta)*cos(inc_phi)+(j+0.5-j_0)*sin(inc_theta)*sin(inc_phi)+(k+0.5-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_2_prime=d+0.5;
//             d_prime=d_2_prime-(int)(d_2_prime);
//             h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];
//
//             h_x_inc=h_inc_d*(sin(polar_psi)*sin(inc_phi)+cos(polar_psi)*cos(inc_theta)*cos(inc_phi));
//
//             if(j != NCELLY-NtfsfY){
//                 ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]+=Cexh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*h_x_inc/dz;
//             }
//
//
//             k=NCELLZ-NtfsfZ;
//             d=(i+0.5-i_0)*sin(inc_theta)*cos(inc_phi)+(j-j_0)*sin(inc_theta)*sin(inc_phi)+(k+0.5-k_0)*cos(inc_theta);
//             d=(dx/d_1D)*d;
//             d_2_prime=d+0.5;
//             d_prime=d_2_prime-(int)(d_2_prime);
//             h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];
//
//             h_y_inc=h_inc_d*(-sin(polar_psi)*cos(inc_phi)+cos(polar_psi)*cos(inc_theta)*sin(inc_phi));
//
//              if(i != NCELLX-NtfsfX ){
//                 ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*h_y_inc/dz;
//             }
// }
//     }
//  }
// }
//
// void SOURCE_IN(void){
//   int i,j,k;
//   k=inc_plane;
//
//   ////#pragma omp parallel for collapse(2) private(i,j) schedule(guided)
//   for(i=0;i<NCELLX-1;i++){
//        for(j=0;j<NCELLY-1;j++){
//            if(TEz){
//              //hy[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)] += (Chye[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]/dz)*PULSE(t)*cexp(-I*(i+0.5)*dx*k_x)*cexp(-I*(j)*dy*k_y);
//              hx[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)] -= (Chxe[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]/dz)*PULSE(t)*cexp(-I*(i)*dx*k_x)*cexp(-I*(j+0.5)*dy*k_y);
//
//            }
//            if(TMz){
//              ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)] -= (Cexh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]/dz)*PULSE(t)*cexp(-I*(i+0.5)*dx*k_x)*cexp(-I*(j)*dy*k_y);
//            }
//
//        }
//
//    }
// }
//
// void SOURCE_IN_B(void){
//    int i,j,k;
//    k=inc_plane;
//
//    ////#pragma omp parallel for collapse(2) private(i,j) schedule(guided)
//    for(i=0;i<NCELLX;i++){
//         for(j=0;j<NCELLY;j++){
//             if(TEz){
//               hy[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)] += (Chye[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]/dz)*Ex_t_FT[t]*cexp(-I*(i+0.5)*dx*k_x)*cexp(-I*(j)*dy*k_y);
//               hx[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)] -= (Chxe[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]/dz)*Ey_t_FT[t]*cexp(-I*(i)*dx*k_x)*cexp(-I*(j+0.5)*dy*k_y);
//
//             }
//             if(TMz){
//               hy[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)] -= (Chye[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]/dz)*Ex_t_FT[t]*cexp(-I*(i+0.5)*dx*k_x)*cexp(-I*(j)*dy*k_y);
//             }
//
//         }
//
//     }
// }
//
// void SOURCE_IN_E(void){
//   int i,j,k;
//   k=inc_plane;
//
//
//   ////#pragma omp parallel for collapse(2) private(i,j) schedule(guided)
//   for(i=0;i<NCELLX;i++){
//        for(j=0;j<NCELLY;j++){
//            if(TEz){
//               ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)] -= (Ceyh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]/dz)*0.5*(Hx_t_FT[t]+Hx_t_FT[t+1])*cexp(-I*(i)*dx*k_x)*cexp(-I*(j+0.5)*dy*k_y);
//            }
//            if(TMz){
//               ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)] += (Ceyh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]/dz)*0.5*(Hx_t_FT[t+1]+Hx_t_FT[t])*cexp(-I*(i)*dx*k_x)*cexp(-I*(j+0.5)*dy*k_y);
//               ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)] += (Cexh[ThreeDMap(i,j,k,NCELLZ,NCELLY)]/dz)*0.5*(Hy_t_FT[t+1]+Hy_t_FT[t])*cexp(-I*(i+0.5)*dx*k_x)*cexp(-I*(j)*dy*k_y);
//            }
//
//        }
//
//    }
// }
//
// comp PULSE(int t){
//   comp pulse;
//   //pulse=exp(-pow((real)(t-delay)/(real)width,2)/2.0)*cexp(I*2*pi*f_0*(t)*dt);
//   if(t < 3*(delay+2*width)) pulse = exp(-pow((real)(t-delay)/(real)width,2)/2.0)*cos(2*pi*f_0*(t)*dt);
//   else pulse = 0;
//   // if(t>3*(delay+2*width)){
//   //   return 0;
//   // }
// return (pulse);
// }
