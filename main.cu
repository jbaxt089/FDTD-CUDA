#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "global_var.h"
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>
// #include <>
//#include<conio.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
// #include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <cudaProfiler.h>
__device__  int ThreeDMapD(int i,int j,int k,int SizeZ,int SizeY){
 int num = k + SizeZ*j +SizeY*SizeZ*i;
 return num;
}


__device__  int FourDMapD(int i,int j,int k,int n,int SizeN,int SizeZ,int SizeY){
 int num = n + SizeN*( k + SizeZ*j +SizeY*SizeZ*i);
 return num;
}

__device__  int TwoDMapD(int i,int j,int size){
 int num = j + i*size;
 return num;
}




    __global__ void ScattAbs(real *ex,real *ey,real *ez,real *hx,real *hy,real *hz,int NUM_freq,int t,real dt,real* freq,real pi,int XSTARTAbs,int XENDAbs,int YSTARTAbs,int YENDAbs,int ZSTARTAbs,int ZENDAbs,int XSTARTSca,int XENDSca,int YSTARTSca,int YENDSca,int ZSTARTSca,int ZENDSca,int XNEARAbs,int XFARAbs,int YNEARAbs,int YFARAbs,int ZNEARAbs,int ZFARAbs,
    int XNEARSca,int XFARSca,int YNEARSca,int YFARSca,int ZNEARSca,int ZFARSca,real2 *ExTransformNearZAbsRe,real2 *ExTransformNearZAbsIm,real2 *EyTransformNearZAbsRe,real2 *EyTransformNearZAbsIm,real2 *HxTransformNearZAbsRe,real2 *HxTransformNearZAbsIm,real2 *HyTransformNearZAbsRe,real2 *HyTransformNearZAbsIm,
    real2 *ExTransformFarZAbsRe,real2 *ExTransformFarZAbsIm,real2 *EyTransformFarZAbsRe,real2 *EyTransformFarZAbsIm,real2 *HxTransformFarZAbsRe,real2 *HxTransformFarZAbsIm,real2 *HyTransformFarZAbsRe,real2 *HyTransformFarZAbsIm,
    real2 *ExTransformNearYAbsRe,real2 *ExTransformNearYAbsIm,real2 *EzTransformNearYAbsRe,real2 *EzTransformNearYAbsIm,real2 *HxTransformNearYAbsRe,real2 *HxTransformNearYAbsIm,real2 *HzTransformNearYAbsRe,real2 *HzTransformNearYAbsIm,
    real2 *ExTransformFarYAbsRe,real2 *ExTransformFarYAbsIm,real2 *EzTransformFarYAbsRe,real2 *EzTransformFarYAbsIm,real2 *HxTransformFarYAbsRe,real2 *HxTransformFarYAbsIm,real2 *HzTransformFarYAbsRe,real2 *HzTransformFarYAbsIm,
    real2 *EyTransformNearXAbsRe,real2 *EyTransformNearXAbsIm,real2 *EzTransformNearXAbsRe,real2 *EzTransformNearXAbsIm,real2 *HyTransformNearXAbsRe,real2 *HyTransformNearXAbsIm,real2 *HzTransformNearXAbsRe,real2 *HzTransformNearXAbsIm,
    real2 *EyTransformFarXAbsRe,real2 *EyTransformFarXAbsIm,real2 *EzTransformFarXAbsRe,real2 *EzTransformFarXAbsIm,real2 *HyTransformFarXAbsRe,real2 *HyTransformFarXAbsIm,real2 *HzTransformFarXAbsRe,real2 *HzTransformFarXAbsIm,
    real2 *ExTransformNearZScaRe,real2 *ExTransformNearZScaIm,real2 *EyTransformNearZScaRe,real2 *EyTransformNearZScaIm,real2 *HxTransformNearZScaRe,real2 *HxTransformNearZScaIm,real2 *HyTransformNearZScaRe,real2 *HyTransformNearZScaIm,
    real2 *ExTransformFarZScaRe,real2 *ExTransformFarZScaIm,real2 *EyTransformFarZScaRe,real2 *EyTransformFarZScaIm,real2 *HxTransformFarZScaRe,real2 *HxTransformFarZScaIm,real2 *HyTransformFarZScaRe,real2 *HyTransformFarZScaIm,
    real2 *ExTransformNearYScaRe,real2 *ExTransformNearYScaIm,real2 *EzTransformNearYScaRe,real2 *EzTransformNearYScaIm,real2 *HxTransformNearYScaRe,real2 *HxTransformNearYScaIm,real2 *HzTransformNearYScaRe,real2 *HzTransformNearYScaIm,
    real2 *ExTransformFarYScaRe,real2 *ExTransformFarYScaIm,real2 *EzTransformFarYScaRe,real2 *EzTransformFarYScaIm,real2 *HxTransformFarYScaRe,real2 *HxTransformFarYScaIm,real2 *HzTransformFarYScaRe,real2 *HzTransformFarYScaIm,
    real2 *EyTransformNearXScaRe,real2 *EyTransformNearXScaIm,real2 *EzTransformNearXScaRe,real2 *EzTransformNearXScaIm,real2 *HyTransformNearXScaRe,real2 *HyTransformNearXScaIm,real2 *HzTransformNearXScaRe,real2 *HzTransformNearXScaIm,
    real2 *EyTransformFarXScaRe,real2 *EyTransformFarXScaIm,real2 *EzTransformFarXScaRe,real2 *EzTransformFarXScaIm,real2 *HyTransformFarXScaRe,real2 *HyTransformFarXScaIm,real2 *HzTransformFarXScaRe,real2 *HzTransformFarXScaIm,int NCELLX,int NCELLY,int NCELLZ){


    int freq_count,i,j,k,II,JJ,KK;
    real TransVecERe;
    real TransVecHRe;
    real TransVecEIm;
    real TransVecHIm;

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
  //  freq_count = blockIdx.y;

    i = idx / (NCELLZ*NCELLY);
    j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
    k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
    for(freq_count=0;freq_count<NUM_freq;freq_count++){
    //if(freq_count < NUM_freq){
    #ifdef FlOATPRECISION
    TransVecERe = cosf(2.0*pi*(t+1.0)*dt*freq[freq_count]);
    TransVecEIm = sinf(2.0*pi*(t+1.0)*dt*freq[freq_count]);
    TransVecHRe = cosf(2.0*pi*(t+0.5)*dt*freq[freq_count]);
    TransVecHIm = sinf(2.0*pi*(t+0.5)*dt*freq[freq_count]);
    #endif

    #ifdef DOUBLEPRECISION
    TransVecERe = cos((real2)2.0*pi*(t+1.0)*dt*freq[freq_count]);
    TransVecEIm = sin((real2)2.0*pi*(t+1.0)*dt*freq[freq_count]);
    TransVecHRe = cos((real2)2.0*pi*(t+0.5)*dt*freq[freq_count]);
    TransVecHIm = sin((real2)2.0*pi*(t+0.5)*dt*freq[freq_count]);
    #endif
    //#pragma omp parallel for private(i,j,k,II,JJ,KK) collapse(2)
      // for(i=XSTARTAbs;i<XENDAbs;i++){
      //   for(j=YSTARTAbs;j<YENDAbs;j++){
          if(i>=XSTARTAbs && i<XENDAbs && j>=YSTARTAbs && j<YENDAbs && k == ZNEARAbs){
        //  printf("%d\t%d\t%d\t%d\n",i,j,XENDAbs,YENDAbs);
          //k = ZNEARAbs;
          II = i - XSTARTAbs;
          JJ = j - YSTARTAbs;

          ExTransformNearZAbsRe[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2) ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EyTransformNearZAbsRe[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
        //  EzTransformNearZAbsRe[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;

          ExTransformNearZAbsIm[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EyTransformNearZAbsIm[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
      //    EzTransformNearZAbsIm[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;

          HxTransformNearZAbsRe[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HyTransformNearZAbsRe[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
        //  HzTransformNearZAbsRe[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;

          HxTransformNearZAbsIm[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HyTransformNearZAbsIm[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
      //    HzTransformNearZAbsIm[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
        }
          if(i>=XSTARTAbs && i<XENDAbs && j>=YSTARTAbs && j<YENDAbs && k == ZFARAbs){
          // k = ZFARAbs;
          II = i - XSTARTAbs;
          JJ = j - YSTARTAbs;

          ExTransformFarZAbsRe[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EyTransformFarZAbsRe[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
      //    EzTransformFarZAbsRe[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;

          ExTransformFarZAbsIm[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] +=(real2) ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EyTransformFarZAbsIm[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
    //      EzTransformFarZAbsIm[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;

          HxTransformFarZAbsRe[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HyTransformFarZAbsRe[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
      //    HzTransformFarZAbsRe[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;

          HxTransformFarZAbsIm[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HyTransformFarZAbsIm[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
      //    HzTransformFarZAbsIm[ThreeDMapD(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
    }

      //#pragma omp parallel for private(i,j,k,II,JJ,KK) collapse(2)
      // for(i=XSTARTAbs;i<XENDAbs;i++){
      //   for(k=ZSTARTAbs;k<ZENDAbs;k++){
          if(i>=XSTARTAbs && i<XENDAbs && k>=ZSTARTAbs && k<ZENDAbs && j == YNEARAbs){

          II = i - XSTARTAbs;
          KK = k - ZSTARTAbs;

          // j=YNEARAbs;

          ExTransformNearYAbsRe[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
      //    EyTransformNearYAbsRe[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EzTransformNearYAbsRe[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;

          ExTransformNearYAbsIm[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
    //      EyTransformNearYAbsIm[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EzTransformNearYAbsIm[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] +=(real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;

          HxTransformNearYAbsRe[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] +=(real2) hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
      //    HyTransformNearYAbsRe[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HzTransformNearYAbsRe[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] +=(real2) hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;

          HxTransformNearYAbsIm[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
      //    HyTransformNearYAbsIm[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HzTransformNearYAbsIm[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
        }

        if(i>=XSTARTAbs && i<XENDAbs && k>=ZSTARTAbs && k<ZENDAbs && j == YFARAbs){
          II = i - XSTARTAbs;
          KK = k - ZSTARTAbs;
          // j=YFARAbs;

          ExTransformFarYAbsRe[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
    //      EyTransformFarYAbsRe[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EzTransformFarYAbsRe[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;

          ExTransformFarYAbsIm[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
    //      EyTransformFarYAbsIm[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EzTransformFarYAbsIm[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;

          HxTransformFarYAbsRe[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
    //      HyTransformFarYAbsRe[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HzTransformFarYAbsRe[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;

          HxTransformFarYAbsIm[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
      //    HyTransformFarYAbsIm[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HzTransformFarYAbsIm[ThreeDMapD(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += (real2)hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
        }
    //  }

      //#pragma omp parallel for private(i,j,k,II,JJ,KK) collapse(2)
      if(j>=YSTARTAbs && j<YENDAbs && k>=ZSTARTAbs && k<ZENDAbs && i == XNEARAbs){

      // for(j=YSTARTAbs;j<YENDAbs;j++){
      //   for(k=ZSTARTAbs;k<ZENDAbs;k++){
          KK = k - ZSTARTAbs;
          JJ = j - YSTARTAbs;

          // i=XNEARAbs;

      //    ExTransformNearXAbsRe[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EyTransformNearXAbsRe[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] +=(real2) ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EzTransformNearXAbsRe[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;

    //        ExTransformNearXAbsIm[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EyTransformNearXAbsIm[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EzTransformNearXAbsIm[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;

    //      HxTransformNearXAbsRe[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HyTransformNearXAbsRe[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] +=(real2) hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HzTransformNearXAbsRe[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;

      //    HxTransformNearXAbsIm[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HyTransformNearXAbsIm[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HzTransformNearXAbsIm[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
        }

          if(j>=YSTARTAbs && j<YENDAbs && k>=ZSTARTAbs && k<ZENDAbs && i == XFARAbs){
            KK = k - ZSTARTAbs;
            JJ = j - YSTARTAbs;
          // i=XFARAbs;

      //    ExTransformFarXAbsRe[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EyTransformFarXAbsRe[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EzTransformFarXAbsRe[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;

    //      ExTransformFarXAbsIm[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EyTransformFarXAbsIm[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EzTransformFarXAbsIm[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;

      //    HxTransformFarXAbsRe[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HyTransformFarXAbsRe[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] +=(real2) hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HzTransformFarXAbsRe[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;

        //  HxTransformFarXAbsIm[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HyTransformFarXAbsIm[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HzTransformFarXAbsIm[ThreeDMapD(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += (real2)hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
        }
    //  }

    if(i>=XSTARTSca && i<XENDSca && j>=YSTARTSca && j<YENDSca && k == ZNEARSca){
      // for(i=XSTARTSca;i<XENDSca;i++){
      //   for(j=YSTARTSca;j<YENDSca;j++){
      //     k = ZNEARSca;
          II = i - XSTARTSca;
          JJ = j - YSTARTSca;

          ExTransformNearZScaRe[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EyTransformNearZScaRe[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
      //    EzTransformNearZScaRe[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;

          ExTransformNearZScaIm[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EyTransformNearZScaIm[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
      //    EzTransformNearZScaIm[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;

          HxTransformNearZScaRe[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HyTransformNearZScaRe[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
      //    HzTransformNearZScaRe[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;

          HxTransformNearZScaIm[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HyTransformNearZScaIm[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
      //    HzTransformNearZScaIm[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
    }
    if(i>=XSTARTSca && i<XENDSca && j>=YSTARTSca && j<YENDSca && k == ZFARSca){
      II = i - XSTARTSca;
      JJ = j - YSTARTSca;
          // k = ZFARSca;

          ExTransformFarZScaRe[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EyTransformFarZScaRe[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
      //    EzTransformFarZScaRe[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;

          ExTransformFarZScaIm[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EyTransformFarZScaIm[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
      //    EzTransformFarZScaIm[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;

          HxTransformFarZScaRe[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HyTransformFarZScaRe[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
      //    HzTransformFarZScaRe[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;

          HxTransformFarZScaIm[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HyTransformFarZScaIm[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
      //    HzTransformFarZScaIm[ThreeDMapD(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
        }
    //  }

      //#pragma omp parallel for private(i,j,k,II,JJ,KK) collapse(2)
      if(i>=XSTARTSca && i<XENDSca && k>=ZSTARTSca && k<ZENDSca && j == YNEARSca){
      // for(i=XSTARTSca;i<XENDSca;i++){
      //   for(k=ZSTARTSca;k<ZENDSca;k++){

          // j=YNEARSca;
          II = i - XSTARTSca;
          KK = k - ZSTARTSca;
          ExTransformNearYScaRe[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
        //  EyTransformNearYScaRe[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EzTransformNearYScaRe[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;

          ExTransformNearYScaIm[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
      //    EyTransformNearYScaIm[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EzTransformNearYScaIm[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;

          HxTransformNearYScaRe[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
        //  HyTransformNearYScaRe[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HzTransformNearYScaRe[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;

          HxTransformNearYScaIm[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
      //    HyTransformNearYScaIm[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HzTransformNearYScaIm[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
    }
    if(i>=XSTARTSca && i<XENDSca && k>=ZSTARTSca && k<ZENDSca && j == YFARSca){

          // j=YFARSca;
          II = i - XSTARTSca;
          KK = k - ZSTARTSca;
          ExTransformFarYScaRe[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
      //    EyTransformFarYScaRe[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EzTransformFarYScaRe[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;

          ExTransformFarYScaIm[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
      //    EyTransformFarYScaIm[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EzTransformFarYScaIm[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;

          HxTransformFarYScaRe[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
      //    HyTransformFarYScaRe[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HzTransformFarYScaRe[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;

          HxTransformFarYScaIm[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
      //    HyTransformFarYScaIm[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HzTransformFarYScaIm[ThreeDMapD(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] +=(real2) hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
      //  }
      }

      //#pragma omp parallel for private(i,j,k,II,JJ,KK) collapse(2)
      if(j>=YSTARTSca && j<YENDSca && k>=ZSTARTSca && k<ZENDSca && i == XNEARSca){

      // for(j=YSTARTSca;j<YENDSca;j++){
      //   for(k=ZSTARTSca;k<ZENDSca;k++){
          KK = k - ZSTARTSca;
          JJ = j - YSTARTSca;
          // i=XNEARSca;

      //    ExTransformNearXScaRe[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EyTransformNearXScaRe[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] +=(real2) ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EzTransformNearXScaRe[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] +=(real2) ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;

    //        ExTransformNearXScaIm[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EyTransformNearXScaIm[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] +=(real2) ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EzTransformNearXScaIm[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;

    //        HxTransformNearXScaRe[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HyTransformNearXScaRe[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HzTransformNearXScaRe[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;

    //        HxTransformNearXScaIm[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HyTransformNearXScaIm[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HzTransformNearXScaIm[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
        }
          if(j>=YSTARTSca && j<YENDSca && k>=ZSTARTSca && k<ZENDSca && i == XFARSca){

          // i=XFARSca;
          KK = k - ZSTARTSca;
          JJ = j - YSTARTSca;
    //      ExTransformFarXScaRe[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EyTransformFarXScaRe[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;
          EzTransformFarXScaRe[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecERe;

      //    ExTransformFarXScaIm[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EyTransformFarXScaIm[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;
          EzTransformFarXScaIm[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecEIm;

      //    HxTransformFarXScaRe[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HyTransformFarXScaRe[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;
          HzTransformFarXScaRe[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHRe;

      //    HxTransformFarXScaIm[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HyTransformFarXScaIm[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
          HzTransformFarXScaIm[ThreeDMapD(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += (real2)hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*TransVecHIm;
        }
    //  }
    }
    return;
    }

//correcting
__global__ void CORRECT_Y(int NtfsfX,int NtfsfY,int NtfsfZ,int NCELLX,int NCELLY,int NCELLZ,real *e_inc,real *h_inc,real *ex,real *ey,real *ez,real *hx,real *hy,real *hz,real inc_theta,
  real inc_phi,real polar_psi,real polar_theta,real dx,real dy,real dz,real dt,int i_0,int j_0,int k_0,real d_1D,int m0,real *Cexe,real *Ceye,real *Ceze,real *Cexh,real *Ceyh,real *Cezh,real *Chxe,real *Chye,real *Chze,real *Chxh,real *Chyh,real *Chzh,int Periodic_XY){

 int i,j,k;

 int idx = blockDim.x * blockIdx.x + threadIdx.x;

 i = idx / (NCELLZ*NCELLY);
 j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
 k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
 real d,d_prime,d_2_prime,e_inc_d,h_inc_d,e_x_inc,e_z_inc,h_z_inc,h_x_inc;

////#pragma omp parallel for collapse(2)
 // for(i=NtfsfX;i<=NCELLX-NtfsfX;i++){
 //    for(k=NtfsfZ;k<=NCELLZ-NtfsfZ;k++){
      if(i>=NtfsfX && i<=NCELLX-NtfsfX && k>=NtfsfZ && k<=NCELLZ-NtfsfZ){
        // j=NtfsfY;
        if(j==NtfsfY){
        d=(i-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k+0.5-k_0)*cos((float)inc_theta);
        d=(dx/d_1D)*d;
        d_prime=d-(int)d;
        e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];

        e_z_inc=e_inc_d*(sin((float)polar_psi)*sin((float)inc_theta));

        if(k != NCELLZ-NtfsfZ){
           hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)]+=Chxe[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)]*e_z_inc/dy;
        }

        // j=NtfsfY;
        d=(i-i_0+0.5)*sin((float)inc_theta)*cos((float)inc_phi)+(j-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-k_0)*cos((float)inc_theta);
        d=(dx/d_1D)*d;
        d_prime=d-(int)d;
        e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];

        e_x_inc=e_inc_d*(cos((float)polar_psi)*sin((float)inc_phi)-sin((float)polar_psi)*cos((float)inc_theta)*cos((float)inc_phi));

         if(i != NCELLX-NtfsfX ){
           hz[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)]-=Chze[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)]*e_x_inc/dy;
        }

        // j=NtfsfY;
        d=(i+0.5-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-0.5-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-k_0)*cos((float)inc_theta);
        d=(dx/d_1D)*d;
        d_2_prime=d+0.5;
        d_prime=d_2_prime-(int)(d_2_prime);
        h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];

        h_z_inc=h_inc_d*(-cos((float)polar_psi)*sin((float)inc_theta));

        if(i != NCELLX-NtfsfX){
            ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*h_z_inc/dy;
        }


        // j=NtfsfY;
        d=(i-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-0.5-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k+0.5-k_0)*cos((float)inc_theta);
        d=(dx/d_1D)*d;
        d_2_prime=d+0.5;
        d_prime=d_2_prime-(int)(d_2_prime);
        h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];

        h_x_inc=h_inc_d*(sin((float)polar_psi)*sin((float)inc_phi)+cos((float)polar_psi)*cos((float)inc_theta)*cos((float)inc_phi));

        if(k != NCELLZ-NtfsfZ){
            ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*h_x_inc/dy;
        }

      }
      if(j == NCELLY-NtfsfY){
        // j=NCELLY-NtfsfY;
        d=(i-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k+0.5-k_0)*cos((float)inc_theta);
        d=(dx/d_1D)*d;
        d_prime=d-(int)d;
        e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];

        e_z_inc=e_inc_d*(sin((float)polar_psi)*sin((float)inc_theta));

        if(k != NCELLZ-NtfsfZ){
           hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*e_z_inc/dy;
        }


        // j=NCELLY-NtfsfY;
        d=(i+0.5-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-k_0)*cos((float)inc_theta);
        d=(dx/d_1D)*d;
        d_prime=d-(int)d;
        e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];

        e_x_inc=e_inc_d*(cos((float)polar_psi)*sin((float)inc_phi)-sin((float)polar_psi)*cos((float)inc_theta)*cos((float)inc_phi));

        if(i != NCELLX-NtfsfX ){
           hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*e_x_inc/dy;
        }


        // j=NCELLY-NtfsfY;
        d=(i+0.5-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j+0.5-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-k_0)*cos((float)inc_theta);
        d=(dx/d_1D)*d;
        d_2_prime=d+0.5;
        d_prime=d_2_prime-(int)(d_2_prime);
        h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];

        h_z_inc=h_inc_d*(-cos((float)polar_psi)*sin((float)inc_theta));

        if(i != NCELLX-NtfsfX){
            ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*h_z_inc/dy;
        }

        // j=NCELLY-NtfsfY;
        d=(i-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j+0.5-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k+0.5-k_0)*cos((float)inc_theta);
        d=(dx/d_1D)*d;
        d_2_prime=d+0.5;
        d_prime=d_2_prime-(int)(d_2_prime);
        h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];

        h_x_inc=h_inc_d*(sin((float)polar_psi)*sin((float)inc_phi)+cos((float)polar_psi)*cos((float)inc_theta)*cos((float)inc_phi));

        if(k != NCELLZ-NtfsfZ){
            ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*h_x_inc/dy;
        }
}
    // }
 }
}

__global__ void CORRECT_X(int NtfsfX,int NtfsfY,int NtfsfZ,int NCELLX,int NCELLY,int NCELLZ,real *e_inc,real *h_inc,real *ex,real *ey,real *ez,real *hx,real *hy,real *hz,real inc_theta,
  real inc_phi,real polar_psi,real polar_theta,real dx,real dy,real dz,real dt,int i_0,int j_0,int k_0,real d_1D,int m0,real *Cexe,real *Ceye,real *Ceze,real *Cexh,real *Ceyh,real *Cezh,real *Chxe,real *Chye,real *Chze,real *Chxh,real *Chyh,real *Chzh,int Periodic_XY){
 int i,j,k;
 ////#pragma omp parallel for collapse(2)
 int idx = blockDim.x * blockIdx.x + threadIdx.x;

 i = idx / (NCELLZ*NCELLY);
 j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
 k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
 real d,d_prime,d_2_prime,e_inc_d,h_inc_d,e_y_inc,e_z_inc,h_z_inc,h_y_inc;

 // for(j=NtfsfY;j<=NCELLY-NtfsfY;j++){
 //        for(k=NtfsfZ;k<=NCELLZ-NtfsfZ; k++){
          if(j>=NtfsfY && j<=NCELLY-NtfsfY && k>=NtfsfZ && k<=NCELLZ-NtfsfZ){
            if(i==NtfsfX){
            // i=NtfsfX;
            d=(i-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k+0.5-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_prime=d-(int)d;
            e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];

            e_z_inc=e_inc_d*(sin((float)polar_psi)*sin((float)inc_theta));

            if(k != NCELLZ-NtfsfZ){
                hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)]*e_z_inc/dx;
            }


            // i=NtfsfX;
            d=(i-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j+0.5-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_prime=d-(int)d;
            e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];

            e_y_inc=e_inc_d*(-cos((float)polar_psi)*cos((float)inc_phi)-sin((float)polar_psi)*cos((float)inc_theta)*sin((float)inc_phi));

            if(j != NCELLY-NtfsfY ){
                hz[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)]+=Chze[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)]*e_y_inc/dx;
            }


            // i=NtfsfX;
            d=(i-0.5-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j+0.5-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_2_prime=d+0.5;
            d_prime=d_2_prime-(int)(d_2_prime);
            h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];

            h_z_inc=h_inc_d*(-cos((float)polar_psi)*sin((float)inc_theta));

            if(j != NCELLY-NtfsfY){
                ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*h_z_inc/dx;
            }


            // i=NtfsfX;
            d=(i-0.5-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k+0.5-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_2_prime=d+0.5;
            d_prime=d_2_prime-(int)(d_2_prime);
            h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];

            h_y_inc=h_inc_d*(-sin((float)polar_psi)*cos((float)inc_phi)+cos((float)polar_psi)*cos((float)inc_theta)*sin((float)inc_phi));

            if(k != NCELLZ-NtfsfZ){
                ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*h_y_inc/dx;
            }


}
          if(i==NCELLX-NtfsfX){
            // i=NCELLX-NtfsfX;
            d=(i-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k+0.5-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_prime=d-(int)d;
            e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];

            e_z_inc=e_inc_d*(sin((float)polar_psi)*sin((float)inc_theta));

            if(k != NCELLZ-NtfsfZ){
                hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*e_z_inc/dx;
            }


            // i=NCELLX-NtfsfX;
            d=(i-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j+0.5-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_prime=d-(int)d;
            e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];

            e_y_inc=e_inc_d*(-cos((float)polar_psi)*cos((float)inc_phi)-sin((float)polar_psi)*cos((float)inc_theta)*sin((float)inc_phi));

            if(j != NCELLY-NtfsfY ){
                hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*e_y_inc/dx;
            }


            // i=NCELLX-NtfsfX;
            d=(i+0.5-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j+0.5-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_2_prime=d+0.5;
            d_prime=d_2_prime-(int)(d_2_prime);
            h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];

            h_z_inc=h_inc_d*(-cos((float)polar_psi)*sin((float)inc_theta));

            if(j != NCELLY-NtfsfY){
                ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*h_z_inc/dx;
            }


            // i=NCELLX-NtfsfX;
            d=(i+0.5-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k+0.5-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_2_prime=d+0.5;
            d_prime=d_2_prime-(int)(d_2_prime);
            h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];

            h_y_inc=h_inc_d*(-sin((float)polar_psi)*cos((float)inc_phi)+cos((float)polar_psi)*cos((float)inc_theta)*sin((float)inc_phi));

            if(k != NCELLZ-NtfsfZ){
                ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*h_y_inc/dx;
            }
}
    // }
 }
}

__global__ void CORRECT_Z(int NtfsfX,int NtfsfY,int NtfsfZ,int NCELLX,int NCELLY,int NCELLZ,real *e_inc,real *h_inc,real *ex,real *ey,real *ez,real *hx,real *hy,real *hz,real inc_theta,
  real inc_phi,real polar_psi,real polar_theta,real dx,real dy,real dz,real dt,int i_0,int j_0,int k_0,real d_1D,int m0,real *Cexe,real *Ceye,real *Ceze,real *Cexh,real *Ceyh,real *Cezh,real *Chxe,real *Chye,real *Chze,real *Chxh,real *Chyh,real *Chzh,int Periodic_XY){

 int i,j,k;
// //#pragma omp parallel for collapse(2)
 // for(i=NtfsfX;i<=NCELLX-NtfsfX;i++){
 //    for(j=NtfsfY;j<=NCELLY-NtfsfY;j++){

      int idx = blockDim.x * blockIdx.x + threadIdx.x;

      i = idx / (NCELLZ*NCELLY);
      j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
      k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
      real d,d_prime,d_2_prime,e_inc_d,h_inc_d,e_y_inc,e_x_inc,h_x_inc,h_y_inc;

       if(j>=NtfsfY && j<=NCELLY-NtfsfY && i>=NtfsfX && i<=NCELLX-NtfsfX){

//    if(t==2001)printf("%d\t%d\n",i,j);
          if(k==NtfsfZ){
            // k=NtfsfZ;
            d=(i-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j+0.5-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_prime=d-(int)d;
          //  if(t==2001) printf("%f,%d",d,(int)d);

            e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];
            e_y_inc=e_inc_d*(-cos((float)polar_psi)*cos((float)inc_phi)-sin((float)polar_psi)*cos((float)inc_theta)*sin((float)inc_phi));

            if(j != NCELLY-NtfsfY){
                hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]-=Chxe[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]*e_y_inc/dz;
            }


            // k=NtfsfZ;
            d=(i+0.5-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_prime=d-(int)d;
            e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];

            e_x_inc=e_inc_d*(cos((float)polar_psi)*sin((float)inc_phi)-sin((float)polar_psi)*cos((float)inc_theta)*cos((float)inc_phi));

            if(i != NCELLX-NtfsfX ){
                hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]+=Chye[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]*e_x_inc/dz;
            }



            // k=NtfsfZ;
            d=(i-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j+0.5-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-0.5-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_2_prime=d+0.5;
            d_prime=d_2_prime-(int)(d_2_prime);
            h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];

            h_x_inc=h_inc_d*(sin((float)polar_psi)*sin((float)inc_phi)+cos((float)polar_psi)*cos((float)inc_theta)*cos((float)inc_phi));

            if(j != NCELLY-NtfsfY){
                ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*h_x_inc/dz;
            }


            // k=NtfsfZ;
            d=(i+0.5-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-0.5-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_2_prime=d+0.5;
            d_prime=d_2_prime-(int)(d_2_prime);
            h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];

            h_y_inc=h_inc_d*(-sin((float)polar_psi)*cos((float)inc_phi)+cos((float)polar_psi)*cos((float)inc_theta)*sin((float)inc_phi));

            if(i != NCELLX-NtfsfX ){
                ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*h_y_inc/dz;
            }
          }

if(Periodic_XY == 0){
            if(k==NCELLZ-NtfsfZ){
            // k=NCELLZ-NtfsfZ;
            d=(i-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j+0.5-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_prime=d-(int)d;
            e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];

            e_y_inc=e_inc_d*(-cos((float)polar_psi)*cos((float)inc_phi)-sin((float)polar_psi)*cos((float)inc_theta)*sin((float)inc_phi));

            if(j != NCELLY-NtfsfY){
                hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*e_y_inc/dz;
            }
            // if(i != NCELLX-NtfsfX ){
            //     hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*e_x_inc/dz;
            // }


            // k=NCELLZ-NtfsfZ;
            d=(i+0.5-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_prime=d-(int)d;
            e_inc_d=(1-d_prime)*e_inc[m0+(int)d]+d_prime*e_inc[m0+(int)d+1];

            e_x_inc=e_inc_d*(cos((float)polar_psi)*sin((float)inc_phi)-sin((float)polar_psi)*cos((float)inc_theta)*cos((float)inc_phi));

            if(i != NCELLX-NtfsfX ){
                hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*e_x_inc/dz;
            }


            // k=NCELLZ-NtfsfZ;
            d=(i-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j+0.5-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k+0.5-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_2_prime=d+0.5;
            d_prime=d_2_prime-(int)(d_2_prime);
            h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];

            h_x_inc=h_inc_d*(sin((float)polar_psi)*sin((float)inc_phi)+cos((float)polar_psi)*cos((float)inc_theta)*cos((float)inc_phi));

            if(j != NCELLY-NtfsfY){
                ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*h_x_inc/dz;
            }


            // k=NCELLZ-NtfsfZ;
            d=(i+0.5-i_0)*sin((float)inc_theta)*cos((float)inc_phi)+(j-j_0)*sin((float)inc_theta)*sin((float)inc_phi)+(k+0.5-k_0)*cos((float)inc_theta);
            d=(dx/d_1D)*d;
            d_2_prime=d+0.5;
            d_prime=d_2_prime-(int)(d_2_prime);
            h_inc_d=(1-d_prime)*h_inc[m0-1+(int)d_2_prime]+d_prime*h_inc[m0+(int)d_2_prime];

            h_y_inc=h_inc_d*(-sin((float)polar_psi)*cos((float)inc_phi)+cos((float)polar_psi)*cos((float)inc_theta)*sin((float)inc_phi));

             if(i != NCELLX-NtfsfX ){
                ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*h_y_inc/dz;
            }
}
    }
 }
}



  __global__ void UPDATE_hx(real *hx,real *hxPrev,real *ez,real *ey,real *Chxh,real *Chxe,real *psi_Hx_z_N,real *psi_Hx_z_F,real *psi_Hx_y_N,real *psi_Hx_y_F,real *khdy,real
    *khdz,real *bh_z_N,real *bh_z_F,real *ch_z_N,real *ch_z_F,real *bh_y_N,real *bh_y_F,real *ch_y_N,real *ch_y_F,int NCELLX,int NCELLY,int NCELLZ,int Periodic_XY,real dx,real dy,real dz,real dt,int cpml_N_Z,int cpml_F_Z,int cpml_N_Y,int cpml_F_Y,int cpml_z_lim,int cpml_y_lim,int cpml_x_lim,int NcpmlZ,int NcpmlY,int Hydrodynamics){
//void UPDATE_hx(void){
// hipProfilerStart();

      int i,j,k,j2,k2;
      comp Curl_E;
      // __shared__ double Ey[110][110][110];
      // __shared__ double Ez[110][110][110];

      int idx = blockDim.x * blockIdx.x + threadIdx.x;

      i = idx / (NCELLZ*NCELLY);
      j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
      k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
      // Ey[i][j][k] = ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
      // Ez[i][j][k] = ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];


      if(Periodic_XY){
  	////#pragma omp parallel for collapse(3) private(i,j,k,Curl_E,j2,k2) // schedule(static)
  	// for(k=0;k<NCELLZ;k++){
  	// 	for(i=0;i<NCELLX;i++){
  	//         for(j=0;j<NCELLY;j++){

              if(i<NCELLX && j<NCELLY && k<NCELLZ){
  	           //     for(k=0;k<NCELLZ-1;k++){
  									 //if(i==1) printf("%d %d %d\n",i,j,k);
  		                	if(j==NCELLY-1){
  												#ifdef DOUBLECOMPLEX
  		                				Curl_E=(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k]-(ez[ThreeDMapD(i,0,k,NCELLZ,NCELLY)]*cexp(-I*k_y*period_y)-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j];
  												#endif
  												#ifdef DOUBLEPRECISION
  		                				Curl_E=(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k]-(ez[ThreeDMapD(i,0,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j];
  												#endif
  		                   	    hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chxh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;

  		                	}
  		                	 else{
  														Curl_E=(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k]-(ez[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j];
  		                   	    hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chxh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
  		                	}
  										 //Z-CPML
  										 if(k<cpml_N_Z && i<cpml_x_lim && j<cpml_y_lim){
  											 	//Near Z-PML
  											 		psi_Hx_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]=bh_z_N[k]*psi_Hx_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]+ch_z_N[k]*(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
  													hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hx_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)];
  										 }
  										 if(k>=cpml_F_Z && j<cpml_y_lim && i<cpml_x_lim){
  											 //Far Z-PML
  											 		k2 = k - cpml_F_Z;
  													psi_Hx_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]=bh_z_F[k2]*psi_Hx_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]+ch_z_F[k2]*(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
  													hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hx_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)];
  										 }
                     }
  	    //             }
  	    //     }
  	    // }
      }

      else{
  			//  //#pragma omp target device(0) MapD(Chxe[:NCELLX-1][:NCELLY-1][:NCELLZ-1],Chxh[:NCELLX-1][:NCELLY-1][:NCELLZ-1],ez[:NCELLX-1][:NCELLY-1][:NCELLZ-1],ey[:NCELLX-1][:NCELLY-1][:NCELLZ-1],khdy[:NCELLY-1],khdz[:NCELLZ-1],bh_z_N[:NcpmlZ-1],bh_z_F[:NcpmlZ-1],ch_z_N[:NcpmlZ-1],ch_z_F[:NcpmlZ-1],bh_y_N[:NcpmlY-1],bh_y_F[:NcpmlY-1],ch_y_N[:NcpmlY-1],ch_y_F[:NcpmlY-1]) 		MapD(tofrom:hx[:NCELLX-1][:NCELLY-1][:NCELLZ-1],psi_Hx_z_N[:NCELLX-1][:NCELLY-1][:cpml_N_Z-1],psi_Hx_z_F[:NCELLX-1][:NCELLY-1][:cpml_N_Z-1],psi_Hx_y_N[:NCELLX-1][:cpml_N_Y-1][:NCELLZ-1],psi_Hx_y_F[:NCELLX-1][:cpml_N_Y-1][:NCELLZ-1])
  			//  {
  			// //#pragma omp parallel for collapse(3) private(i,j,k,Curl_E,j2,k2) // schedule(static)
  	    // for(i=0;i<NCELLX;i++){
  	    //     for(j=0;j<NCELLY-1;j++){
  	    //             for(k=0;k<NCELLZ-1;k++){
                      if(i<NCELLX && j<(NCELLY-1) && k<(NCELLZ-1)){
                        if(Hydrodynamics >= 1) hxPrev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  	                    Curl_E=(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k]-(ez[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j];
  	                    hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chxh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
  											//Z-CPML
  											if(k<cpml_N_Z && i<cpml_x_lim && j<cpml_y_lim){
  												 //Near Z-PML
  													 psi_Hx_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]=bh_z_N[k]*psi_Hx_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]+ch_z_N[k]*(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
  													 hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hx_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)];
  											}
  											if(k>=cpml_F_Z && j<cpml_y_lim && i<cpml_x_lim){
  												//Far Z-PML
  													 k2 = k - cpml_F_Z;
  													 psi_Hx_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]=bh_z_F[k2]*psi_Hx_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]+ch_z_F[k2]*(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
  													 hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hx_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)];
  											}
  										   //Y- PML
  											if(j<cpml_N_Y && i<cpml_x_lim && j<cpml_y_lim){
  													 psi_Hx_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY)]=bh_y_N[j]*psi_Hx_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY)]+ch_y_N[j]*(ez[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dy;
  													 hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hx_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY)];
  											}
  											if(j>=cpml_F_Y && i<cpml_x_lim && k<cpml_z_lim){
  													j2 = j - cpml_F_Y;
  													psi_Hx_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY)]=bh_y_F[j2]*psi_Hx_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY)]+ch_y_F[j2]*(ez[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dy;
  													hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hx_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY)];
  											}
                      }
  	  //               }
  	  //       }
  	  //  // }
  		// }
    }
    // hipProfilerStop();

    return;
  }

  __global__   void UPDATE_hy(real *hy,real *hyPrev,real *ez,real *ex,real *Chyh,real *Chye,real *psi_Hy_z_N,real *psi_Hy_z_F,real *psi_Hy_x_N,real *psi_Hy_x_F,real *khdx,real
    *khdz,real *bh_z_N,real *bh_z_F,real *ch_z_N,real *ch_z_F,real *bh_x_N,real *bh_x_F,real *ch_x_N,real *ch_x_F,int NCELLX,int NCELLY,int NCELLZ,int Periodic_XY,real dx,real dy,real dz,real dt,int cpml_N_Z,int cpml_F_Z,int cpml_N_X,int cpml_F_X,int cpml_z_lim,int cpml_y_lim,int cpml_x_lim,int NcpmlZ,int NcpmlX,int Hydrodynamics){

      int i,j,k,n,i2,k2;
      comp Curl_E;
      int idx = blockDim.x * blockIdx.x + threadIdx.x;


      i = idx / (NCELLZ*NCELLY);
      j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
      k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
      if(Periodic_XY){
  		////#pragma omp parallel for collapse(3) private(Curl_E,i,i2,j,k,k2) // schedule(static)
  		// for(k=0;k<NCELLZ;k++){
  		// for(i=0;i<NCELLX;i++){
  	  //       for(j=0;j<NCELLY;j++){
              if(i<NCELLX && j<NCELLY && k<NCELLZ){
  	             //   for(k=0;k<NCELLZ-1;k++){
  	                	if(i==NCELLX-1){
  											#ifdef DOUBLECOMPLEX
  	                		 Curl_E=(ez[ThreeDMapD(0,j,k,NCELLZ,NCELLY)]*cexp(-I*k_x*period_x)-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i]-(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k];
  											 #endif
  											 #ifdef DOUBLEPRECISION
  											 Curl_E=(ez[ThreeDMapD(0,j,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i]-(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k];
  											 #endif
  	                    	 hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
  	                	}
  	                	else{
  	                		 Curl_E=(ez[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i]-(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k];
  	                    	 hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
  	                }
  										//Z-PML
  										if(k<cpml_N_Z && j<cpml_y_lim && k<cpml_z_lim){
  											psi_Hy_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]=bh_z_N[k]*psi_Hy_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]+ch_z_N[k]*(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
  											hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hy_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)];
  										}
  										if(k>=cpml_F_Z && j<cpml_y_lim && k<cpml_z_lim){
  											k2 = k - cpml_F_Z;
  											psi_Hy_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]=bh_z_F[k2]*psi_Hy_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]+ch_z_F[k2]*(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
  											hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hy_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)];
  										}
                    }
  	    //             }
  	    //     }
  	    // }
      }

      else{
  			////#pragma omp parallel for collapse(3) private(Curl_E,i,i2,j,k,k2) // schedule(static)
  	    // for(i=0;i<NCELLX-1;i++){
  	    //     for(j=0;j<NCELLY;j++){
  	    //             for(k=0;k<NCELLZ-1;k++){
                      if(i<(NCELLX-1) && j<NCELLY && k<(NCELLZ-1)){
                        if(Hydrodynamics >= 1)hyPrev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  	                    Curl_E=(ez[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i]-(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k];
  	                    hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;

  											//Z-PML
  											if(k<cpml_N_Z && j<cpml_y_lim && k<cpml_z_lim){
  												psi_Hy_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]=bh_z_N[k]*psi_Hy_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]+ch_z_N[k]*(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
  												hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hy_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)];
  											}
  											if(k>=cpml_F_Z && j<cpml_y_lim && k<cpml_z_lim){
  												k2 = k - cpml_F_Z;
  												psi_Hy_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]=bh_z_F[k2]*psi_Hy_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]+ch_z_F[k2]*(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
  												hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hy_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)];
  											}
  											//X-PML
  											if(i<cpml_N_X && j<cpml_y_lim && k<cpml_z_lim){
  												psi_Hy_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=bh_x_N[i]*psi_Hy_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ch_x_N[i]*(ez[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dx;
  												hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hy_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  											}
  											if(i>=cpml_F_X && j<cpml_y_lim && k<cpml_z_lim){
  												i2 = i - cpml_F_X;
  												psi_Hy_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]=bh_x_F[i2]*psi_Hy_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]+ch_x_F[i2]*(ez[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dx;
  												hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hy_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)];
  											}
                      }
  	    //             }
  	    //     }
  	    // }
      }
    return;
  }

  //void UPDATE_hz(void){
  __global__   void UPDATE_hz(real *hz,real *hzPrev,real *ey,real *ex,real *Chzh,real *Chze,real *psi_Hz_x_N,real *psi_Hz_x_F,real *psi_Hz_y_N,real *psi_Hz_y_F,real *khdx,real
    *khdy,real *bh_x_N,real *bh_x_F,real *ch_x_N,real *ch_x_F,real *bh_y_N,real *bh_y_F,real *ch_y_N,real *ch_y_F,int NCELLX,int NCELLY,int NCELLZ,int Periodic_XY,real dx,real dy,real dz,real dt,int cpml_N_X,int cpml_F_X,int cpml_N_Y,int cpml_F_Y,int cpml_z_lim,int cpml_y_lim,int cpml_x_lim,int NcpmlY,int NcpmlX,int Hydrodynamics){

      int i,j,k,j2,i2;
      comp Curl_E;
      int idx = blockDim.x * blockIdx.x + threadIdx.x;

      i = idx / (NCELLZ*NCELLY);
      j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
      k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
      if(Periodic_XY){
    //  //#pragma omp parallel for collapse(3) private(Curl_E,i,j,k,j2,i2) // schedule(static)
  		// for(k=0;k<NCELLZ;k++){
  	  //   for(i=0;i<NCELLX;i++){
  	  //       for(j=0;j<NCELLY;j++){
          if(i<NCELLX && j<NCELLY && k<NCELLZ){
  	          //      for(k=0;k<NCELLZ;k++){
  	                	if(i==NCELLX-1 || j== NCELLY-1){
  	                		if(i==NCELLX-1 && j==NCELLY-1){
  	                			//printf("%d,%d,%d\n",i,j,k);
  												#ifdef DOUBLECOMPLEX
  	                			 Curl_E=(ex[ThreeDMapD(i,0,k,NCELLZ,NCELLY)]*cexp(-I*k_y*period_y)-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(0,j,k,NCELLZ,NCELLY)]*cexp(-I*k_x*period_x)-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
  												 #endif
  												 #ifdef DOUBLEPRECISION
  													Curl_E=(ex[ThreeDMapD(i,0,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(0,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
  													#endif
  	                   			 hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chzh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
  	                		}
  	                		else if(i==NCELLX-1){
  												#ifdef DOUBLECOMPLEX
  	                			 Curl_E=(ex[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(0,j,k,NCELLZ,NCELLY)]*cexp(-I*k_x*period_x)-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
  												 #endif
  												 #ifdef DOUBLEPRECISION
  												 Curl_E=(ex[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(0,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
  												 #endif
  	                    		 hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chzh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
  	                		}
  	                		else{
  												#ifdef DOUBLECOMPLEX
  	                			 Curl_E=(ex[ThreeDMapD(i,0,k,NCELLZ,NCELLY)]*cexp(-I*k_y*period_y)-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
  												 #endif
  												 #ifdef DOUBLEPRECISION
  												 Curl_E=(ex[ThreeDMapD(i,0,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
  												 #endif
  	                    		 hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chzh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
  	                		}
  	                	}
  	                	else{
  							 Curl_E=(ex[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
  	                    	 hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chzh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
  	               	}
                  }
  	    //             }
  	    //     }
  	    // }
      }

      else{
  		//	//#pragma omp parallel for collapse(3) private(Curl_E,i,j,k,j2,i2) // schedule(static)
  	    // for(i=0;i<NCELLX-1;i++){
  	    //     for(j=0;j<NCELLY-1;j++){
  	    //             for(k=0;k<NCELLZ;k++){
                      if(i<(NCELLX-1) && j<(NCELLY-1) && k<NCELLZ){
                        if(Hydrodynamics>=1)  hzPrev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] =  hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  	                    Curl_E=(ex[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
  	                    hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chzh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
  											//X-PML
  											if(i<cpml_N_X && j<cpml_y_lim && k<cpml_z_lim){
  												psi_Hz_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=bh_x_N[i]*psi_Hz_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ch_x_N[i]*(ey[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dx;
  												hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hz_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  											}
  											if(i>=cpml_F_X && j<cpml_y_lim && k<cpml_z_lim){
  												i2 = i - cpml_F_X;
  												psi_Hz_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]=bh_x_F[i2]*psi_Hz_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]+ch_x_F[i2]*(ey[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dx;
  												hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hz_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)];
  											}
  											//Y-PML
  											if(j<cpml_N_Y && i<cpml_x_lim && k<cpml_z_lim){
  												psi_Hz_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY)]=bh_y_N[j]*psi_Hz_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY)]+ch_y_N[j]*(ex[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dy;
  												hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hz_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY)];
  											}
  											if(j>=cpml_F_Y && i<cpml_x_lim && k<cpml_z_lim){
  												j2 = j - cpml_F_Y;
  												psi_Hz_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY)]=bh_y_F[j2]*psi_Hz_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY)]+ch_y_F[j2]*(ex[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dy;
  												hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hz_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY)];
  											}
                      }
  	    //             }
  	    //     }
  	    // }
      }
    return;
  }



    __global__ void UPDATE_ex(real *ex,real *ex_n,real *ex_n_1,real *hy,real *hz,real *Cexe,real *Cexh,real *kedy,real *kedz,int *mat_matrix,int *mat_matrixX,int first_medium_max,real *psi_Ex_z_N,
    real *psi_Ex_z_F,real *psi_Ex_y_N,real *psi_Ex_y_F,real *Px_cp,real *Px_cp_n,real *Px_cp_n_1,real *Px_d,real *Px_d_n,real *Px_d_n_1,real *Px_d_n_2,real *Py_d,real *Py_d_n,real *Py_d_n_1,real *Py_d_n_2,real *Pz_d,real *Pz_d_n,real *Pz_d_n_1,real *Pz_d_n_2,real *C_1_cp,real *C_2_cp,real *C_3_cp,real *C_4_cp,real *C_5_cp,real *d_1_d,
    real *d_2_d,real *d_3_d,real *d_4_d,real *d_5_d,real *d_NL,real C_E,real z0,int N_CP_poles,int N_drude_poles,real *ce_z_N,real *ce_z_F,real *be_z_N,real *be_z_F,real *ce_y_N,real *ce_y_F,real *be_y_N,real *be_y_F,
    real dx,real dy,real dz,real dt,int NCELLX,int NCELLY,int NCELLZ,int Hydrodynamics,int cpml_x_lim,int cpml_y_lim,int cpml_z_lim,int cpml_N_Y,int cpml_F_Y,int cpml_N_Z,int cpml_F_Z,int NcpmlY,int NcpmlZ,real C_E_1,real C_E_2,int Periodic_XY,
  real *NDx,real *NDy,real *NDz,real *NDx_prev,real *NDy_prev,real *NDz_prev,real e0,real N_EQ){
        int i,j,k,n,k2,j2;
        comp Curl_H, Div_Grad=0.0,J_T,dummy_var;
    		comp Vx1,Vx2,Vy1,Vy2,Vz1,Vz2,Nx1,Nx2,Ny1,Ny2,Nz1,Nz2,NxHold;
    			    comp C_P_1,C_P_2,C_P_3,C_P_4,C_P_NL;
    					real INV_DX = 1.0/dx;
    					real INV_DY = 1.0/dy;
    					real INV_DZ = 1.0/dz;

              int idx = blockDim.x * blockIdx.x + threadIdx.x;

              i = idx / (NCELLZ*NCELLY);
              j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
              k = idx - i*NCELLZ*NCELLY - j*NCELLZ;

        if(Periodic_XY){
    			////#pragma omp parallel for collapse(3) private(Curl_H,i,j,j2,k,k2,n,dummy_var,J_T,Div_Grad) // schedule(static)
    			// for(k=1;k<NCELLZ-1;k++){
    			// 	for(i=0;i<NCELLX;i++){
    		  //       for(j=0;j<NCELLY;j++){
                  if(i<NCELLX && j<NCELLY && k>0 && k<(NCELLZ-1)){
    	                //for(k=1;k<NCELLZ-1;k++){
    	                	if(j==0){
    											#ifdef DOUBLECOMPLEX
    												Curl_H=(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i,NCELLY-1,k,NCELLZ,NCELLY)]*cexp(I*k_y*period_y))/kedy[j]-(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k];
    											#endif
    											#ifdef DOUBLEPRECISION
    												Curl_H=(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i,NCELLY-1,k,NCELLZ,NCELLY)])/kedy[j]-(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k];
    											#endif

    	                	}
    	                	else{
    	                		Curl_H=(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/kedy[j]-(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k];
    	                	}

    										if(mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max && mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] < 6){


    											//  Div_Grad = Calc_DIV_GRADx(i,j,k);
                          Div_Grad = 0.0;

                          C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;

                          for(n=0;n<N_drude_poles;n++){
                              C_P_1+=(d_1_d[n]-1)*Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                              C_P_3+=(d_2_d[n])*Px_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                             C_P_NL += d_NL[n]*Div_Grad;
                          }
                          for(n=0;n<N_CP_poles;n++){
                              C_P_2+=(C_1_cp[n]-1)*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                              C_P_4+=(C_2_cp[n])*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                          }
                          ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                          ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                          ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4 - C_P_NL);


    												//printf("%e\n",Div_Grad);
    												//Z-CPML
    												if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
    													//Near-Z-PML
    													psi_Ex_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]=be_z_N[k]*psi_Ex_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]+ce_z_N[k]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    													ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)];
    												}
    												if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
    													k2 = k - cpml_F_Z ;
    													psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    													ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=(1/C_E)*dt*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
    												}

    											 	for(n=0;n<N_CP_poles;n++){
    											 							Px_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    											 	}

    												for(n=0;n<N_drude_poles;n++){
    													Px_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Px_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_NL[n]*Div_Grad;

    												}
                            for(n=0;n<N_drude_poles;n++){
                              Px_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Px_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                              Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Px_d[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];


                            }
                            for(n=0;n<N_CP_poles;n++){
                              Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                              Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Px_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];

                            }
    										}

    										else{
    												ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Cexe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_H;
    												//Z-CPML
    												if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
    													//Near-Z-PML
    													psi_Ex_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=be_z_N[k]*psi_Ex_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ce_z_N[k]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    													ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    												}
    												if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
    													k2 = k - cpml_F_Z ;
    													psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    													ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)];
    												}
    										}
                      }



    	  //       }
    	  //   }
        // }
    	}
        //No PBCs
        else{
    			////#pragma omp target teams distribute parallel for collapse(3) schedule(static,1) private(Curl_H,i,j,j2,k,k2,n,dummy_var,J_T,Div_Grad)
    		//	//#pragma omp parallel for collapse(3) private(Curl_H,i,j,j2,k,k2,n,dummy_var,J_T,Div_Grad) // schedule(static)
    	    // for(i=0;i<NCELLX-1;i++){
    	    //     for(j=1;j<NCELLY-1;j++){
    	    //             for(k=1;k<NCELLZ-1;k++){
                        if(i<(NCELLX-1) && j>0 && j<(NCELLY-1) && k>0 && k<(NCELLZ-1)){
    	                    Curl_H=(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/kedy[j]-(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k];

    											if(mat_matrixX[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max && mat_matrixX[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] < 6){


    													if(Hydrodynamics == 0)
    													{
    														//Div_Grad = Calc_DIV_GRADx(i,j,k);
                                Div_Grad = 0.0;
    														// CP_D_ex(i,j,k,Curl_H,Div_Grad);

                                 C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;

                                 for(n=0;n<N_drude_poles;n++){
                                     C_P_1+=(d_1_d[n]-1)*Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                     C_P_3+=(d_2_d[n])*Px_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                    C_P_NL += d_NL[n]*Div_Grad;
                                 }
                                 for(n=0;n<N_CP_poles;n++){
                                     C_P_2+=(C_1_cp[n]-1)*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                     C_P_4+=(C_2_cp[n])*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                 }
                                 ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                                 ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                                 ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4 - C_P_NL);




    														for(n=0;n<N_CP_poles;n++){
    																				Px_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    														}

    														for(n=0;n<N_drude_poles;n++){
    																				Px_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Px_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_NL[n]*Div_Grad;

    														}

                                for(n=0;n<N_drude_poles;n++){
                                  Px_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                  Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Px_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                }
                                for(n=0;n<N_CP_poles;n++){
                                  Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                  Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Px_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                }
    													}
    													else{



    														Vx1 = Px_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)];
    														Vx2 = Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)];
    														Nx1 = NDx_prev[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)];
    														Nx2 = NDx_prev[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)];

    														Vy1 = 0.5*(Py_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
    														Vy2 = 0.5*(Py_d_n[FourDMapD(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
    														Ny1 = 0.5*(NDy_prev[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)] + NDy_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
    														Ny2 = 0.5*(NDy_prev[ThreeDMapD(i+1,j-1,k,NCELLZ,NCELLY)] + NDy_prev[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)]);

    														Vz1 = 0.5*(Pz_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
    														Vz2 = 0.5*(Pz_d_n[FourDMapD(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
    														Nz1 = 0.5*(NDz_prev[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)] + NDz_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
    														Nz2 = 0.5*(NDz_prev[ThreeDMapD(i+1,j,k-1,NCELLZ,NCELLY)] + NDz_prev[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]);

    														NDx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = NDx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] - 2.0*dt*INV_DX*(0.5*(Nx1*Vx1-Nx2*Vx2) + (Ny1*Vy1-Ny2*Vy2) + (Nz1*Vz1-Nz2*Vz2) + (0.5*(Vx1-Vx2) + (Vy1-Vy2) + (Vz1-Vz2))*N_EQ);



    												    C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;


    												    for(n=0;n<N_CP_poles;n++){
    												        C_P_2+=(C_1_cp[n]-1)*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
    												        C_P_4+=(C_2_cp[n])*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
    												    }
    												    ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    												    ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    												    ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0 + dt*Px_d[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*e0*(N_EQ + NDx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]) + C_E_1*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] -C_E_2*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_2-C_P_4);

    														for(n=0;n<N_CP_poles;n++){
    																				Px_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    														}
                                for(n=0;n<N_drude_poles;n++){
                                  Px_d_n_2[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Px_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                  Px_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                  Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Px_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                }
                                for(n=0;n<N_CP_poles;n++){
                                  Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                  Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Px_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                }

                                NxHold = NDx_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                                NDx_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = NDx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                                NDx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = NxHold;

    													}

    												//	printf("%e\n",Div_Grad);
    													//Z-CPML
    													// if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
    													// 	//Near-Z-PML
    													// 	psi_Ex_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=be_z_N[k]*psi_Ex_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ce_z_N[k]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    													// 	ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    													// }
    													// if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
    													// 	k2 = k - cpml_F_Z ;
    													// 	psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    													// 	ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=(1/C_E)*dt*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
    													// }



    											}

    	                    else{
    	                        ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Cexe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_H;
    	                    }
    											//Z-CPML
    											if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
    												//Near-Z-PML
    												psi_Ex_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]=be_z_N[k]*psi_Ex_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]+ce_z_N[k]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    												ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)];
    											}
    											if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){ //Far Z PML
    												k2 = k - cpml_F_Z;
    												psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    												// if(mat_matrixX[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max){
    												// 	ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=(1/C_E)*dt*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
    												// }
    												// else{
    													ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)];
    												//}
    											}
    											//Y PML
    											if(j<cpml_N_Y+1 && i<cpml_x_lim && k<cpml_z_lim){ //Near Y PML
    												psi_Ex_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY+1)]=be_y_N[j]*psi_Ex_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY+1)]+ce_y_N[j]*(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/dy;
    												ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY+1)];
    											}
    											if(j>=cpml_F_Y && i<cpml_x_lim && k<cpml_z_lim){ //Far Y PML
    												j2 = j - cpml_F_Y;
    												psi_Ex_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY+1)]=be_y_F[j2]*psi_Ex_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY+1)]+ce_y_F[j2]*(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/dy;
    												ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY+1)];
    											}
    									// 		if(mat_matrixX[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max){
    									// 				for(n=0;n<N_CP_poles;n++){
    									// 										Px_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    									// 				}
    									// 				for(n=0;n<N_drude_poles;n++){
    									// 										Px_d[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=d_1_d[n]*Px_d_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+d_2_d[n]*Px_d_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+d_3_d[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    									// 				}
    	                // }
                    }
    	//         }
    	//     }
    	// }
    }

      return;
    }






    __global__ void UPDATE_ey(real *ey,real *ey_n,real *ey_n_1,real *hx,real *hz,real *Ceye,real *Ceyh,real *kedx,real *kedz,int *mat_matrix,int *mat_matrixY,int first_medium_max,real *psi_Ey_z_N,
    real *psi_Ey_z_F,real *psi_Ey_x_N,real *psi_Ey_x_F,real *Py_cp,real *Py_cp_n,real *Py_cp_n_1, real *Px_d,real *Px_d_n,real *Px_d_n_1,real *Px_d_n_2,real *Py_d,real *Py_d_n,real *Py_d_n_1,real *Py_d_n_2,real *Pz_d,real *Pz_d_n,real *Pz_d_n_1,real *Pz_d_n_2,real *C_1_cp,real *C_2_cp,real *C_3_cp,real *C_4_cp,real *C_5_cp,real *d_1_d,
    real *d_2_d,real *d_3_d,real *d_4_d,real *d_5_d,real *d_NL,real C_E,real z0,int N_CP_poles,int N_drude_poles,real *ce_z_N,real *ce_z_F,real *be_z_N,real *be_z_F,real *ce_x_N,real *ce_x_F,real *be_x_N,real *be_x_F,
    real dx,real dy,real dz,real dt,int NCELLX,int NCELLY,int NCELLZ,int Hydrodynamics,int cpml_x_lim,int cpml_y_lim,int cpml_z_lim,int cpml_N_X,int cpml_F_X,int cpml_N_Z,int cpml_F_Z,int NcpmlX,int NcpmlZ,real C_E_1,real C_E_2,int Periodic_XY,
    real *NDx,real *NDy,real *NDz,real *NDx_prev,real *NDy_prev,real *NDz_prev,real e0,real N_EQ){
        int i,j,k,i2,k2,n;
        comp Curl_H,Div_Grad=0.0,dummy_var,J_T;
    		comp C_P_1,C_P_2,C_P_3,C_P_4,C_P_NL;
    		real INV_DX = 1.0/dx;
    		real INV_DY = 1.0/dy;
    		real INV_DZ = 1.0/dz;
    		comp Vx1,Vx2,Vy1,Vy2,Vz1,Vz2,Nx1,Nx2,Ny1,Ny2,Nz1,Nz2,NyHold;
        int idx = blockDim.x * blockIdx.x + threadIdx.x;

        i = idx / (NCELLZ*NCELLY);
        j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
        k = idx - i*NCELLZ*NCELLY - j*NCELLZ;

        if(Periodic_XY){
        	////#pragma omp parallel for collapse(3) private(Curl_H,i,j,i2,k,k2,n,dummy_var,J_T,Div_Grad) // schedule(static)
    			// for(k=0;k<NCELLZ-1;k++){
    			//   for(i=0;i<NCELLX;i++){
    		  //       for(j=0;j<NCELLY;j++){
                  if(i<NCELLX && j<NCELLY && k>0 && k<(NCELLZ-1)){
    	               // for(k=1;k<NCELLZ-1;k++){
    	                	if(i==0){
    											#ifdef DOUBLECOMPLEX
    	                		Curl_H=(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k]-(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)]*cexp(I*period_x*k_x))/kedx[i];
    											#endif
    											#ifdef DOUBLEPRECISION
    											Curl_H=(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k]-(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)])/kedx[i];
    											#endif
    											  //printf("%d,%d,%d \t %f\t%f\t%f\n",i,j,k,creal(Curl_H),creal(hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]), creal(hz[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)]));

    	                	}

    	                	else{
    	                		Curl_H=(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k]-(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/kedx[i];
    	                	}
    										if(mat_matrixY[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max && mat_matrixY[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] <6){

    												//Div_Grad = Calc_DIV_GRADy(i,j,k);
                            Div_Grad = 0.0;
    											// printf("%e\n",d_NL[0]*Div_Grad);
    												// CP_D_ey(i,j,k,Curl_H,Div_Grad);
                            C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;
                            //printf("here");
                            for(n=0;n<N_drude_poles;n++){
                                C_P_1+=(d_1_d[n]-1.0)*Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                C_P_3+=(d_2_d[n])*Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                C_P_NL += d_NL[n]*Div_Grad;
                            }
                            for(n=0;n<N_CP_poles;n++){
                                C_P_2+=(C_1_cp[n]-1.0)*Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                C_P_4+=(C_2_cp[n])*Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                            }
                            ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                            ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                            ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4-C_P_NL);


    											//	printf("%e\n",ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
    												//Z-CPML
    												if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
    													//Here we are in the near Z-PML
    													psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]=be_z_N[k]*psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]+ce_z_N[k]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    													ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)];
    												}
    												if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
    													//Here we are in the far Z-PML
    														k2 = k - cpml_F_Z ;
    														psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    															ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=(1/C_E)*dt*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
    												}

    														for(n=0;n<N_CP_poles;n++){
    																				Py_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    														}

    														//if(Hydrodynamics == 0){
    														for(n=0;n<N_drude_poles;n++){
    															Py_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_NL[n]*Div_Grad;

    														}

                                for(n=0;n<N_drude_poles;n++){
                                  Py_d_n_2[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                  Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                  Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Py_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];


                                }
                                for(n=0;n<N_CP_poles;n++){

                                  Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                  Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Py_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];

                                }
    													//}

    										}

    										else{
    												ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Ceye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_H;
    												//Z-CPML
    												if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
    													//Here we are in the near Z-PML
    													psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]=be_z_N[k]*psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]+ce_z_N[k]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    													ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)];
    												}
    												if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
    													//Here we are in the far Z-PML
    														k2 = k - cpml_F_Z ;
    														psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    														if(mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max){
    															ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=(1/C_E)*dt*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
    														}
    														else{
    															ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)];
    														}
    												}
    										}

                      }

    	      //       }
    	      //   }
       	    // }
        }

        else{
    		//	//#pragma omp parallel for collapse(3) private(Curl_H,i,j,i2,k,k2,n,dummy_var,J_T,Div_Grad) // schedule(static)
    	    // for(i=1;i<NCELLX-1;i++){
    	    //     for(j=0;j<NCELLY-1;j++){
    	    //             for(k=1;k<NCELLZ-1;k++){
                        if(i>0 && i<(NCELLX-1) && j<(NCELLY-1) && k>0 && k<(NCELLZ-1)){
    	                    Curl_H=(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k]-(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/kedx[i];
    											if(mat_matrixY[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max && mat_matrixY[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] <6){

    												if(Hydrodynamics == 0){
    													//Div_Grad = Calc_DIV_GRADy(i,j,k);
                              Div_Grad= 0.0;
    												// printf("%e\n",d_NL[0]*Div_Grad);
    													// CP_D_ey(i,j,k,Curl_H,Div_Grad);
                              C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;
                          		//printf("here");
                              for(n=0;n<N_drude_poles;n++){
                                  C_P_1+=(d_1_d[n]-1.0)*Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                  C_P_3+=(d_2_d[n])*Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                          				C_P_NL += d_NL[n]*Div_Grad;
                              }
                              for(n=0;n<N_CP_poles;n++){
                                  C_P_2+=(C_1_cp[n]-1.0)*Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                  C_P_4+=(C_2_cp[n])*Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                              }
                              ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                              ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                              ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4-C_P_NL);

    												//	printf("%e\n",ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
    													//Z-CPML
    													// if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
    													// 	//Here we are in the near Z-PML
    													// 	psi_Ey_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=be_z_N[k]*psi_Ey_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ce_z_N[k]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    													// 	ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    													// }
    													// if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
    													// 	//Here we are in the far Z-PML
    													// 		k2 = k - cpml_F_Z ;
    													// 		psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    													// 			ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=(1/C_E)*dt*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
    													// }

    															for(n=0;n<N_CP_poles;n++){
    																					Py_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    															}

    															for(n=0;n<N_drude_poles;n++){
    																Py_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_NL[n]*Div_Grad;

    															}
                                  for(n=0;n<N_drude_poles;n++){

                                    Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                    Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Py_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];


                                  }
                                  for(n=0;n<N_CP_poles;n++){
                                    // Py_cp_n_2[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];

                                    Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                    Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Py_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];

                                  }
    													}
    													else{

    														Vy1 = Py_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)];
    														Vy2 = Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)];
    														Ny1 = NDy_prev[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)];
    														Ny2 = NDy_prev[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)];

    														Vx1 = 0.5*(Px_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
    														Vx2 = 0.5*(Px_d_n[FourDMapD(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
    														Nx1 = 0.5*(NDx_prev[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)] + NDx_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
    														Nx2 = 0.5*(NDx_prev[ThreeDMapD(i-1,j+1,k,NCELLZ,NCELLY)] + NDx_prev[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)]);

    														Vz1 = 0.5*(Pz_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
    														Vz2 = 0.5*(Pz_d_n[FourDMapD(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
    														Nz1 = 0.5*(NDz_prev[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)] + NDz_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
    														Nz2 = 0.5*(NDz_prev[ThreeDMapD(i,j+1,k-1,NCELLZ,NCELLY)] + NDz_prev[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]);

    														 NDy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = NDy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] - 2.0*dt*INV_DX*((Nx1*Vx1-Nx2*Vx2) + 0.5*(Ny1*Vy1-Ny2*Vy2) + (Nz1*Vz1-Nz2*Vz2) + ((Vx1-Vx2) + 0.5*(Vy1-Vy2) + (Vz1-Vz2))*N_EQ);


    											    C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;

    											    for(n=0;n<N_CP_poles;n++){
    											        C_P_2+=(C_1_cp[n]-1.0)*Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
    											        C_P_4+=(C_2_cp[n])*Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
    											    }
    											    ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    											    ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    											    ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0 + dt*Py_d[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*e0*(N_EQ + NDy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]) + C_E_1*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_2-C_P_4);
    													for(n=0;n<N_CP_poles;n++){
    																			Py_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    													}

                              for(n=0;n<N_CP_poles;n++){
                                Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Py_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];

                              }
                              for(n=0;n<N_drude_poles;n++){
                                Py_d_n_2[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Py_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];


                              }


                                                              NyHold = NDy_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                                                              NDy_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = NDy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                                                              NDy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = NyHold;

    											}
    										}


    										else{
    												ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Ceye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_H;

    										}

    										if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
    											//Here we are in the near Z-PML
    											psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]=be_z_N[k]*psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]+ce_z_N[k]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    											ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)];
    										}
    										if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
    												//Here we are in the far Z-PML
    												k2 = k - cpml_F_Z;
    												psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
    												// if(mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max){
    												// 	ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=(1/C_E)*dt*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
    												// }
    												// else{
    													ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)];
    												//}
    										}
    										//X-CPML
    										if(i<cpml_N_X+1 && j<cpml_y_lim && k<cpml_z_lim){
    											//Here we are in the near-X-PML
    											psi_Ey_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=be_x_N[i]*psi_Ey_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ce_x_N[i]*(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/dx;
    											ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    										}
    										if(i>=cpml_F_X && j<cpml_y_lim && k<cpml_z_lim){
    											//Here we are in the far-X-PML
    											i2 = i - cpml_F_X;
    											psi_Ey_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]=be_x_F[i2]*psi_Ey_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]+ce_x_F[i2]*(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/dx;
    											ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)];
    										}


                      }

    								}
    	  //       }
        //
    	  //   }
        // }

      return;
    }


    //void UPDATE_ez(void){
      __global__ void UPDATE_ez(real *ez,real *ez_n,real *ez_n_1,real *hx,real *hy,real *Ceze,real *Cezh,real *kedx,real *kedy,int *mat_matrix,int *mat_matrixZ,int first_medium_max,real *psi_Ez_y_N,
      real *psi_Ez_y_F,real *psi_Ez_x_N,real *psi_Ez_x_F,real *Pz_cp,real *Pz_cp_n,real *Pz_cp_n_1,real *Px_d,real *Px_d_n,real *Px_d_n_1,real *Px_d_n_2,real *Py_d,real *Py_d_n,real *Py_d_n_1,real *Py_d_n_2,real *Pz_d,real *Pz_d_n,real *Pz_d_n_1,real *Pz_d_n_2,real *C_1_cp,real *C_2_cp,real *C_3_cp,real *C_4_cp,real *C_5_cp,real *d_1_d,
      real *d_2_d,real *d_3_d,real *d_4_d,real *d_5_d,real *d_NL,real C_E,real z0,int N_CP_poles,int N_drude_poles,real *ce_y_N,real *ce_y_F,real *be_y_N,real *be_y_F,real *ce_x_N,real *ce_x_F,real *be_x_N,real *be_x_F,
      real dx,real dy,real dz,real dt,int NCELLX,int NCELLY,int NCELLZ,int Hydrodynamics,int cpml_x_lim,int cpml_y_lim,int cpml_z_lim,int cpml_N_X,int cpml_F_X,int cpml_N_Y,int cpml_F_Y,int NcpmlX,int NcpmlY,real C_E_1,real C_E_2,int Periodic_XY,
    real *NDx,real *NDy,real *NDz,real *NDx_prev,real *NDy_prev,real *NDz_prev,real e0,real N_EQ){

        int i,j,k;//,i2,j2,n;
        // comp Curl_H,Div_Grad=0.0,dummy_var,J_T;
    		// comp C_P_1,C_P_2,C_P_3,C_P_4,C_P_NL;
    		// comp Vx1,Vx2,Vy1,Vy2,Vz1,Vz2,Nx1,Nx2,Ny1,Ny2,Nz1,Nz2;

    		real INV_DX = 1.0/dx;
    		real INV_DY = 1.0/dy;
    		real INV_DZ = 1.0/dz;
        int idx = blockDim.x * blockIdx.x + threadIdx.x;
        comp Curl_H,Div_Grad=0.0,dummy_var,J_T;
        comp C_P_1,C_P_2,C_P_3,C_P_4,C_P_NL;
        comp Vx1,Vx2,Vy1,Vy2,Vz1,Vz2,Nx1,Nx2,Ny1,Ny2,Nz1,Nz2,NzHold;
        int i2,j2,k2,n;
        i = idx / (NCELLZ*NCELLY);
        j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
        k = idx - i*NCELLZ*NCELLY - j*NCELLZ;

        if(Periodic_XY){
    	//	//#pragma omp parallel for collapse(3) private(Curl_H,i,j,k,i2,j2,n,dummy_var,J_T,Div_Grad) // schedule(static)
    		// for(k=0;k<NCELLZ-1;k++){
    		// for(i=0;i<NCELLX;i++){
    		//         for(j=0;j<NCELLY;j++){
                  if(i<NCELLX,j<NCELLY,k<NCELLZ){
    		           //     for(k=0;k<NCELLZ-1;k++){
    											//printf("Thread %d, ready to work\n",omp_get_thread_num());

    		                	if(i==0 || j==0){
    		                		if(i==0 && j==0){
    													#ifdef DOUBLECOMPLEX
    				        						Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)]*cexp(I*k_x*period_x))/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,NCELLY-1,k,NCELLZ,NCELLY)]*cexp(I*k_y*period_y))/kedy[j];
    														#endif
    														#ifdef DOUBLEPRECISION
    														Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)])/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,NCELLY-1,k,NCELLZ,NCELLY)])/kedy[j];
    														#endif


    			               	    }
    			               	    else if(i==0){
    													#ifdef DOUBLECOMPLEX
    				                 	Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)]*cexp(I*k_x*period_x))/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/kedy[j];
    													#endif
    													#ifdef DOUBLEPRECISION
    													Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)])/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/kedy[j];
    													#endif


    				                }
    				                else{
    													#ifdef DOUBLECOMPLEX
    				                	Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,NCELLY-1,k,NCELLZ,NCELLY)]*cexp(I*k_y*period_y))/kedy[j];
    													#endif
    													#ifdef DOUBLEPRECISION
    													Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,NCELLY-1,k,NCELLZ,NCELLY)])/kedy[j];
    													#endif


    				                }
    		        			}

    		        			else{
    		        				Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/kedy[j];

    		        			}
    									if(mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max && mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]<6){

    											// Div_Grad  = Calc_DIV_GRADz(i,j,k);
                          Div_Grad = 0.0;

    											// CP_D_ez(i,j,k,Curl_H,Div_Grad);
                          C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;

                          for(n=0;n<N_drude_poles;n++){
                              C_P_1+=(d_1_d[n]-1)*Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                              C_P_3+=(d_2_d[n])*Pz_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                              C_P_NL += d_NL[n]*Div_Grad;
                          }
                          for(n=0;n<N_CP_poles;n++){
                              C_P_2+=(C_1_cp[n]-1)*Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                              C_P_4+=(C_2_cp[n])*Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                          }
                          ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                          ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                          ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4-C_P_NL);


    												for(n=0;n<N_CP_poles;n++){
    																		Pz_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    												}

    												for(n=0;n<N_drude_poles;n++){
    													Pz_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Pz_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_NL[n]*Div_Grad;
    												}

                            for(n=0;n<N_drude_poles;n++){

                              Pz_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                              Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Pz_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];

                            }
                            for(n=0;n<N_CP_poles;n++){

                              Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                              Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Pz_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                            }
    									}

    									else{
    											ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Ceze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_H;
    									}

                    }
    		      //     }
    		      //   }
    					// }
        }

        else{
    	//		//#pragma omp parallel for collapse(3) private(Curl_H,i,j,k,i2,j2,n,dummy_var,J_T,Div_Grad) // schedule(static)
    	    // for(i=1;i<NCELLX-1;i++){
    	    //     for(j=1;j<NCELLY-1;j++){
    	    //             for(k=0;k<NCELLZ-1;k++){
                        if(i>0 && i<(NCELLX-1) && j>0 && j<(NCELLY-1) && k<(NCELLZ-1)){
    	                    Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/kedy[j];
    											if(mat_matrixZ[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max && mat_matrixZ[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]<6){

    												if(Hydrodynamics == 0){
    			 									 //Div_Grad  = Calc_DIV_GRADz(i,j,k);
                             Div_Grad = 0.0;
    			 									// CP_D_ez(i,j,k,Curl_H,Div_Grad);
                            C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;

                            // Vz1 = d_1_d[0];
                            // Vy2 = d_2_d[0];
                            // Vy1 = d_3_d[0];
                            // Vx1 = d_4_d[0];
                            // Vx2 = d_5_d[0];


                            for(n=0;n<N_drude_poles;n++){
                                C_P_1+=(d_1_d[n]-1)*Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                C_P_3+=(d_2_d[n])*Pz_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                C_P_NL += d_NL[n]*Div_Grad;
                            }
                            for(n=0;n<N_CP_poles;n++){
                                C_P_2+=(C_1_cp[n]-1)*Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                C_P_4+=(C_2_cp[n])*Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                            }
                            ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                            ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                            ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4-C_P_NL);


    			 										 for(n=0;n<N_CP_poles;n++){
    			 																 Pz_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    			 										 }
    			 										 for(n=0;n<N_drude_poles;n++){
    			 											 Pz_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Pz_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_NL[n]*Div_Grad;
    			 										 }

                               for(n=0;n<N_drude_poles;n++){

                                 Pz_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                                 Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Pz_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];

                               }
                               for(n=0;n<N_CP_poles;n++){

                                 Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                                 Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Pz_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                               }

    											 }
    											 else{

    												 C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;

    												 Vz1 = Pz_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)];
    												 Vz2 = Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)];
    												 Nz1 = NDz_prev[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)];
    												 Nz2 = NDz_prev[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)];

    												 Vx1 = 0.5*(Px_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
    												 Vx2 = 0.5*(Px_d_n[FourDMapD(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
    												 Nx1 = 0.5*(NDx_prev[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)] + NDx_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
    												 Nx2 = 0.5*(NDx_prev[ThreeDMapD(i-1,j,k+1,NCELLZ,NCELLY)] + NDx_prev[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)]);

    												 Vy1 = 0.5*(Py_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
    												 Vy2 = 0.5*(Py_d_n[FourDMapD(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
    												 Ny1 = 0.5*(NDy_prev[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)] + NDy_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
    												 Ny2 = 0.5*(NDy_prev[ThreeDMapD(i,j-1,k+1,NCELLZ,NCELLY)] + NDy_prev[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)]);

    												  NDz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = NDz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] - 2.0*dt*INV_DX*((Nx1*Vx1-Nx2*Vx2) + (Ny1*Vy1-Ny2*Vy2) + 0.5*(Vz1-Vz2) + ((Vx1-Vx2) + (Vy1-Vy2) + 0.5*(Vz1-Vz2))*N_EQ);


    												 for(n=0;n<N_CP_poles;n++){
    														 C_P_2+=(C_1_cp[n]-1)*Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
    														 C_P_4+=(C_2_cp[n])*Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
    												 }
    												 ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    												 ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    												 ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0 + dt*Pz_d[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*e0*(NDz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + N_EQ)+C_E_1*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_2-C_P_4);

    												 for(n=0;n<N_CP_poles;n++){
    																		 Pz_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    												 }

                             for(n=0;n<N_drude_poles;n++){
                               Pz_d_n_2[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Pz_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                               Pz_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
                               Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Pz_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];

                             }
                             for(n=0;n<N_CP_poles;n++){

                               Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                               Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Pz_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
                             }

                                                             NzHold = NDz_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                                                             NDz_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = NDz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                                                             NDz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = NzHold;

    			 							 }
                       }


    	                    else{
    	                        ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Ceze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_H;
    	                    }
    											//Y CPML
    											if(j<cpml_N_Y && i<cpml_x_lim && k<cpml_z_lim){ //Near Y PML
    												psi_Ez_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY+1)]=be_y_N[j]*psi_Ez_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY+1)]+ce_y_N[j]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/dy;
    												ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ez_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY+1)];
    											}
    											if(j>=cpml_F_Y && i<cpml_x_lim &&  k<cpml_z_lim){
    												j2 = j - cpml_F_Y;
    												psi_Ez_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY+1)]=be_y_F[j2]*psi_Ez_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY+1)]+ce_y_F[j2]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/dy;
    												ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ez_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY+1)];
    											}
    											//X PML
    											if(i<cpml_N_X+1 && j<cpml_y_lim && k<cpml_z_lim){//Near X-PML
    												psi_Ez_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=be_x_N[i]*psi_Ez_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ce_x_N[i]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/dx;
    												ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ez_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    											}
    											if(i>=cpml_F_X && j<cpml_y_lim && k<cpml_z_lim){//far X-PML
    												i2 = i - cpml_F_X;
    												psi_Ez_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]=be_x_F[i2]*psi_Ez_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]+ce_x_F[i2]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/dx;
    												ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ez_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)];
    											}
    											// if(mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max){
    											// 		 for(n=0;n<N_CP_poles;n++){
    											// 				Pz_cp[ThreeDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Pz_cp_n[ThreeDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Pz_cp_n_1[ThreeDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    											// 				}
    											// 		for(n=0;n<N_drude_poles;n++){
    											// 				Pz_d[ThreeDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Pz_d_n[ThreeDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Pz_d_n_1[ThreeDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
    											// 				}
    											// }

    	                }
          //           }
    	    //     }
          //
    	    // }
        }

      return;
    }



    __global__ void UpdateHydroPx(real *ex,real *ex_n,real *Px_d,real *Px_d_n,real *Px_d_n_1,real *Px_d_n_2,real *Py_d,real *Py_d_n,real *Py_d_n_1,real *Py_d_n_2,real *Pz_d,real *Pz_d_n,real *Pz_d_n_1,real *Pz_d_n_2,real *NDx,real *NDx_prev,real *NDy,real *NDy_prev,real *NDz,real *NDz_prev,
			real *hx,real *hy,real *hz,real *hxPrev,real *hyPrev,real *hzPrev,int WithConvection,int WithMagField,real N_EQ,int *mat_matrixX,int *mat_matrixY,int *mat_matrixZ,real dt,real dx,real dy,real dz,int NCELLX,int NCELLY,int NCELLZ,int first_medium,real *d_1_d,real *d_2_d,real *d_3_d,real *d_4_d,real *d_5_d,real *d_NL,int N_drude_poles,real mu0,real e0, real me,real z0){
      int i,j,k;

      real Vx1,Vx2,Vx3,Vy1,Vy2,Vy3,Vz1,Vz2,Vz3,Hx1,Hz1,Hy1,Hx2,Hz2,Hy2,Ex1,Ey1,Ez1,VdotGrad,VdotGrad2,VdotGrad3,DivV,VcrossH,VcrossH2,Pressure,ND1,ND2,ND3,Grad_Div,Grad_Div2;
      real INV_DX,INV_DY,INV_DZ;
      INV_DX = 1.0/dx;
      INV_DY = 1.0/dy;
      INV_DZ = 1.0/dz;
      Grad_Div = 0.0;
      Grad_Div2 =0.0;

      ////#pragma omp parallel for collapse(3)  // schedule(static)
      // for(i=0;i<NCELLX-1;i++){
      //     for(j=1;j<NCELLY-1;j++){
      //             for(k=1;k<NCELLZ-1;k++){

      int idx = blockDim.x * blockIdx.x + threadIdx.x;

      i = idx / (NCELLZ*NCELLY);
      j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
      k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
                    if(i<NCELLX-1 && j>0 && j<NCELLY-1 && k>0 && k<NCELLZ-1){
                    if(mat_matrixX[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]> first_medium && mat_matrixX[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] < 6){
                      ND1 = N_EQ + NDx_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
                      //
                      // Grad_Div = Calc_DIV_GRADx(i,j,k);
                      // Grad_Div2 = Calc_DIV_GRADx2(i,j,k);


                      Grad_Div2 = INV_DX*INV_DX*(Px_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])
                                 + INV_DX*INV_DY*(Py_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMapD(i+1,j-1,k,NCELLZ,NCELLY)]+Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])
                                 + INV_DX*INV_DZ*(Pz_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMapD(i+1,j,k-1,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]);

                                 if(i==0 && j==0){
                                     Grad_Div = INV_DX*INV_DX*(Px_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                                + INV_DX*INV_DY*(Py_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i+1,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMapD(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                                + INV_DX*INV_DZ*(Pz_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                                 }

                                 else if(i==NCELLX-1 && j==0){
                                     Grad_Div = INV_DX*INV_DX*(Px_d_n[FourDMapD(0,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                                + INV_DX*INV_DY*(Py_d_n[FourDMapD(0,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(0,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMapD(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                                + INV_DX*INV_DZ*(Pz_d_n[FourDMapD(0,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(0,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                                 }
                                 else if(i==NCELLX-1){
                                     Grad_Div = INV_DX*INV_DX*(Px_d_n[FourDMapD(0,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                                + INV_DX*INV_DY*(Py_d_n[FourDMapD(0,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(0,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                                + INV_DX*INV_DZ*(Pz_d_n[FourDMapD(0,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(0,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                                 }
                                 else if(i==0){
                                     Grad_Div = INV_DX*INV_DX*(Px_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                                + INV_DX*INV_DY*(Py_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                                + INV_DX*INV_DZ*(Pz_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                                 }
                                 else if(j==0){
                                     Grad_Div = INV_DX*INV_DX*(Px_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                                + INV_DX*INV_DY*(Py_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i+1,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMapD(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                                + INV_DX*INV_DZ*(Pz_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                                 }
                                 else{
                                     Grad_Div = INV_DX*INV_DX*(Px_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                                + INV_DX*INV_DY*(Py_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                                + INV_DX*INV_DZ*(Pz_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                                 }
                      // Grad_Div = 0.0;
                      // Grad_Div2 = 0.0;

                      // Vx1 = d_1_d[0];
                      // Vx2 = d_2_d[0];
                      // Vy1 = d_3_d[0];
                      // Vy2 = d_4_d[0];
                      // Vz1 = d_5_d[0];
                      //




                      Vx1 = 0.5 * (Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                      Vx2 = 0.5 * (Px_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                      Vx2 = 0.5 * (Px_d_n_2[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_2[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);


                      Vy1 = 0.25 * (Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                      Vy2 = 0.25 * (Py_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMapD(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                      Vy3 = 0.25 * (Py_d_n_2[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_2[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_2[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_2[FourDMapD(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);

                      //Vy1 = 0.5 * (Vy1 + Vy2);
                      Vz1 = 0.25 * (Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                      Vz2 = 0.25 * (Pz_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMapD(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                      Vz3 = 0.25 * (Pz_d_n_2[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_2[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_2[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_2[FourDMapD(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);

                      //Vz1 = 0.5 * (Vz1 + Vz2);
                      Hy2 = 0.5 * (hyPrev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + hyPrev[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]);
                      Hz2 = 0.5 * (hzPrev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + hzPrev[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)]);
                      Hy1 = 0.5 * (hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]);
                      Hz1 = 0.5 * (hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + hz[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)]);

                      if(WithConvection==1) {
                        VdotGrad = 0.5*(Vx1*(Px_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy1*(Px_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz1*(Px_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
                        VdotGrad2 = 0.5*(Vx2*(Px_d_n_1[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n_1[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy2*(Px_d_n_1[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n_1[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz2*(Px_d_n_1[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n_1[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
                        VdotGrad3= 0.5*(Vx3*(Px_d_n_2[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n_2[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy3*(Px_d_n_2[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n_2[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz3*(Px_d_n_2[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n_2[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);

                        // VdotGrad = 1.0*(Vx1*(Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy1*(Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz1*(Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
                        // VdotGrad2 = 1.0*(Vx2*(Px_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n_1[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy2*(Px_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n_1[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz2*(Px_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n_1[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);

                        // VdotGrad = 0.5*(3.0*VdotGrad - 4.0*VdotGrad2 + VdotGrad3)/dt;
                        VdotGrad = (VdotGrad-VdotGrad2)/dt;
                      }
                      else VdotGrad = 0.0;

                      if(WithMagField==1){
                        VcrossH = Vy1*Hz1 - Vz1*Hy1;
                        VcrossH2 = Vy2*Hz2 - Vz2*Hy2;
                        VcrossH = (VcrossH - VcrossH2)/dt;
                      }
                      else VcrossH = 0.0;

                         Px_d[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] = d_1_d[0]*Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + d_2_d[0]*Px_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] +d_3_d[0]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]
                         + d_4_d[0]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_5_d[0]*(VdotGrad + (mu0*e0/me)*VcrossH/z0) + d_NL[0]*(Grad_Div + Grad_Div2/N_EQ)/powf(ND1,1.0/3.0);


                    }

             }
              //   }
              // }
    }

    __global__ void UpdateHydroPy(real *ey,real *ey_n,real *Px_d,real *Px_d_n,real *Px_d_n_1,real *Px_d_n_2,real *Py_d,real *Py_d_n,real *Py_d_n_1,real *Py_d_n_2,real *Pz_d,real *Pz_d_n,real *Pz_d_n_1,real *Pz_d_n_2,real *NDx,real *NDx_prev,real *NDy,real *NDy_prev,real *NDz,real *NDz_prev,
			real *hx,real *hy,real *hz,real *hxPrev,real *hyPrev,real *hzPrev,int WithConvection,int WithMagField,real N_EQ,int *mat_matrixX,int *mat_matrixY,int *mat_matrixZ,real dt,real dx,real dy,real dz,int NCELLX,int NCELLY,int NCELLZ,int first_medium,real *d_1_d,real *d_2_d,real *d_3_d,real *d_4_d,real *d_5_d,real *d_NL,int N_drude_poles,real mu0,real e0, real me,real z0){
      int i,j,k;
      real Vx1,Vx2,Vx3,Vy1,Vy2,Vy3,Vz1,Vz2,Vz3,Hx1,Hz1,Hy1,Hx2,Hz2,Hy2,Ex1,Ey1,Ez1,VdotGrad,VdotGrad2,VdotGrad3,DivV,VcrossH,VcrossH2,Pressure,ND1,ND2,ND3,Grad_Div,Grad_Div2;
      real INV_DX,INV_DY,INV_DZ;
      INV_DX = 1.0/dx;
      INV_DY = 1.0/dy;
      INV_DZ = 1.0/dz;
      Grad_Div = 0.0;
      Grad_Div2 =0.0;
      ////#pragma omp parallel for collapse(3) // schedule(static)
      // for(i=1;i<NCELLX-1;i++){
      //     for(j=0;j<NCELLY-1;j++){
      //             for(k=1;k<NCELLZ-1;k++){

  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  i = idx / (NCELLZ*NCELLY);
  j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
  k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
                if(j<NCELLY-1 && i>0 && i<NCELLX-1 && k>0 && k<NCELLZ-1){
                    if(mat_matrixY[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]> first_medium && mat_matrixY[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] < 6){

                      // Grad_Div = Calc_DIV_GRADy(i,j,k);
                      // Grad_Div2 = Calc_DIV_GRADy2(i,j,k);


                      Grad_Div2 = INV_DY*INV_DY*(Py_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])
                                 + INV_DX*INV_DY*(Px_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMapD(i-1,j+1,k,NCELLZ,NCELLY)]+Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])
                                 + INV_DY*INV_DZ*(Pz_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMapD(i,j+1,k-1,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]);

                      if(i==0 && j==0){
                          Grad_Div = INV_DY*INV_DY*(Py_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                     + INV_DX*INV_DY*(Px_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(NCELLX-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMapD(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                     + INV_DY*INV_DZ*(Pz_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                        }

                        else if(i==0 && j==NCELLY-1){
                          Grad_Div = INV_DY*INV_DY*(Py_d_n[FourDMapD(i,0,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                     + INV_DX*INV_DY*(Px_d_n[FourDMapD(i,0,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(NCELLX-1,0,k,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMapD(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                     + INV_DY*INV_DZ*(Pz_d_n[FourDMapD(i,0,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,0,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                          }
                          else if(j==NCELLY-1){
                            Grad_Div = INV_DY*INV_DY*(Py_d_n[FourDMapD(i,0,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                       + INV_DX*INV_DY*(Px_d_n[FourDMapD(i,0,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i-1,0,k,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                       + INV_DY*INV_DZ*(Pz_d_n[FourDMapD(i,0,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,0,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                          }
                        else if(j==0){
                          Grad_Div = INV_DY*INV_DY*(Py_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                     + INV_DX*INV_DY*(Px_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                     + INV_DY*INV_DZ*(Pz_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                        }
                        else if(i==0){
                          Grad_Div = INV_DY*INV_DY*(Py_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                     + INV_DX*INV_DY*(Px_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(NCELLX-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMapD(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                     + INV_DY*INV_DZ*(Pz_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                        }
                        else{
                          Grad_Div = INV_DY*INV_DY*(Py_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                     + INV_DX*INV_DY*(Px_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                     + INV_DY*INV_DZ*(Pz_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMapD(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                        }

                      //
                      // Grad_Div = 0.0;
                      // Grad_Div2 = 0.0;

                      ND1 = N_EQ + NDy_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];

                      Vy1 = 0.5 * (Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                      Vy2 = 0.5 * (Py_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                      Vy3 = 0.5 * (Py_d_n_2[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_2[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);


                      Vx1 = 0.25 * (Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                      Vx2 = 0.25 * (Px_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMapD(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                      Vx3 = 0.25 * (Px_d_n_2[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_2[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_2[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_2[FourDMapD(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]);

                     //	Vx1 = 0.5 * (Vx1 + Vx2);
                      Vz1 = 0.25 * (Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                      Vz2 = 0.25 * (Pz_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMapD(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
                      Vz3 = 0.25 * (Pz_d_n_2[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_2[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_2[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_2[FourDMapD(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);

                      //Vz1 = 0.5 * (Vz1 + Vz2);
                      Hx1 = 0.5 * (hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]);
                      Hz1 = 0.5 * (hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + hz[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)]);
                      Hx2 = 0.5 * (hxPrev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + hxPrev[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]);
                      Hz2 = 0.5 * (hzPrev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + hzPrev[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)]);


                       if(WithMagField==1){
                         VcrossH = Vz1*Hx1 - Vx1*Hz1;
                         VcrossH2 = Vz2*Hx2 - Vx2*Hz2;
                         VcrossH = (VcrossH - VcrossH2)/dt;
                       }
                       else VcrossH = 0.0;
                       if(WithConvection==1) {
                         VdotGrad = 0.5*(Vx1*(Py_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy1*(Py_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz1*(Py_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
                         VdotGrad2 = 0.5*(Vx2*(Py_d_n_1[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n_1[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy2*(Py_d_n_1[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n_1[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz2*(Py_d_n_1[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n_1[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
                         VdotGrad3 = 0.5*(Vx3*(Py_d_n_2[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n_2[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy3*(Py_d_n_2[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n_2[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz3*(Py_d_n_2[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n_2[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);

                         // VdotGrad = 1.0*(Vx1*(Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy1*(Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz1*(Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
                         // VdotGrad2 = 1.0*(Vx2*(Py_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n_1[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy2*(Py_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n_1[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz2*(Py_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n_1[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);

                         // VdotGrad = (VdotGrad - VdotGrad2)/dt;
                         // VdotGrad = 0.5*(3.0*VdotGrad - 4.0*VdotGrad2 + VdotGrad3)/dt;
                         VdotGrad = (VdotGrad-VdotGrad2)/dt;


                       }
                       else VdotGrad = 0.0;

                     Py_d[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] = d_1_d[0]*Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + d_2_d[0]*Py_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]
                     + d_3_d[0]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_4_d[0]*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ d_5_d[0]*(VdotGrad + (mu0*e0/me)*VcrossH/z0) + d_NL[0]*(Grad_Div + Grad_Div2/N_EQ)/powf(ND1,1.0/3.0);

                    }


                //   }
                // }
              }
    }

    __global__ void UpdateHydroPz(real *ez,real *ez_n,real *Px_d,real *Px_d_n,real *Px_d_n_1,real *Px_d_n_2,real *Py_d,real *Py_d_n,real *Py_d_n_1,real *Py_d_n_2,real *Pz_d,real *Pz_d_n,real *Pz_d_n_1,real *Pz_d_n_2,real *NDx,real *NDx_prev,real *NDy,real *NDy_prev,real *NDz,real *NDz_prev,
      real *hx,real *hy,real *hz,real *hxPrev,real *hyPrev,real *hzPrev,int WithConvection,int WithMagField,real N_EQ,int *mat_matrixX,int *mat_matrixY,int *mat_matrixZ,real dt,real dx,real dy,real dz,int NCELLX,int NCELLY,int NCELLZ,int first_medium,real *d_1_d,real *d_2_d,real *d_3_d,real *d_4_d,real *d_5_d,real *d_NL,int N_drude_poles,real mu0,real e0, real me,real z0){
      int i,j,k;
      real Vx1,Vx2,Vx3,Vy1,Vy2,Vy3,Vz1,Vz2,Vz3,Hx1,Hz1,Hy1,Hx2,Hz2,Hy2,Ex1,Ey1,Ez1,VdotGrad,VdotGrad2,VdotGrad3,DivV,VcrossH,VcrossH2,Pressure,ND1,ND2,ND3,Grad_Div,Grad_Div2;
      real INV_DX,INV_DY,INV_DZ;
      INV_DX = 1.0/dx;
      INV_DY = 1.0/dy;
      INV_DZ = 1.0/dz;
      Grad_Div = 0.0;
      Grad_Div2 =0.0;
      ////#pragma omp parallel for collapse(3) // schedule(static)
      // for(i=1;i<NCELLX-1;i++){
      //     for(j=1;j<NCELLY-1;j++){
      //             for(k=0;k<NCELLZ-1;k++){

                      int idx = blockDim.x * blockIdx.x + threadIdx.x;

                      i = idx / (NCELLZ*NCELLY);
                      j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
                      k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
                    if(k<NCELLZ-1 && i>0 && i<NCELLX-1 && j>0 && j<NCELLY-1){

                    if(mat_matrixZ[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]> first_medium && mat_matrixZ[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] < 6){


                        Grad_Div2 = INV_DZ*INV_DZ*(Pz_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)] - 2.0*Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])
                                   + INV_DZ*INV_DY*(Py_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMapD(i,j-1,k+1,NCELLZ,NCELLY)]+Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])
                                   + INV_DX*INV_DZ*(Px_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMapD(i-1,j,k+1,NCELLZ,NCELLY)]+Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)]);

                          if(i==0 && j==0){
                            Grad_Div = INV_DZ*INV_DZ*(Pz_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])
                                       + INV_DZ*INV_DY*(Py_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,NCELLY-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMapD(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                       + INV_DX*INV_DZ*(Px_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(NCELLX-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMapD(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                          }
                          else if(i==0){
                            Grad_Div = INV_DZ*INV_DZ*(Pz_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])
                                       + INV_DZ*INV_DY*(Py_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                       + INV_DX*INV_DZ*(Px_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(NCELLX-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMapD(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                          }
                          else if(j==0){
                            Grad_Div = INV_DZ*INV_DZ*(Pz_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])
                                       + INV_DZ*INV_DY*(Py_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,NCELLY-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMapD(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                       + INV_DX*INV_DZ*(Px_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                          }
                          else{
                            Grad_Div = INV_DZ*INV_DZ*(Pz_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])
                                       + INV_DZ*INV_DY*(Py_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMapD(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
                                       + INV_DX*INV_DZ*(Px_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMapD(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);

                          }


                      // Grad_Div = 0.0;
                      // Grad_Div2 =0.0;
                      ND1 = N_EQ + NDz_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];

                     Vz1 = 0.5 * (Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                     Vz2 = 0.5 * (Pz_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                     Vz3 = 0.5 * (Pz_d_n_2[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_2[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);


                     Vx1 = 0.25 * (Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                     Vx2 = 0.25 * (Px_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMapD(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                     Vx3 = 0.25 * (Px_d_n_2[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_2[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_2[FourDMapD(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_2[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);

                     //Vx1 = 0.5 * (Vx1 + Vx2);
                     Vy1 = 0.25 * (Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                     Vy2 = 0.25 * (Py_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMapD(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
                     Vy3 = 0.25 * (Py_d_n_2[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_2[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_2[FourDMapD(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_2[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);

                     //	Vy1 = 0.5 * (Vy1 + Vy2);
                     Hy1 = 0.5 * (hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)] + hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
                     Hx1 = 0.5 * (hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)]);
                     Hy2 = 0.5 * (hyPrev[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)] + hyPrev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
                     Hx2 = 0.5 * (hxPrev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + hxPrev[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)]);

                     if(WithMagField==1){
                      VcrossH = Vx1*Hy1 - Vy1*Hx1;
                      VcrossH2 = Vx2*Hy2 - Vy2*Hx2;
                      VcrossH = (VcrossH - VcrossH2)/dt;
                     }
                     else VcrossH = 0.0;

                     if(WithConvection==1){
                      VdotGrad = 0.5*(Vx1*(Pz_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy1*(Pz_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz1*(Pz_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
                      VdotGrad2 = 0.5*(Vx2*(Pz_d_n_1[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n_1[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy2*(Pz_d_n_1[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n_1[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz2*(Pz_d_n_1[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n_1[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
                      VdotGrad3 = 0.5*(Vx3*(Pz_d_n_2[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n_2[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy3*(Pz_d_n_2[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n_2[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz3*(Pz_d_n_2[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n_2[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);

                      // VdotGrad = 1.0*(Vx1*(Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy1*(Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz1*(Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
                      // VdotGrad2 = 1.0*(Vx2*(Pz_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n_1[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy2*(Pz_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n_1[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz2*(Pz_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n_1[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);

                      // VdotGrad = (VdotGrad - VdotGrad2)/dt;
                      // VdotGrad = 0.5*(3.0*VdotGrad - 4.0*VdotGrad2 + VdotGrad3)/dt;
                      VdotGrad = (VdotGrad-VdotGrad2)/dt;


                     }
                     else VdotGrad = 0.0;

                     Pz_d[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] = d_1_d[0]*Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + d_2_d[0]*Pz_d_n_1[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]
                     + d_3_d[0]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_4_d[0]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ d_5_d[0]*(VdotGrad + (mu0*e0/me)*VcrossH/z0) + d_NL[0]*(Grad_Div + Grad_Div2/N_EQ)/powf(ND1,1.0/3.0);
                  //
                  //   }
                  //
                  //
                  // }
                }
              }
    }



  __global__  void UPDATE_e_inc(real* e_inc,real* h_inc,int inc_Length,real d_1D,real c0,real dt,real z0,real ep0,int t,real delay,real width,real pi,real f_0,int m0){
        real pulse;
        real factor = c0*dt;
        real e1,e2;
        int m;

        //1st Order Mur ABC buffers


        int i = blockDim.x * blockIdx.x + threadIdx.x;
        if(i<inc_Length){
        e1=e_inc[1];

        e2=e_inc[inc_Length-2];
      }
        //#pragma omp parallel for private(i)// schedule(guided)
    //    for(i=1;i<inc_Length-1;i++){
    __syncthreads();
    if(i>0 && i<inc_Length-1){
            e_inc[i]=e_inc[i]-(dt/(z0*ep0*d_1D))*(h_inc[i]-h_inc[i-1]);
      }
    //    }
      //  printf("%f\t%f\n",(dt/(z0*ep0*d_1D))*(h_inc[i]-h_inc[i-1]),((c0*dt-d_1D)/(c0*dt+d_1D)));
        //1st order Mur ABC
      //  __syncthreads();
      __syncthreads();
      //
      // if(i==inc_Length-1){
      //   e_inc[0]=e1+((factor-d_1D)/(factor+d_1D))*(e_inc[1]-e_inc[0]);
      //
      //   e_inc[inc_Length-1]=e2+((factor-d_1D)/(factor+d_1D))*(e_inc[inc_Length-2]-e_inc[inc_Length-1]);
      //
      //
      //   }
      //   if(i==m0-50){
      //     //introduce source
      //     #ifdef DOUBLECOMPLEX
      //     pulse=exp(-pow((real)(t-delay)/(real)width,2)/2.0)*cexp(I*2*pi*f_0*(t)*dt);
      //     #endif
      //     #ifndef DOUBLECOMPLEX
      //     pulse=exp(-powf((real)(t-delay)/(real)width,2)/2.0)*sin(2*pi*f_0*(t)*dt);
      //     #endif
      //   //  pulse =0.0;
      //     e_inc[m0-50]+=pulse;
      //   }
      //  printf("%f\n",pulse);
    }

  __global__  void UPDATE_h_inc(real* e_inc,real* h_inc,int inc_Length,real d_1D,real c0,real dt,real z0,real ep0,int t,real delay,real width,real pi,real f_0,real mu0){
        // int i;

    //    //#pragma omp parallel for private(i)
      //  for(i=0;i<inc_Length-1;i++){
      int i = blockDim.x * blockIdx.x + threadIdx.x;
      if(i<inc_Length-1){
            h_inc[i]=h_inc[i]-(z0*dt/(mu0*d_1D))*(e_inc[i+1]-e_inc[i]);
        }
    }

comp *Incident_spec;

int main(void)
{

  real  StaticBuild = 1000.0;
  real Transmit;
  int CALC_REFL = 0;
  // if(argc<=1){
  //   printf("Please Input max and min trials \n");
  //   exit(1);
  // }
  //
  //  int trials, max_trials, min_trials, num_trials;
   clock_t begin, End;
   real time_spent;
   int i;
// printf("Max threads:%d\n", omp_get_max_threads() );
//   //  printf("argc=%d\n", argc);
//   //  printf("min_trials=%d \t max_trials=%d\n",(int)atoi(argv[1]),(int)atoi(argv[1]));
//   //  min_trials = atoi(argv[1]);
//   //  max_trials = atoi(argv[2]);
//
//   //omp_set_num_threads(100);
//    #pragma omp parallel
//    {
//         if(omp_get_thread_num() == 0){
//            printf("Number of parallel cores: %d\n", omp_get_num_threads());
//         }
// //         printf("Thread %d, ready to work\n",omp_get_thread_num());
// //
// // if(omp_in_parallel()) printf("In Parallel Construct\n");
// 	//else printf("NOT\n");
//     }
//
//     printf("NUMBER OF DEVICES = %d\n",omp_get_num_devices());
    begin=clock();

    READ_DATA_FILE();

    if(StaticField == 1){
      printf("Static Field\n");
    }

    #ifdef DOUBLEPRECISION
    printf("Double Precision, Real Number Simulation\n");
    PBC_CTW = 0;
    if(num_trials != 1) {
      printf("Error, Only One Trial Allowed with Double Precision\n");
    exit(-1) ;
  }
    #endif
    #ifdef DOUBLECOMPLEX
    printf("Complex Double Precision, Complex Number Simulation\n");
    PBC_CTW = 1;
    if(num_trials == 1) printf("Double Check number of trials. Only 1 selected.\n");
    #endif


    Periodic_XY=0;
    Periodic_XZ=0;
    Periodic_YZ=0;

    if(Periodic_XY == 1){
      CALC_REFL = 1;
    }

    if(TE_TM == 0){
      TEz=1;
      TMz=0;
      printf("TE Polarization");
    }
    else if(TE_TM == 1){
      TEz=0;
      TMz=1;
      printf("TM Polarization");

    }
    else{
      printf("Invalid Polarization!!\n");
      exit(-1);
    }



    real polar_theta=0;
    // num_trials=15;
    polar_psi=polar_theta*(3.14159265359)/180;
    //Infinite dispersive slab (1-yes 0-no)
    //inf_disp_slab=1;
    printf("Setup Constants\n");
    SETUP_CONST();
    printf("Setup Material Matrix\n");
    MATERIAL_MATRIX();
    printf("Entering Trials Iterations\n");

    for(trials=min_trials;trials<=max_trials;trials++){

    k_rho=trials*3.1416e7/num_trials;
    k_x=k_rho*cos(polar_psi);
    k_y=k_rho*sin(polar_psi);
    printf("kx=%f, ky=%f\n", k_x,k_y);

    freq=MALLOC1D_double(freq,NUM_freq);
    real *freqdev;
    hipMalloc(&freqdev,NUM_freq*sizeof(real));
real lam_max,lam_min;
lam_max = 700e-9;
lam_min = 200e-9;

  if(WL_or_freq == 1){ //Wavelength plot
    for(i=0;i<NUM_freq;i++){
      freq[i] = c0/(lam_min + i*(lam_max- lam_min)/(NUM_freq-1.0));
      }
      // if(NONLOCAL >= 3){
      //   // freq[0] = f_0;
      //   freq[1] = f_0*2.0;
      //   if(NUM_freq >= 3) freq[2] = f_0*3.0;
      // }
  }
  else{ //frequency plot
    for(i=0;i<NUM_freq;i++){
    freq[i] = c0/700e-9 + i*(c0/200e-9 - c0/700e-9)/(NUM_freq - 1.0);
  }
  }
hipMemcpy(freqdev,freq,sizeof(real)*NUM_freq,hipMemcpyHostToDevice);

    SETUP_Drude_CP();

    SETUP_SNAPSHOT();

    ALLOCATE_MEM();

    SETUP_TFSF();
    printf("fmin = %e, WL_max = %e\n",f_min, c0/f_min);


    //make the field vectors all zeros
    ex=ZERO_VECTORS3D_Complex(ex,NCELLX,NCELLY,NCELLZ);
    ey=ZERO_VECTORS3D_Complex(ey,NCELLX,NCELLY,NCELLZ);
    ez=ZERO_VECTORS3D_Complex(ez,NCELLX,NCELLY,NCELLZ);
    hx=ZERO_VECTORS3D_Complex(hx,NCELLX,NCELLY,NCELLZ);
    hy=ZERO_VECTORS3D_Complex(hy,NCELLX,NCELLY,NCELLZ);
    hz=ZERO_VECTORS3D_Complex(hz,NCELLX,NCELLY,NCELLZ);
hipMemcpy(exdev,ex,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(eydev,ey,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(ezdev,ez,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(hxdev,hx,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(hydev,hy,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(hzdev,hz,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);


printf("E and H fields zeroed\n");


    if(Hydrodynamics == 1){
    hxPrev=ZERO_VECTORS3D_Complex(hxPrev,NCELLX,NCELLY,NCELLZ);
    hyPrev=ZERO_VECTORS3D_Complex(hyPrev,NCELLX,NCELLY,NCELLZ);
    hzPrev=ZERO_VECTORS3D_Complex(hzPrev,NCELLX,NCELLY,NCELLZ);
    hipMemcpy(hxPrevdev,hxPrev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(hyPrevdev,hyPrev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(hzPrevdev,hzPrev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    NDx=ZERO_VECTORS3D_Complex(NDx,NCELLX,NCELLY,NCELLZ);
    NDy=ZERO_VECTORS3D_Complex(NDy,NCELLX,NCELLY,NCELLZ);
    NDz=ZERO_VECTORS3D_Complex(NDz,NCELLX,NCELLY,NCELLZ);
    hipMemcpy(NDxdev,NDx,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(NDydev,NDy,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(NDzdev,NDz,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    NDx_prev=ZERO_VECTORS3D_Complex(NDx_prev,NCELLX,NCELLY,NCELLZ);
    NDy_prev=ZERO_VECTORS3D_Complex(NDy_prev,NCELLX,NCELLY,NCELLZ);
    NDz_prev=ZERO_VECTORS3D_Complex(NDz_prev,NCELLX,NCELLY,NCELLZ);
    hipMemcpy(NDx_prevdev,NDx_prev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(NDy_prevdev,NDy_prev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(NDz_prevdev,NDz_prev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
  }

    ex_n=ZERO_VECTORS3D_Complex(ex_n,NCELLX,NCELLY,NCELLZ);
    ey_n=ZERO_VECTORS3D_Complex(ey_n,NCELLX,NCELLY,NCELLZ);
    ez_n=ZERO_VECTORS3D_Complex(ez_n,NCELLX,NCELLY,NCELLZ);
    hipMemcpy(ex_ndev,ex_n,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ey_ndev,ey_n,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ez_ndev,ez_n,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    ex_n_1=ZERO_VECTORS3D_Complex(ex_n_1,NCELLX,NCELLY,NCELLZ);
    ey_n_1=ZERO_VECTORS3D_Complex(ey_n_1,NCELLX,NCELLY,NCELLZ);
    ez_n_1=ZERO_VECTORS3D_Complex(ez_n_1,NCELLX,NCELLY,NCELLZ);
    hipMemcpy(ex_n_1dev,ex_n_1,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ey_n_1dev,ey_n_1,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ez_n_1dev,ez_n_1,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    printf("E and H fields zeroed\n");

    E_incident=ZERO_VECTORS1D_Complex2(E_incident,NUM_freq);
    E_reflected=ZERO_VECTORS1D_Complex2(E_reflected,NUM_freq);
    E_transmitted=ZERO_VECTORS1D_Complex2(E_transmitted,NUM_freq);


    Incident_spec=MALLOC1D_Complex(Incident_spec,NUM_freq);
    Incident_spec=ZERO_VECTORS1D_Complex(Incident_spec,NUM_freq);

    Ex_Reflected = ZERO_VECTORS3D_Complex2(Ex_Reflected, NUM_freq, NCELLX,NCELLY);
    Hx_Reflected = ZERO_VECTORS3D_Complex2(Hx_Reflected, NUM_freq, NCELLX,NCELLY);
    Ey_Reflected = ZERO_VECTORS3D_Complex2(Ey_Reflected, NUM_freq, NCELLX,NCELLY);
    Hy_Reflected = ZERO_VECTORS3D_Complex2(Hy_Reflected, NUM_freq, NCELLX,NCELLY);

    Ex_Transmitted = ZERO_VECTORS3D_Complex2(Ex_Transmitted, NUM_freq, NCELLX,NCELLY);
    Hx_Transmitted = ZERO_VECTORS3D_Complex2(Hx_Transmitted, NUM_freq, NCELLX,NCELLY);
    Ey_Transmitted = ZERO_VECTORS3D_Complex2(Ey_Transmitted, NUM_freq, NCELLX,NCELLY);
    Hy_Transmitted = ZERO_VECTORS3D_Complex2(Hy_Transmitted, NUM_freq, NCELLX,NCELLY);

    E_Incident = ZERO_VECTORS3D_Complex2(E_Incident, NUM_freq, NCELLX, NCELLY);
    H_Incident = ZERO_VECTORS3D_Complex2(H_Incident, NUM_freq, NCELLX, NCELLY);

    printf("Spectral Vectors Zeroed\n");
     e_inc=ZERO_VECTORS1D_Complex(e_inc,inc_Length);
     h_inc=ZERO_VECTORS1D_Complex(h_inc,inc_Length);
    //
    // ex_inc=ZERO_VECTORS1D_Complex(ex_inc,inc_Length);
    // ey_inc=ZERO_VECTORS1D_Complex(ey_inc,inc_Length);
    // ez_inc=ZERO_VECTORS1D_Complex(ez_inc,inc_Length);
    // hx_inc=ZERO_VECTORS1D_Complex(hx_inc,inc_Length);
    // hy_inc=ZERO_VECTORS1D_Complex(hy_inc,inc_Length);
    // hz_inc=ZERO_VECTORS1D_Complex(hz_inc,inc_Length);

    //Define permitivity, permeability, and conductivities (electric and magnetic)
    DEF_EPS();
    DEF_MU();
    DEF_SIGMA_E();
    DEF_SIGMA_M();
  //  DIELECTRIC_SLAB();

    //Define update coefficients
    printf("Setting Up update coefficients for E\n");
    Cexe=DEF_UPDATE_COEFF_EonE(Cexe);
    Cexh=DEF_UPDATE_COEFF_EonH(Cexh);
    Ceye=DEF_UPDATE_COEFF_EonE(Ceye);
    Ceyh=DEF_UPDATE_COEFF_EonH(Ceyh);
    Ceze=DEF_UPDATE_COEFF_EonE(Ceze);
    Cezh=DEF_UPDATE_COEFF_EonH(Cezh);
    hipMemcpy(Cexedev,Cexe,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(Cexhdev,Cexh,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(Ceyedev,Ceye,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(Ceyhdev,Ceyh,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(Cezedev,Ceze,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(Cezhdev,Cezh,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    printf("Setting Up update coefficients for H\n");

    Chxe=DEF_UPDATE_COEFF_HonE(Chxe);
//    printf("here\n");
    Chxh=DEF_UPDATE_COEFF_HonH(Chxh);
  //  printf("here1\n");
    Chye=DEF_UPDATE_COEFF_HonE(Chye);
  //  printf("here2\n");
    Chyh=DEF_UPDATE_COEFF_HonH(Chyh);
  //  printf("here3\n");
    Chze=DEF_UPDATE_COEFF_HonE(Chze);
  //  printf("here4\n");
    Chzh=DEF_UPDATE_COEFF_HonH(Chzh);

    hipMemcpy(Chxedev,Chxe,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(Chxhdev,Chxh,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(Chyedev,Chye,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(Chyhdev,Chyh,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(Chzedev,Chze,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(Chzhdev,Chzh,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);



    printf("Setting Up CPML\n");
    SETUP_CPML();
  //  SETUP_t_inc();
  //  FT_source_calc();
trigger = 0;

// real FOURIERVEC_RE[NUM_freq*Tend];
// real FOURIERVEC_IE[NUM_freq*Tend];
// real FOURIERVEC_RH[NUM_freq*Tend];
// real FOURIERVEC_IH[NUM_freq*Tend];
// // real *FOURIERVEC_REdev,*FOURIERVEC_RHdev,*FOURIERVEC_IHdev,*FOURIERVEC_IEdev;
// // hipMalloc(FOURIERVEC_REdev,sizeof()
// int mm;
//
// for(mm=0;mm<NUM_freq;mm++){
// for(t=0;t<Tend;t++){
//   FOURIERVEC_RE[TwoDMap(mm,t,Tend)] = cos(2.0*pi*(t+1)*dt*freq[mm]);
//   FOURIERVEC_IE[TwoDMap(mm,t,Tend)] = sin(2.0*pi*(t+1)*dt*freq[mm]);
//   FOURIERVEC_RH[TwoDMap(mm,t,Tend)] = cos(2.0*pi*(t+0.5)*dt*freq[mm]);
//   FOURIERVEC_IH[TwoDMap(mm,t,Tend)] = sin(2.0*pi*(t+0.5)*dt*freq[mm]);
// }
// }
comp pulse;
// if(StaticField == 0){
FILE *SourceFile;
SourceFile = fopen("SourceFile.txt","w");
 int m;
for(t=0;t<Tend;t++){
  #ifdef DOUBLECOMPLEX
  pulse=exp(-pow((real)(t-delay)/(real)width,2)/2.0)*cexp(I*2*pi*f_0*(t)*dt);
  #endif
  #ifndef DOUBLECOMPLEX
  pulse=MAX_AMP*exp(-powf((real)(t-delay)/(real)width,2)/2.0)*sin(2*pi*f_0*(t)*dt);
  #endif
fprintf(SourceFile,"%e\t%e\n",t*dt,pulse);
for(m=0;m<NUM_freq;m++){
  E_incident[m] += pulse*cexp(I*2*pi*t*dt*freq[m]);
}
}
fclose(SourceFile);
// }

hipMalloc(&e_incdev,inc_Length*sizeof(comp));
hipMalloc(&h_incdev,inc_Length*sizeof(comp));
hipMemcpy(e_incdev,e_inc,inc_Length*sizeof(comp),hipMemcpyHostToDevice);
hipMemcpy(h_incdev,h_inc,inc_Length*sizeof(comp),hipMemcpyHostToDevice);

//   printf("Malloc H sources\n");
//   Hx_source = MALLOC3D_Complex(Hx_source,NCELLX,NCELLY,Tend_inc);
//   Hy_source = MALLOC3D_Complex(Hy_source,NCELLX,NCELLY,Tend_inc);
// //  Hz_source = MALLOC3D_Complex(Hz_source,NCELLX,NCELLY,Tend_inc);
//   printf("Malloc E sources\n");
//   Ex_source = MALLOC3D_Complex(Ex_source,NCELLX,NCELLY,Tend_inc);
//   Ey_source = MALLOC3D_Complex(Ey_source,NCELLX,NCELLY,Tend_inc);
// //  Ez_source = MALLOC3D_Complex(Ez_source,NCELLX,NCELLY,Tend_inc);
//
//   printf("Zero H sources\n");
//   Hx_source = ZERO_VECTORS3D_Complex(Hx_source,NCELLX,NCELLY,Tend_inc);
//   Hy_source = ZERO_VECTORS3D_Complex(Hy_source,NCELLX,NCELLY,Tend_inc);
// //  Hz_source = ZERO_VECTORS3D_Complex(Hz_source,NCELLX,NCELLY,Tend_inc);
//
//   printf("Zero E Sources\n");
//   Ex_source = ZERO_VECTORS3D_Complex(Ex_source,NCELLX,NCELLY,Tend_inc);
//   Ey_source = ZERO_VECTORS3D_Complex(Ey_source,NCELLX,NCELLY,Tend_inc);
// //  Ez_source = ZERO_VECTORS3D_Complex(Ez_source,NCELLX,NCELLY,Tend_inc);
  //printf("Setting Up Source");


//if(PBC_CTW == 1) SOURCE_SETUP();
printf("Source Setup Finished \n");
Source=fopen("source.txt","w");
trigger = 1;
//    int m;
//    for(m=0;m<NcpmlZ+1;m++){
//
//        printf("%e\t%e\n",kedz[m],kedz[NCELLZ-m-1]);
//    }

FILE *Test;
Test = fopen("Test.txt","w");
Test_offset;


    char filename[100];
    FILE* Reflected;
    //Reflected=fopen(filename,"w");
    real Reflectivity;
    int i;
    real Amplitude;
real *EzInf,*JzInf,*EzInfdev,*JzInfdev;
hipMalloc(&EzInfdev,sizeof(real));
hipMalloc(&JzInfdev,sizeof(real));
EzInf = (real *)malloc(sizeof(real));
JzInf = (real *)malloc(sizeof(real));


    centerx = floor(NCELLX/2);
    centery = floor(NCELLY/2);
    int Number;
  	int threadsPerBlock = 350;
    Number = NCELLX * NCELLY *NCELLZ;
  	int blocksPerGrid = Number/threadsPerBlock + 1;
    dim3 blocksPerGrid2(blocksPerGrid,NUM_freq);
    int threadsPerBlock1D = threadsPerBlock;
    int blocksPerGrid1D = inc_Length/threadsPerBlock1D + 1;
    //Time Stepping
    real factor = c0*dt;

     hipError_t err = hipSuccess;
      hipProfilerStart();
      err = hipGetLastError();
      if( err != hipSuccess)
      {
          printf( "Cuda error: %s.\n",hipGetErrorString( err) );
          exit(-1);
      }

    FILE *OUTFile;
    OUTFile = fopen("OUT.txt","w");
    for(t=0;t<=Tend;t++)
    {
        printf("%d\t%d\t%e\t%e\n",t,trials,ey[ThreeDMap((int)NCELLX/2,(int)NCELLY/2,(int)NCELLZ/2,NCELLZ,NCELLY)],pulse);

        if(isnan(ex[ThreeDMap((int)NCELLX/2,(int)NCELLY/2,(int)NCELLZ/2 - int(NCELLZ/4),NCELLZ,NCELLY)]) ) {
          fprintf(OUTFile, "NAN Detected\n");
          break;
        }
        fprintf(OUTFile,"%e\t%e\t%e\t%e\n",t*dt,ex[ThreeDMap((int)NCELLX/2,(int)NCELLY/2,(int)NCELLZ/2 ,NCELLZ,NCELLY)],ey[ThreeDMap((int)NCELLX/2,(int)NCELLY/2,(int)NCELLZ/2,NCELLZ,NCELLY)],ez[ThreeDMap((int)NCELLX/2,(int)NCELLY/2,(int)NCELLZ/2,NCELLZ,NCELLY)]);
        //UPDATE_B();
// printf("here\n");
      UPDATE_hx <<<blocksPerGrid, threadsPerBlock>>> (hxdev,hxPrevdev,ezdev,eydev,Chxhdev,Chxedev,psi_Hx_z_Ndev,psi_Hx_z_Fdev,psi_Hx_y_Ndev,psi_Hx_y_Fdev,khdydev,khdzdev,bh_z_Ndev,bh_z_Fdev,ch_z_Ndev,ch_z_Fdev,bh_y_Ndev,bh_y_Fdev,ch_y_Ndev,ch_y_Fdev,NCELLX,NCELLY,NCELLZ,Periodic_XY,dx,dy,dz,dt,cpml_N_Z,cpml_F_Z,cpml_N_Y,cpml_F_Y,cpml_z_lim,cpml_y_lim,cpml_x_lim,NcpmlZ,NcpmlY,Hydrodynamics);
        err = hipGetLastError();
      if( hipSuccess != err)
      {
          printf( "Cuda error: %s.\n",hipGetErrorString( err) );
          exit(-1);
      }
      // printf("here\n");

      // err = hipDeviceSynchronize();
      // if( hipSuccess != err)
      // {
      //     printf( "Cuda error: %s.\n",hipGetErrorString( err) );
      //     exit(-1);
      // }
      //

      UPDATE_hy <<<blocksPerGrid, threadsPerBlock>>> (hydev,hyPrevdev,ezdev,exdev,Chyhdev,Chyedev,psi_Hy_z_Ndev,psi_Hy_z_Fdev,psi_Hy_x_Ndev,psi_Hy_x_Fdev,khdxdev,khdzdev,bh_z_Ndev,bh_z_Fdev,ch_z_Ndev,ch_z_Fdev,bh_x_Ndev,bh_x_Fdev,ch_x_Ndev,ch_x_Fdev,NCELLX,NCELLY,NCELLZ,Periodic_XY,dx,dy,dz,dt,cpml_N_Z,cpml_F_Z,cpml_N_X,cpml_F_X,cpml_z_lim,cpml_y_lim,cpml_x_lim,NcpmlZ,NcpmlX,Hydrodynamics);
        err = hipGetLastError();

      if( hipSuccess != err)
      {
          printf( "Cuda error: %s.\n",hipGetErrorString( err) );
          exit(-1);
      }

      // err = hipDeviceSynchronize();
      //
      // if( hipSuccess != err)
      // {
      //     printf( "Cuda error: %s.\n",hipGetErrorString( err) );
      //     exit(-1);
      // }

       UPDATE_hz <<<blocksPerGrid, threadsPerBlock>>> (hzdev,hzPrevdev,eydev,exdev,Chzhdev,Chzedev,psi_Hz_x_Ndev,psi_Hz_x_Fdev,psi_Hz_y_Ndev,psi_Hz_y_Fdev,khdxdev,khdydev,bh_x_Ndev,bh_x_Fdev,ch_x_Ndev,ch_x_Fdev,bh_y_Ndev,bh_y_Fdev,ch_y_Ndev,ch_y_Fdev,NCELLX,NCELLY,NCELLZ,Periodic_XY,dx,dy,dz,dt,cpml_N_X,cpml_F_X,cpml_N_Y,cpml_F_Y,cpml_z_lim,cpml_y_lim,cpml_x_lim,NcpmlY,NcpmlX,Hydrodynamics);
        err = hipGetLastError();
      if( hipSuccess != err)
      {
          printf( "Cuda error: %s.\n",hipGetErrorString( err) );
          exit(-1);
      }

    // err = hipDeviceSynchronize();
        if( hipSuccess != err)
        {
            printf( "Cuda error: %s.\n",hipGetErrorString( err) );
            exit(-1);
        }

      //   hipMemcpy(ex,exdev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
      //   hipMemcpy(ey,eydev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
      //   hipMemcpy(ez,ezdev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
      // hipMemcpy(hx,hxdev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
      // hipMemcpy(hy,hydev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
      // hipMemcpy(hz,hzdev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
     // TFSF_CORRECT();
     UPDATE_h_inc<<<blocksPerGrid1D, threadsPerBlock1D>>>(e_incdev,h_incdev,inc_Length,d_1D,c0,dt,z0,ep0,t,delay,width,pi,f_0,mu0);
     err = hipGetLastError();
   if( hipSuccess != err)
   {
       printf( "Cuda error: %s.\n",hipGetErrorString( err) );
       exit(-1);
   }
     CORRECT_X<<<blocksPerGrid, threadsPerBlock>>>(NtfsfX,NtfsfY,NtfsfZ,NCELLX,NCELLY,NCELLZ,e_incdev,h_incdev,exdev,eydev,ezdev,hxdev,hydev,hzdev,inc_theta,inc_phi,polar_psi,polar_theta,dx,dy,dz,dt,i_0,j_0,k_0,d_1D,m0,Cexedev,Ceyedev,Cezedev,Cexhdev,Ceyhdev,Cezhdev,Chxedev,Chyedev,Chzedev,Chxhdev,Chyhdev,Chzhdev,Periodic_XY);
     err = hipGetLastError();
   if( hipSuccess != err)
   {
       printf( "Cuda error: %s.\n",hipGetErrorString( err) );
       exit(-1);
   }
     CORRECT_Y<<<blocksPerGrid, threadsPerBlock>>>(NtfsfX,NtfsfY,NtfsfZ,NCELLX,NCELLY,NCELLZ,e_incdev,h_incdev,exdev,eydev,ezdev,hxdev,hydev,hzdev,inc_theta,inc_phi,polar_psi,polar_theta,dx,dy,dz,dt,i_0,j_0,k_0,d_1D,m0,Cexedev,Ceyedev,Cezedev,Cexhdev,Ceyhdev,Cezhdev,Chxedev,Chyedev,Chzedev,Chxhdev,Chyhdev,Chzhdev,Periodic_XY);
     err = hipGetLastError();
   if( hipSuccess != err)
   {
       printf( "Cuda error: %s.\n",hipGetErrorString( err) );
       exit(-1);
   }

     CORRECT_Z<<<blocksPerGrid, threadsPerBlock>>>(NtfsfX,NtfsfY,NtfsfZ,NCELLX,NCELLY,NCELLZ,e_incdev,h_incdev,exdev,eydev,ezdev,hxdev,hydev,hzdev,inc_theta,inc_phi,polar_psi,polar_theta,dx,dy,dz,dt,i_0,j_0,k_0,d_1D,m0,Cexedev,Ceyedev,Cezedev,Cexhdev,Ceyhdev,Cezhdev,Chxedev,Chyedev,Chzedev,Chxhdev,Chyhdev,Chzhdev,Periodic_XY);
     err = hipGetLastError();
   if( hipSuccess != err)
   {
       printf( "Cuda error: %s.\n",hipGetErrorString( err) );
       exit(-1);
   }


     UPDATE_e_inc<<<blocksPerGrid1D, threadsPerBlock1D>>>(e_incdev,h_incdev,inc_Length,d_1D,c0,dt,z0,ep0,t,delay,width,pi,f_0,m0);
     //
     err = hipGetLastError();
   if( hipSuccess != err)
   {
       printf( "Cuda error: %s.\n",hipGetErrorString( err) );
       exit(-1);
   }

   hipMemcpy(e_inc,e_incdev,inc_Length*sizeof(comp),hipMemcpyDeviceToHost);

     e_inc[0]=e1+((factor-d_1D)/(factor+d_1D))*(e_inc[1]-e_inc[0]);

     e_inc[inc_Length-1]=e2+((factor-d_1D)/(factor+d_1D))*(e_inc[inc_Length-2]-e_inc[inc_Length-1]);

     e1=e_inc[1];

     e2=e_inc[inc_Length-2];


     if(StaticField == 0){
       //introduce source
       #ifdef DOUBLECOMPLEX
       pulse=exp(-pow((real)(t-delay)/(real)width,2)/2.0)*cexp(I*2*pi*f_0*(t)*dt);
       #endif
       #ifndef DOUBLECOMPLEX
       pulse=MAX_AMP*exp(-powf((real)(t-delay)/(real)width,2)/2.0)*sin(2*pi*f_0*(t)*dt);
       #endif
     //  pulse =0.0;
       e_inc[m0-50]+=pulse;
     }
     else{
          if(t<StaticBuild) pulse = (real) MAX_AMP * t/StaticBuild;
          else pulse = MAX_AMP * 1.0;
          e_inc[m0-50]+=pulse;

     }
       hipMemcpy(e_incdev,e_inc,inc_Length*sizeof(comp),hipMemcpyHostToDevice);

   // hipDeviceSynchronize();

     // hipMemcpy(exdev,ex,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
     // hipMemcpy(eydev,ey,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
     // hipMemcpy(ezdev,ez,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
     // hipMemcpy(hxdev,hx,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
     // hipMemcpy(hydev,hy,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
     // hipMemcpy(hzdev,hz,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
     if(Hydrodynamics == 1){
     UpdateHydroPx<<<blocksPerGrid, threadsPerBlock>>>(exdev,ex_ndev,Px_ddev,Px_d_ndev,Px_d_n_1dev,Px_d_n_2dev,Py_ddev,Py_d_ndev,Py_d_n_1dev,Py_d_n_2dev,Pz_ddev,Pz_d_ndev,Pz_d_n_1dev,Pz_d_n_2dev,NDxdev,NDx_prevdev,NDydev,NDy_prevdev,NDzdev,NDz_prevdev,
       hxdev,hydev,hzdev,hxPrevdev,hyPrevdev,hzPrevdev,WithConvection,WithMagField,N_EQ,mat_matrixXdev,mat_matrixYdev,mat_matrixZdev,dt,dx,dy,dz,NCELLX,NCELLY,NCELLZ,first_medium,d_1_ddev,d_2_ddev,d_3_ddev,d_4_ddev,d_5_ddev,d_NLdev,N_drude_poles, mu0, e0,  me,z0);
       UpdateHydroPy<<<blocksPerGrid, threadsPerBlock>>>(eydev,ey_ndev,Px_ddev,Px_d_ndev,Px_d_n_1dev,Px_d_n_2dev,Py_ddev,Py_d_ndev,Py_d_n_1dev,Py_d_n_2dev,Pz_ddev,Pz_d_ndev,Pz_d_n_1dev,Pz_d_n_2dev,NDxdev,NDx_prevdev,NDydev,NDy_prevdev,NDzdev,NDz_prevdev,
         hxdev,hydev,hzdev,hxPrevdev,hyPrevdev,hzPrevdev,WithConvection,WithMagField,N_EQ,mat_matrixXdev,mat_matrixYdev,mat_matrixZdev,dt,dx,dy,dz,NCELLX,NCELLY,NCELLZ,first_medium,d_1_ddev,d_2_ddev,d_3_ddev,d_4_ddev,d_5_ddev,d_NLdev,N_drude_poles, mu0, e0,  me,z0);

         UpdateHydroPz<<<blocksPerGrid, threadsPerBlock>>>(ezdev,ez_ndev,Px_ddev,Px_d_ndev,Px_d_n_1dev,Px_d_n_2dev,Py_ddev,Py_d_ndev,Py_d_n_1dev,Py_d_n_2dev,Pz_ddev,Pz_d_ndev,Pz_d_n_1dev,Pz_d_n_2dev,NDxdev,NDx_prevdev,NDydev,NDy_prevdev,NDzdev,NDz_prevdev,
           hxdev,hydev,hzdev,hxPrevdev,hyPrevdev,hzPrevdev,WithConvection,WithMagField,N_EQ,mat_matrixXdev,mat_matrixYdev,mat_matrixZdev,dt,dx,dy,dz,NCELLX,NCELLY,NCELLZ,first_medium,d_1_ddev,d_2_ddev,d_3_ddev,d_4_ddev,d_5_ddev,d_NLdev,N_drude_poles, mu0, e0,  me,z0);
}


        UPDATE_ex <<<blocksPerGrid, threadsPerBlock>>> (exdev,ex_ndev,ex_n_1dev,hydev,hzdev,Cexedev,Cexhdev,kedydev,kedzdev,mat_matrixdev,mat_matrixXdev,first_medium_max,psi_Ex_z_Ndev,psi_Ex_z_Fdev,psi_Ex_y_Ndev,psi_Ex_y_Fdev,Px_cpdev,Px_cp_ndev,Px_cp_n_1dev,Px_ddev,Px_d_ndev,Px_d_n_1dev,Px_d_n_2dev,Py_ddev,Py_d_ndev,Py_d_n_1dev,Py_d_n_2dev,Pz_ddev,Pz_d_ndev,Pz_d_n_1dev,Pz_d_n_2dev,
       C_1_cpdev,C_2_cpdev,C_3_cpdev,C_4_cpdev,C_5_cpdev,d_1_ddev,d_2_ddev,d_3_ddev,d_4_ddev,d_5_ddev,d_NLdev,C_E,z0,N_CP_poles,N_drude_poles,ce_z_Ndev,ce_z_Fdev,be_z_Ndev,be_z_Fdev,ce_y_Ndev,ce_y_Fdev,be_y_Ndev,be_y_Fdev,dx,dy,dz,dt,NCELLX,NCELLY,NCELLZ,
       Hydrodynamics,cpml_x_lim,cpml_y_lim,cpml_z_lim,cpml_N_Y,cpml_F_Y,cpml_N_Z,cpml_F_Z,NcpmlY,NcpmlZ, C_E_1,C_E_2,Periodic_XY,NDxdev,NDydev,NDzdev,NDx_prevdev,NDy_prevdev,NDz_prevdev, e0, N_EQ);
       err = hipGetLastError();
     if( hipSuccess != err)
     {
         printf( "Cuda error: %s.\n",hipGetErrorString( err) );
         exit(-1);
     }
        //UPDATE_ey();
        UPDATE_ey <<<blocksPerGrid, threadsPerBlock>>> (eydev,ey_ndev,ey_n_1dev,hxdev,hzdev,Ceyedev,Ceyhdev,kedxdev,kedzdev,mat_matrixdev,mat_matrixYdev,first_medium_max,psi_Ey_z_Ndev,psi_Ey_z_Fdev,psi_Ey_x_Ndev,psi_Ey_x_Fdev,Py_cpdev,Py_cp_ndev,Py_cp_n_1dev,Px_ddev,Px_d_ndev,Px_d_n_1dev,Px_d_n_2dev,Py_ddev,Py_d_ndev,Py_d_n_1dev,Py_d_n_2dev,Pz_ddev,Pz_d_ndev,Pz_d_n_1dev,Pz_d_n_2dev,
       C_1_cpdev,C_2_cpdev,C_3_cpdev,C_4_cpdev,C_5_cpdev,d_1_ddev,d_2_ddev,d_3_ddev,d_4_ddev,d_5_ddev,d_NLdev,C_E,z0,N_CP_poles,N_drude_poles,ce_z_Ndev,ce_z_Fdev,be_z_Ndev,be_z_Fdev,ce_y_Ndev,ce_x_Fdev,be_x_Ndev,be_x_Fdev,dx,dy,dz,dt,NCELLX,NCELLY,NCELLZ,
       Hydrodynamics,cpml_x_lim,cpml_y_lim,cpml_z_lim,cpml_N_X,cpml_F_X,cpml_N_Z,cpml_F_Z,NcpmlX,NcpmlZ, C_E_1,C_E_2,Periodic_XY,NDxdev,NDydev,NDzdev,NDx_prevdev,NDy_prevdev,NDz_prevdev, e0, N_EQ);
       err = hipGetLastError();
     if( hipSuccess != err)
     {
         printf( "Cuda error: %s.\n",hipGetErrorString( err) );
         exit(-1);
     }

        // UPDATE_ez();
        UPDATE_ez <<<blocksPerGrid, threadsPerBlock>>> (ezdev,ez_ndev,ez_n_1dev,hxdev,hydev,Cezedev,Cezhdev,kedxdev,kedydev,mat_matrixdev,mat_matrixZdev,first_medium_max,psi_Ez_y_Ndev,psi_Ez_y_Fdev,psi_Ez_x_Ndev,psi_Ez_x_Fdev,Pz_cpdev,Pz_cp_ndev,Pz_cp_n_1dev,Px_ddev,Px_d_ndev,Px_d_n_1dev,Px_d_n_2dev,Py_ddev,Py_d_ndev,Py_d_n_1dev,Py_d_n_2dev,Pz_ddev,Pz_d_ndev,Pz_d_n_1dev,Pz_d_n_2dev,
       C_1_cpdev,C_2_cpdev,C_3_cpdev,C_4_cpdev,C_5_cpdev,d_1_ddev,d_2_ddev,d_3_ddev,d_4_ddev,d_5_ddev,d_NLdev,C_E,z0,N_CP_poles,N_drude_poles,ce_y_Ndev,ce_y_Fdev,be_y_Ndev,be_y_Fdev,ce_x_Ndev,ce_x_Fdev,be_x_Ndev,be_x_Fdev,dx,dy,dz,dt,NCELLX,NCELLY,NCELLZ,
       Hydrodynamics,cpml_x_lim,cpml_y_lim,cpml_z_lim,cpml_N_X,cpml_F_X,cpml_N_Y,cpml_F_Y,NcpmlX,NcpmlY, C_E_1,C_E_2,Periodic_XY,NDxdev,NDydev,NDzdev,NDx_prevdev,NDy_prevdev,NDz_prevdev, e0, N_EQ);

       err = hipGetLastError();
     if( hipSuccess != err)
     {
         printf( "Cuda error: %s.\n",hipGetErrorString( err) );
         exit(-1);
     }

//      hipMemcpy(EzInf,EzInfdev,sizeof(real),hipMemcpyDeviceToHost);
//      hipMemcpy(JzInf,JzInfdev,sizeof(real),hipMemcpyDeviceToHost);
// printf("%e\t%e\n",JzInf[0],EzInf[0]);

// hipDeviceSynchronize();


  //  Fourier_Transform();
 if(StaticField == 0 && t % 4 == 0){
  ScattAbs <<<blocksPerGrid, threadsPerBlock>>>  (exdev,eydev,ezdev,hxdev,hydev,hzdev,NUM_freq,t,dt,freqdev,pi,XSTARTAbs,XENDAbs,YSTARTAbs,YENDAbs,ZSTARTAbs,ZENDAbs,XSTARTSca,XENDSca,YSTARTSca,YENDSca,ZSTARTSca,ZENDSca,XNEARAbs,XFARAbs,YNEARAbs,YFARAbs,ZNEARAbs,ZFARAbs,
   XNEARSca,XFARSca,YNEARSca,YFARSca,ZNEARSca,ZFARSca,ExTransformNearZAbsRedev,ExTransformNearZAbsImdev,EyTransformNearZAbsRedev,EyTransformNearZAbsImdev,HxTransformNearZAbsRedev,HxTransformNearZAbsImdev,HyTransformNearZAbsRedev,HyTransformNearZAbsImdev,
   ExTransformFarZAbsRedev,ExTransformFarZAbsImdev,EyTransformFarZAbsRedev,EyTransformFarZAbsImdev,HxTransformFarZAbsRedev,HxTransformFarZAbsImdev,HyTransformFarZAbsRedev,HyTransformFarZAbsImdev,
   ExTransformNearYAbsRedev,ExTransformNearYAbsImdev,EzTransformNearYAbsRedev,EzTransformNearYAbsImdev,HxTransformNearYAbsRedev,HxTransformNearYAbsImdev,HzTransformNearYAbsRedev,HzTransformNearYAbsImdev,
   ExTransformFarYAbsRedev,ExTransformFarYAbsImdev,EzTransformFarYAbsRedev,EzTransformFarYAbsImdev,HxTransformFarYAbsRedev,HxTransformFarYAbsImdev,HzTransformFarYAbsRedev,HzTransformFarYAbsImdev,
   EyTransformNearXAbsRedev,EyTransformNearXAbsImdev,EzTransformNearXAbsRedev,EzTransformNearXAbsImdev,HyTransformNearXAbsRedev,HyTransformNearXAbsImdev,HzTransformNearXAbsRedev,HzTransformNearXAbsImdev,
   EyTransformFarXAbsRedev,EyTransformFarXAbsImdev,EzTransformFarXAbsRedev,EzTransformFarXAbsImdev,HyTransformFarXAbsRedev,HyTransformFarXAbsImdev,HzTransformFarXAbsRedev,HzTransformFarXAbsImdev,
   ExTransformNearZScaRedev,ExTransformNearZScaImdev,EyTransformNearZScaRedev,EyTransformNearZScaImdev,HxTransformNearZScaRedev,HxTransformNearZScaImdev,HyTransformNearZScaRedev,HyTransformNearZScaImdev,
   ExTransformFarZScaRedev,ExTransformFarZScaImdev,EyTransformFarZScaRedev,EyTransformFarZScaImdev,HxTransformFarZScaRedev,HxTransformFarZScaImdev,HyTransformFarZScaRedev,HyTransformFarZScaImdev,
   ExTransformNearYScaRedev,ExTransformNearYScaImdev,EzTransformNearYScaRedev,EzTransformNearYScaImdev,HxTransformNearYScaRedev,HxTransformNearYScaImdev,HzTransformNearYScaRedev,HzTransformNearYScaImdev,
   ExTransformFarYScaRedev,ExTransformFarYScaImdev,EzTransformFarYScaRedev,EzTransformFarYScaImdev,HxTransformFarYScaRedev,HxTransformFarYScaImdev,HzTransformFarYScaRedev,HzTransformFarYScaImdev,
   EyTransformNearXScaRedev,EyTransformNearXScaImdev,EzTransformNearXScaRedev,EzTransformNearXScaImdev,HyTransformNearXScaRedev,HyTransformNearXScaImdev,HzTransformNearXScaRedev,HzTransformNearXScaImdev,
   EyTransformFarXScaRedev,EyTransformFarXScaImdev,EzTransformFarXScaRedev,EzTransformFarXScaImdev,HyTransformFarXScaRedev,HyTransformFarXScaImdev,HzTransformFarXScaRedev,HzTransformFarXScaImdev,NCELLX,NCELLY,NCELLZ);
}
// hipDeviceSynchronize();
        //
        // #ifdef DOUBLECOMPLEX
        // fprintf(Test, "%e\t%e\t%e\t%e\t%e\t%e\n",creal(ex[centerx][centery][inc_plane - Test_offset]),creal(ey[centerx][centery][inc_plane - Test_offset]),creal(ez[centerx][centery][inc_plane - Test_offset]),creal(hx[centerx][centery][inc_plane - Test_offset]),creal(hy[centerx][centery][inc_plane - Test_offset]),creal(hz[centerx][centery][inc_plane - Test_offset] ));
        // #endif
        //
        // #ifdef DOUBLEPRECISION
        // fprintf(Test, "%e\t%e\t%e\t%e\t%e\t%e\n",ex[centerx][centery][25],ey[centerx][centery][25],ez[centerx][centery][25],hx[centerx][centery][25],hy[centerx][centery][25],hz[centerx][centery][25]);
        // #endif
        // hipMemcpy(ex,exdev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
        hipMemcpy(ey,eydev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
        // hipMemcpy(ez,ezdev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);

      if(t%t_skip==0){
          //snapshot_count++;
//          //  SNAPSHOT_2D();
//            //aux field:
      if(Snap_in == 1) SNAPSHOT_1D();
      if(Snap_in == 2){
        hipDeviceSynchronize();

          // hipMemcpy(ex,NDxdev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
          // hipMemcpy(ey,NDydev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
          // hipMemcpy(ez,NDzdev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);

          hipMemcpy(ex,Px_ddev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
          hipMemcpy(ey,Py_ddev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
          hipMemcpy(ez,Pz_ddev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);

          hipMemcpy(NDx,NDxdev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
          hipMemcpy(NDy,NDydev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
          hipMemcpy(NDz,NDzdev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);

           SNAPSHOT_2D();
           SNAPSHOT_2D_N();

           hipDeviceSynchronize();

      }
      if(Snap_in == 3){
        hipDeviceSynchronize();
        hipMemcpy(ex,exdev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
        hipMemcpy(ey,eydev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);
        hipMemcpy(ez,ezdev,NCELLX*NCELLY*NCELLZ*sizeof(real),hipMemcpyDeviceToHost);

        // hipMemcpy(e_inc,e_incdev,inc_Length*sizeof(comp),hipMemcpyDeviceToHost);
          err = hipGetLastError();
        if( hipSuccess != err)
        {
            printf( "Cuda error: %s.\n",hipGetErrorString( err) );
            exit(-1);
        }
hipDeviceSynchronize();
        SNAPSHOT_1D();
        SNAPSHOT_2D();
      }
    }


  if(CALC_REFL==1 && t>0 && t%(Tend/10) == 0 && StaticField == 0){
	Reflectance_XZ();
	 if(TMz) sprintf(filename,"ReflectivityTM.%d.%d.txt",trials,t);
   if(TEz) sprintf(filename,"ReflectivityTE.%d.%d.txt",trials,t);

    Reflected=fopen(filename,"w");
    double theta;
    printf("%f",f_min);
    for(i=1;i<NUM_freq;i++){

      if(PBC_CTW == 1){
      if(freq[i]>f_min){

       Amplitude = creal(E_incident[i]);
        if(TEz) Reflectivity = -1*creal(E_reflected[i]);
        if(TMz) Reflectivity = 1*creal(E_reflected[i]);
        theta = asin(c0*k_x/(2*pi*freq[i]));
        theta = 180*theta/pi;

              if(WL_or_freq == 1)  fprintf(Reflected,"%e\t%e\t%e\t%e\t%e\t%e\n",c0/freq[i],(Reflectivity/Amplitude),Amplitude,Reflectivity,(Reflectivity/(NCELLX*dx*NCELLZ*dz))/(cabs(Incident_spec[i])*cabs(Incident_spec[i])),cabs(Incident_spec[i])*cabs(Incident_spec[i]));
              else fprintf(Reflected,"%e\t%e\t%e\t%e\t%e\t%e\n",freq[i],(Reflectivity/Amplitude),Amplitude,Reflectivity,t_inc[i],theta);

        }
        else{
            Amplitude=0.0;
            Reflectivity=1.0;
            theta = 0.0;
            if(WL_or_freq == 1) fprintf(Reflected,"%e\t%f\t%f\t%f\t%e\t%e\n",c0/freq[i],Reflectivity,Amplitude,Reflectivity,t_inc[i],theta);
            else fprintf(Reflected,"%e\t%f\t%f\t%f\t%e\t%e\n",freq[i],Reflectivity,Amplitude,Reflectivity,t_inc[i],theta);
        }
    }

  else{
    Amplitude = sqrt(creal(E_incident[i])*creal(E_incident[i]) + cimag(E_incident[i])*cimag(E_incident[i]));
    Amplitude = Amplitude*Amplitude;
    if(TEz) Reflectivity = 1*creal(E_reflected[i]);
    else if(TMz) Reflectivity = 1*creal(E_reflected[i]);
    else Reflectivity = creal(E_reflected[i]);
    Transmit = creal(E_transmitted[i]);
    if(WL_or_freq == 1)  fprintf(Reflected,"%e\t%e\t%e\t%e\t%e\t%e\n",c0/freq[i],(Reflectivity/Amplitude),(Transmit/Amplitude),Amplitude,Reflectivity,Transmit);
    else fprintf(Reflected,"%e\t%e\t%e\t%e\t%e\t%e\n",freq[i],(Reflectivity/Amplitude),(Transmit/Amplitude),Amplitude,Reflectivity,Transmit);
  }
}
    fclose(Reflected);

    }


  if((Scattering == 1 || Absorption == 1) && t%(Tend/10) == 0 && StaticField == 0){
    hipDeviceSynchronize();

    hipMemcpy(ExTransformNearZScaRe,ExTransformNearZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);
    hipMemcpy(EyTransformNearZScaRe,EyTransformNearZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);
    hipMemcpy(HxTransformNearZScaRe,HxTransformNearZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);
    hipMemcpy(HyTransformNearZScaRe,HyTransformNearZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);

    hipMemcpy(ExTransformNearYScaRe,ExTransformNearYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
    hipMemcpy(EzTransformNearYScaRe,EzTransformNearYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
    hipMemcpy(HxTransformNearYScaRe,HxTransformNearYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
    hipMemcpy(HzTransformNearYScaRe,HzTransformNearYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);



  hipMemcpy(EyTransformNearXScaRe,EyTransformNearXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
  hipMemcpy(EzTransformNearXScaRe,EzTransformNearXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
  hipMemcpy(HyTransformNearXScaRe,HyTransformNearXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
  hipMemcpy(HzTransformNearXScaRe,HzTransformNearXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);

  hipMemcpy(ExTransformNearZScaIm,ExTransformNearZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);
  hipMemcpy(EyTransformNearZScaIm,EyTransformNearZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);
  hipMemcpy(HxTransformNearZScaIm,HxTransformNearZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);
  hipMemcpy(HyTransformNearZScaIm,HyTransformNearZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);

  hipMemcpy(ExTransformNearYScaIm,ExTransformNearYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
  hipMemcpy(EzTransformNearYScaIm,EzTransformNearYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
  hipMemcpy(HxTransformNearYScaIm,HxTransformNearYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
  hipMemcpy(HzTransformNearYScaIm,HzTransformNearYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);

  hipMemcpy(EyTransformNearXScaIm,EyTransformNearXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
  hipMemcpy(EzTransformNearXScaIm,EzTransformNearXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
  hipMemcpy(HyTransformNearXScaIm,HyTransformNearXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
  hipMemcpy(HzTransformNearXScaIm,HzTransformNearXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);

  hipMemcpy(ExTransformNearZAbsRe,ExTransformNearZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);
  hipMemcpy(EyTransformNearZAbsRe,EyTransformNearZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);
  hipMemcpy(HxTransformNearZAbsRe,HxTransformNearZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);
  hipMemcpy(HyTransformNearZAbsRe,HyTransformNearZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);

  hipMemcpy(ExTransformNearYAbsRe,ExTransformNearYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
  hipMemcpy(EzTransformNearYAbsRe,EzTransformNearYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
  hipMemcpy(HxTransformNearYAbsRe,HxTransformNearYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
  hipMemcpy(HzTransformNearYAbsRe,HzTransformNearYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);

  hipMemcpy(EyTransformNearXAbsRe,EyTransformNearXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
  hipMemcpy(EzTransformNearXAbsRe,EzTransformNearXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
  hipMemcpy(HyTransformNearXAbsRe,HyTransformNearXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
  hipMemcpy(HzTransformNearXAbsRe,HzTransformNearXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);


    hipMemcpy(ExTransformNearZAbsIm,ExTransformNearZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);
    hipMemcpy(EyTransformNearZAbsIm,EyTransformNearZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);
    hipMemcpy(HxTransformNearZAbsIm,HxTransformNearZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);
    hipMemcpy(HyTransformNearZAbsIm,HyTransformNearZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);

    hipMemcpy(ExTransformNearYAbsIm,ExTransformNearYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
    hipMemcpy(EzTransformNearYAbsIm,EzTransformNearYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
    hipMemcpy(HxTransformNearYAbsIm,HxTransformNearYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
    hipMemcpy(HzTransformNearYAbsIm,HzTransformNearYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);

    hipMemcpy(EyTransformNearXAbsIm,EyTransformNearXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
    hipMemcpy(EzTransformNearXAbsIm,EzTransformNearXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
    hipMemcpy(HyTransformNearXAbsIm,HyTransformNearXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
    hipMemcpy(HzTransformNearXAbsIm,HzTransformNearXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);

      hipMemcpy(ExTransformFarZScaRe,ExTransformFarZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(EyTransformFarZScaRe,EyTransformFarZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(HxTransformFarZScaRe,HxTransformFarZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(HyTransformFarZScaRe,HyTransformFarZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);

      hipMemcpy(ExTransformFarYScaRe,ExTransformFarYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(EzTransformFarYScaRe,EzTransformFarYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(HxTransformFarYScaRe,HxTransformFarYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(HzTransformFarYScaRe,HzTransformFarYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);

      hipMemcpy(EyTransformFarXScaRe,EyTransformFarXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(EzTransformFarXScaRe,EzTransformFarXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(HyTransformFarXScaRe,HyTransformFarXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(HzTransformFarXScaRe,HzTransformFarXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);

      hipMemcpy(ExTransformFarZScaIm,ExTransformFarZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(EyTransformFarZScaIm,EyTransformFarZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(HxTransformFarZScaIm,HxTransformFarZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(HyTransformFarZScaIm,HyTransformFarZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyDeviceToHost);

      hipMemcpy(ExTransformFarYScaIm,ExTransformFarYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(EzTransformFarYScaIm,EzTransformFarYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(HxTransformFarYScaIm,HxTransformFarYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(HzTransformFarYScaIm,HzTransformFarYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);

      hipMemcpy(EyTransformFarXScaIm,EyTransformFarXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(EzTransformFarXScaIm,EzTransformFarXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(HyTransformFarXScaIm,HyTransformFarXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);
      hipMemcpy(HzTransformFarXScaIm,HzTransformFarXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyDeviceToHost);

      hipMemcpy(ExTransformFarZAbsRe,ExTransformFarZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(EyTransformFarZAbsRe,EyTransformFarZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(HxTransformFarZAbsRe,HxTransformFarZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(HyTransformFarZAbsRe,HyTransformFarZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);

      hipMemcpy(ExTransformFarYAbsRe,ExTransformFarYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(EzTransformFarYAbsRe,EzTransformFarYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(HxTransformFarYAbsRe,HxTransformFarYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(HzTransformFarYAbsRe,HzTransformFarYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);


      hipMemcpy(EyTransformFarXAbsRe,EyTransformFarXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(EzTransformFarXAbsRe,EzTransformFarXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(HyTransformFarXAbsRe,HyTransformFarXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(HzTransformFarXAbsRe,HzTransformFarXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);


      hipMemcpy(ExTransformFarZAbsIm,ExTransformFarZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(EyTransformFarZAbsIm,EyTransformFarZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(HxTransformFarZAbsIm,HxTransformFarZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(HyTransformFarZAbsIm,HyTransformFarZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyDeviceToHost);

      hipMemcpy(ExTransformFarYAbsIm,ExTransformFarYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(EzTransformFarYAbsIm,EzTransformFarYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(HxTransformFarYAbsIm,HxTransformFarYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(HzTransformFarYAbsIm,HzTransformFarYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);

      hipMemcpy(EyTransformFarXAbsIm,EyTransformFarXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(EzTransformFarXAbsIm,EzTransformFarXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(HyTransformFarXAbsIm,HyTransformFarXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipMemcpy(HzTransformFarXAbsIm,HzTransformFarXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyDeviceToHost);
      hipDeviceSynchronize();

    CalculateAbsScatt();
  }


}
hipProfilerStop();
fclose(OUTFile);
fclose(Test);
    fclose(Source);
    printf("\n TOTAL SNAPSHOTS=%d",snapshot_count);




//    FREE_MEM();

    }
    End=clock();
    time_spent=(real)(End-begin)/CLOCKS_PER_SEC;
    printf("Time Spent= %e",time_spent);
    return 0;
}


void Reflectance_XZ()
{
    real  P_r,P_t,P_r_inc, Area, dsurf1, dsurf2,k_rho,k_z;
    real u,v,u2,v2;
    double complex  zz,zz_trans;
    int i, k, m;
    double complex cEx, cEy, ccHx, ccHy;
    double complex cEx_trans, cEy_trans, ccHx_trans, ccHy_trans;

    double complex cEx_inc, cEy_inc, ccHx_inc, cc_Hz_inc;

    FILE *Spectrum;
    Spectrum = fopen("Spectrum_ref.txt","w");

    for (m = 0; m < NUM_freq; m++) {
    		Area = 0.0;
    		P_r = 0.0;
        P_r_inc = 0.0;



      //  fprintf(Spectrum,"%e\t%e\t%e\t%e\t%e\n",freq[m],creal(E_Reflected[m][3][3]),cimag(E_Reflected[m][3][3]),creal(H_Reflected[m][3][3]),cimag(H_Reflected[m][3][3]));

    		for (i = 0; i < NCELLX; i++) {
    			dsurf1 = dx;
    			for (k = 0; k < NCELLY; k++) {
    			   dsurf2 = dy;

      				//XZ-NEAR = Reflected
      				if(PBC_CTW == 1 || PBC_CTW == 0) Area += dsurf1*dsurf2;
            //  if(PBC_CTW == 0) Area += 1;


      				// u = ReTFEx_XZ[m][i][0][k];
      				// v = ImTFEx_XZ[m][i][0][k];
              //
      				// cEx = u + I_UNIT*v;
              if(PBC_CTW == 1){
                u = creal(Ey_Reflected[ThreeDMap(m,i,k,NCELLY,NCELLX)]-E_Incident[ThreeDMap(m,i,k,NCELLY,NCELLX)]);
                v = cimag(Ey_Reflected[ThreeDMap(m,i,k,NCELLY,NCELLX)]-E_Incident[ThreeDMap(m,i,k,NCELLY,NCELLX)]);
              }
              else{
                u = creal(Ey_Reflected[ThreeDMap(m,i,k,NCELLY,NCELLX)]);
                v = cimag(Ey_Reflected[ThreeDMap(m,i,k,NCELLY,NCELLX)]);
                u2 = creal(Ex_Reflected[ThreeDMap(m,i,k,NCELLY,NCELLX)]);
                v2 = cimag(Ex_Reflected[ThreeDMap(m,i,k,NCELLY,NCELLX)]);
              }


      				cEy = u + I*v;
              cEx = u2 + I*v2;

              u = creal(E_Incident[ThreeDMap(m,i,k,NCELLY,NCELLX)]);
              v = cimag(E_Incident[ThreeDMap(m,i,k,NCELLY,NCELLX)]);

              cEy_inc = u + I*v;

              if(!PBC_CTW){
                u = creal(Ey_Transmitted[ThreeDMap(m,i,k,NCELLY,NCELLX)]);
                v = cimag(Ey_Transmitted[ThreeDMap(m,i,k,NCELLY,NCELLX)]);
                u2 = creal(Ex_Transmitted[ThreeDMap(m,i,k,NCELLY,NCELLX)]);
                v2 = cimag(Ex_Transmitted[ThreeDMap(m,i,k,NCELLY,NCELLX)]);
              }

              cEy_trans = u + I*v;
              cEx_trans = u2 + I*v2;

              //
      				// u = ReTFHx_XZ[m][i][0][k];// + ReTFHx_xz_n[ThreeDMap(m,i,k,NCELLY,NCELLX)];
      				// v = ImTFHx_XZ[m][i][0][k];// + ImTFHx_xz_n[ThreeDMap(m,i,k,NCELLY,NCELLX)];
              //
      				// ccHx = u - I_UNIT*v;
              if(PBC_CTW == 1){
                u = creal(Hx_Reflected[ThreeDMap(m,i,k,NCELLY,NCELLX)] - H_Incident[ThreeDMap(m,i,k,NCELLY,NCELLX)]);// + ReTFHz_xz_n[ThreeDMap(m,i,k,NCELLY,NCELLX)];
        				v = cimag(Hx_Reflected[ThreeDMap(m,i,k,NCELLY,NCELLX)] - H_Incident[ThreeDMap(m,i,k,NCELLY,NCELLX)]);// + ImTFHz_xz_n[ThreeDMap(m,i,k,NCELLY,NCELLX)];
              }
              else{
                u = creal(Hx_Reflected[ThreeDMap(m,i,k,NCELLY,NCELLX)] );// + ReTFHz_xz_n[ThreeDMap(m,i,k,NCELLY,NCELLX)];
                v = cimag(Hx_Reflected[ThreeDMap(m,i,k,NCELLY,NCELLX)] );// + ImTFHz_xz_n[ThreeDMap(m,i,k,NCELLY,NCELLX)];
                u2 = creal(Hy_Reflected[ThreeDMap(m,i,k,NCELLY,NCELLX)] );// + ReTFHz_xz_n[ThreeDMap(m,i,k,NCELLY,NCELLX)];
                v2 = cimag(Hy_Reflected[ThreeDMap(m,i,k,NCELLY,NCELLX)] );// + ImTFHz_xz_n[ThreeDMap(m,i,k,NCELLY,NCELLX)];
              }


      				ccHx = u - I*v;
              ccHy = u2 - I*v2;
              u = creal(H_Incident[ThreeDMap(m,i,k,NCELLY,NCELLX)]);
              v = cimag(H_Incident[ThreeDMap(m,i,k,NCELLY,NCELLX)]);

              ccHx_inc = u - I*v;

              if(!PBC_CTW){
                u = creal(Hx_Transmitted[ThreeDMap(m,i,k,NCELLY,NCELLX)] );// + ReTFHz_xz_n[ThreeDMap(m,i,k,NCELLY,NCELLX)];
                v = cimag(Hx_Transmitted[ThreeDMap(m,i,k,NCELLY,NCELLX)] );// + ImTFHz_xz_n[ThreeDMap(m,i,k,NCELLY,NCELLX)];
                u2 = creal(Hy_Transmitted[ThreeDMap(m,i,k,NCELLY,NCELLX)] );// + ReTFHz_xz_n[ThreeDMap(m,i,k,NCELLY,NCELLX)];
                v2 = cimag(Hy_Transmitted[ThreeDMap(m,i,k,NCELLY,NCELLX)] );// + ImTFHz_xz_n[ThreeDMap(m,i,k,NCELLY,NCELLX)];
              }
              ccHx_trans = u - I*v;
              ccHy_trans = u2 - I*v2;

      				zz = cEy*ccHx - cEx*ccHy;
              zz_trans = cEy_trans*ccHx_trans - cEx_trans*ccHy_trans;


      				P_r += creal(zz)*dsurf1*dsurf2;
              P_t += creal(zz_trans)*dsurf1*dsurf2;



              zz = cEy_inc*ccHx_inc;

              P_r_inc += creal(zz)*dsurf1*dsurf2;

    				}
    		}

        //printf("%e\n", P_r/Area);
       E_reflected[m] = P_r/Area;
       if(!PBC_CTW) E_transmitted[m] = P_t/Area;
       if(PBC_CTW) E_incident[m] = P_r_inc/Area;

    }
    //Area_loc = Area;
  fclose(Spectrum);
	return;
}

// void SOURCE_SETUP(void){
//
//     int *** mat_matrix_safe;
//     printf("Creating Second MAT Matrix\n");
//
//     mat_matrix_safe = MALLOC3D_int(mat_matrix_safe,NCELLX,NCELLY,NCELLZ);
//
//     double complex ex_tot=0.0;
//     double complex ex_source=0.0;
//     double complex ey_tot=0.0;
//     double complex ey_source=0.0;
//     double complex ez_tot=0.0;
//     double complex ez_source=0.0;
//
//     double complex hx_tot=0.0;
//     double complex hx_source=0.0;
//     double complex hy_tot=0.0;
//     double complex hy_source=0.0;
//     double complex hz_tot=0.0;
//     double complex hz_source=0.0;
//
//
//     int n,w,i,j,k=inc_plane;
//
//     printf("Switching Material Matrices\n");
//
//     for(i=0;i<NCELLX;i++){
//       for(j=0;j<NCELLY;j++){
//         for(k=0;k<NCELLZ;k++){
//
//           mat_matrix_safe[i][j][k] = mat_matrix[i][j][k];
//
//           mat_matrix[i][j][k] = first_medium;
//         }
//       }
//     }
//
//     FILE *Test2;
//
//    Test2 = fopen("Test2.txt","w");
//
//    k = inc_plane;
//
//     for(t=0;t<=Tend_inc;t++)
//     {
//       printf("Source Setup\t%d\n",t);
//
//       UPDATE_B();
//     //  printf("B updated\n");
//       SOURCE_IN();
//
//
//       UPDATE_E();
//     //  printf("E updated\n");
//
//             for(i=0;i<NCELLX;i++){
//               for(j=0;j<NCELLY;j++){
//             //   //  printf("%d\n",Spect_loc);
//             //     Ex_source[i][j][t] = ex[i][j][inc_plane-Spect_loc];
//             //     Ey_source[i][j][t] = ey[i][j][inc_plane-Spect_loc];
//             // //    Ez_source[i][j][t] = ez[i][j][inc_plane-Spect_loc];
//             //
//             //     Hx_source[i][j][t] = hx[i][j][inc_plane-Spect_loc];
//             //     Hy_source[i][j][t] = hy[i][j][inc_plane-Spect_loc];
//             // //    Hz_source[i][j][t] = hz[i][j][inc_plane-Spect_loc];
//
//                 ex_tot = (ex[i][j][inc_plane-Spect_loc])*cexp(I*(i+0.5)*dx*k_x)*cexp(I*j*dy*k_y);
//                 ey_tot = (ey[i][j][inc_plane-Spect_loc])*cexp(I*i*dx*k_x)*cexp(I*(j+0.5)*dy*k_y);
//
//                 hx_tot = (hx[i][j][inc_plane-Spect_loc])*cexp(I*(i)*dx*k_x)*cexp(I*(j+0.5)*dy*k_y);
//                 hy_tot = (hy[i][j][inc_plane-Spect_loc])*cexp(I*(i+0.5)*dx*k_x)*cexp(I*(j)*dy*k_y);
//
//
//                 for(w=0;w<NUM_freq;w++){
//                   if(TEz){
//                     H_Incident[w][i][j] += hx_tot*cexp(-I*2*pi*t*dt*freq[w]);//*cexp(-I*(i+0.5)*dx*k_x)*cexp(-I*(j)*dy*k_y);
//                     E_Incident[w][i][j] += ey_tot*cexp(-I*2*pi*t*dt*freq[w]);//*cexp(-I*(i+0.5)*dx*k_x)*cexp(-I*(j)*dy*k_y);
//                   }
//                   if(TMz){
//                     H_Incident[w][i][j] += hy_tot*cexp(-I*2*pi*t*dt*freq[w]);//*cexp(-I*(i+0.5)*dx*k_x)*cexp(-I*(j)*dy*k_y);
//                     E_Incident[w][i][j] += ex_tot*cexp(-I*2*pi*t*dt*freq[w]);//*cexp(-I*(i+0.5)*dx*k_x)*cexp(-I*(j)*dy*k_y);
//                   }
//                 // if(i == 0 && j ==0) Incident_spec[w] += PULSE(t)*cexp(-I*2*pi*t*dt*freq[w]);
//                 }
//
//
//               }
//             }
//       fprintf(Test2, "%e\t%e\t%e\t%e\t%e\t%e\n",creal(ex[centerx][centery][inc_plane - Test_offset]),creal(ey[centerx][centery][inc_plane - Test_offset]),creal(ez[centerx][centery][inc_plane - Test_offset]),creal(hx[centerx][centery][inc_plane - Test_offset]),creal(hy[centerx][centery][inc_plane - Test_offset]),creal(hz[centerx][centery][inc_plane - Test_offset]));
//
//       if(t%t_skip==0 && t>NUM_freq_inc ){
//             snapshot_count++;
// //          //  SNAPSHOT_2D();
// //            //aux field:
//       if(Snap_in == 1) SNAPSHOT_1D();
// //
// //
// //
//     }
//     }
//
//     FREE3D_Complex(ez,NCELLX,NCELLY);
//     FREE3D_Complex(ey,NCELLX,NCELLY);
//     FREE3D_Complex(ex,NCELLX,NCELLY);
//     FREE3D_Complex(hx,NCELLX,NCELLY);
//     FREE3D_Complex(hy,NCELLX,NCELLY);
//     FREE3D_Complex(hz,NCELLX,NCELLY);
//
//     FREE3D_Complex(psi_Ex_y_N,NCELLX,NcpmlY+1);
//     FREE3D_Complex(psi_Ex_z_N,NCELLX,NCELLY);
//     FREE3D_Complex(psi_Ey_x_N,NcpmlX+1,NCELLY);
//     FREE3D_Complex(psi_Ey_z_N,NCELLX,NCELLY);
//     FREE3D_Complex(psi_Ez_y_N,NCELLX,NcpmlY+1);
//     FREE3D_Complex(psi_Ez_x_N,NcpmlX+1,NCELLY);
//     FREE3D_Complex(psi_Hx_z_N,NCELLX,NCELLY);
//     FREE3D_Complex(psi_Hx_y_N,NCELLX,NcpmlY);
//     FREE3D_Complex(psi_Hy_x_N,NcpmlX,NCELLY);
//     FREE3D_Complex(psi_Hy_z_N,NCELLX,NCELLY);
//     FREE3D_Complex(psi_Hz_x_N,NcpmlX,NCELLY);
//     FREE3D_Complex(psi_Hz_y_N,NCELLX,NcpmlY);
//
//     FREE3D_Complex(psi_Ex_y_F,NCELLX,NcpmlY+1);
//     FREE3D_Complex(psi_Ex_z_F,NCELLX,NCELLY);
//     FREE3D_Complex(psi_Ey_x_F,NcpmlX+1,NCELLY);
//     FREE3D_Complex(psi_Ey_z_F,NCELLX,NCELLY);
//     FREE3D_Complex(psi_Ez_y_F,NCELLX,NcpmlY+1);
//     FREE3D_Complex(psi_Ez_x_F,NcpmlX+1,NCELLY);
//     FREE3D_Complex(psi_Hx_z_F,NCELLX,NCELLY);
//     FREE3D_Complex(psi_Hx_y_F,NCELLX,NcpmlY);
//     FREE3D_Complex(psi_Hy_x_F,NcpmlX,NCELLY);
//     FREE3D_Complex(psi_Hy_z_F,NCELLX,NCELLY);
//     FREE3D_Complex(psi_Hz_x_F,NcpmlX,NCELLY);
//     FREE3D_Complex(psi_Hz_y_F,NCELLX,NcpmlY);
//
//     FREE4D_Complex(Pz_cp,NCELLX,NCELLY,NCELLZ);
//     FREE4D_Complex(Py_cp,NCELLX,NCELLY,NCELLZ);
//     FREE4D_Complex(Px_cp,NCELLX,NCELLY,NCELLZ);
//
//     FREE4D_Complex(Pz_cp_n,NCELLX,NCELLY,NCELLZ);
//     FREE4D_Complex(Py_cp_n,NCELLX,NCELLY,NCELLZ);
//     FREE4D_Complex(Px_cp_n,NCELLX,NCELLY,NCELLZ);
//
//     FREE4D_Complex(Pz_cp_n_1,NCELLX,NCELLY,NCELLZ);
//     FREE4D_Complex(Py_cp_n_1,NCELLX,NCELLY,NCELLZ);
//     FREE4D_Complex(Px_cp_n_1,NCELLX,NCELLY,NCELLZ);
//
//     FREE4D_Complex(Px_d,NCELLX,NCELLY,NCELLZ);
//     FREE4D_Complex(Px_d_n,NCELLX,NCELLY,NCELLZ);
//     FREE4D_Complex(Px_d_n_1,NCELLX,NCELLY,NCELLZ);
//
//     FREE4D_Complex(Py_d,NCELLX,NCELLY,NCELLZ);
//     FREE4D_Complex(Py_d_n,NCELLX,NCELLY,NCELLZ);
//     FREE4D_Complex(Py_d_n_1,NCELLX,NCELLY,NCELLZ);
//
//     FREE4D_Complex(Pz_d,NCELLX,NCELLY,NCELLZ);
//     FREE4D_Complex(Pz_d_n,NCELLX,NCELLY,NCELLZ);
//     FREE4D_Complex(Pz_d_n_1,NCELLX,NCELLY,NCELLZ);
//
//     psi_Ex_y_N=MALLOC3D_Complex(psi_Ex_y_N,NCELLX,NcpmlY+1,NCELLZ);
//     psi_Ez_y_N=MALLOC3D_Complex(psi_Ez_y_N,NCELLX,NcpmlY+1,NCELLZ);
//     psi_Ex_y_F=MALLOC3D_Complex(psi_Ex_y_F,NCELLX,NcpmlY+1,NCELLZ);
//     psi_Ez_y_F=MALLOC3D_Complex(psi_Ez_y_F,NCELLX,NcpmlY+1,NCELLZ);
//
//     psi_Ex_z_N=MALLOC3D_Complex(psi_Ex_z_N,NCELLX,NCELLY,NcpmlZ+1);
//     psi_Ey_z_N=MALLOC3D_Complex(psi_Ey_z_N,NCELLX,NCELLY,NcpmlZ+1);
//     psi_Ex_z_F=MALLOC3D_Complex(psi_Ex_z_F,NCELLX,NCELLY,NcpmlZ+1);
//     psi_Ey_z_F=MALLOC3D_Complex(psi_Ey_z_F,NCELLX,NCELLY,NcpmlZ+1);
//
//     psi_Ey_x_N=MALLOC3D_Complex(psi_Ey_x_N,NcpmlX+1,NCELLY,NCELLZ);
//     psi_Ez_x_N=MALLOC3D_Complex(psi_Ez_x_N,NcpmlX+1,NCELLY,NCELLZ);
//     psi_Ey_x_F=MALLOC3D_Complex(psi_Ey_x_F,NcpmlX+1,NCELLY,NCELLZ);
//     psi_Ez_x_F=MALLOC3D_Complex(psi_Ez_x_F,NcpmlX+1,NCELLY,NCELLZ);
//
//     psi_Hx_y_F=MALLOC3D_Complex(psi_Hx_y_F,NCELLX,NcpmlY,NCELLZ);
//     psi_Hz_y_F=MALLOC3D_Complex(psi_Hz_y_F,NCELLX,NcpmlY,NCELLZ);
//     psi_Hx_y_N=MALLOC3D_Complex(psi_Hx_y_N,NCELLX,NcpmlY,NCELLZ);
//     psi_Hz_y_N=MALLOC3D_Complex(psi_Hz_y_N,NCELLX,NcpmlY,NCELLZ);
//
//     psi_Hx_z_F=MALLOC3D_Complex(psi_Hx_z_F,NCELLX,NCELLY,NcpmlZ);
//     psi_Hy_z_F=MALLOC3D_Complex(psi_Hy_z_F,NCELLX,NCELLY,NcpmlZ);
//     psi_Hx_z_N=MALLOC3D_Complex(psi_Hx_z_N,NCELLX,NCELLY,NcpmlZ);
//     psi_Hy_z_N=MALLOC3D_Complex(psi_Hy_z_N,NCELLX,NCELLY,NcpmlZ);
//
//     psi_Hz_x_F=MALLOC3D_Complex(psi_Hz_x_F,NcpmlX,NCELLY,NCELLZ);
//     psi_Hy_x_F=MALLOC3D_Complex(psi_Hy_x_F,NcpmlX,NCELLY,NCELLZ);
//     psi_Hy_x_N=MALLOC3D_Complex(psi_Hy_x_N,NcpmlX,NCELLY,NCELLZ);
//     psi_Hz_x_N=MALLOC3D_Complex(psi_Hz_x_N,NcpmlX,NCELLY,NCELLZ);
//
//
//     ex=MALLOC3D_Complex(ex,NCELLX,NCELLY,NCELLZ);
//     ey=MALLOC3D_Complex(ey,NCELLX,NCELLY,NCELLZ);
//     ez=MALLOC3D_Complex(ez,NCELLX,NCELLY,NCELLZ);
// //    Dx=MALLOC3D_Complex(Dx,NCELLX,NCELLY,NCELLZ);
// //    Dy=MALLOC3D_Complex(Dy,NCELLX,NCELLY,NCELLZ);
// //    Dz=MALLOC3D_Complex(Dz,NCELLX,NCELLY,NCELLZ);
//     hx=MALLOC3D_Complex(hx,NCELLX,NCELLY,NCELLZ);
//     hy=MALLOC3D_Complex(hy,NCELLX,NCELLY,NCELLZ);
//     hz=MALLOC3D_Complex(hz,NCELLX,NCELLY,NCELLZ);
//
//     Px_cp=MALLOC4D_Complex(Px_cp,NCELLX,NCELLY,NCELLZ,N_CP_poles);
//     Px_cp_n=MALLOC4D_Complex(Px_cp_n,NCELLX,NCELLY,NCELLZ,N_CP_poles);
//     Px_cp_n_1=MALLOC4D_Complex(Px_cp_n_1,NCELLX,NCELLY,NCELLZ,N_CP_poles);
//     Py_cp=MALLOC4D_Complex(Py_cp,NCELLX,NCELLY,NCELLZ,N_CP_poles);
//     Py_cp_n=MALLOC4D_Complex(Py_cp_n,NCELLX,NCELLY,NCELLZ,N_CP_poles);
//     Py_cp_n_1=MALLOC4D_Complex(Py_cp_n_1,NCELLX,NCELLY,NCELLZ,N_CP_poles);
//     Pz_cp=MALLOC4D_Complex(Pz_cp,NCELLX,NCELLY,NCELLZ,N_CP_poles);
//     Pz_cp_n=MALLOC4D_Complex(Pz_cp_n,NCELLX,NCELLY,NCELLZ,N_CP_poles);
//     Pz_cp_n_1=MALLOC4D_Complex(Pz_cp_n_1,NCELLX,NCELLY,NCELLZ,N_CP_poles);
//
//     Px_d=MALLOC4D_Complex(Px_d,NCELLX,NCELLY,NCELLZ,N_drude_poles);
//     Px_d_n=MALLOC4D_Complex(Px_d_n,NCELLX,NCELLY,NCELLZ,N_drude_poles);
//     Px_d_n_1=MALLOC4D_Complex(Px_d_n_1,NCELLX,NCELLY,NCELLZ,N_drude_poles);
//     Py_d=MALLOC4D_Complex(Py_d,NCELLX,NCELLY,NCELLZ,N_drude_poles);
//     Py_d_n=MALLOC4D_Complex(Py_d_n,NCELLX,NCELLY,NCELLZ,N_drude_poles);
//     Py_d_n_1=MALLOC4D_Complex(Py_d_n_1,NCELLX,NCELLY,NCELLZ,N_drude_poles);
//     Pz_d=MALLOC4D_Complex(Pz_d,NCELLX,NCELLY,NCELLZ,N_drude_poles);
//     Pz_d_n=MALLOC4D_Complex(Pz_d_n,NCELLX,NCELLY,NCELLZ,N_drude_poles);
//     Pz_d_n_1=MALLOC4D_Complex(Pz_d_n_1,NCELLX,NCELLY,NCELLZ,N_drude_poles);
//
//
//     ex=ZERO_VECTORS3D_Complex(ex,NCELLX,NCELLY,NCELLZ);
//     ey=ZERO_VECTORS3D_Complex(ey,NCELLX,NCELLY,NCELLZ);
//     ez=ZERO_VECTORS3D_Complex(ez,NCELLX,NCELLY,NCELLZ);
//     hx=ZERO_VECTORS3D_Complex(hx,NCELLX,NCELLY,NCELLZ);
//     hy=ZERO_VECTORS3D_Complex(hy,NCELLX,NCELLY,NCELLZ);
//     hz=ZERO_VECTORS3D_Complex(hz,NCELLX,NCELLY,NCELLZ);
//
//  //    for(i=0;i<NCELLX;i++){
//  //      for(j=0;j<NCELLY;j++){
//  //        for(k=0;k<NCELLZ;k++){
//  //          ex[i][j][k] = 0.0;
//  //          ey[i][j][k] = 0.0;
//  //          ez[i][j][k] = 0.0;
//  //          hx[i][j][k] = 0.0;
//  //          hy[i][j][k] = 0.0;
//  //          hz[i][j][k] = 0.0;
//  //
//  //        }
//  //      }
//  //    }
//  //
//  //    for(i=0; i<NCELLX;i++){
//  //        for(j=0;j<NCELLY;j++){
//  //            for(k=0;k<NcpmlZ+1;k++){
//  //
//  //                psi_Ex_z_F[i][j][k]=0.0;
//  //                psi_Ex_z_N[i][j][k]=0.0;
//  //                psi_Ey_z_F[i][j][k]=0.0;
//  //                psi_Ey_z_N[i][j][k]=0.0;
//  //
//  //
//  //        }
//  //      }
//  //    }
//  //
//  // for(i=0; i<NCELLX;i++){
//  //     for(j=0;j<NCELLY;j++){
//  //         for(k=0;k<NcpmlZ;k++){
//  //
//  //                psi_Hx_z_F[i][j][k]=0.0;
//  //                psi_Hx_z_N[i][j][k]=0.0;
//  //                psi_Hy_z_F[i][j][k]=0.0;
//  //                psi_Hy_z_N[i][j][k]=0.0;
//  //
//  //
//  //        }
//  //      }
//  //    }
//
//  for(i=0; i<NCELLX;i++){
//      for(j=0;j<NCELLY;j++){
//          for(k=0;k<NcpmlZ+1;k++){
//
//              psi_Ex_z_F[i][j][k]=0.0;
//              psi_Ex_z_N[i][j][k]=0.0;
//              psi_Ey_z_F[i][j][k]=0.0;
//              psi_Ey_z_N[i][j][k]=0.0;
//
//
//      }
//    }
//  }
//  printf("Here\n");
//
// for(i=0; i<NCELLX;i++){
//   for(j=0;j<NCELLY;j++){
//       for(k=0;k<NcpmlZ;k++){
//
//              psi_Hx_z_F[i][j][k]=0.0;
//              psi_Hx_z_N[i][j][k]=0.0;
//              psi_Hy_z_F[i][j][k]=0.0;
//              psi_Hy_z_N[i][j][k]=0.0;
//
//
//      }
//    }
//  }
//  printf("Here\n");
//
//  for(i=0;i<NCELLX;i++){
//      for(j=0;j<NCELLY;j++){
//          for(k=0;k<NCELLZ;k++){
//              for(n=0;n<N_CP_poles;n++){
//                  Px_cp[i][j][k][n]=0.0;
//                  Px_cp_n[i][j][k][n]=0.0;
//                  Px_cp_n_1[i][j][k][n]=0.0;
//                  Py_cp[i][j][k][n]=0.0;
//                  Py_cp_n[i][j][k][n]=0.0;
//                  Py_cp_n_1[i][j][k][n]=0.0;
//                  Pz_cp[i][j][k][n]=0.0;
//                  Pz_cp_n[i][j][k][n]=0.0;
//                  Pz_cp_n_1[i][j][k][n]=0.0;
//
//                  printf("%d,%d,%d,%d\n",i,j,k,n);
//
//          }
//        }
//      }
//  }
//  printf("Here\n");
//
//  for(i=0;i<NCELLX;i++){
//      for(j=0;j<NCELLY;j++){
//         for(k=0;k<NCELLZ;k++){
//           for(n=0;n<N_drude_poles;n++){
//              Px_d[i][j][k][n]=0.0;
//              Px_d_n[i][j][k][n]=0.0;
//              Px_d_n_1[i][j][k][n]=0.0;
//              Py_d[i][j][k][n]=0.0;
//              Py_d_n[i][j][k][n]=0.0;
//              Py_d_n_1[i][j][k][n]=0.0;
//              Pz_d[i][j][k][n]=0.0;
//              Pz_d_n[i][j][k][n]=0.0;
//              Pz_d_n_1[i][j][k][n]=0.0;
//          }
//        }
//      }
//  }
//  printf("Here\n");
//
//     for(i=0;i<NCELLX;i++){
//       for(j=0;j<NCELLY;j++){
//         for(k=0;k<NCELLZ;k++){
//
//           mat_matrix[i][j][k] = mat_matrix_safe[i][j][k];
//
//         }
//       }
//     }
//     printf("Here\n");
//
//     FILE *Spectrum;
//     Spectrum = fopen("Spectrum.txt","w");
//
//     i=1;
//     j=1;
//
//     for(w=0;w<NUM_freq;w++){
//       fprintf(Spectrum,"%e\t%e\t%e\n",freq[w],cabs(E_Incident[w][1][1]),cabs(H_Incident[w][1][1]));
//     }
//
//     fclose(Spectrum);
//
//     fclose(Test2);
//
// }



__host__ __device__  int ThreeDMap(int i,int j,int k,int SizeZ,int SizeY){
  int num = k + SizeZ*j +SizeY*SizeZ*i;
  return num;
}


__host__ __device__  int FourDMap(int i,int j,int k,int n,int SizeN,int SizeZ,int SizeY){
  int num = n + SizeN*( k + SizeZ*j +SizeY*SizeZ*i);
  return num;
}

__host__ __device__  int TwoDMap(int i,int j,int size){
  int num = j + i*size;
  return num;
}

//
// int ThreeDMap(int i,int j,int k,int SizeZ,int SizeY){
//   int num = k + SizeZ*j +SizeY*SizeZ*i;
//   return num;
// }
//
//
// int FourDMap(int i,int j,int k,int n,int SizeN,int SizeZ,int SizeY){
//   int num = n + SizeN*( k + SizeZ*j +SizeY*SizeZ*i);
//   return num;
// }
//
// int TwoDMap(int i,int j,int size){
//   int num = j + i*size;
//   return num;
// }
