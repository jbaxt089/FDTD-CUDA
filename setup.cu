#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "extern_var.h"
#include <math.h>
#include "define.h"
#include <hip/hip_runtime.h>



void READ_DATA_FILE(void){
  FILE *Input;
  Input=fopen("Input.txt", "r");
  char string[300];
  real lam_0;
//printf("HERE\n");
#ifndef FlOATPRECISION
  fscanf(Input,"%s", &string);
  fscanf(Input,"%d %s",&Tend, &string);
  fscanf(Input,"%lf %s",&dx, &string);
  fscanf(Input,"%lf %s",&dy, &string);
  fscanf(Input,"%lf %s",&dz, &string);
  fscanf(Input,"%d %s",&num_trials, &string);
  fscanf(Input,"%d %s",&min_trials, &string);
  fscanf(Input,"%d %s",&max_trials, &string);
  fscanf(Input,"%d %s",&TE_TM, &string);
  fscanf(Input,"%d %s",&NCELLX, &string);
  fscanf(Input,"%d %s",&NCELLY, &string);
  fscanf(Input,"%d %s",&NCELLZ, &string);
  fscanf(Input,"%d %s",&NcpmlX, &string);
  fscanf(Input,"%d %s",&NcpmlY, &string);
  fscanf(Input,"%d %s",&NcpmlZ, &string);
  fscanf(Input,"%d %s",&inf_disp_slab, &string);
  fscanf(Input,"%d %s",&nano_sphere, &string);
  fscanf(Input,"%lf %s",&nano_sphere_radius, &string);
  fscanf(Input,"%d %s",&material, &string);
  fscanf(Input,"%d %s",&first_medium, &string);
  fscanf(Input,"%d %s",&inc_plane, &string);
  fscanf(Input,"%d %s",&dispersive_slab, &string);
  fscanf(Input,"%d %s", &NUM_freq, &string);
  fscanf(Input,"%d %s", &WL_or_freq, &string);
  fscanf(Input,"%d %s", &NUM_freq_inc,&string);
  fscanf(Input,"%d %s", &Tend_inc, &string);
  fscanf(Input,"%d %s", &Snap_in, &string);
  fscanf(Input,"%d %s", &Freq_start, &string);
  fscanf(Input,"%d %s", &Test_offset, &string);
  fscanf(Input,"%d %s", &Spect_loc, &string);
  fscanf(Input,"%lf %s", &BandWidth, &string);
  fscanf(Input,"%d %s", &NONLOCAL, &string);
  fscanf(Input,"%d %s", &Diverge_Gradient, &string);
  fscanf(Input,"%d %s", &t_skip, &string);
  fscanf(Input,"%d %s", &x_skip, &string);
  fscanf(Input,"%d %s", &y_skip, &string);
  fscanf(Input,"%d %s", &z_skip, &string);
  fscanf(Input,"%lf %s", &Nonlocalend, &string);
  fscanf(Input,"%lf %s", &MAX_AMP, &string);
  fscanf(Input,"%lf %s", &TimeFactor, &string);
  fscanf(Input,"%lf %s", &lam_0, &string);
  fscanf(Input,"%d %s", &StaticField, &string);

  printf("%d\n",StaticField);

  f_0 = C0/lam_0;
  printf("Domain Size: %d,%d,%d\n", NCELLX,NCELLY,NCELLZ);
  printf("Sim Time: %d\n", Tend);
  printf("dx: %e\tdy: %e\tdz: %e\n",dx,dy,dz);
  if(inf_disp_slab && material == 2) printf("Infinite dispersive slab: material: Silver\n");
  if(inf_disp_slab && material == 3) printf("Infinite dispersive slab: material: Gold\n");
  if(!inf_disp_slab) printf("No Dispersive Slab\n");
  printf("Number of Trials: %d, Min Trial: %d, Max Trial %d\n", num_trials,min_trials,max_trials);
  printf("Freq Count = %d\n",NUM_freq);
  printf("Inc Freq Count = %d\n",NUM_freq_inc);
  printf("Inc Tend: %d\n",Tend_inc);
  if(WL_or_freq == 1) printf("Wavelength Plot\n");
  else printf("Frequency Plot\n");
  // printf("Sphere Radius: %e\n",nano_sphere_radius);
#else
fscanf(Input,"%s", &string);
fscanf(Input,"%d %s",&Tend, &string);
fscanf(Input,"%f %s",&dx, &string);
fscanf(Input,"%f %s",&dy, &string);
fscanf(Input,"%f %s",&dz, &string);
fscanf(Input,"%d %s",&num_trials, &string);
fscanf(Input,"%d %s",&min_trials, &string);
fscanf(Input,"%d %s",&max_trials, &string);
fscanf(Input,"%d %s",&TE_TM, &string);
fscanf(Input,"%d %s",&NCELLX, &string);
fscanf(Input,"%d %s",&NCELLY, &string);
fscanf(Input,"%d %s",&NCELLZ, &string);
fscanf(Input,"%d %s",&NcpmlX, &string);
fscanf(Input,"%d %s",&NcpmlY, &string);
fscanf(Input,"%d %s",&NcpmlZ, &string);
fscanf(Input,"%d %s",&inf_disp_slab, &string);
fscanf(Input,"%d %s",&nano_sphere, &string);
fscanf(Input,"%f %s",&nano_sphere_radius, &string);
fscanf(Input,"%d %s",&material, &string);
fscanf(Input,"%d %s",&first_medium, &string);
fscanf(Input,"%d %s",&inc_plane, &string);
fscanf(Input,"%d %s",&dispersive_slab, &string);
fscanf(Input,"%d %s", &NUM_freq, &string);
fscanf(Input,"%d %s", &WL_or_freq, &string);
fscanf(Input,"%d %s", &NUM_freq_inc,&string);
fscanf(Input,"%d %s", &Tend_inc, &string);
fscanf(Input,"%d %s", &Snap_in, &string);
fscanf(Input,"%d %s", &Freq_start, &string);
fscanf(Input,"%d %s", &Test_offset, &string);
fscanf(Input,"%d %s", &Spect_loc, &string);
fscanf(Input,"%f %s", &BandWidth, &string);
fscanf(Input,"%d %s", &NONLOCAL, &string);
fscanf(Input,"%d %s", &Diverge_Gradient, &string);
fscanf(Input,"%d %s", &t_skip, &string);
fscanf(Input,"%d %s", &x_skip, &string);
fscanf(Input,"%d %s", &y_skip, &string);
fscanf(Input,"%d %s", &z_skip, &string);
fscanf(Input,"%f %s", &Nonlocalend, &string);
fscanf(Input,"%lf %s", &MAX_AMP, &string);
fscanf(Input,"%lf %s", &TimeFactor, &string);
fscanf(Input,"%lf %s", &lam_0, &string);
fscanf(Input,"%d %s", &StaticField, &string);
#endif

  if(Diverge_Gradient == 1)
{
  Laplacian =0;
}
else if(Diverge_Gradient==0) Laplacian =1;
  //printf("HERE2\n");

  fclose(Input);

  //printf("HERE3\n");


}

void MATERIAL_MATRIX(void){
hipError_t err;
  mat_matrix = MALLOC3D_int(mat_matrix, NCELLX,NCELLY,NCELLZ);
  mat_matrixX = MALLOC3D_int(mat_matrixX, NCELLX,NCELLY,NCELLZ);
  mat_matrixY = MALLOC3D_int(mat_matrixY, NCELLX,NCELLY,NCELLZ);
  mat_matrixZ = MALLOC3D_int(mat_matrixZ, NCELLX,NCELLY,NCELLZ);

err=  hipMalloc(&mat_matrixdev,NCELLX*NCELLY*NCELLZ*sizeof(int));
err=  hipMalloc(&mat_matrixXdev,NCELLX*NCELLY*NCELLZ*sizeof(int));
err=   hipMalloc(&mat_matrixYdev,NCELLX*NCELLY*NCELLZ*sizeof(int));
err=  hipMalloc(&mat_matrixZdev,NCELLX*NCELLY*NCELLZ*sizeof(int));
if( hipSuccess != err)
{
    printf( "Cuda error: %s.\n",hipGetErrorString( err) );
    exit(-1);
}

  int i,j,k;

  printf("Material: %d\n",material);

  if(inf_disp_slab==1){
    if(PBC_CTW==1) dispersive_slab=inc_plane+dispersive_slab;
    if(PBC_CTW==0) dispersive_slab=dispersive_slab + NtfsfY;
    for(i=0; i<NCELLX; i++){
      for(j=0;j<NCELLY; j++){
        for(k=0; k<NCELLZ; k++){

            if(k>=dispersive_slab){
              mat_matrix[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
              mat_matrixX[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
              mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
              mat_matrixZ[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
            }
            else{
              mat_matrix[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = first_medium;
              mat_matrixX[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = first_medium;
              mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = first_medium;
              mat_matrixZ[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = first_medium;


            }
        }
      }
    }
  }
  else if(inf_disp_slab == 2){
    if(PBC_CTW==1) dispersive_slab=inc_plane+dispersive_slab;
    if(PBC_CTW==0) dispersive_slab=dispersive_slab + NtfsfY;
    for(i=0; i<NCELLX; i++){
      for(j=0;j<NCELLY; j++){
        for(k=0; k<NCELLZ; k++){
            if(k>=dispersive_slab && k <(dispersive_slab + (int)(nano_sphere_radius/dz))){
              mat_matrix[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
              mat_matrixX[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
              mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
              mat_matrixZ[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;

            }
            else{
              mat_matrix[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = first_medium;
              mat_matrixX[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = first_medium;
              mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = first_medium;
              mat_matrixZ[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = first_medium;

            }
        }
      }
    }
  }

  else{
    for(i=0; i<NCELLX; i++){
      for(j=0;j<NCELLY; j++){
        for(k=0; k<NCELLZ; k++){
              mat_matrix[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = first_medium;
              mat_matrixX[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = first_medium;
              mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = first_medium;
              mat_matrixZ[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = first_medium;
        }
      }
    }
  }

  if(nano_sphere==1){
    int nano_sphere_radius_int,center_point;
    nano_sphere_radius_int = (int)(nano_sphere_radius/dx);
    double center_point_z;
    if(Periodic_XY)  center_point_z = dispersive_slab - nano_sphere_radius_int;
    else  center_point_z = NCELLZ/2;
    double center_point_x = NCELLX/2;
    double center_point_y = NCELLY/2;
    printf("%f\t%f\t%f\n",center_point_x,center_point_y,center_point_z);
    for(i=0; i<NCELLX; i++){
      for(j=0;j<NCELLY; j++){
        for(k=0; k<NCELLZ; k++){
          if(pow((i-center_point_x)*(i-center_point_x)+(j-center_point_y)*(j-center_point_y)+(k-center_point_z)*(k-center_point_z),0.5) < nano_sphere_radius_int){
            mat_matrix[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
          }
          if(pow((i+0.5-center_point_x)*(i+0.5-center_point_x)+(j-center_point_y)*(j-center_point_y)+(k-center_point_z)*(k-center_point_z),0.5) < nano_sphere_radius_int){
            mat_matrixX[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
          }

          if(pow((i-center_point_x)*(i-center_point_x)+(j+0.5-center_point_y)*(j+0.5-center_point_y)+(k-center_point_z)*(k-center_point_z),0.5) < nano_sphere_radius_int){
            mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
          }

          if(pow((i-center_point_x)*(i-center_point_x)+(j-center_point_y)*(j-center_point_y)+(k+0.5-center_point_z)*(k+0.5-center_point_z),0.5) < nano_sphere_radius_int){
            mat_matrixZ[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
          }
        }
      }
    }
  }
  else if(nano_sphere==2){
    for(i=0; i<NCELLX; i++){
      for(j=0;j<NCELLY; j++){
        for(k=0; k<NCELLZ; k++){
          double center_point_z = dispersive_slab;
          double center_point_x = NCELLX/2;
          double center_point_y = NCELLY/2;
          if(i<center_point_x+2 && i>center_point_x-2 && j<center_point_y+2 && j>center_point_y-2 && k>center_point_z-20 ){
            mat_matrix[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
          }
        }
      }
    }
  }

  if(nano_sphere==3){
    int nano_sphere_radius_int,center_point;
    nano_sphere_radius_int = (int)(nano_sphere_radius/dx);
    double center_point_z;
    if(Periodic_XY)  center_point_z = dispersive_slab - nano_sphere_radius_int;
    else  center_point_z = NCELLZ/2;
    double center_point_x = NCELLX/2;
    double center_point_y = NCELLY/2;
    printf("%f\t%f\t%f\n",center_point_x,center_point_y,center_point_z);
    for(i=0; i<NCELLX; i++){
      for(j=0;j<NCELLY; j++){
        for(k=0; k<NCELLZ; k++){
          if((i-center_point_x)*(i-center_point_x) < nano_sphere_radius_int*nano_sphere_radius_int && (j-center_point_y)*(j-center_point_y) < nano_sphere_radius_int*nano_sphere_radius_int && (k-center_point_z)*(k-center_point_z) < nano_sphere_radius_int*nano_sphere_radius_int){
            mat_matrix[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
          }
          if((i+0.5-center_point_x)*(i+0.5-center_point_x) < nano_sphere_radius_int*nano_sphere_radius_int&& (j-center_point_y)*(j-center_point_y) < nano_sphere_radius_int*nano_sphere_radius_int && (k-center_point_z)*(k-center_point_z) < nano_sphere_radius_int*nano_sphere_radius_int){
            mat_matrixX[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
          }

          if((i-center_point_x)*(i-center_point_x)  < nano_sphere_radius_int*nano_sphere_radius_int && (j+0.5-center_point_y)*(j+0.5-center_point_y) < nano_sphere_radius_int*nano_sphere_radius_int && (k-center_point_z)*(k-center_point_z)< nano_sphere_radius_int*nano_sphere_radius_int){
            mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
          }

          if((i-center_point_x)*(i-center_point_x) < nano_sphere_radius_int*nano_sphere_radius_int && (j-center_point_y)*(j-center_point_y)  < nano_sphere_radius_int*nano_sphere_radius_int&& (k+0.5-center_point_z)*(k+0.5-center_point_z)  < nano_sphere_radius_int*nano_sphere_radius_int){
            mat_matrixZ[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = material;
          }
        }
      }
    }
  }

  //Output MaterialPlot
  FILE *MaterialPlot;
  MaterialPlot = fopen("Material_XZ.txt","w");
  j = (int)NCELLY/2;
  for(i=0;i<NCELLX;i++){
      for(k=0;k<NCELLZ;k++){
        fprintf(MaterialPlot,"%d\t",mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)]);
      }
      fprintf(MaterialPlot,"\n");
    }
    fclose(MaterialPlot);

    MaterialPlot = fopen("Material_XY.txt","w");
    k = (int)NCELLZ/2;
    for(i=0;i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
          fprintf(MaterialPlot,"%d\t",mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)]);
        }
        fprintf(MaterialPlot,"\n");
      }
      fclose(MaterialPlot);


      MaterialPlot = fopen("Material_YZ.txt","w");
      i = (int)NCELLX/2;

          for(j=0;j<NCELLY;j++){
            for(k=0;k<NCELLZ;k++){
            fprintf(MaterialPlot,"%d\t",mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)]);
          }
          fprintf(MaterialPlot,"\n");
        }
        fclose(MaterialPlot);





    Nonlocalend_int = dispersive_slab + ceil(Nonlocalend/dz);
    printf("Nonlocal Barrier: %d\n",Nonlocalend_int);


    hipMemcpy(mat_matrixdev,mat_matrix,NCELLX*NCELLY*NCELLZ*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(mat_matrixXdev,mat_matrixX,NCELLX*NCELLY*NCELLZ*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(mat_matrixYdev,mat_matrixY,NCELLX*NCELLY*NCELLZ*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(mat_matrixZdev,mat_matrixZ,NCELLX*NCELLY*NCELLZ*sizeof(int),hipMemcpyHostToDevice);

}



//Sets variables
void SETUP_CONST(){

    source_end=1230;
    inc_Length=NCELLZ*6;

    int i;
    //only if dx=dy=dz
    if(NCELLX==1 || NCELLY==1 || NCELLZ==1){
        dt=0.9*dy/(C0*pow(2,0.5));
    }
    else{

        dt=0.5*dy/C0/(real)TimeFactor;
         printf("dt = %e\n",dt);
    }

    //constants
    z0=Z0;
  //  z0 = 1.0;
    ep0=EP0;
    mu0=MU0;
    c0=C0;
    pi=PI;
    me = ME;
    e0 = E0;
    //CPML constants
    // NcpmlX=0;
    // NcpmlY=0;
    // NcpmlZ=100;
    //add the CPML cells to the regular domain
    NCELLX+=2*NcpmlX;
    NCELLY+=2*NcpmlY;
    NCELLZ+=2*NcpmlZ;

    cpml_N_X=NcpmlX;
    cpml_F_X=NCELLX-NcpmlX;
    cpml_N_Y=NcpmlY;
    cpml_F_Y=NCELLY-NcpmlY;
    cpml_N_Z=NcpmlZ;
    cpml_F_Z=NCELLZ-NcpmlZ;

    //exceitation plane
    inc_plane=NcpmlZ+inc_plane;
    //inc_plane=NCELLZ/2;
    NCELLZ+=1;
    NCELLY+=1;
    NCELLX+=1;
    cpml_x_lim=NCELLX-1;
    cpml_y_lim=NCELLY-1;
    cpml_z_lim=NCELLZ-1;
    //Lateral PML Boundaries
    Absorption = 0;
    Scattering = 0;

    if(Periodic_XY){
        NCELLX-=1;
        NCELLY-=1;
        cpml_x_lim=NCELLX;
        cpml_y_lim=NCELLY;
    }

    else if(Periodic_XZ){
        NCELLX-=1;
        NCELLZ-=1;
        cpml_x_lim=NCELLX;
        cpml_z_lim=NCELLZ;
    }

    else if(Periodic_YZ){
        NCELLY-=1;
        NCELLZ-=1;
        cpml_y_lim=NCELLY;
        cpml_z_lim=NCELLZ;
    }
    else{
      Absorption = 1;
      Scattering = 1;
    }

    if(PBC_CTW == 0)
    {
    NtfsfX=NtfsfY=NtfsfZ=10;
    if(Periodic_XY) NtfsfX = NtfsfY = 0;
    NtfsfX+=NcpmlX;
    NtfsfY+=NcpmlY;
    NtfsfZ+=NcpmlZ;
    printf("TFSF:%d\t%d\t%d\n",NtfsfX,NtfsfY,NtfsfZ);
    if(Absorption && Scattering){
      XSTARTAbs = NtfsfX + 5;
      YSTARTAbs = NtfsfY + 5;
      ZSTARTAbs = NtfsfZ + 5;
      XENDAbs = NCELLX-NtfsfX - 5;
      YENDAbs = NCELLY-NtfsfY - 5;
      ZENDAbs = NCELLZ-NtfsfZ - 5;

      printf("%d\t%d\n",XENDAbs ,XSTARTAbs);
      printf("%d\t%d\n",YENDAbs , YSTARTAbs);
      printf("%d\t%d\n",ZENDAbs , ZSTARTAbs);


      XSTARTSca = NtfsfX - 5;
      YSTARTSca = NtfsfY - 5;
      ZSTARTSca = NtfsfZ - 5;
      XENDSca = NCELLX-NtfsfX + 5;
      YENDSca = NCELLY-NtfsfY + 5;
      ZENDSca = NCELLZ-NtfsfZ + 5;

      XNEARAbs = XSTARTAbs;
      XFARAbs = XENDAbs-1;
      YNEARAbs = YSTARTAbs;
      YFARAbs = YENDAbs-1;
      ZNEARAbs = ZSTARTAbs;
      ZFARAbs = ZENDAbs-1;

      XNEARSca = XSTARTSca;
      XFARSca = XENDSca-1;
      YNEARSca = YSTARTSca;
      YFARSca = YENDSca-1;
      ZNEARSca = ZSTARTSca;
      ZFARSca = ZENDSca-1;

      printf("%d\t%d\n",XENDSca ,XSTARTSca);
      printf("%d\t%d\n",YENDSca , YSTARTSca);
      printf("%d\t%d\n",ZENDSca , ZSTARTSca);

    }
    if(Scattering){

    }
    }
    else if(PBC_CTW == 1){
      NtfsfX=NtfsfY=NtfsfZ=0;
    }
    //polynomial grading exponent (recommended 3 or 4)
    cpml_exp=3;
    //k_max
  //  max_stretch_factor_x=max_stretch_factor_y=max_stretch_factor_z=15;
    max_stretch_factor_x=max_stretch_factor_y=max_stretch_factor_z=5;

    //sigma_max (see eqn 7.66)
    // max_sigma_cpml_x=0.8*(cpml_exp+1)/(Z0*dx*pow(10,0.5));
    // max_sigma_cpml_y=0.8*(cpml_exp+1)/(Z0*dy*pow(10,0.5));
    // max_sigma_cpml_z=0.8*(cpml_exp+1)/(Z0*dz*pow(10,0.5));
    max_sigma_cpml_x=0.8*(cpml_exp+1)/(Z0*dx);
    max_sigma_cpml_y=0.8*(cpml_exp+1)/(Z0*dy);
    max_sigma_cpml_z=0.8*(cpml_exp+1)/(Z0*dz);


    //Suggested by CONVOLUTION PML CPML : ANEFFICIENT FDTD IMPLEMENTATION OFTHE CFS
    //� PML FOR ARBITRARY MEDIA by Roden et. Al
    max_alpha_x=max_alpha_y=max_alpha_z=0.05;

    //See eqn (7.79)
    exp_alpha_x=exp_alpha_y=exp_alpha_z=1;
     //PBC parameters
    period_x=(NCELLX-1)*dx;
    period_y=(NCELLY-1)*dy;

    //Fourier Analysis
    e_reflected=e_incident=0.0;


    first_medium_max = 1;
}

//Everything nessecary for CPML to work
void SETUP_CPML(void){

    int i;

    //set the ked_ and khd_ vectors to d_ (will further update later)
    for(i=0;i<NCELLX;i++){
        kedx[i]=dx;
        khdx[i]=dx;
    }
    for(i=0;i<NCELLY;i++){
        kedy[i]=dy;
        khdy[i]=dy;
    }
    for(i=0;i<NCELLZ;i++){
        kedz[i]=dz;
        khdz[i]=dz;
    }

   SETUP_CPML_X();
   printf("Setting Up CPMPLZ\n");
   SETUP_CPML_Z();
   printf("Done\n");
   SETUP_CPML_Y();

    return;
}

void SETUP_CPML_X(void){

    int i,j,k;
    real S,A,K;

    for(i=0;i<NcpmlX+1;i++){
        for(j=0;j<NCELLY;j++){
            for(k=0;k<NCELLZ;k++){

                psi_Ey_x_F[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=0;
                psi_Ey_x_N[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=0;
                psi_Ez_x_F[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=0;
                psi_Ez_x_N[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=0;

        }
      }
    }

 for(i=0; i<NcpmlX;i++){
     for(j=0;j<NCELLY;j++){
         for(k=0;k<NCELLZ;k++){

                psi_Hy_x_F[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=0;
                psi_Hy_x_N[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=0;
                psi_Hz_x_F[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=0;
                psi_Hz_x_N[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=0;


        }
      }
    }

    hipMemcpy(psi_Ey_x_Fdev,psi_Ey_x_F,(NcpmlX+1)*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Ey_x_Ndev,psi_Ey_x_N,(NcpmlX+1)*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Ez_x_Fdev,psi_Ez_x_F,(NcpmlX+1)*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Ez_x_Ndev,psi_Ez_x_N,(NcpmlX+1)*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);


    hipMemcpy(psi_Hy_x_Fdev,psi_Hy_x_F,(NcpmlX)*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Hy_x_Ndev,psi_Hy_x_N,(NcpmlX)*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Hz_x_Fdev,psi_Hz_x_F,(NcpmlX)*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Hz_x_Ndev,psi_Hz_x_N,(NcpmlX)*NCELLY*NCELLZ*sizeof(real),hipMemcpyHostToDevice);

    //Y COMPONENT for E field
    for(i=0;i<NcpmlX+1;i++){
      //PML grading in the x direction (eqn 7.60a)
      S=pow(((real)(NcpmlX-i)/((real)(NcpmlX))),cpml_exp)*max_sigma_cpml_x;
      // equation (7.79)
      A=max_alpha_x*pow((real)i/((real)(NcpmlX)),exp_alpha_x);
      // (eqn 7.60b)
      K=1+(max_stretch_factor_x-1)*pow((real)(NcpmlX-i)/((real)(NcpmlX)),cpml_exp);
      // (eqn 7.102) part of the c_w term. Also see (7.114a)
      be_x_N[i]=exp(-(S/K+A)*(dt/EP0));
      //7.114(b)

    if(K*A+S==0.0){
        ce_x_N[i]=0.0;
      }
      else{
        ce_x_N[i]=S*(be_x_N[i]-1)/(K*(S+K*A));
      }
      kedx[i]=kedx[i]*K;
    }

    for(i=0;i<NcpmlX+1;i++){
      //PML grading in the x direction (eqn 7.60a)
      S=pow(((real)(i)/((real)(NcpmlX))),cpml_exp)*(max_sigma_cpml_x);
      // equation (7.79)
      A=max_alpha_x*pow((real)(NcpmlX-i)/((real)(NcpmlX)),exp_alpha_x);
      // (eqn 7.60b)
      K=1+(max_stretch_factor_x-1)*pow((real)(i)/((real)(NcpmlX)),cpml_exp);
      // (eqn 7.102) part of the c_w term. Also see (7.114a)
      be_x_F[i]=exp(-(S/K+A)*(dt/EP0));
      //7.114(b)
      if(K*A+S==0.0){
        ce_x_F[i]=0.0;
      }
      else{
        ce_x_F[i]=S*(be_x_F[i]-1)/(K*(S+K*A));
      }

      kedx[i+cpml_F_X]=kedx[i+cpml_F_X]*K;
      //printf("%f\t%f\t%f\n",be_z_F[i],ce_z_F[i],kedz[i+cpml_F_Z]);
    }

    //Y COMPONENT for H-field
    for(i=0;i<NcpmlX;i++){
      //PML grading in the x direction (eqn 7.60a)
      S=pow((((real)NcpmlX-(real)i-0.5)/((real)NcpmlX)),cpml_exp)*max_sigma_cpml_x;
      // equation (7.79)
      A=max_alpha_x*pow(((real)i+0.5)/((real)NcpmlX),exp_alpha_x);
      // (eqn 7.60b)
      K=1+(max_stretch_factor_x-1)*pow(((real)NcpmlX-(real)i-0.5)/((real)NcpmlX),cpml_exp);
      // (eqn 7.102) part of the c_w term. Also see (7.114a)
      bh_x_N[i]=exp(-(S/K+A)*(dt/EP0));
      //7.114(b)
      if(K*A+S==0.0){
        ch_x_N[i]=0;
      }
      else{
        ch_x_N[i]=S*(bh_x_N[i]-1)/(K*(S+K*A));
      }

      khdx[i]=khdx[i]*K;
    }
    /*bh_y_N[NcpmlY-1]=be_y_N[NcpmlY-1];
    ch_y_N[NcpmlY-1]=ce_y_N[NcpmlY-1];
    khdy[NcpmlY-1]=kedy[NcpmlY-1];*/

    for(i=0;i<NcpmlX;i++){
      //PML grading in the x direction (eqn 7.60a)
      S=pow((((real)i+0.5)/((real)NcpmlX)),cpml_exp)*(max_sigma_cpml_x);
      // equation (7.79)
      A=max_alpha_x*pow(((real)NcpmlX-(real)i-0.5)/((real)NcpmlX),exp_alpha_x);
      // (eqn 7.60b)
      K=1+(max_stretch_factor_x-1)*pow(((real)i+0.5)/((real)NcpmlX),cpml_exp);
      // (eqn 7.102) part of the c_w term. Also see (7.114a)
      bh_x_F[i]=exp(-(S/K+A)*(dt/EP0));
      //7.114(b)

    if(K*A+S==0.0){
        ch_x_F[i]=0;
      }
      else{
        ch_x_F[i]=S*(bh_x_F[i]-1)/(K*(S+K*A));
      }

      khdx[i+cpml_F_X]=khdx[i+cpml_F_X]*K;
    }
    /*bh_y_F[NcpmlY-1]=be_y_F[NcpmlY-1];
    ch_y_F[NcpmlY-1]=ce_y_F[NcpmlY-1];
    khdy[cpml_F_Y]=kedy[cpml_F_Y];*/
    hipMemcpy(kedxdev,kedx,NCELLX*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(khdxdev,khdx,NCELLX*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ce_x_Ndev,ce_x_N,(NcpmlX+1)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ce_x_Fdev,ce_x_F,(NcpmlX+1)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(be_x_Ndev,be_x_N,(NcpmlX+1)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(be_x_Fdev,be_x_F,(NcpmlX+1)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ch_x_Ndev,ch_x_N,(NcpmlX)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ch_x_Fdev,ch_x_F,(NcpmlX)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(bh_x_Ndev,bh_x_N,(NcpmlX)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(bh_x_Fdev,bh_z_F,(NcpmlX)*sizeof(real),hipMemcpyHostToDevice);
}

void SETUP_CPML_Y(void){

    int i,j,k;
    real S,A,K;

    for(i=0; i<NCELLX;i++){
        for(j=0;j<NcpmlY+1;j++){
            for(k=0;k<NCELLZ;k++){

                psi_Ex_y_F[ThreeDMap(i,j,k,NCELLZ,NcpmlY+1)]=0;
                psi_Ex_y_N[ThreeDMap(i,j,k,NCELLZ,NcpmlY+1)]=0;
                psi_Ez_y_F[ThreeDMap(i,j,k,NCELLZ,NcpmlY+1)]=0;
                psi_Ez_y_N[ThreeDMap(i,j,k,NCELLZ,NcpmlY+1)]=0;

        }
      }
    }

 for(i=0; i<NCELLX;i++){
     for(j=0;j<NcpmlY;j++){
         for(k=0;k<NCELLZ;k++){

                psi_Hx_y_F[ThreeDMap(i,j,k,NCELLZ,NcpmlY)]=0;
                psi_Hx_y_N[ThreeDMap(i,j,k,NCELLZ,NcpmlY)]=0;
                psi_Hz_y_F[ThreeDMap(i,j,k,NCELLZ,NcpmlY)]=0;
                psi_Hz_y_N[ThreeDMap(i,j,k,NCELLZ,NcpmlY)]=0;

        }
      }
    }
    hipMemcpy(psi_Ex_y_Fdev,psi_Ex_y_F,(NcpmlY+1)*NCELLX*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Ex_y_Ndev,psi_Ex_y_N,(NcpmlY+1)*NCELLX*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Ez_y_Fdev,psi_Ez_y_F,(NcpmlY+1)*NCELLX*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Ez_y_Ndev,psi_Ez_y_N,(NcpmlY+1)*NCELLX*NCELLZ*sizeof(real),hipMemcpyHostToDevice);


    hipMemcpy(psi_Hx_y_Fdev,psi_Hx_y_F,(NcpmlY)*NCELLX*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Hx_y_Ndev,psi_Hx_y_N,(NcpmlY)*NCELLX*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Hz_y_Fdev,psi_Hz_y_F,(NcpmlY)*NCELLX*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Hz_y_Ndev,psi_Hz_y_N,(NcpmlY)*NCELLX*NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    //Y COMPONENT for E field
    for(i=0;i<NcpmlY+1;i++){
      //PML grading in the x direction (eqn 7.60a)
      S=pow(((real)(NcpmlY-i)/((real)(NcpmlY))),cpml_exp)*max_sigma_cpml_y;
      // equation (7.79)
      A=max_alpha_y*pow((real)i/((real)(NcpmlY)),exp_alpha_y);
      // (eqn 7.60b)
      K=1+(max_stretch_factor_y-1)*pow((real)(NcpmlY-i)/((real)(NcpmlY)),cpml_exp);
      // (eqn 7.102) part of the c_w term. Also see (7.114a)
      be_y_N[i]=exp(-(S/K+A)*(dt/EP0));
      //7.114(b)

    if(K*A+S==0.0){
        ce_y_N[i]=0.0;
      }
      else{
        ce_y_N[i]=S*(be_y_N[i]-1)/(K*(S+K*A));
      }
      kedy[i]=kedy[i]*K;
    }

    for(i=0;i<NcpmlY+1;i++){
      //PML grading in the x direction (eqn 7.60a)
      S=pow(((real)(i)/((real)(NcpmlY))),cpml_exp)*(max_sigma_cpml_y);
      // equation (7.79)
      A=max_alpha_y*pow((real)(NcpmlY-i)/((real)(NcpmlY)),exp_alpha_y);
      // (eqn 7.60b)
      K=1+(max_stretch_factor_y-1)*pow((real)(i)/((real)(NcpmlY)),cpml_exp);
      // (eqn 7.102) part of the c_w term. Also see (7.114a)
      be_y_F[i]=exp(-(S/K+A)*(dt/EP0));
      //7.114(b)
      if(K*A+S==0.0){
        ce_y_F[i]=0.0;
      }
      else{
        ce_y_F[i]=S*(be_y_F[i]-1)/(K*(S+K*A));
      }

      kedy[i+cpml_F_Y]=kedy[i+cpml_F_Y]*K;
      //printf("%f\t%f\t%f\n",be_z_F[i],ce_z_F[i],kedz[i+cpml_F_Z]);
    }

    //Y COMPONENT for H-field
    for(i=0;i<NcpmlY;i++){
      //PML grading in the x direction (eqn 7.60a)
      S=pow((((real)NcpmlY-(real)i-0.5)/((real)NcpmlY)),cpml_exp)*max_sigma_cpml_y;
      // equation (7.79)
      A=max_alpha_y*pow(((real)i+0.5)/((real)NcpmlY),exp_alpha_y);
      // (eqn 7.60b)
      K=1+(max_stretch_factor_y-1)*pow(((real)NcpmlY-(real)i-0.5)/((real)NcpmlY),cpml_exp);
      // (eqn 7.102) part of the c_w term. Also see (7.114a)
      bh_y_N[i]=exp(-(S/K+A)*(dt/EP0));
      //7.114(b)
      if(K*A+S==0.0){
        ch_y_N[i]=0;
      }
      else{
        ch_y_N[i]=S*(bh_y_N[i]-1)/(K*(S+K*A));
      }

      khdy[i]=khdy[i]*K;
    }
    /*bh_y_N[NcpmlY-1]=be_y_N[NcpmlY-1];
    ch_y_N[NcpmlY-1]=ce_y_N[NcpmlY-1];
    khdy[NcpmlY-1]=kedy[NcpmlY-1];*/

    for(i=0;i<NcpmlY;i++){
      //PML grading in the x direction (eqn 7.60a)
      S=pow((((real)i+0.5)/((real)NcpmlY)),cpml_exp)*(max_sigma_cpml_y);
      // equation (7.79)
      A=max_alpha_y*pow(((real)NcpmlY-(real)i-0.5)/((real)NcpmlY),exp_alpha_y);
      // (eqn 7.60b)
      K=1+(max_stretch_factor_y-1)*pow(((real)i+0.5)/((real)NcpmlY),cpml_exp);
      // (eqn 7.102) part of the c_w term. Also see (7.114a)
      bh_y_F[i]=exp(-(S/K+A)*(dt/EP0));
      //7.114(b)

    if(K*A+S==0.0){
        ch_y_F[i]=0;
      }
      else{
        ch_y_F[i]=S*(bh_y_F[i]-1)/(K*(S+K*A));
      }

      khdy[i+cpml_F_Y]=khdy[i+cpml_F_Y]*K;
    }
    /*bh_y_F[NcpmlY-1]=be_y_F[NcpmlY-1];
    ch_y_F[NcpmlY-1]=ce_y_F[NcpmlY-1];
    khdy[cpml_F_Y]=kedy[cpml_F_Y];*/

    hipMemcpy(kedydev,kedy,NCELLY*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(khdydev,khdy,NCELLY*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ce_y_Ndev,ce_y_N,(NcpmlY+1)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ce_y_Fdev,ce_y_F,(NcpmlY+1)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(be_y_Ndev,be_y_N,(NcpmlY+1)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(be_y_Fdev,be_y_F,(NcpmlY+1)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ch_y_Ndev,ch_y_N,(NcpmlY)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ch_y_Fdev,ch_y_F,(NcpmlY)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(bh_y_Ndev,bh_y_N,(NcpmlY)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(bh_y_Fdev,bh_y_F,(NcpmlY)*sizeof(real),hipMemcpyHostToDevice);



}



void SETUP_CPML_Z(void){

    int i,j,k;
    real S,A,K;

    for(i=0; i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
            for(k=0;k<NcpmlZ+1;k++){

                psi_Ex_z_F[ThreeDMap(i,j,k,NcpmlZ+1,NCELLY)]=0.0;
                psi_Ex_z_N[ThreeDMap(i,j,k,NcpmlZ+1,NCELLY)]=0.0;
                psi_Ey_z_F[ThreeDMap(i,j,k,NcpmlZ+1,NCELLY)]=0.0;
                psi_Ey_z_N[ThreeDMap(i,j,k,NcpmlZ+1,NCELLY)]=0.0;


        }
      }
    }
// printf("here\n" );
 for(i=0; i<NCELLX;i++){
     for(j=0;j<NCELLY;j++){
         for(k=0;k<NcpmlZ;k++){

                psi_Hx_z_F[ThreeDMap(i,j,k,NcpmlZ,NCELLY)]=0.0;
                psi_Hx_z_N[ThreeDMap(i,j,k,NcpmlZ,NCELLY)]=0.0;
                psi_Hy_z_F[ThreeDMap(i,j,k,NcpmlZ,NCELLY)]=0.0;
                psi_Hy_z_N[ThreeDMap(i,j,k,NcpmlZ,NCELLY)]=0.0;


        }
      }
    }

    // printf("here\n" );
    hipMemcpy(psi_Ex_z_Fdev,psi_Ex_z_F,(NcpmlZ+1)*NCELLX*NCELLY*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Ex_z_Ndev,psi_Ex_z_N,(NcpmlZ+1)*NCELLX*NCELLY*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Ey_z_Fdev,psi_Ey_z_F,(NcpmlZ+1)*NCELLX*NCELLY*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Ey_z_Ndev,psi_Ey_z_N,(NcpmlZ+1)*NCELLX*NCELLY*sizeof(real),hipMemcpyHostToDevice);


    hipMemcpy(psi_Hx_z_Fdev,psi_Hx_z_F,(NcpmlZ)*NCELLX*NCELLY*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Hx_z_Ndev,psi_Hx_z_N,(NcpmlZ)*NCELLX*NCELLY*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Hy_z_Fdev,psi_Hy_z_F,(NcpmlZ)*NCELLX*NCELLY*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(psi_Hy_z_Ndev,psi_Hy_z_N,(NcpmlZ)*NCELLX*NCELLY*sizeof(real),hipMemcpyHostToDevice);
    //Y COMPONENT for E field
    for(i=0;i<NcpmlZ+1;i++){
      //PML grading in the x direction (eqn 7.60a)
      S=pow(((real)(NcpmlZ-i)/((real)(NcpmlZ))),cpml_exp)*max_sigma_cpml_z;
      // equation (7.79)
      A=max_alpha_z*pow((real)i/((real)(NcpmlZ)),exp_alpha_z);
      // (eqn 7.60b)
      K=1+(max_stretch_factor_z-1)*pow((real)(NcpmlZ-i)/((real)(NcpmlZ)),cpml_exp);
      // (eqn 7.102) part of the c_w term. Also see (7.114a)
      be_z_N[i]=exp(-(S/K+A)*(dt/EP0));
      //7.114(b)

    if(K*A+S==0.0){
        ce_z_N[i]=0.0;
      }
      else{
        ce_z_N[i]=S*(be_z_N[i]-1)/(K*(S+K*A));
      }
      kedz[i]=kedz[i]*K;
    }
    //printf("here\n" );

    for(i=0;i<NcpmlZ+1;i++){
      //PML grading in the x direction (eqn 7.60a)
      S=pow(((real)(i)/((real)(NcpmlZ))),cpml_exp)*(max_sigma_cpml_z);
      // equation (7.79)
      A=max_alpha_z*pow((real)(NcpmlZ-i)/((real)(NcpmlZ)),exp_alpha_z);
      // (eqn 7.60b)
      K=1+(max_stretch_factor_z-1)*pow((real)(i)/((real)(NcpmlZ)),cpml_exp);
      // (eqn 7.102) part of the c_w term. Also see (7.114a)
      be_z_F[i]=exp(-(S/K+A)*(dt/EP0));
      //7.114(b)
      if(K*A+S==0.0){
        ce_z_F[i]=0.0;
      }
      else{
        ce_z_F[i]=S*(be_z_F[i]-1)/(K*(S+K*A));
      }

      kedz[i+cpml_F_Z]=kedz[i+cpml_F_Z]*K;
      //printf("%f\t%f\t%f\n",be_z_F[i],ce_z_F[i],kedz[i+cpml_F_Z]);
    }

    //Y COMPONENT for H-field
    for(i=0;i<NcpmlZ;i++){
      //PML grading in the x direction (eqn 7.60a)
      S=pow((((real)NcpmlZ-(real)i-0.5)/((real)NcpmlZ)),cpml_exp)*max_sigma_cpml_z;
      // equation (7.79)
      A=max_alpha_z*pow(((real)i+0.5)/((real)NcpmlZ),exp_alpha_z);
      // (eqn 7.60b)
      K=1+(max_stretch_factor_z-1)*pow(((real)NcpmlZ-(real)i-0.5)/((real)NcpmlZ),cpml_exp);
      // (eqn 7.102) part of the c_w term. Also see (7.114a)
      bh_z_N[i]=exp(-(S/K+A)*(dt/EP0));
      //7.114(b)
      if(K*A+S==0.0){
        ch_z_N[i]=0;
      }
      else{
        ch_z_N[i]=S*(bh_z_N[i]-1)/(K*(S+K*A));
      }

      khdz[i]=khdz[i]*K;
    }
    /*bh_y_N[NcpmlY-1]=be_y_N[NcpmlY-1];
    ch_y_N[NcpmlY-1]=ce_y_N[NcpmlY-1];
    khdy[NcpmlY-1]=kedy[NcpmlY-1];*/
    printf("here\n" );

    for(i=0;i<NcpmlZ;i++){
      //PML grading in the x direction (eqn 7.60a)
      S=pow((((real)i+0.5)/((real)NcpmlZ)),cpml_exp)*(max_sigma_cpml_z);
      // equation (7.79)
      A=max_alpha_z*pow(((real)NcpmlZ-(real)i-0.5)/((real)NcpmlZ),exp_alpha_z);
      // (eqn 7.60b)
      K=1+(max_stretch_factor_z-1)*pow(((real)i+0.5)/((real)NcpmlZ),cpml_exp);
      // (eqn 7.102) part of the c_w term. Also see (7.114a)
      bh_z_F[i]=exp(-(S/K+A)*(dt/EP0));
      //7.114(b)

    if(K*A+S==0.0){
        ch_z_F[i]=0;
      }
      else{
        ch_z_F[i]=S*(bh_z_F[i]-1)/(K*(S+K*A));
      }

      khdz[i+cpml_F_Z]=khdz[i+cpml_F_Z]*K;
    }
    /*bh_y_F[NcpmlY-1]=be_y_F[NcpmlY-1];
    ch_y_F[NcpmlY-1]=ce_y_F[NcpmlY-1];
    khdy[cpml_F_Y]=kedy[cpml_F_Y];*/
    printf("here\n" );
    hipMemcpy(kedzdev,kedz,NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(khdzdev,khdz,NCELLZ*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ce_z_Ndev,ce_z_N,(NcpmlZ+1)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ce_z_Fdev,ce_z_F,(NcpmlZ+1)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(be_z_Ndev,be_z_N,(NcpmlZ+1)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(be_z_Fdev,be_z_F,(NcpmlZ+1)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ch_z_Ndev,ch_z_N,(NcpmlZ)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(ch_z_Fdev,ch_z_F,(NcpmlZ)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(bh_z_Ndev,bh_z_N,(NcpmlZ)*sizeof(real),hipMemcpyHostToDevice);
    hipMemcpy(bh_z_Fdev,bh_z_F,(NcpmlZ)*sizeof(real),hipMemcpyHostToDevice);
}

void SETUP_Drude_CP(void){
  real FERMI_VELOCITY = 1.39e8 / 100.0; //for silver and gold (m/s)
  real D = 2.0;
  real C = 3.0*D/(D+2.0);
  real NL_COEFF = sqrt(C/D);

//  NL_COEFF = 1.0;
  real NONLOC;
    N_drude_poles=1;
    N_CP_poles=2;
    N_lorentz_poles = 2;
    int i,k,j,n;
    real something=0.0;
    Hydrodynamics = 0;
    WithMagField =0;
    WithConvection = 0;
    if(NONLOCAL == 3){
       Hydrodynamics = 1;
       WithMagField = 0;
       WithConvection = 0;
     }
     size_t extentD = NCELLX*NCELLY*NCELLZ*N_drude_poles*sizeof(real);
     size_t extentCP = NCELLX*NCELLY*NCELLZ*N_CP_poles*sizeof(real);
hipError_t err;
     printf("HERE\n");
    //Allocate memory for parameter vectors
  err=  hipMalloc(&C_1_cpdev,N_CP_poles*sizeof(real));
  err=  hipMalloc(&C_2_cpdev,N_CP_poles*sizeof(real));
  err=  hipMalloc(&C_3_cpdev,N_CP_poles*sizeof(real));
  err=  hipMalloc(&C_4_cpdev,N_CP_poles*sizeof(real));
  err=  hipMalloc(&C_5_cpdev,N_CP_poles*sizeof(real));
  if( hipSuccess != err)
  {
      printf( "Cuda error: %s.\n",hipGetErrorString( err) );
      exit(-1);
  }
    C_1_cp=MALLOC1D(C_1_cp,N_CP_poles);
    C_2_cp=MALLOC1D(C_2_cp,N_CP_poles);
    C_3_cp=MALLOC1D(C_3_cp,N_CP_poles);
    C_4_cp=MALLOC1D(C_4_cp,N_CP_poles);
    C_5_cp=MALLOC1D(C_5_cp,N_CP_poles);
    C_cp=MALLOC1D(C_cp,N_CP_poles);

    err=  hipMalloc(&d_1_ddev,N_drude_poles*sizeof(real));
    err=  hipMalloc(&d_2_ddev,N_drude_poles*sizeof(real));
    err=  hipMalloc(&d_3_ddev,N_drude_poles*sizeof(real));
    err=  hipMalloc(&d_4_ddev,N_drude_poles*sizeof(real));
    err=  hipMalloc(&d_5_ddev,N_drude_poles*sizeof(real));
    err=  hipMalloc(&d_NLdev,N_drude_poles*sizeof(real));
    if( hipSuccess != err)
    {
        printf( "Cuda error: %s.\n",hipGetErrorString( err) );
        exit(-1);
    }
    printf("HERE\n");

    d_1_d=MALLOC1D(d_1_d,N_drude_poles);
    d_2_d=MALLOC1D(d_2_d,N_drude_poles);
    d_3_d=MALLOC1D(d_3_d,N_drude_poles);
    d_4_d=MALLOC1D(d_4_d,N_drude_poles);
    d_5_d=MALLOC1D(d_5_d,N_drude_poles);
    d_d=MALLOC1D(d_d,N_drude_poles);
   d_NL = MALLOC1D(d_NL,N_drude_poles);

    psi_L = MALLOC1D(psi_L,N_lorentz_poles);
    psi_HD = MALLOC1D(psi_HD,N_drude_poles);
    alpha_HD1 = MALLOC1D(alpha_HD1,N_drude_poles);
    alpha_HD2 = MALLOC1D(alpha_HD2,N_drude_poles);
    alpha_L = MALLOC1D(alpha_L,N_lorentz_poles);
    eta_L = MALLOC1D(eta_L,N_lorentz_poles);
    eta_HD = MALLOC1D(eta_HD,N_drude_poles);

    w_D=MALLOC1D(w_D,N_drude_poles);
    gamma_d=MALLOC1D(gamma_d,N_drude_poles);

    A_cp=MALLOC1D(A_cp,N_CP_poles);
    OMEGA_cp=MALLOC1D(OMEGA_cp,N_CP_poles);
    phi_cp=MALLOC1D(phi_cp,N_CP_poles);
    GAMMA_cp=MALLOC1D(GAMMA_cp,N_CP_poles);

    d_eps_L = MALLOC1D(d_eps_L,N_lorentz_poles);
    delta_L = MALLOC1D(delta_L,N_lorentz_poles);
    omg_L = MALLOC1D(omg_L,N_lorentz_poles);
    printf("HERE\n");



    a_0_cp=MALLOC1D(a_0_cp,N_CP_poles);
    a_1_cp=MALLOC1D(a_1_cp,N_CP_poles);
    b_0_cp=MALLOC1D(b_0_cp,N_CP_poles);
    b_1_cp=MALLOC1D(b_1_cp,N_CP_poles);
    b_2_cp=MALLOC1D(b_2_cp,N_CP_poles);


    err=  hipMalloc(&Px_cpdev,extentCP);
    err=  hipMalloc(&Px_cp_ndev,extentCP);
    err=  hipMalloc(&Px_cp_n_1dev,extentCP);
  err=    hipMalloc(&Py_cpdev,extentCP);
    err=  hipMalloc(&Py_cp_ndev,extentCP);
    err=  hipMalloc(&Py_cp_n_1dev,extentCP);
    err=  hipMalloc(&Pz_cpdev,extentCP);
    err=  hipMalloc(&Pz_cp_ndev,extentCP);
    err=  hipMalloc(&Pz_cp_n_1dev,extentCP);
    if( hipSuccess != err)
    {
        printf( "Cuda error: %s.\n",hipGetErrorString( err) );
        exit(-1);
    }
    printf("HERE\n");

    Px_cp=MALLOC4D_Complex(Px_cp,NCELLX,NCELLY,NCELLZ,N_CP_poles);
    Px_cp_n=MALLOC4D_Complex(Px_cp_n,NCELLX,NCELLY,NCELLZ,N_CP_poles);
    Px_cp_n_1=MALLOC4D_Complex(Px_cp_n_1,NCELLX,NCELLY,NCELLZ,N_CP_poles);
    Py_cp=MALLOC4D_Complex(Py_cp,NCELLX,NCELLY,NCELLZ,N_CP_poles);
    Py_cp_n=MALLOC4D_Complex(Py_cp_n,NCELLX,NCELLY,NCELLZ,N_CP_poles);
    Py_cp_n_1=MALLOC4D_Complex(Py_cp_n_1,NCELLX,NCELLY,NCELLZ,N_CP_poles);
    Pz_cp=MALLOC4D_Complex(Pz_cp,NCELLX,NCELLY,NCELLZ,N_CP_poles);
    Pz_cp_n=MALLOC4D_Complex(Pz_cp_n,NCELLX,NCELLY,NCELLZ,N_CP_poles);
    Pz_cp_n_1=MALLOC4D_Complex(Pz_cp_n_1,NCELLX,NCELLY,NCELLZ,N_CP_poles);

    err= hipMalloc(&Px_ddev,extentD);
    err= hipMalloc(&Px_d_ndev,extentD);
    err= hipMalloc(&Px_d_n_1dev,extentD);
    err= hipMalloc(&Py_ddev,extentD);
    err= hipMalloc(&Py_d_ndev,extentD);
    err= hipMalloc(&Py_d_n_1dev,extentD);
    err= hipMalloc(&Pz_ddev,extentD);
    err= hipMalloc(&Pz_d_ndev,extentD);
    err= hipMalloc(&Pz_d_n_1dev,extentD);
    err= hipMalloc(&Px_d_n_2dev,extentD);
    err= hipMalloc(&Py_d_n_2dev,extentD);
    err= hipMalloc(&Pz_d_n_2dev,extentD);

    if( hipSuccess != err)
    {
        printf( "Cuda error: %s.\n",hipGetErrorString( err) );
        exit(-1);
    }
    printf("HERE\n");


    Px_d=MALLOC4D_Complex(Px_d,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Px_d_n=MALLOC4D_Complex(Px_d_n,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Px_d_n_1=MALLOC4D_Complex(Px_d_n_1,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Py_d=MALLOC4D_Complex(Py_d,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Py_d_n=MALLOC4D_Complex(Py_d_n,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Py_d_n_1=MALLOC4D_Complex(Py_d_n_1,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Pz_d=MALLOC4D_Complex(Pz_d,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Pz_d_n=MALLOC4D_Complex(Pz_d_n,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Pz_d_n_1=MALLOC4D_Complex(Pz_d_n_1,NCELLX,NCELLY,NCELLZ,N_drude_poles);

    Px_NL=MALLOC4D_Complex(Px_NL,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Px_NL_n=MALLOC4D_Complex(Px_NL_n,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Px_NL_n_1=MALLOC4D_Complex(Px_NL_n_1,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Py_NL=MALLOC4D_Complex(Py_NL,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Py_NL_n=MALLOC4D_Complex(Py_NL_n,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Py_NL_n_1=MALLOC4D_Complex(Py_NL_n_1,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Pz_NL=MALLOC4D_Complex(Pz_NL,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Pz_NL_n=MALLOC4D_Complex(Pz_NL_n,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Pz_NL_n_1=MALLOC4D_Complex(Pz_NL_n_1,NCELLX,NCELLY,NCELLZ,N_drude_poles);

    Jx_NL=MALLOC4D_Complex(Jx_NL,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Jx_NL_n=MALLOC4D_Complex(Jx_NL_n,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Jx_NL_n_1=MALLOC4D_Complex(Jx_NL_n_1,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Jy_NL=MALLOC4D_Complex(Jy_NL,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Jy_NL_n=MALLOC4D_Complex(Jy_NL_n,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Jy_NL_n_1=MALLOC4D_Complex(Jy_NL_n_1,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Jz_NL=MALLOC4D_Complex(Jz_NL,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Jz_NL_n=MALLOC4D_Complex(Jz_NL_n,NCELLX,NCELLY,NCELLZ,N_drude_poles);
    Jz_NL_n_1=MALLOC4D_Complex(Jz_NL_n_1,NCELLX,NCELLY,NCELLZ,N_drude_poles);

    Jx_Lo=MALLOC4D_Complex(Jx_Lo,NCELLX,NCELLY,NCELLZ,N_lorentz_poles);
    Jx_Lo_n=MALLOC4D_Complex(Jx_Lo_n,NCELLX,NCELLY,NCELLZ,N_lorentz_poles);
    Jx_Lo_n_1=MALLOC4D_Complex(Jx_Lo_n_1,NCELLX,NCELLY,NCELLZ,N_lorentz_poles);
    Jy_Lo=MALLOC4D_Complex(Jy_Lo,NCELLX,NCELLY,NCELLZ,N_lorentz_poles);
    Jy_Lo_n=MALLOC4D_Complex(Jy_Lo_n,NCELLX,NCELLY,NCELLZ,N_lorentz_poles);
    Jy_Lo_n_1=MALLOC4D_Complex(Jy_Lo_n_1,NCELLX,NCELLY,NCELLZ,N_lorentz_poles);
    Jz_Lo=MALLOC4D_Complex(Jz_Lo,NCELLX,NCELLY,NCELLZ,N_lorentz_poles);
    Jz_Lo_n=MALLOC4D_Complex(Jz_Lo_n,NCELLX,NCELLY,NCELLZ,N_lorentz_poles);
    Jz_Lo_n_1=MALLOC4D_Complex(Jz_Lo_n_1,NCELLX,NCELLY,NCELLZ,N_lorentz_poles);

    printf("HERE\n");


    //SILVER
  if(material == 2){
    eps_inf=1.4447;
    w_D[0]=1.3280e16;
    gamma_d[0]=9.1269e13;

    A_cp[0]=-1.5951;
    phi_cp[0]=3.1288;
    OMEGA_cp[0]=8.2749e15;
    GAMMA_cp[0]=5.1770e15;

    A_cp[1]=0.25261;
    phi_cp[1]=-1.5066;
    OMEGA_cp[1]=6.1998e15;
    GAMMA_cp[1]=5.4126e14;

    if(NONLOCAL == 1) NONLOC = FERMI_VELOCITY*NL_COEFF;
    else NONLOC = 0;

    if(Hydrodynamics == 1){
      N_EQ = 5.541318196761572e28;
      printf("NEQ=%e\n",N_EQ);
       NONLOC = FERMI_VELOCITY*NL_COEFF;
    }

  }
    //Gold
  if(material == 3){
    eps_inf=1.1431;
    w_D[0]=1.3202e16;
    gamma_d[0]=1.0805e14;

    A_cp[0]=0.26698;
    phi_cp[0]=-1.2371;
    OMEGA_cp[0]=3.8711e15;
    GAMMA_cp[0]=4.4642e14;

    A_cp[1]=3.0834;
    phi_cp[1]=-1.0968;
    OMEGA_cp[1]=4.1684e15;
    GAMMA_cp[1]=2.3555e15;

    if(NONLOCAL == 1) NONLOC = FERMI_VELOCITY*NL_COEFF;
    else NONLOC = 0;
    if(Hydrodynamics == 1){
      N_EQ = 5.476415562682574e28;
       NONLOC = FERMI_VELOCITY*NL_COEFF;
    }
  }

  //Nonlocal SILVER
if(material == 4){
  // eps_inf=1.4447;
  // w_D[0]=1.3280e16;
  // gamma_d[0]=9.1269e13;
  //
  // A_cp[0]=-1.5951;
  // phi_cp[0]=3.1288;
  // OMEGA_cp[0]=8.2749e15;
  // GAMMA_cp[0]=5.1770e15;
  //
  // A_cp[1]=0.25261;
  // phi_cp[1]=-1.5066;
  // OMEGA_cp[1]=6.1998e15;
  // GAMMA_cp[1]=5.4126e14;
  //
  // NONLOC = FERMI_VELOCITY*NL_COEFF;
}
  //Nonlocal Gold
if(material == 5){


  // eps_inf=1.1431;
  // w_D[0]=1.3202e16;
  // gamma_d[0]=1.0805e14;
  //
  // A_cp[0]=0.26698;
  // phi_cp[0]=-1.2371;
  // OMEGA_cp[0]=3.8711e15;
  // GAMMA_cp[0]=4.4642e14;
  //
  // A_cp[1]=3.0834;
  // phi_cp[1]=-1.0968;
  // OMEGA_cp[1]=4.1684e15;
  // GAMMA_cp[1]=2.3555e15;
  //
  //
  //
  // NONLOC = FERMI_VELOCITY*NL_COEFF;
}
if(material == 6){

}
if(material == 7){
  eps_inf = 3.559;
  d_eps_L[0] = 2.912;
  d_eps_L[1] = 1.272;
  delta_L[0] = (2.0*PI/PLANKS)*1.541;
  delta_L[1] = (2.0*PI/PLANKS)*0.525;
  omg_L[0] = (2.0*PI/PLANKS)*4.693;
  omg_L[1] = (2.0*PI/PLANKS)*3.112;
  // delta_L[0] = 1.541;
  // delta_L[1] = 0.525;
  // omg_L[0] = 4.693;
  // omg_L[1] = 3.112;

  w_D[0] = (2.0*PI/PLANKS)*8.812;
  gamma_d[0] = (2.0*PI/PLANKS)*0.0752;
  // w_D[0] = 8.812;
  // gamma_d[0] = 0.0752;
  if(NONLOCAL==1) NONLOC = FERMI_VELOCITY*NL_COEFF;
  else NONLOC = 0;
}
printf("HERE\n");


  if(material >= 2 && material <=5){

    //set-up update coefficients
    for(i=0;i<N_CP_poles;i++){

        a_0_cp[i]=2.0*ep0*A_cp[i]*OMEGA_cp[i]*(OMEGA_cp[i]*cos(phi_cp[i])-GAMMA_cp[i]*sin(phi_cp[i]));
        a_1_cp[i]=-2.0*ep0*A_cp[i]*OMEGA_cp[i]*sin(phi_cp[i]);
        b_0_cp[i]=GAMMA_cp[i]*GAMMA_cp[i]+OMEGA_cp[i]*OMEGA_cp[i];
        b_1_cp[i]=2.0*GAMMA_cp[i];
        b_2_cp[i]=1.0;
        C_cp[i]=b_2_cp[i]/(dt*dt)+b_1_cp[i]/(2.0*dt)+b_0_cp[i]/4.0;
        C_1_cp[i]=(2*b_2_cp[i]/(dt*dt)-b_0_cp[i]/2.0)/C_cp[i];
        C_2_cp[i]=(b_1_cp[i]/(2.0*dt)-b_2_cp[i]/(dt*dt)-b_0_cp[i]/4.0)/C_cp[i];
        C_3_cp[i]=(a_0_cp[i]/4.0+a_1_cp[i]/(2.0*dt))/C_cp[i];
        C_4_cp[i]=a_0_cp[i]/(2.0*C_cp[i]);
        C_5_cp[i]=(a_0_cp[i]/4.0-a_1_cp[i]/(2.0*dt))/C_cp[i];

    }


    for(i=0;i<N_drude_poles;i++){
      if(Hydrodynamics==0){
        d_d[i]=1.0/(dt*dt)+gamma_d[i]/(2.0*dt);
        d_1_d[i]=2.0/(d_d[i]*dt*dt);
        d_2_d[i]=(gamma_d[i]/(2.0*dt)-1.0/(dt*dt))/d_d[i];
        d_3_d[i]=d_5_d[i]=ep0*w_D[i]*w_D[i]/(4.0*d_d[i]);
        d_4_d[i]=ep0*w_D[i]*w_D[i]/(2.0*d_d[i]);
        d_NL[i] = NONLOC*NONLOC/d_d[i];

        printf("%e,%e,%e,%e,%e,%e,%e,%e,%e\n",e0,me, C_E,d_d[i],d_1_d[i], d_2_d[i], d_3_d[i], d_4_d[i], d_5_d[i]);
        // printf("%e,%e,%e,%e,%e\n", cimag(d_1_d[i]), cimag(d_2_d[i]), cimag(d_3_d[i]), cimag(d_4_d[i]), cimag(d_5_d[i]));
        printf("%e\n",d_NL[i]);


        for(n=0;n<N_drude_poles;n++){
            something+=d_3_d[n];
        }
        for(n=0;n<N_CP_poles;n++){
            something+=C_3_cp[n];
        }
        C_E=(ep0*eps_inf+something);


        something=0.0;
        for(n=0;n<N_drude_poles;n++){
            something+=d_4_d[n];
        }
        for(n=0;n<N_CP_poles;n++){
            something+=C_4_cp[n];
        }
        C_E_1=-something+ep0*eps_inf;

        something=0.0;
        for(n=0;n<N_drude_poles;n++){
            something+=d_5_d[n];
        }
        for(n=0;n<N_CP_poles;n++){
            something+=C_5_cp[n];
        }
        C_E_2=something;

      }
      else{
        d_d[i] = 1.0/(dt*dt) + gamma_d[i]/(2.0*dt);
        d_1_d[i]=2.0/(d_d[i]*dt*dt);
        d_2_d[i]=(gamma_d[i]/(2.0*dt)-1.0/(dt*dt))/d_d[i];
        d_3_d[i]= -1.0*e0/(me)/d_d[i]/dt;
        d_4_d[i]= 1.0*e0/(me)/d_d[i]/dt;
        d_5_d[i] = -1.0/d_d[i];
        d_NL[i] = pow(N_EQ,1.0/3.0)*NONLOC*NONLOC/d_d[i];
        printf("%e,%e,%e,%e,%e,%e,%e,%e,%e\n",e0,me, C_E,d_d[i],d_1_d[i], d_2_d[i], d_3_d[i], d_4_d[i], d_5_d[i]);
        // printf("%e,%e,%e,%e,%e\n", cimag(d_1_d[i]), cimag(d_2_d[i]), cimag(d_3_d[i]), cimag(d_4_d[i]), cimag(d_5_d[i]));
        printf("%e\n",d_NL[i]);

        for(n=0;n<N_CP_poles;n++){
            something+=C_3_cp[n];
        }
        C_E=(ep0*eps_inf+something);


        something=0.0;

        for(n=0;n<N_CP_poles;n++){
            something+=C_4_cp[n];
        }
        C_E_1=-something+ep0*eps_inf;

        something=0.0;

        for(n=0;n<N_CP_poles;n++){
            something+=C_5_cp[n];
        }
        C_E_2=something;

      }

      printf("%d,%e,%e,%e,%e,%e,%e,%e,%e,%e\n",i,e0,me, C_E,d_d[i],d_1_d[i], d_2_d[i], d_3_d[i], d_4_d[i], d_5_d[i]);
      // printf("%e,%e,%e,%e,%e\n", cimag(d_1_d[i]), cimag(d_2_d[i]), cimag(d_3_d[i]), cimag(d_4_d[i]), cimag(d_5_d[i]));
      printf("%e\n",d_NL[i]);
    }

}



if(material >= 6){
    for(n=0;n<N_lorentz_poles;n++){
      alpha_L[n] = (2.0 - omg_L[n]*omg_L[n]*dt*dt)/(1.0 + delta_L[n]*dt);
      psi_L[n] = -1.0*(1.0 - delta_L[n]*dt)/(1.0 + delta_L[n]*dt);
      eta_L[n] = ep0*d_eps_L[n]*omg_L[n]*omg_L[n]*dt*dt/(1.0 + delta_L[n]*dt);
    }
    for(n=0;n<N_drude_poles;n++){
      alpha_HD1[n] = 4.0/(2.0 + gamma_d[n]*dt);
      alpha_HD2[n] = 2.0*dt*dt*NONLOC*NONLOC/(2.0 + gamma_d[n]*dt);
      psi_HD[n] = -1.0*(2.0 - gamma_d[n]*dt)/(2.0 + gamma_d[n]*dt);
      eta_HD[n] = 2.0*ep0*w_D[n]*w_D[n]*dt*dt/(2.0 + gamma_d[n]*dt);
    }
    C1_NL = eps_inf*ep0/dt;
    C2_NL = (1.0/(4.0*dt))*(eta_L[0]+eta_L[1]+eta_HD[0]);
    // printf("CNLs: %f\t%f\n",C1_NL,C2_NL);
    // printf("etas: %f\t%f\t%f\n",eta_L[0]/ep0,eta_L[1]/ep0,eta_HD[0]/ep0);
    printf("%e\n",alpha_HD2[0]);

}
printf("HERE\n");

    for(i=0;i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
            for(k=0;k<NCELLZ;k++){
                for(n=0;n<N_CP_poles;n++){
                    Px_cp[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=0.0;
                    Px_cp_n[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=0.0;
                    Px_cp_n_1[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=0.0;
                    Py_cp[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=0.0;
                    Py_cp_n[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=0.0;
                    Py_cp_n_1[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=0.0;
                    Pz_cp[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=0.0;
                    Pz_cp_n[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=0.0;
                    Pz_cp_n_1[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=0.0;

            }
          }
        }
    }
    printf("HERE1\n");

    for(i=0;i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
           for(k=0;k<NCELLZ;k++){
             for(n=0;n<N_drude_poles;n++){
                Px_NL[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Px_NL_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Px_NL_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Py_NL[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Py_NL_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Py_NL_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Pz_NL[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Pz_NL_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Pz_NL_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;

                Px_d[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Px_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Px_d_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Py_d[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Py_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Py_d_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Pz_d[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Pz_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Pz_d_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;

                Jx_NL[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Jx_NL_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Jx_NL_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Jy_NL[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Jy_NL_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Jy_NL_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Jz_NL[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Jz_NL_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
                Jz_NL_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=0.0;
            }
            }
          }
        }
        for(i=0;i<NCELLX;i++){
            for(j=0;j<NCELLY;j++){
               for(k=0;k<NCELLZ;k++){
                 for(n=0;n<N_lorentz_poles;n++){

                    Jx_Lo[FourDMap(i,j,k,n,N_lorentz_poles,NCELLZ,NCELLY)]=0.0;
                    Jx_Lo_n[FourDMap(i,j,k,n,N_lorentz_poles,NCELLZ,NCELLY)]=0.0;
                    Jx_Lo_n_1[FourDMap(i,j,k,n,N_lorentz_poles,NCELLZ,NCELLY)]=0.0;
                    Jy_Lo[FourDMap(i,j,k,n,N_lorentz_poles,NCELLZ,NCELLY)]=0.0;
                    Jy_Lo_n[FourDMap(i,j,k,n,N_lorentz_poles,NCELLZ,NCELLY)]=0.0;
                    Jy_Lo_n_1[FourDMap(i,j,k,n,N_lorentz_poles,NCELLZ,NCELLY)]=0.0;
                    Jz_Lo[FourDMap(i,j,k,n,N_lorentz_poles,NCELLZ,NCELLY)]=0.0;
                    Jz_Lo_n[FourDMap(i,j,k,n,N_lorentz_poles,NCELLZ,NCELLY)]=0.0;
                    Jz_Lo_n_1[FourDMap(i,j,k,n,N_lorentz_poles,NCELLZ,NCELLY)]=0.0;
                }
                }
              }
            }


            printf("HERE1\n");


    FREE1D(w_D);
    FREE1D(gamma_d);
    FREE1D(A_cp);
    FREE1D(OMEGA_cp);
    FREE1D(phi_cp);
    FREE1D(GAMMA_cp);

    FREE1D(a_0_cp);
    FREE1D(a_1_cp);
    FREE1D(b_0_cp);
    FREE1D(b_1_cp);
    FREE1D(b_2_cp);
    printf("CE=%e\n",C_E);

hipMemcpy(d_1_ddev,d_1_d,N_drude_poles*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(d_2_ddev,d_2_d,N_drude_poles*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(d_3_ddev,d_3_d,N_drude_poles*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(d_4_ddev,d_4_d,N_drude_poles*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(d_5_ddev,d_5_d,N_drude_poles*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(d_NLdev,d_NL,N_drude_poles*sizeof(real),hipMemcpyHostToDevice);

hipMemcpy(C_1_cpdev,C_1_cp,N_CP_poles*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(C_2_cpdev,C_2_cp,N_CP_poles*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(C_3_cpdev,C_3_cp,N_CP_poles*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(C_4_cpdev,C_4_cp,N_CP_poles*sizeof(real),hipMemcpyHostToDevice);
hipMemcpy(C_5_cpdev,C_5_cp,N_CP_poles*sizeof(real),hipMemcpyHostToDevice);

    err=  hipMemcpy(Px_cpdev,Px_cp,extentCP,hipMemcpyHostToDevice);
    err=  hipMemcpy(Px_cp_ndev,Px_cp_n,extentCP,hipMemcpyHostToDevice);
    err=  hipMemcpy(Px_cp_n_1dev,Px_cp_n_1,extentCP,hipMemcpyHostToDevice);

    err=  hipMemcpy(Py_cpdev,Py_cp,extentCP,hipMemcpyHostToDevice);
    err=  hipMemcpy(Py_cp_ndev,Py_cp_n,extentCP,hipMemcpyHostToDevice);
    err=  hipMemcpy(Py_cp_n_1dev,Py_cp_n_1,extentCP,hipMemcpyHostToDevice);


    err=  hipMemcpy(Pz_cpdev,Pz_cp,extentCP,hipMemcpyHostToDevice);
    err=  hipMemcpy(Pz_cp_ndev,Pz_cp_n,extentCP,hipMemcpyHostToDevice);
    err=  hipMemcpy(Pz_cp_n_1dev,Pz_cp_n_1,extentCP,hipMemcpyHostToDevice);

    err=  hipMemcpy(Px_ddev,Px_d,extentD,hipMemcpyHostToDevice);
    err=  hipMemcpy(Px_d_ndev,Px_d_n,extentD,hipMemcpyHostToDevice);
    err=  hipMemcpy(Px_d_n_1dev,Px_d_n_1,extentD,hipMemcpyHostToDevice);
    err=  hipMemcpy(Px_d_n_2dev,Px_NL,extentD,hipMemcpyHostToDevice);


    err=  hipMemcpy(Py_ddev,Py_d,extentD,hipMemcpyHostToDevice);
    err=  hipMemcpy(Py_d_ndev,Py_d_n,extentD,hipMemcpyHostToDevice);
    err=  hipMemcpy(Py_d_n_1dev,Py_d_n_1,extentD,hipMemcpyHostToDevice);
    err=  hipMemcpy(Py_d_n_2dev,Py_NL,extentD,hipMemcpyHostToDevice);


    err=  hipMemcpy(Pz_ddev,Pz_d,extentD,hipMemcpyHostToDevice);
    err=  hipMemcpy(Pz_d_ndev,Pz_d_n,extentD,hipMemcpyHostToDevice);
    err=  hipMemcpy(Pz_d_n_1dev,Pz_d_n_1,extentD,hipMemcpyHostToDevice);
    err=  hipMemcpy(Pz_d_n_2dev,Pz_NL,extentD,hipMemcpyHostToDevice);

    // err=  hipMemcpy(Py_cpdev,extentCP);
    // err=  hipMemcpy(Py_cp_ndev,extentCP);
    // err=  hipMemcpy(Py_cp_n_1dev,extentCP);
    // err=  hipMemcpy(Pz_cpdev,extentCP);
    // err=  hipMemcpy(Pz_cp_ndev,extentCP);
    // err=  hipMemcpy(Pz_cp_n_1dev,extentCP);
    //
    // err= hipMemcpy(Px_ddev,extentD);
    // err= hipMemcpy(Px_d_ndev,extentD);
    // err= hipMemcpy(Px_d_n_1dev,extentD);
    // err= hipMemcpy(Py_ddev,extentD);
    // err= hipMemcpy(Py_d_ndev,extentD);
    // err= hipMemcpy(Py_d_n_1dev,extentD);
    // err= hipMemcpy(Pz_ddev,extentD);
    // err= hipMemcpy(Pz_d_ndev,extentD);
    // err= hipMemcpy(Pz_d_n_1dev,extentD);
}



//1D memory allocation
real* MALLOC1D(real *grid, int SIZE){
  grid=(real *)malloc((SIZE)*sizeof(real));
  return grid;
}

real* MALLOC1D_double(real *grid, int SIZE){
  grid=(real *)malloc((SIZE)*sizeof(real));
  return grid;
}

comp* MALLOC1D_Complex(comp *grid, int SIZE){
  grid=(comp *)malloc((SIZE)*sizeof(comp));
  return grid;
}
real2* MALLOC1D_Real2(real2 *grid, int SIZE){
  grid=(real2 *)malloc((SIZE)*sizeof(real2));
  return grid;
}

double complex* MALLOC1D_Complex2(double complex *grid, int SIZE){
  grid=(double complex *)malloc((SIZE)*sizeof(double complex));
  return grid;
}

//2-D Memory Allocation
real* MALLOC2D(real *grid, int sizeX, int sizeZ){
    int i;
    grid=(real *)malloc((sizeX*sizeZ)*sizeof(real));
    if(!grid){
        printf("\n\nERROR IN MEMORY ALLOCATION\n\n");
    }
    return grid;

}

real* MALLOC2D_double(real *grid, int sizeX, int sizeZ){
    int i;
    grid=(real *)malloc((sizeX*sizeZ)*sizeof(real));
    if(!grid){
        printf("\n\nERROR IN MEMORY ALLOCATION\n\n");

    }

    return grid;

}

comp* MALLOC2D_Complex(comp *grid, int sizeX, int sizeZ){
    int i;
    grid=(comp *)malloc((sizeX*sizeZ)*sizeof(comp));
    if(!grid){
        printf("\n\nERROR IN MEMORY ALLOCATION\n\n");

    }
    return grid;

}



// 3-D array memory allocation
real* MALLOC3D(real *grid, int sizeX, int sizeY, int sizeZ){
    int i,j;
    grid=(real *)malloc((sizeX*sizeY*sizeZ)*sizeof(real));
    if(!grid){
        printf("\n\nERROR IN MEMORY ALLOCATION\n\n");

    }

    return grid;

}

real* MALLOC3D_double(real *grid, int sizeX, int sizeY, int sizeZ){
    int i,j;
    grid=(real *)malloc((sizeX*sizeY*sizeZ)*sizeof(real));
    if(!grid){
        printf("\n\nERROR IN MEMORY ALLOCATION\n\n");

    }

    return grid;

}
real2* MALLOC3D_Real2(real2 *grid, int sizeX, int sizeY, int sizeZ){
    int i,j;
    grid=(real2 *)malloc((sizeX*sizeY*sizeZ)*sizeof(real2));
    if(!grid){
        printf("\n\nERROR IN MEMORY ALLOCATION\n\n");

    }

    return grid;

}

comp* MALLOC3D_Complex(comp *grid, int sizeX, int sizeY,int sizeZ){
    int i,j;
    grid=(comp *)malloc((sizeX*sizeY*sizeZ)*sizeof(comp));
    if(!grid){
        printf("\n\nERROR IN MEMORY ALLOCATION\n\n");

    }
    return grid;

}

double complex * MALLOC3D_Complex2(double complex *grid, int sizeX, int sizeY,int sizeZ){
    int i,j;
    grid=(double complex  *)malloc((sizeX*sizeY*sizeZ)*sizeof(double complex ));
    if(!grid){
        printf("\n\nERROR IN MEMORY ALLOCATION\n\n");

    }
    return grid;
}

int * MALLOC3D_int(int *grid,int sizeX, int sizeY, int sizeZ){
  int i,j;
  grid=(int *)malloc((sizeX*sizeY*sizeZ)*sizeof(int));
  if(!grid){
      printf("\n\nERROR IN MEMORY ALLOCATION\n\n");

  }
  return grid;

}

//4-D Memory Allocation

real* MALLOC4D(real *grid, int sizeX, int sizeY, int sizeZ,int size4){
    int i,j,k;
    grid=(real *)malloc((sizeX*sizeY*sizeZ*size4)*sizeof(real));
    if(!grid){
        printf("\n\nERROR IN MEMORY ALLOCATION\n\n");

    }
      return grid;
}


real* MALLOC4D_double(real *grid, int sizeX, int sizeY, int sizeZ,int size4){
    int i,j,k;
    grid=(real *)malloc((sizeX*sizeY*sizeZ*size4)*sizeof(real));
    if(!grid){
        printf("\n\nERROR IN MEMORY ALLOCATION\n\n");

    }
      return grid;
}

comp* MALLOC4D_Complex(comp *grid, int sizeX, int sizeY,int sizeZ,int size4){
    int i,j,k;
    grid=(comp *)malloc((sizeX*sizeY*sizeZ*size4)*sizeof(comp));
    if(!grid){
        printf("\n\nERROR IN MEMORY ALLOCATION\n\n");

    }

    return grid;

}


//free arrays
void FREE1D(real *grid){
    free(grid);
}

void FREE1D_double(double *grid){
    free(grid);
}

void FREE1D_Complex(comp *grid){
    free(grid);
}

void FREE1D_Complex2(double complex *grid){
    free(grid);
}

void FREE2D(real *grid,int sizeX){
        free(grid);
    return;
}

void FREE2D_double(double *grid,int sizeX){

        free(grid);
    return;
}

void FREE2D_Complex(comp *grid,int sizeX){

    free(grid);
    return;
}

void FREE3D(real *grid,int sizeX,int sizeY){

    free(grid);
    return;
}


void FREE3D_double(double *grid,int sizeX,int sizeY){

    free(grid);
    return;
}


void FREE3D_Complex(comp *grid,int sizeX,int sizeY){
    int i,j;

    free(grid);
    return;
}

void FREE3D_Complex2(double complex *grid,int sizeX,int sizeY){

    free(grid);
    return;
}



void FREE4D(real *grid,int sizeX,int sizeY,int sizeZ){
    free(grid);
    return;
}

void FREE4D_double(double *grid,int sizeX,int sizeY,int sizeZ){

    free(grid);
    return;
}


void FREE4D_Complex(comp *grid,int sizeX,int sizeY,int sizeZ){

    free(grid);
    return;
}

//make the vector zero
real* ZERO_VECTORS2D(real *grid,int SizeX,int SizeY){
    int i,j;
    for (i=0;i<SizeX;i++){
        for(j=0;j<SizeY;j++){

                grid[TwoDMap(i,j,SizeY)]=0.0;

        }
    }
    return grid;
}

real* ZERO_VECTORS2D_double(real *grid,int SizeX,int SizeY){
    int i,j;
    for (i=0;i<SizeX;i++){
        for(j=0;j<SizeY;j++){

                grid[TwoDMap(i,j,SizeY)]=0.0;

        }
    }
    return grid;
}

real* ZERO_VECTORS3D(real *grid,int SizeX,int SizeY,int SizeZ){
    int i,j,k;
    for (i=0;i<SizeX;i++){
        for(j=0;j<SizeY;j++){
           for(k=0;k<SizeZ;k++) {
              grid[ThreeDMap(i,j,k,SizeZ,SizeY)]=0.0;
           }

        }
    }
    return grid;
}

real* ZERO_VECTORS3D_double(real *grid,int SizeX,int SizeY,int SizeZ){
    int i,j,k;
    for (i=0;i<SizeX;i++){
        for(j=0;j<SizeY;j++){
           for(k=0;k<SizeZ;k++) {
              grid[ThreeDMap(i,j,k,SizeZ,SizeY)]=0.0;
           }

        }
    }
    return grid;
}

real2* ZERO_VECTORS3D_Real2 (real2 *grid,int SizeX,int SizeY,int SizeZ){
    int i,j,k;
    for (i=0;i<SizeX;i++){
        for(j=0;j<SizeY;j++){
           for(k=0;k<SizeZ;k++) {
              grid[ThreeDMap(i,j,k,SizeZ,SizeY)]=0.0;
           }

        }
    }
    return grid;
}

comp * ZERO_VECTORS2D_Complex(comp *grid,int SizeX,int SizeY){
    int i,j;
    for (i=0;i<SizeX;i++){
        for(j=0;j<SizeY;j++){
                grid[TwoDMap(i,j,SizeY)]=0.0;

        }
    }
    return grid;
}

comp * ZERO_VECTORS3D_Complex(comp *grid,int SizeX,int SizeY,int SizeZ){
    int i,j,k;
    for (i=0;i<SizeX;i++){
        for(j=0;j<SizeY;j++){
                for(k=0;k<SizeZ;k++){
                    grid[ThreeDMap(i,j,k,SizeZ,SizeY)]=0.0;
                }
        }
    }
    return grid;
}

double complex * ZERO_VECTORS3D_Complex2(double complex *grid,int SizeX,int SizeY,int SizeZ){
    int i,j,k;
    for (i=0;i<SizeX;i++){
        for(j=0;j<SizeY;j++){
                for(k=0;k<SizeZ;k++){
                    grid[ThreeDMap(i,j,k,SizeZ,SizeY)]=0.0;
                }
        }
    }
    return grid;
}


real* ZERO_VECTORS1D(real *grid){
    int i;
    for(i=0;i<inc_Length;i++){
        grid[i]=0;
    }
    return grid;
}

real* ZERO_VECTORS1D_double(real *grid){
    int i;
    for(i=0;i<inc_Length;i++){
        grid[i]=0;
    }
    return grid;
}

comp* ZERO_VECTORS1D_Complex(comp *grid,int length){
    int i;
    for(i=0;i<length;i++){
        grid[i]=0;
    }
    return grid;
}

double complex* ZERO_VECTORS1D_Complex2(double complex *grid,int length){
    int i;
    for(i=0;i<length;i++){
        grid[i]=0;
    }
    return grid;
}



//define the electric conductivity
void DEF_SIGMA_E(void){
    int i,j,k;
    for(i=0;i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
            for(k=0;k<NCELLZ;k++){
                sigma_e[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=0;
            }
        }
    }

    return;
}

//define the magnetic conductivity
void DEF_SIGMA_M(void){
    int i,j,k;
    for(i=0;i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
            for(k=0;k<NCELLZ;k++){
                sigma_m[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=0;
            }
        }
    }

    return;
}

//define the epsilon
void DEF_EPS(void){
    int i,j,k;
    for(i=0;i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
            for(k=0;k<NCELLZ;k++){
                    eps[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=EP0;

            }
        }
    }

    return;
}

//define the mu
void DEF_MU(void){
    int i,j,k;
    for(i=0;i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
            for(k=0;k<NCELLZ;k++){
                mu[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=MU0;
            }
        }
    }

    return;
}

void DIELECTRIC_SLAB(void){
    int i,j,k;
    real eps_r=10;
    //int location=inc_plane+ceil(NCELLZ/2)-2*NcpmlZ;
    int location=cpml_N_Z+200;
  //  int SLAB_SIZE=31;
    for(i=0;i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
            for(k=location;k<NCELLZ;k++){
                   eps[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=eps[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*eps_r;
            }
        }
    }
}

//define the update coefficients for E-field update equations on the H field components
real * DEF_UPDATE_COEFF_EonH(real *grid){
      int i,j,k;
      for(i=0;i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
            for(k=0;k<NCELLZ;k++){
                grid[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=(dt/(z0*eps[ThreeDMap(i,j,k,NCELLZ,NCELLY)]))/((1+(dt*sigma_e[ThreeDMap(i,j,k,NCELLZ,NCELLY)])/(2*eps[ThreeDMap(i,j,k,NCELLZ,NCELLY)])));
            }
        }
      }
    return grid;
}
//define the update coefficients for E-field update equations on the E field components
real * DEF_UPDATE_COEFF_EonE(real *grid){
      int i,j,k;
      for(i=0;i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
            for(k=0;k<NCELLZ;k++){
                grid[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=(1-(dt*sigma_e[ThreeDMap(i,j,k,NCELLZ,NCELLY)])/(2*eps[ThreeDMap(i,j,k,NCELLZ,NCELLY)]))/(1+(dt*sigma_e[ThreeDMap(i,j,k,NCELLZ,NCELLY)])/(2*eps[ThreeDMap(i,j,k,NCELLZ,NCELLY)]));
            }
        }
      }
    return grid;
}

//define the update coefficients for H-field update equations on the E field components
real *DEF_UPDATE_COEFF_HonE(real *grid){
      int i,j,k;
      for(i=0;i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
            for(k=0;k<NCELLZ;k++){
                grid[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=(z0*dt/mu[ThreeDMap(i,j,k,NCELLZ,NCELLY)])/(1+(dt*sigma_m[ThreeDMap(i,j,k,NCELLZ,NCELLY)])/(2*mu[ThreeDMap(i,j,k,NCELLZ,NCELLY)]));
            }
        }
      }
    return grid;
}
//define the update coefficients for H-field update equations on the H field components
real *DEF_UPDATE_COEFF_HonH(real *grid){
      int i,j,k;
      for(i=0;i<NCELLX;i++){
        for(j=0;j<NCELLY;j++){
            for(k=0;k<NCELLZ;k++){
                grid[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=(1-(dt*sigma_m[ThreeDMap(i,j,k,NCELLZ,NCELLY)])/(2*mu[ThreeDMap(i,j,k,NCELLZ,NCELLY)]))/(1+(dt*sigma_m[ThreeDMap(i,j,k,NCELLZ,NCELLY)])/(2*mu[ThreeDMap(i,j,k,NCELLZ,NCELLY)]));
            }
        }
      }
    return grid;
}

void SETUP_t_inc(void){
  int i;
  real n_inc = 1.0;
  real d_silver = (dispersive_slab - inc_plane)* dz;
  real arg, arg_sq;
  for(i=0;i<NUM_freq; i++){
    if(freq[i]>f_min){
      arg = (c0*k_x/(2*pi*freq[i]));
      arg_sq = arg*arg;
      arg = pow(1-arg_sq,0.5);
      t_inc[i] =  (d_silver*n_inc/(c0*arg));
      t_inc[i] =(double) ceil (t_inc[i] / dt + delay + width);
      // if(c0/freq[i] >= 675e-9){
      //     t_inc[i] += 1500;
      // }
      t_inc[i] = 0;
    }
    else{
      t_inc[i] = 1.0;
    }
  }
}
//
// void FT_source_calc(void){
//
//   int i,j,k,n,w;
//
//   int Tend_inc = Tend;
//   int NUM_freq_inc = Tend;
//
//   FILE *Input;
//   FILE *Input_w;
//
//   Input = fopen("Input_Pulse.txt", "w");
//   Input_w = fopen("Input_Pulse_w.txt", "w");
//
//   double max_omega = 2 * pi * freq[0];
//   double min_omega = 1;
//   double min_freq = min_omega/(2*pi);
//   double max_freq = max_omega/(2*pi);
//   double Tend_real = width_real*11;
//   double *omega2;
//   double *freq2;
//   double *time2;
//   double dw_2;
//
//   Ex_t_FT=MALLOC1D_Complex(Ex_t_FT,Tend_inc);
//   Hx_t_FT=MALLOC1D_Complex(Hx_t_FT,Tend_inc);
//   Ey_t_FT=MALLOC1D_Complex(Ey_t_FT,Tend_inc);
//   Hy_t_FT=MALLOC1D_Complex(Hy_t_FT,Tend_inc);
//   Ez_t_FT=MALLOC1D_Complex(Ez_t_FT,Tend_inc);
//   Hz_t_FT=MALLOC1D_Complex(Hz_t_FT,Tend_inc);
//
//   Ex_w_FT=MALLOC1D_Complex(Ex_w_FT,NUM_freq_inc);
//   Hx_w_FT=MALLOC1D_Complex(Hx_w_FT,NUM_freq_inc);
//   Ey_w_FT=MALLOC1D_Complex(Ey_w_FT,NUM_freq_inc);
//   Hy_w_FT=MALLOC1D_Complex(Hy_w_FT,NUM_freq_inc);
//   Ez_w_FT=MALLOC1D_Complex(Ez_w_FT,NUM_freq_inc);
//   Hz_w_FT=MALLOC1D_Complex(Hz_w_FT,NUM_freq_inc);
//
// omega2 = MALLOC1D_double(omega2, NUM_freq_inc);
// freq2 = MALLOC1D_double(freq2,NUM_freq_inc);
// time2 = MALLOC1D_double(time2,Tend_inc);
//
//   //printf("%e\n%e",freq[0], freq[NUM_freq_inc-1]);
//
//   dw_2 = (max_omega - min_omega)/(NUM_freq_inc - 1);
//
//
//   Hx_t_FT = ZERO_VECTORS1D_Complex(Hx_t_FT,Tend_inc);
//   Hy_t_FT = ZERO_VECTORS1D_Complex(Hy_t_FT,Tend_inc);
//   Hz_t_FT = ZERO_VECTORS1D_Complex(Hz_t_FT,Tend_inc);
//
//   Ex_t_FT = ZERO_VECTORS1D_Complex(Ex_t_FT,Tend_inc);
//   Ey_t_FT = ZERO_VECTORS1D_Complex(Ey_t_FT,Tend_inc);
//   Ez_t_FT = ZERO_VECTORS1D_Complex(Ez_t_FT,Tend_inc);
//
//   Hx_w_FT = ZERO_VECTORS1D_Complex(Hx_w_FT,NUM_freq_inc);
//   Hy_w_FT = ZERO_VECTORS1D_Complex(Hy_w_FT,NUM_freq_inc);
//   Hz_w_FT = ZERO_VECTORS1D_Complex(Hz_w_FT,NUM_freq_inc);
//
//   Ex_w_FT = ZERO_VECTORS1D_Complex(Ex_w_FT,NUM_freq_inc);
//   Ey_w_FT = ZERO_VECTORS1D_Complex(Ey_w_FT,NUM_freq_inc);
//   Ez_w_FT = ZERO_VECTORS1D_Complex(Ez_w_FT,NUM_freq_inc);
//
//   if(TEz){
//
//      for(n=0;n<Tend_inc;n++){
//         Ey_t_FT[n] = PULSE(n);
//         time2[n] = n*dt;
//       }
//
//       for(w=0;w<NUM_freq_inc;w++){
//         omega2[w] = min_omega + w*(max_omega - min_omega)/(NUM_freq_inc - 1);
//         freq2[w] = omega2[w]/(2*pi);
//       }
//
//
//       for(n=0;n<Tend_inc;n++){
//         for(w=0;w<NUM_freq_inc;w++){
//           Ey_w_FT[w] += Ey_t_FT[n] * cexp(-I*time2[n]*omega2[w])*dt;
//         }
//       }
//
//       for(w=0;w<NUM_freq_inc;w++){
//         Ey_w_FT[w] /= csqrt(2*pi);
//       }
//
//       for(w=0;w<NUM_freq_inc;w++){
//         Hx_w_FT[w] = Ey_w_FT[w]*csqrt((omega2[w]/c0)*(omega2[w]/c0) - (k_rho*k_rho))/(omega2[w]/c0);
//         fprintf(Input_w,"%e \t %e \t %e \n", omega2[w], cabs(Ey_w_FT[w]), cabs(Hx_w_FT[w]));
//       }
//       //printf("here\n");
//
//       for(n=0;n<Tend_inc;n++){
//           for(w=0;w<NUM_freq_inc;w++){
//           Hx_t_FT[n] += Hx_w_FT[w] * cexp(I*time2[n]*omega2[w])*dw_2;
//           Ey_t_FT[n] += Ey_w_FT[w] * cexp(I*time2[n]*omega2[w])*dw_2;
//
//         }
//         Ey_t_FT[n] /= csqrt(4*pi);
//         Hx_t_FT[n] /= csqrt(2*pi);
//         fprintf(Input,"%e \t %e \t %e \t %e\n", time2[n] , creal(Ey_t_FT[n]), creal(Hx_t_FT[n]), creal(PULSE(n)));
//       }
//       //printf("here\n");
//
//   }
//
//   fclose(Input);
//   fclose(Input_w);
//
//   free(omega2);
//   free(freq2);
//   free(time2);
// }
//


// void FT_source_calc(void){
//
//   int i,j,k,n,w;
//
//   // int Tend_inc = Tend;
//   // int NUM_freq_inc = 5000;
//   // if(Tend>250000) {
//   //   Tend_inc =  250000;
//   //   NUM_freq_inc = 5000;
//   // }
//   FILE *Input;
//   FILE *Input_w;
//
//   Input = fopen("Input_Pulse.txt", "w");
//   Input_w = fopen("Input_Pulse_w.txt", "w");
//
//   double Tend_real = Tend_inc*dt;
//   double *omega2;
//   double *freq2;
//   double *time2;
//   comp k_z,k_0;
//   double dw_2 = 2*pi/Tend_real;
//
//   Ex_t_FT=MALLOC1D_Complex(Ex_t_FT,Tend);
//   Hx_t_FT=MALLOC1D_Complex(Hx_t_FT,Tend);
//   Ey_t_FT=MALLOC1D_Complex(Ey_t_FT,Tend);
//   Hy_t_FT=MALLOC1D_Complex(Hy_t_FT,Tend);
//   Ez_t_FT=MALLOC1D_Complex(Ez_t_FT,Tend);
//   Hz_t_FT=MALLOC1D_Complex(Hz_t_FT,Tend);
//
//   Ex_w_FT=MALLOC1D_Complex(Ex_w_FT,NUM_freq_inc);
//   Hx_w_FT=MALLOC1D_Complex(Hx_w_FT,NUM_freq_inc);
//   Ey_w_FT=MALLOC1D_Complex(Ey_w_FT,NUM_freq_inc);
//   Hy_w_FT=MALLOC1D_Complex(Hy_w_FT,NUM_freq_inc);
//   Ez_w_FT=MALLOC1D_Complex(Ez_w_FT,NUM_freq_inc);
//   Hz_w_FT=MALLOC1D_Complex(Hz_w_FT,NUM_freq_inc);
//
// omega2 = MALLOC1D_double(omega2, NUM_freq_inc);
// freq2 = MALLOC1D_double(freq2,NUM_freq_inc);
// time2 = MALLOC1D_double(time2,Tend_inc);
//
//   //printf("%e\n%e",freq[0], freq[NUM_freq_inc-1]);
//
//   Hx_t_FT = ZERO_VECTORS1D_Complex(Hx_t_FT,Tend);
//   Hy_t_FT = ZERO_VECTORS1D_Complex(Hy_t_FT,Tend);
//   Hz_t_FT = ZERO_VECTORS1D_Complex(Hz_t_FT,Tend);
//
//   Ex_t_FT = ZERO_VECTORS1D_Complex(Ex_t_FT,Tend);
//   Ey_t_FT = ZERO_VECTORS1D_Complex(Ey_t_FT,Tend);
//   Ez_t_FT = ZERO_VECTORS1D_Complex(Ez_t_FT,Tend);
//
//   Hx_w_FT = ZERO_VECTORS1D_Complex(Hx_w_FT,NUM_freq_inc);
//   Hy_w_FT = ZERO_VECTORS1D_Complex(Hy_w_FT,NUM_freq_inc);
//   Hz_w_FT = ZERO_VECTORS1D_Complex(Hz_w_FT,NUM_freq_inc);
//
//   Ex_w_FT = ZERO_VECTORS1D_Complex(Ex_w_FT,NUM_freq_inc);
//   Ey_w_FT = ZERO_VECTORS1D_Complex(Ey_w_FT,NUM_freq_inc);
//   Ez_w_FT = ZERO_VECTORS1D_Complex(Ez_w_FT,NUM_freq_inc);
//
//   // FILE *SOURCE_IN;
//   // char filename[200];
//   // sprintf(filename,"Source.%d.txt",trials);
// 	// SOURCE_IN = fopen(filename,"r");
//   double Source_re1,Source_comp1,Source_re2,Source_comp2;
//
//   if(TEz){
//
//     printf("Define Function\n");
//      for(n=0;n<Tend_inc;n++){
//         //  fscanf(SOURCE_IN,"%lf %lf %lf %lf\n",&Source_re1, &Source_comp1, &Source_re2, &Source_comp2);
//         //  Ey_t_FT[n] = 1.0*(Source_re1 + I*Source_comp1);
//         //  //printf("%e\t%e\t%e\t%e\n",creal(Source_comp1),cimag(Source_comp1), creal(I*Source_comp1),cimag(I*Source_comp1));
//         //  Hx_t_FT[n] = -1.0*(Source_re2 + I*Source_comp2);
//         Ey_t_FT[n] = PULSE(n);
//       }
//
//       printf("DFT\n");
//
//       for(n=0;n<Tend_inc;n++){
//         for(w=0;w<NUM_freq_inc;w++){
//           Ey_w_FT[w] += Ey_t_FT[n] * cexp(-I*2*pi*n*w/Tend_inc);
//         }
//       }
//
//       printf("Transform\n");
//
//       for(w=0;w<NUM_freq_inc;w++){
//         Ey_w_FT[w] /= Tend_inc;
//         if(w>1){
//           k_z = csqrt((dw_2*w/c0)*(dw_2*w/c0) - (k_rho*k_rho));
//           k_0 = dw_2*w/c0;
//           Hx_w_FT[w] = (-1.0)*Ey_w_FT[w]*(k_z/k_0)*cexp(I*k_z*0.5*dz);
//           fprintf(Input_w,"%e \t %e \t %e \n", dw_2*w, cabs(Ey_w_FT[w]), cabs(Hx_w_FT[w]));
//         }
//       }
//       //printf("here\n");
//       printf("IDFT\n");
//
//       for(n=0;n<Tend_inc;n++){
//           for(w=1;w<NUM_freq_inc;w++){
//           Hx_t_FT[n] += Hx_w_FT[w] * cexp(I*2*pi*n*w/Tend_inc);
//         }
//
//         //fprintf(Input,"%e \t %e \t %e \t %e\n", time2[n] , creal(Ey_t_FT[n]), creal(Hx_t_FT[n]), creal(PULSE(n)));
//       }
//       printf("Output File\n");
//     for(n=0;n<Tend_inc;n++){
//       fprintf(Input,"%e \t %e \t %e\n", time2[n] , creal(Ey_t_FT[n]), creal(Hx_t_FT[n]));
//     }
//
//       i=0;
//       j=0;
//       //printf("here\n");
//       printf("Fourier Transform of Input Pulse\n");
//     for(n=0;n<Tend_inc;n++){
//       for(w=0;w<NUM_freq;w++){
//       //  for(i=0;i<NCELLX;i++){
//         //  for(j=0;j<NCELLY;j++){
//             E_Incident[w][i][j] += Ey_t_FT[n]*cexp(-I*2*pi*n*dt*freq[w]);//*cexp(-I*(i)*dx*k_x)*cexp(-I*(j+0.5)*dy*k_y);
//             H_Incident[w][i][j] += Hx_t_FT[n]*cexp(-I*2*pi*n*dt*freq[w]);//*cexp(-I*(i)*dx*k_x)*cexp(-I*(j+0.5)*dy*k_y);
//           //}
//         //}
//       }
//       printf("n=%d\n",n);
//
//     }
//
//   }
//
//   if(TMz){
//     printf("Define Function\n");
//     for(n=0;n<Tend_inc;n++){
//        Ex_t_FT[n] = PULSE(n);
//      }
//
//      printf("DFT\n");
//      for(n=0;n<Tend_inc;n++){
//        for(w=0;w<NUM_freq_inc;w++){
//          Ex_w_FT[w] += Ex_t_FT[n] * cexp(-I*2*pi*n*w/Tend_inc);
//        }
//      }
//
//
//      printf("Transform\n");
//      for(w=0;w<NUM_freq_inc;w++){
//        Ex_w_FT[w] /= Tend_inc;
//        if(w>1){
//          k_z = csqrt((dw_2*w/c0)*(dw_2*w/c0) - (k_rho*k_rho));
//          k_0 = dw_2*w/c0;
//          Hy_w_FT[w] = (-1.0)*Ex_w_FT[w]*(k_0/k_z)*cexp(I*k_z*0.5*dz);
//          fprintf(Input_w,"%e \t %e \t %e \n", dw_2*w, cabs(Ex_w_FT[w]), cabs(Hy_w_FT[w]));
//        }
//      }
//      //printf("here\n");
//      printf("IDFT\n");
//      for(n=0;n<Tend_inc;n++){
//          //Ey_t_FT[n] = 0.0;
//          for(w=1;w<NUM_freq_inc;w++){
//          //Hy_t_FT[n] += Hy_w_FT[w] * cexp(I*2*pi*n*w/Tend_inc);
//          Hy_t_FT[n] += Hy_w_FT[w] * cexp(I*2*pi*n*w/Tend_inc);
//
//        }
//
//        fprintf(Input,"%e \t %e \t %e \t %e\n", time2[n] , creal(Ex_t_FT[n]), creal(Hy_t_FT[n]), creal(PULSE(n)));
//      }
//      //printf("here\n");
//      printf("Fourier Transform of Input Pulse \n");
//      i=0;
//      j=0;
//    for(n=0;n<Tend_inc;n++){
//      for(w=0;w<NUM_freq;w++){
//        //for(i=0;i<NCELLX;i++){
//          //for(j=0;j<NCELLY;j++){
//            H_Incident[w][i][j] += Hy_t_FT[n]*cexp(-I*2*pi*n*dt*freq[w]);//*cexp(-I*(i+0.5)*dx*k_x)*cexp(-I*(j)*dy*k_y);
//            E_Incident[w][i][j] += Ex_t_FT[n]*cexp(-I*2*pi*n*dt*freq[w]);//*cexp(-I*(i+0.5)*dx*k_x)*cexp(-I*(j)*dy*k_y);
//          //}
//       // }
//      }
//    }
//   }
//   printf("HERE1\n");
//
//   fclose(Input);
//   fclose(Input_w);
//   //fclose(SOURCE_IN);
//   printf("HERE2\n");
//
//   free(omega2);
//   free(freq2);
//   free(time2);
//   printf("HERE3\n");
//
// }

void ALLOCATE_MEM(void){

     //allocate memory for 1-D incident fields

    e_inc=MALLOC1D_Complex(e_inc,inc_Length);
    h_inc=MALLOC1D_Complex(h_inc,inc_Length);
    ex_inc=MALLOC1D_Complex(ex_inc,inc_Length);
    ey_inc=MALLOC1D_Complex(ey_inc,inc_Length);
    ez_inc=MALLOC1D_Complex(ez_inc,inc_Length);
    hx_inc=MALLOC1D_Complex(hx_inc,inc_Length);
    hy_inc=MALLOC1D_Complex(hy_inc,inc_Length);
    hz_inc=MALLOC1D_Complex(hz_inc,inc_Length);

    //allocate memory for 3-D vectors
    ex=MALLOC3D_Complex(ex,NCELLX,NCELLY,NCELLZ);
    ey=MALLOC3D_Complex(ey,NCELLX,NCELLY,NCELLZ);
    ez=MALLOC3D_Complex(ez,NCELLX,NCELLY,NCELLZ);
//    Dx=MALLOC3D_Complex(Dx,NCELLX,NCELLY,NCELLZ);
//    Dy=MALLOC3D_Complex(Dy,NCELLX,NCELLY,NCELLZ);
//    Dz=MALLOC3D_Complex(Dz,NCELLX,NCELLY,NCELLZ);
    hx=MALLOC3D_Complex(hx,NCELLX,NCELLY,NCELLZ);
    hy=MALLOC3D_Complex(hy,NCELLX,NCELLY,NCELLZ);
    hz=MALLOC3D_Complex(hz,NCELLX,NCELLY,NCELLZ);


        size_t extent = NCELLX*NCELLY*NCELLZ * sizeof(real);
        size_t extent1 = NCELLX*NCELLY*NcpmlZ * sizeof(real);
        size_t extent2 = NCELLX*NcpmlY*NCELLZ * sizeof(real);
        size_t extent3 = NcpmlX*NCELLY*NCELLZ * sizeof(real);


        hipMalloc(&hxdev,extent);
        hipMalloc(&hydev,extent);
        hipMalloc(&hzdev,extent);
        hipMalloc(&exdev,extent);
        hipMalloc(&ezdev,extent);
        hipMalloc(&eydev,extent);

        hipMalloc(&Chxhdev,extent);
        hipMalloc(&Chxedev,extent);
        hipMalloc(&Chyhdev,extent);
        hipMalloc(&Chyedev,extent);
        hipMalloc(&Chzhdev,extent);
        hipMalloc(&Chzedev,extent);

        hipMalloc(&Cexhdev,extent);
        hipMalloc(&Cexedev,extent);
        hipMalloc(&Ceyhdev,extent);
        hipMalloc(&Ceyedev,extent);
        hipMalloc(&Cezhdev,extent);
        hipMalloc(&Cezedev,extent);

        hipMalloc(&psi_Hx_z_Ndev,extent1);
        hipMalloc(&psi_Hx_z_Fdev,extent1);
        hipMalloc(&psi_Hx_y_Ndev,extent2);
        hipMalloc(&psi_Hx_y_Fdev,extent2);

        hipMalloc(&psi_Hy_z_Ndev,extent1);
        hipMalloc(&psi_Hy_z_Fdev,extent1);
        hipMalloc(&psi_Hy_x_Ndev,extent3);
        hipMalloc(&psi_Hy_x_Fdev,extent3);

        hipMalloc(&psi_Hz_y_Ndev,extent2);
        hipMalloc(&psi_Hz_y_Fdev,extent2);
        hipMalloc(&psi_Hz_x_Ndev,extent3);
        hipMalloc(&psi_Hz_x_Fdev,extent3);

    //    size_t eextent = NCELLX*NCELLY*NCELLZ * sizeof(real);
        size_t eextent1 = NCELLX*NCELLY*(NcpmlZ+1) * sizeof(real);
        size_t eextent2 = NCELLX*(NcpmlY+1)*NCELLZ * sizeof(real);
        size_t eextent3 = (NcpmlX+1)*NCELLY*NCELLZ * sizeof(real);


        hipMalloc(&psi_Ex_z_Ndev,eextent1);
        hipMalloc(&psi_Ex_z_Fdev,eextent1);
        hipMalloc(&psi_Ex_y_Ndev,eextent2);
        hipMalloc(&psi_Ex_y_Fdev,eextent2);

        hipMalloc(&psi_Ey_z_Ndev,eextent1);
        hipMalloc(&psi_Ey_z_Fdev,eextent1);
        hipMalloc(&psi_Ey_x_Ndev,eextent3);
        hipMalloc(&psi_Ey_x_Fdev,eextent3);

        hipMalloc(&psi_Ez_y_Ndev,eextent2);
        hipMalloc(&psi_Ez_y_Fdev,eextent2);
        hipMalloc(&psi_Ez_x_Ndev,eextent3);
        hipMalloc(&psi_Ez_x_Fdev,eextent3);


        hipMalloc(&khdydev,NCELLY*sizeof(real));
        hipMalloc(&khdzdev,NCELLZ*sizeof(real));
        hipMalloc(&khdxdev,NCELLX*sizeof(real));

        hipMalloc(&kedydev,NCELLY*sizeof(real));
        hipMalloc(&kedzdev,NCELLZ*sizeof(real));
        hipMalloc(&kedxdev,NCELLX*sizeof(real));

        hipMalloc(&bh_z_Ndev,NcpmlZ*sizeof(real));
        hipMalloc(&bh_z_Fdev,NcpmlZ*sizeof(real));
        hipMalloc(&ch_z_Ndev,NcpmlZ*sizeof(real));
        hipMalloc(&ch_z_Fdev,NcpmlZ*sizeof(real));
        hipMalloc(&bh_y_Ndev,NcpmlY*sizeof(real));
        hipMalloc(&bh_y_Fdev,NcpmlY*sizeof(real));
        hipMalloc(&ch_y_Ndev,NcpmlY*sizeof(real));
        hipMalloc(&ch_y_Fdev,NcpmlY*sizeof(real));
        hipMalloc(&bh_x_Ndev,NcpmlX*sizeof(real));
        hipMalloc(&bh_x_Fdev,NcpmlX*sizeof(real));
        hipMalloc(&ch_x_Ndev,NcpmlX*sizeof(real));
        hipMalloc(&ch_x_Fdev,NcpmlX*sizeof(real));

        hipMalloc(&be_z_Ndev,(NcpmlZ+1)*sizeof(real));
        hipMalloc(&be_z_Fdev,(NcpmlZ+1)*sizeof(real));
        hipMalloc(&ce_z_Ndev,(NcpmlZ+1)*sizeof(real));
        hipMalloc(&ce_z_Fdev,(NcpmlZ+1)*sizeof(real));
        hipMalloc(&be_y_Ndev,(NcpmlY+1)*sizeof(real));
        hipMalloc(&be_y_Fdev,(NcpmlY+1)*sizeof(real));
        hipMalloc(&ce_y_Ndev,(NcpmlY+1)*sizeof(real));
        hipMalloc(&ce_y_Fdev,(NcpmlY+1)*sizeof(real));
        hipMalloc(&be_x_Ndev,(NcpmlX+1)*sizeof(real));
        hipMalloc(&be_x_Fdev,(NcpmlX+1)*sizeof(real));
        hipMalloc(&ce_x_Ndev,(NcpmlX+1)*sizeof(real));
        hipMalloc(&ce_x_Fdev,(NcpmlX+1)*sizeof(real));

        hipMalloc(&ex_ndev,extent);
        hipMalloc(&ez_ndev,extent);
        hipMalloc(&ey_ndev,extent);

        hipMalloc(&ex_n_1dev,extent);
        hipMalloc(&ez_n_1dev,extent);
        hipMalloc(&ey_n_1dev,extent);
        ex_n=MALLOC3D_Complex(ex_n,NCELLX,NCELLY,NCELLZ);
        ey_n=MALLOC3D_Complex(ey_n,NCELLX,NCELLY,NCELLZ);
        ez_n=MALLOC3D_Complex(ez_n,NCELLX,NCELLY,NCELLZ);

        ex_n_1=MALLOC3D_Complex(ex_n_1,NCELLX,NCELLY,NCELLZ);
        ey_n_1=MALLOC3D_Complex(ey_n_1,NCELLX,NCELLY,NCELLZ);
        ez_n_1=MALLOC3D_Complex(ez_n_1,NCELLX,NCELLY,NCELLZ);


    if(Hydrodynamics == 1){
      NDx=MALLOC3D_Complex(NDx,NCELLX,NCELLY,NCELLZ);
      NDy=MALLOC3D_Complex(NDy,NCELLX,NCELLY,NCELLZ);
      NDz=MALLOC3D_Complex(NDz,NCELLX,NCELLY,NCELLZ);
      hipMalloc(&NDxdev,NCELLX*NCELLY*NCELLZ*sizeof(real));
      hipMalloc(&NDydev,NCELLX*NCELLY*NCELLZ*sizeof(real));
      hipMalloc(&NDzdev,NCELLX*NCELLY*NCELLZ*sizeof(real));
      NDx_prev=MALLOC3D_Complex(NDx_prev,NCELLX,NCELLY,NCELLZ);
      NDy_prev=MALLOC3D_Complex(NDy_prev,NCELLX,NCELLY,NCELLZ);
      NDz_prev=MALLOC3D_Complex(NDz_prev,NCELLX,NCELLY,NCELLZ);
      hipMalloc(&NDx_prevdev,NCELLX*NCELLY*NCELLZ*sizeof(real));
      hipMalloc(&NDy_prevdev,NCELLX*NCELLY*NCELLZ*sizeof(real));
      hipMalloc(&NDz_prevdev,NCELLX*NCELLY*NCELLZ*sizeof(real));
      hxPrev=MALLOC3D_Complex(hxPrev,NCELLX,NCELLY,NCELLZ);
      hyPrev=MALLOC3D_Complex(hyPrev,NCELLX,NCELLY,NCELLZ);
      hzPrev=MALLOC3D_Complex(hzPrev,NCELLX,NCELLY,NCELLZ);
      hipMalloc(&hxPrevdev,NCELLX*NCELLY*NCELLZ*sizeof(real));
      hipMalloc(&hyPrevdev,NCELLX*NCELLY*NCELLZ*sizeof(real));
      hipMalloc(&hzPrevdev,NCELLX*NCELLY*NCELLZ*sizeof(real));


    }

    ExTransformNearZScaRe = MALLOC3D_Real2(ExTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
    EyTransformNearZScaRe = MALLOC3D_Real2(EyTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
    EzTransformNearZScaRe = MALLOC3D_Real2(EzTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
    HxTransformNearZScaRe = MALLOC3D_Real2(HxTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
    HyTransformNearZScaRe = MALLOC3D_Real2(HyTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
    HzTransformNearZScaRe = MALLOC3D_Real2(HzTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);

    hipMalloc(&ExTransformNearZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));
    hipMalloc(&EyTransformNearZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));
    hipMalloc(&HxTransformNearZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));
    hipMalloc(&HyTransformNearZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));

    ExTransformNearYScaRe = MALLOC3D_Real2(ExTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
    EyTransformNearYScaRe = MALLOC3D_Real2(EyTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
    EzTransformNearYScaRe = MALLOC3D_Real2(EzTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
    HxTransformNearYScaRe = MALLOC3D_Real2(HxTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
    HyTransformNearYScaRe = MALLOC3D_Real2(HyTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
    HzTransformNearYScaRe = MALLOC3D_Real2(HzTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);

    hipMalloc(&ExTransformNearYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
    hipMalloc(&EzTransformNearYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
    hipMalloc(&HxTransformNearYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
    hipMalloc(&HzTransformNearYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));

    ExTransformNearXScaRe = MALLOC3D_Real2(ExTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    EyTransformNearXScaRe = MALLOC3D_Real2(EyTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    EzTransformNearXScaRe = MALLOC3D_Real2(EzTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    HxTransformNearXScaRe = MALLOC3D_Real2(HxTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    HyTransformNearXScaRe = MALLOC3D_Real2(HyTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    HzTransformNearXScaRe = MALLOC3D_Real2(HzTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);

    hipMalloc(&EyTransformNearXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
    hipMalloc(&EzTransformNearXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
    hipMalloc(&HyTransformNearXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
    hipMalloc(&HzTransformNearXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));


        ExTransformNearZScaIm = MALLOC3D_Real2(ExTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        EyTransformNearZScaIm = MALLOC3D_Real2(EyTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        EzTransformNearZScaIm = MALLOC3D_Real2(EzTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        HxTransformNearZScaIm = MALLOC3D_Real2(HxTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        HyTransformNearZScaIm = MALLOC3D_Real2(HyTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        HzTransformNearZScaIm = MALLOC3D_Real2(HzTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);

        hipMalloc(&ExTransformNearZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));
        hipMalloc(&EyTransformNearZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));
        hipMalloc(&HxTransformNearZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));
        hipMalloc(&HyTransformNearZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));

        ExTransformNearYScaIm = MALLOC3D_Real2(ExTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        EyTransformNearYScaIm = MALLOC3D_Real2(EyTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        EzTransformNearYScaIm = MALLOC3D_Real2(EzTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HxTransformNearYScaIm = MALLOC3D_Real2(HxTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HyTransformNearYScaIm = MALLOC3D_Real2(HyTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HzTransformNearYScaIm = MALLOC3D_Real2(HzTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);

        hipMalloc(&ExTransformNearYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
        hipMalloc(&EzTransformNearYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
        hipMalloc(&HxTransformNearYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
        hipMalloc(&HzTransformNearYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));

        ExTransformNearXScaIm = MALLOC3D_Real2(ExTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        EyTransformNearXScaIm = MALLOC3D_Real2(EyTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        EzTransformNearXScaIm = MALLOC3D_Real2(EzTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HxTransformNearXScaIm = MALLOC3D_Real2(HxTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HyTransformNearXScaIm = MALLOC3D_Real2(HyTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HzTransformNearXScaIm = MALLOC3D_Real2(HzTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);

        hipMalloc(&EyTransformNearXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
        hipMalloc(&EzTransformNearXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
        hipMalloc(&HyTransformNearXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
        hipMalloc(&HzTransformNearXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));


    ExTransformNearZAbsRe = MALLOC3D_Real2(ExTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    EyTransformNearZAbsRe = MALLOC3D_Real2(EyTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    EzTransformNearZAbsRe = MALLOC3D_Real2(EzTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    HxTransformNearZAbsRe = MALLOC3D_Real2(HxTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    HyTransformNearZAbsRe = MALLOC3D_Real2(HyTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    HzTransformNearZAbsRe = MALLOC3D_Real2(HzTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);

    hipMalloc(&ExTransformNearZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));
    hipMalloc(&EyTransformNearZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));
    hipMalloc(&HxTransformNearZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));
    hipMalloc(&HyTransformNearZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));

    ExTransformNearYAbsRe = MALLOC3D_Real2(ExTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EyTransformNearYAbsRe = MALLOC3D_Real2(EyTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EzTransformNearYAbsRe = MALLOC3D_Real2(EzTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HxTransformNearYAbsRe = MALLOC3D_Real2(HxTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HyTransformNearYAbsRe = MALLOC3D_Real2(HyTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HzTransformNearYAbsRe = MALLOC3D_Real2(HzTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

    hipMalloc(&ExTransformNearYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&EzTransformNearYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HxTransformNearYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HzTransformNearYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));

    ExTransformNearXAbsRe = MALLOC3D_Real2(ExTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EyTransformNearXAbsRe = MALLOC3D_Real2(EyTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EzTransformNearXAbsRe = MALLOC3D_Real2(EzTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HxTransformNearXAbsRe = MALLOC3D_Real2(HxTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HyTransformNearXAbsRe = MALLOC3D_Real2(HyTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HzTransformNearXAbsRe = MALLOC3D_Real2(HzTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

    hipMalloc(&EyTransformNearXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&EzTransformNearXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HyTransformNearXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HzTransformNearXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));

    ExTransformNearZAbsIm = MALLOC3D_Real2(ExTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    EyTransformNearZAbsIm = MALLOC3D_Real2(EyTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    EzTransformNearZAbsIm = MALLOC3D_Real2(EzTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    HxTransformNearZAbsIm = MALLOC3D_Real2(HxTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    HyTransformNearZAbsIm = MALLOC3D_Real2(HyTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    HzTransformNearZAbsIm = MALLOC3D_Real2(HzTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);

    hipMalloc(&ExTransformNearZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));
    hipMalloc(&EyTransformNearZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));
    hipMalloc(&HxTransformNearZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));
    hipMalloc(&HyTransformNearZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));

    ExTransformNearYAbsIm = MALLOC3D_Real2(ExTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EyTransformNearYAbsIm = MALLOC3D_Real2(EyTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EzTransformNearYAbsIm = MALLOC3D_Real2(EzTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HxTransformNearYAbsIm = MALLOC3D_Real2(HxTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HyTransformNearYAbsIm = MALLOC3D_Real2(HyTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HzTransformNearYAbsIm = MALLOC3D_Real2(HzTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

    hipMalloc(&ExTransformNearYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&EzTransformNearYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HxTransformNearYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HzTransformNearYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));

    ExTransformNearXAbsIm = MALLOC3D_Real2(ExTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EyTransformNearXAbsIm = MALLOC3D_Real2(EyTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EzTransformNearXAbsIm = MALLOC3D_Real2(EzTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HxTransformNearXAbsIm = MALLOC3D_Real2(HxTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HyTransformNearXAbsIm = MALLOC3D_Real2(HyTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HzTransformNearXAbsIm = MALLOC3D_Real2(HzTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

    hipMalloc(&EyTransformNearXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&EzTransformNearXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HyTransformNearXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HzTransformNearXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));

    // ExTransformFarZScaRe = MALLOC3D_Real2(ExTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
    // EyTransformFarZScaRe = MALLOC3D_Real2(EyTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
    // EzTransformFarZScaRe = MALLOC3D_Real2(EzTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
    // HxTransformFarZScaRe = MALLOC3D_Real2(HxTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
    // HyTransformFarZScaRe = MALLOC3D_Real2(HyTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
    // HzTransformFarZScaRe = MALLOC3D_Real2(HzTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
    //
    // ExTransformFarYScaRe = MALLOC3D_Real2(ExTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // EyTransformFarYScaRe = MALLOC3D_Real2(EyTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // EzTransformFarYScaRe = MALLOC3D_Real2(EzTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // HxTransformFarYScaRe = MALLOC3D_Real2(HxTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // HyTransformFarYScaRe = MALLOC3D_Real2(HyTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // HzTransformFarYScaRe = MALLOC3D_Real2(HzTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    //
    // ExTransformFarXScaRe = MALLOC3D_Real2(ExTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // EyTransformFarXScaRe = MALLOC3D_Real2(EyTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // EzTransformFarXScaRe = MALLOC3D_Real2(EzTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // HxTransformFarXScaRe = MALLOC3D_Real2(HxTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // HyTransformFarXScaRe = MALLOC3D_Real2(HyTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // HzTransformFarXScaRe = MALLOC3D_Real2(HzTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    //
    //
    //
    // ExTransformFarZScaIm = MALLOC3D_Real2(ExTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
    // EyTransformFarZScaIm = MALLOC3D_Real2(EyTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
    // EzTransformFarZScaIm = MALLOC3D_Real2(EzTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
    // HxTransformFarZScaIm = MALLOC3D_Real2(HxTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
    // HyTransformFarZScaIm = MALLOC3D_Real2(HyTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
    // HzTransformFarZScaIm = MALLOC3D_Real2(HzTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
    //
    // ExTransformFarYScaIm = MALLOC3D_Real2(ExTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // EyTransformFarYScaIm = MALLOC3D_Real2(EyTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // EzTransformFarYScaIm = MALLOC3D_Real2(EzTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // HxTransformFarYScaIm = MALLOC3D_Real2(HxTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // HyTransformFarYScaIm = MALLOC3D_Real2(HyTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // HzTransformFarYScaIm = MALLOC3D_Real2(HzTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    //
    // ExTransformFarXScaIm = MALLOC3D_Real2(ExTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // EyTransformFarXScaIm = MALLOC3D_Real2(EyTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // EzTransformFarXScaIm = MALLOC3D_Real2(EzTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // HxTransformFarXScaIm = MALLOC3D_Real2(HxTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // HyTransformFarXScaIm = MALLOC3D_Real2(HyTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    // HzTransformFarXScaIm = MALLOC3D_Real2(HzTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    //
    //
    // ExTransformFarZAbsRe = MALLOC3D_Real2(ExTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    // EyTransformFarZAbsRe = MALLOC3D_Real2(EyTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    // EzTransformFarZAbsRe = MALLOC3D_Real2(EzTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    // HxTransformFarZAbsRe = MALLOC3D_Real2(HxTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    // HyTransformFarZAbsRe = MALLOC3D_Real2(HyTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    // HzTransformFarZAbsRe = MALLOC3D_Real2(HzTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    //
    // ExTransformFarYAbsRe = MALLOC3D_Real2(ExTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    // EyTransformFarYAbsRe = MALLOC3D_Real2(EyTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    // EzTransformFarYAbsRe = MALLOC3D_Real2(EzTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    // HxTransformFarYAbsRe = MALLOC3D_Real2(HxTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    // HyTransformFarYAbsRe = MALLOC3D_Real2(HyTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    // HzTransformFarYAbsRe = MALLOC3D_Real2(HzTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //
    // ExTransformFarXAbsRe = MALLOC3D_Real2(ExTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    // EyTransformFarXAbsRe = MALLOC3D_Real2(EyTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    // EzTransformFarXAbsRe = MALLOC3D_Real2(EzTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    // HxTransformFarXAbsRe = MALLOC3D_Real2(HxTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    // HyTransformFarXAbsRe = MALLOC3D_Real2(HyTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    // HzTransformFarXAbsRe = MALLOC3D_Real2(HzTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //
    //
    //
    //     ExTransformFarZAbsIm = MALLOC3D_Real2(ExTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    //     EyTransformFarZAbsIm = MALLOC3D_Real2(EyTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    //     EzTransformFarZAbsIm = MALLOC3D_Real2(EzTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    //     HxTransformFarZAbsIm = MALLOC3D_Real2(HxTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    //     HyTransformFarZAbsIm = MALLOC3D_Real2(HyTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    //     HzTransformFarZAbsIm = MALLOC3D_Real2(HzTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    //
    //     ExTransformFarYAbsIm = MALLOC3D_Real2(ExTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //     EyTransformFarYAbsIm = MALLOC3D_Real2(EyTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //     EzTransformFarYAbsIm = MALLOC3D_Real2(EzTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //     HxTransformFarYAbsIm = MALLOC3D_Real2(HxTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //     HyTransformFarYAbsIm = MALLOC3D_Real2(HyTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //     HzTransformFarYAbsIm = MALLOC3D_Real2(HzTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //
    //     ExTransformFarXAbsIm = MALLOC3D_Real2(ExTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //     EyTransformFarXAbsIm = MALLOC3D_Real2(EyTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //     EzTransformFarXAbsIm = MALLOC3D_Real2(EzTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //     HxTransformFarXAbsIm = MALLOC3D_Real2(HxTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //     HyTransformFarXAbsIm = MALLOC3D_Real2(HyTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //     HzTransformFarXAbsIm = MALLOC3D_Real2(HzTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    //




    ExTransformFarZScaRe = MALLOC3D_Real2(ExTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
    EyTransformFarZScaRe = MALLOC3D_Real2(EyTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
    EzTransformFarZScaRe = MALLOC3D_Real2(EzTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
    HxTransformFarZScaRe = MALLOC3D_Real2(HxTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
    HyTransformFarZScaRe = MALLOC3D_Real2(HyTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
    HzTransformFarZScaRe = MALLOC3D_Real2(HzTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);

    hipMalloc(&ExTransformFarZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));
    hipMalloc(&EyTransformFarZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));
    hipMalloc(&HxTransformFarZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));
    hipMalloc(&HyTransformFarZScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));

    ExTransformFarYScaRe = MALLOC3D_Real2(ExTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
    EyTransformFarYScaRe = MALLOC3D_Real2(EyTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
    EzTransformFarYScaRe = MALLOC3D_Real2(EzTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
    HxTransformFarYScaRe = MALLOC3D_Real2(HxTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
    HyTransformFarYScaRe = MALLOC3D_Real2(HyTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
    HzTransformFarYScaRe = MALLOC3D_Real2(HzTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);

    hipMalloc(&ExTransformFarYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
    hipMalloc(&EzTransformFarYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
    hipMalloc(&HxTransformFarYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
    hipMalloc(&HzTransformFarYScaRedev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));

    ExTransformFarXScaRe = MALLOC3D_Real2(ExTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    EyTransformFarXScaRe = MALLOC3D_Real2(EyTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    EzTransformFarXScaRe = MALLOC3D_Real2(EzTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    HxTransformFarXScaRe = MALLOC3D_Real2(HxTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    HyTransformFarXScaRe = MALLOC3D_Real2(HyTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
    HzTransformFarXScaRe = MALLOC3D_Real2(HzTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);

    hipMalloc(&EyTransformFarXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
    hipMalloc(&EzTransformFarXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
    hipMalloc(&HyTransformFarXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
    hipMalloc(&HzTransformFarXScaRedev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));


        ExTransformFarZScaIm = MALLOC3D_Real2(ExTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        EyTransformFarZScaIm = MALLOC3D_Real2(EyTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        EzTransformFarZScaIm = MALLOC3D_Real2(EzTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        HxTransformFarZScaIm = MALLOC3D_Real2(HxTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        HyTransformFarZScaIm = MALLOC3D_Real2(HyTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        HzTransformFarZScaIm = MALLOC3D_Real2(HzTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);

        hipMalloc(&ExTransformFarZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));
        hipMalloc(&EyTransformFarZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));
        hipMalloc(&HxTransformFarZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));
        hipMalloc(&HyTransformFarZScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1));

        ExTransformFarYScaIm = MALLOC3D_Real2(ExTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        EyTransformFarYScaIm = MALLOC3D_Real2(EyTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        EzTransformFarYScaIm = MALLOC3D_Real2(EzTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HxTransformFarYScaIm = MALLOC3D_Real2(HxTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HyTransformFarYScaIm = MALLOC3D_Real2(HyTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HzTransformFarYScaIm = MALLOC3D_Real2(HzTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);

        hipMalloc(&ExTransformFarYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
        hipMalloc(&EzTransformFarYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
        hipMalloc(&HxTransformFarYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
        hipMalloc(&HzTransformFarYScaImdev,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1));

        ExTransformFarXScaIm = MALLOC3D_Real2(ExTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        EyTransformFarXScaIm = MALLOC3D_Real2(EyTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        EzTransformFarXScaIm = MALLOC3D_Real2(EzTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HxTransformFarXScaIm = MALLOC3D_Real2(HxTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HyTransformFarXScaIm = MALLOC3D_Real2(HyTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HzTransformFarXScaIm = MALLOC3D_Real2(HzTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);

        hipMalloc(&EyTransformFarXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
        hipMalloc(&EzTransformFarXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
        hipMalloc(&HyTransformFarXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));
        hipMalloc(&HzTransformFarXScaImdev,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1));


    ExTransformFarZAbsRe = MALLOC3D_Real2(ExTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    EyTransformFarZAbsRe = MALLOC3D_Real2(EyTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    EzTransformFarZAbsRe = MALLOC3D_Real2(EzTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    HxTransformFarZAbsRe = MALLOC3D_Real2(HxTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    HyTransformFarZAbsRe = MALLOC3D_Real2(HyTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    HzTransformFarZAbsRe = MALLOC3D_Real2(HzTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);

    hipMalloc(&ExTransformFarZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));
    hipMalloc(&EyTransformFarZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));
    hipMalloc(&HxTransformFarZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));
    hipMalloc(&HyTransformFarZAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));

    ExTransformFarYAbsRe = MALLOC3D_Real2(ExTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EyTransformFarYAbsRe = MALLOC3D_Real2(EyTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EzTransformFarYAbsRe = MALLOC3D_Real2(EzTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HxTransformFarYAbsRe = MALLOC3D_Real2(HxTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HyTransformFarYAbsRe = MALLOC3D_Real2(HyTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HzTransformFarYAbsRe = MALLOC3D_Real2(HzTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

    hipMalloc(&ExTransformFarYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&EzTransformFarYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HxTransformFarYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HzTransformFarYAbsRedev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));

    ExTransformFarXAbsRe = MALLOC3D_Real2(ExTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EyTransformFarXAbsRe = MALLOC3D_Real2(EyTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EzTransformFarXAbsRe = MALLOC3D_Real2(EzTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HxTransformFarXAbsRe = MALLOC3D_Real2(HxTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HyTransformFarXAbsRe = MALLOC3D_Real2(HyTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HzTransformFarXAbsRe = MALLOC3D_Real2(HzTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

    hipMalloc(&EyTransformFarXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&EzTransformFarXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HyTransformFarXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HzTransformFarXAbsRedev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));

    ExTransformFarZAbsIm = MALLOC3D_Real2(ExTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    EyTransformFarZAbsIm = MALLOC3D_Real2(EyTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    EzTransformFarZAbsIm = MALLOC3D_Real2(EzTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    HxTransformFarZAbsIm = MALLOC3D_Real2(HxTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    HyTransformFarZAbsIm = MALLOC3D_Real2(HyTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
    HzTransformFarZAbsIm = MALLOC3D_Real2(HzTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);

    hipMalloc(&ExTransformFarZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));
    hipMalloc(&EyTransformFarZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));
    hipMalloc(&HxTransformFarZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));
    hipMalloc(&HyTransformFarZAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1));

    ExTransformFarYAbsIm = MALLOC3D_Real2(ExTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EyTransformFarYAbsIm = MALLOC3D_Real2(EyTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EzTransformFarYAbsIm = MALLOC3D_Real2(EzTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HxTransformFarYAbsIm = MALLOC3D_Real2(HxTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HyTransformFarYAbsIm = MALLOC3D_Real2(HyTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HzTransformFarYAbsIm = MALLOC3D_Real2(HzTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

    hipMalloc(&ExTransformFarYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&EzTransformFarYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HxTransformFarYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HzTransformFarYAbsImdev,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));

    ExTransformFarXAbsIm = MALLOC3D_Real2(ExTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EyTransformFarXAbsIm = MALLOC3D_Real2(EyTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    EzTransformFarXAbsIm = MALLOC3D_Real2(EzTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HxTransformFarXAbsIm = MALLOC3D_Real2(HxTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HyTransformFarXAbsIm = MALLOC3D_Real2(HyTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
    HzTransformFarXAbsIm = MALLOC3D_Real2(HzTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

    hipMalloc(&EyTransformFarXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&EzTransformFarXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HyTransformFarXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));
    hipMalloc(&HzTransformFarXAbsImdev,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1));




        ExTransformNearZScaRe = ZERO_VECTORS3D_Real2(ExTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
        EyTransformNearZScaRe = ZERO_VECTORS3D_Real2(EyTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
        EzTransformNearZScaRe = ZERO_VECTORS3D_Real2(EzTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
        HxTransformNearZScaRe = ZERO_VECTORS3D_Real2(HxTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
        HyTransformNearZScaRe = ZERO_VECTORS3D_Real2(HyTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
        HzTransformNearZScaRe = ZERO_VECTORS3D_Real2(HzTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);


        hipMemcpy(ExTransformNearZScaRedev,ExTransformNearZScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);
        hipMemcpy(EyTransformNearZScaRedev,EyTransformNearZScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);
        hipMemcpy(HxTransformNearZScaRedev,HxTransformNearZScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);
        hipMemcpy(HyTransformNearZScaRedev,HyTransformNearZScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);

        ExTransformNearYScaRe = ZERO_VECTORS3D_Real2(ExTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
        EyTransformNearYScaRe = ZERO_VECTORS3D_Real2(EyTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
        EzTransformNearYScaRe = ZERO_VECTORS3D_Real2(EzTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
        HxTransformNearYScaRe = ZERO_VECTORS3D_Real2(HxTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
        HyTransformNearYScaRe = ZERO_VECTORS3D_Real2(HyTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
        HzTransformNearYScaRe = ZERO_VECTORS3D_Real2(HzTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);

        hipMemcpy(ExTransformNearYScaRedev,ExTransformNearYScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
        hipMemcpy(EzTransformNearYScaRedev,EzTransformNearYScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
        hipMemcpy(HxTransformNearYScaRedev,HxTransformNearYScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
        hipMemcpy(HzTransformNearYScaRedev,HzTransformNearYScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);

        ExTransformNearXScaRe = ZERO_VECTORS3D_Real2(ExTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        EyTransformNearXScaRe = ZERO_VECTORS3D_Real2(EyTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        EzTransformNearXScaRe = ZERO_VECTORS3D_Real2(EzTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HxTransformNearXScaRe = ZERO_VECTORS3D_Real2(HxTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HyTransformNearXScaRe = ZERO_VECTORS3D_Real2(HyTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        HzTransformNearXScaRe = ZERO_VECTORS3D_Real2(HzTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);

        hipMemcpy(EyTransformNearXScaRedev,EyTransformNearXScaRe,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
        hipMemcpy(EzTransformNearXScaRedev,EzTransformNearXScaRe,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
        hipMemcpy(HyTransformNearXScaRedev,HyTransformNearXScaRe,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
        hipMemcpy(HzTransformNearXScaRedev,HzTransformNearXScaRe,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);

            ExTransformNearZScaIm = ZERO_VECTORS3D_Real2(ExTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
            EyTransformNearZScaIm = ZERO_VECTORS3D_Real2(EyTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
            EzTransformNearZScaIm = ZERO_VECTORS3D_Real2(EzTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
            HxTransformNearZScaIm = ZERO_VECTORS3D_Real2(HxTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
            HyTransformNearZScaIm = ZERO_VECTORS3D_Real2(HyTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
            HzTransformNearZScaIm = ZERO_VECTORS3D_Real2(HzTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);

            hipMemcpy(ExTransformNearZScaImdev,ExTransformNearZScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);
            hipMemcpy(EyTransformNearZScaImdev,EyTransformNearZScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);
            hipMemcpy(HxTransformNearZScaImdev,HxTransformNearZScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);
            hipMemcpy(HyTransformNearZScaImdev,HyTransformNearZScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);

            ExTransformNearYScaIm = ZERO_VECTORS3D_Real2(ExTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
            EyTransformNearYScaIm = ZERO_VECTORS3D_Real2(EyTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
            EzTransformNearYScaIm = ZERO_VECTORS3D_Real2(EzTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
            HxTransformNearYScaIm = ZERO_VECTORS3D_Real2(HxTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
            HyTransformNearYScaIm = ZERO_VECTORS3D_Real2(HyTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
            HzTransformNearYScaIm = ZERO_VECTORS3D_Real2(HzTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);

            hipMemcpy(ExTransformNearYScaImdev,ExTransformNearYScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
            hipMemcpy(EzTransformNearYScaImdev,EzTransformNearYScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
            hipMemcpy(HxTransformNearYScaImdev,HxTransformNearYScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
            hipMemcpy(HzTransformNearYScaImdev,HzTransformNearYScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);

            ExTransformNearXScaIm = ZERO_VECTORS3D_Real2(ExTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
            EyTransformNearXScaIm = ZERO_VECTORS3D_Real2(EyTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
            EzTransformNearXScaIm = ZERO_VECTORS3D_Real2(EzTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
            HxTransformNearXScaIm = ZERO_VECTORS3D_Real2(HxTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
            HyTransformNearXScaIm = ZERO_VECTORS3D_Real2(HyTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
            HzTransformNearXScaIm = ZERO_VECTORS3D_Real2(HzTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);

            hipMemcpy(EyTransformNearXScaImdev,EyTransformNearXScaIm,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
            hipMemcpy(EzTransformNearXScaImdev,EzTransformNearXScaIm,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
            hipMemcpy(HyTransformNearXScaImdev,HyTransformNearXScaIm,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
            hipMemcpy(HzTransformNearXScaImdev,HzTransformNearXScaIm,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);

        // ExTransformNearZAbsRe = ZERO_VECTORS3D_Real2(ExTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // EyTransformNearZAbsRe = ZERO_VECTORS3D_Real2(EyTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // EzTransformNearZAbsRe = ZERO_VECTORS3D_Real2(EzTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // HxTransformNearZAbsRe = ZERO_VECTORS3D_Real2(HxTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // HyTransformNearZAbsRe = ZERO_VECTORS3D_Real2(HyTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // HzTransformNearZAbsRe = ZERO_VECTORS3D_Real2(HzTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        //
        // ExTransformNearYAbsRe = ZERO_VECTORS3D_Real2(ExTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // EyTransformNearYAbsRe = ZERO_VECTORS3D_Real2(EyTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // EzTransformNearYAbsRe = ZERO_VECTORS3D_Real2(EzTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HxTransformNearYAbsRe = ZERO_VECTORS3D_Real2(HxTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HyTransformNearYAbsRe = ZERO_VECTORS3D_Real2(HyTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HzTransformNearYAbsRe = ZERO_VECTORS3D_Real2(HzTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //
        // ExTransformNearXAbsRe = ZERO_VECTORS3D_Real2(ExTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // EyTransformNearXAbsRe = ZERO_VECTORS3D_Real2(EyTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // EzTransformNearXAbsRe = ZERO_VECTORS3D_Real2(EzTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HxTransformNearXAbsRe = ZERO_VECTORS3D_Real2(HxTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HyTransformNearXAbsRe = ZERO_VECTORS3D_Real2(HyTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HzTransformNearXAbsRe = ZERO_VECTORS3D_Real2(HzTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //
        //
        // ExTransformNearZAbsIm = ZERO_VECTORS3D_Real2(ExTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // EyTransformNearZAbsIm = ZERO_VECTORS3D_Real2(EyTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // EzTransformNearZAbsIm = ZERO_VECTORS3D_Real2(EzTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // HxTransformNearZAbsIm = ZERO_VECTORS3D_Real2(HxTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // HyTransformNearZAbsIm = ZERO_VECTORS3D_Real2(HyTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // HzTransformNearZAbsIm = ZERO_VECTORS3D_Real2(HzTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        //
        // ExTransformNearYAbsIm = ZERO_VECTORS3D_Real2(ExTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // EyTransformNearYAbsIm = ZERO_VECTORS3D_Real2(EyTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // EzTransformNearYAbsIm = ZERO_VECTORS3D_Real2(EzTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HxTransformNearYAbsIm = ZERO_VECTORS3D_Real2(HxTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HyTransformNearYAbsIm = ZERO_VECTORS3D_Real2(HyTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HzTransformNearYAbsIm = ZERO_VECTORS3D_Real2(HzTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //
        // ExTransformNearXAbsIm = ZERO_VECTORS3D_Real2(ExTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // EyTransformNearXAbsIm = ZERO_VECTORS3D_Real2(EyTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // EzTransformNearXAbsIm = ZERO_VECTORS3D_Real2(EzTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HxTransformNearXAbsIm = ZERO_VECTORS3D_Real2(HxTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HyTransformNearXAbsIm = ZERO_VECTORS3D_Real2(HyTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HzTransformNearXAbsIm = ZERO_VECTORS3D_Real2(HzTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);













        ExTransformNearZAbsRe = ZERO_VECTORS3D_Real2(ExTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,YENDAbs - YSTARTAbs+1);
        EyTransformNearZAbsRe = ZERO_VECTORS3D_Real2(EyTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,YENDAbs - YSTARTAbs+1);
        EzTransformNearZAbsRe = ZERO_VECTORS3D_Real2(EzTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,YENDAbs - YSTARTAbs+1);
        HxTransformNearZAbsRe = ZERO_VECTORS3D_Real2(HxTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,YENDAbs - YSTARTAbs+1);
        HyTransformNearZAbsRe = ZERO_VECTORS3D_Real2(HyTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,YENDAbs - YSTARTAbs+1);
        HzTransformNearZAbsRe = ZERO_VECTORS3D_Real2(HzTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,YENDAbs - YSTARTAbs+1);


        hipMemcpy(ExTransformNearZAbsRedev,ExTransformNearZAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);
        hipMemcpy(EyTransformNearZAbsRedev,EyTransformNearZAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);
        hipMemcpy(HxTransformNearZAbsRedev,HxTransformNearZAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);
        hipMemcpy(HyTransformNearZAbsRedev,HyTransformNearZAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);

        ExTransformNearYAbsRe = ZERO_VECTORS3D_Real2(ExTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,ZENDAbs - ZSTARTAbs+1);
        EyTransformNearYAbsRe = ZERO_VECTORS3D_Real2(EyTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,ZENDAbs - ZSTARTAbs+1);
        EzTransformNearYAbsRe = ZERO_VECTORS3D_Real2(EzTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,ZENDAbs - ZSTARTAbs+1);
        HxTransformNearYAbsRe = ZERO_VECTORS3D_Real2(HxTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,ZENDAbs - ZSTARTAbs+1);
        HyTransformNearYAbsRe = ZERO_VECTORS3D_Real2(HyTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,ZENDAbs - ZSTARTAbs+1);
        HzTransformNearYAbsRe = ZERO_VECTORS3D_Real2(HzTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,ZENDAbs - ZSTARTAbs+1);

        hipMemcpy(ExTransformNearYAbsRedev,ExTransformNearYAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
        hipMemcpy(EzTransformNearYAbsRedev,EzTransformNearYAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
        hipMemcpy(HxTransformNearYAbsRedev,HxTransformNearYAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
        hipMemcpy(HzTransformNearYAbsRedev,HzTransformNearYAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);

        ExTransformNearXAbsRe = ZERO_VECTORS3D_Real2(ExTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        EyTransformNearXAbsRe = ZERO_VECTORS3D_Real2(EyTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        EzTransformNearXAbsRe = ZERO_VECTORS3D_Real2(EzTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        HxTransformNearXAbsRe = ZERO_VECTORS3D_Real2(HxTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        HyTransformNearXAbsRe = ZERO_VECTORS3D_Real2(HyTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        HzTransformNearXAbsRe = ZERO_VECTORS3D_Real2(HzTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

        hipMemcpy(EyTransformNearXAbsRedev,EyTransformNearXAbsRe,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
        hipMemcpy(EzTransformNearXAbsRedev,EzTransformNearXAbsRe,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
        hipMemcpy(HyTransformNearXAbsRedev,HyTransformNearXAbsRe,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
        hipMemcpy(HzTransformNearXAbsRedev,HzTransformNearXAbsRe,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);

            ExTransformNearZAbsIm = ZERO_VECTORS3D_Real2(ExTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
            EyTransformNearZAbsIm = ZERO_VECTORS3D_Real2(EyTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
            EzTransformNearZAbsIm = ZERO_VECTORS3D_Real2(EzTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
            HxTransformNearZAbsIm = ZERO_VECTORS3D_Real2(HxTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
            HyTransformNearZAbsIm = ZERO_VECTORS3D_Real2(HyTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
            HzTransformNearZAbsIm = ZERO_VECTORS3D_Real2(HzTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);

            hipMemcpy(ExTransformNearZAbsImdev,ExTransformNearZAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);
            hipMemcpy(EyTransformNearZAbsImdev,EyTransformNearZAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);
            hipMemcpy(HxTransformNearZAbsImdev,HxTransformNearZAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);
            hipMemcpy(HyTransformNearZAbsImdev,HyTransformNearZAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);

            ExTransformNearYAbsIm = ZERO_VECTORS3D_Real2(ExTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
            EyTransformNearYAbsIm = ZERO_VECTORS3D_Real2(EyTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
            EzTransformNearYAbsIm = ZERO_VECTORS3D_Real2(EzTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
            HxTransformNearYAbsIm = ZERO_VECTORS3D_Real2(HxTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
            HyTransformNearYAbsIm = ZERO_VECTORS3D_Real2(HyTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
            HzTransformNearYAbsIm = ZERO_VECTORS3D_Real2(HzTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

            hipMemcpy(ExTransformNearYAbsImdev,ExTransformNearYAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
            hipMemcpy(EzTransformNearYAbsImdev,EzTransformNearYAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
            hipMemcpy(HxTransformNearYAbsImdev,HxTransformNearYAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
            hipMemcpy(HzTransformNearYAbsImdev,HzTransformNearYAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);

            ExTransformNearXAbsIm = ZERO_VECTORS3D_Real2(ExTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
            EyTransformNearXAbsIm = ZERO_VECTORS3D_Real2(EyTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
            EzTransformNearXAbsIm = ZERO_VECTORS3D_Real2(EzTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
            HxTransformNearXAbsIm = ZERO_VECTORS3D_Real2(HxTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
            HyTransformNearXAbsIm = ZERO_VECTORS3D_Real2(HyTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
            HzTransformNearXAbsIm = ZERO_VECTORS3D_Real2(HzTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

            hipMemcpy(EyTransformNearXAbsImdev,EyTransformNearXAbsIm,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
            hipMemcpy(EzTransformNearXAbsImdev,EzTransformNearXAbsIm,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
            hipMemcpy(HyTransformNearXAbsImdev,HyTransformNearXAbsIm,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
            hipMemcpy(HzTransformNearXAbsImdev,HzTransformNearXAbsIm,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);










        // ExTransformFarZScaRe = ZERO_VECTORS3D_Real2(ExTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        // EyTransformFarZScaRe = ZERO_VECTORS3D_Real2(EyTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        // EzTransformFarZScaRe = ZERO_VECTORS3D_Real2(EzTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        // HxTransformFarZScaRe = ZERO_VECTORS3D_Real2(HxTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        // HyTransformFarZScaRe = ZERO_VECTORS3D_Real2(HyTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        // HzTransformFarZScaRe = ZERO_VECTORS3D_Real2(HzTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        //
        // ExTransformFarYScaRe = ZERO_VECTORS3D_Real2(ExTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // EyTransformFarYScaRe = ZERO_VECTORS3D_Real2(EyTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // EzTransformFarYScaRe = ZERO_VECTORS3D_Real2(EzTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // HxTransformFarYScaRe = ZERO_VECTORS3D_Real2(HxTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // HyTransformFarYScaRe = ZERO_VECTORS3D_Real2(HyTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // HzTransformFarYScaRe = ZERO_VECTORS3D_Real2(HzTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        //
        // ExTransformFarXScaRe = ZERO_VECTORS3D_Real2(ExTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // EyTransformFarXScaRe = ZERO_VECTORS3D_Real2(EyTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // EzTransformFarXScaRe = ZERO_VECTORS3D_Real2(EzTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // HxTransformFarXScaRe = ZERO_VECTORS3D_Real2(HxTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // HyTransformFarXScaRe = ZERO_VECTORS3D_Real2(HyTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // HzTransformFarXScaRe = ZERO_VECTORS3D_Real2(HzTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        //
        //
        //
        // ExTransformFarZScaIm = ZERO_VECTORS3D_Real2(ExTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        // EyTransformFarZScaIm = ZERO_VECTORS3D_Real2(EyTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        // EzTransformFarZScaIm = ZERO_VECTORS3D_Real2(EzTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        // HxTransformFarZScaIm = ZERO_VECTORS3D_Real2(HxTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        // HyTransformFarZScaIm = ZERO_VECTORS3D_Real2(HyTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        // HzTransformFarZScaIm = ZERO_VECTORS3D_Real2(HzTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
        //
        // ExTransformFarYScaIm = ZERO_VECTORS3D_Real2(ExTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // EyTransformFarYScaIm = ZERO_VECTORS3D_Real2(EyTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // EzTransformFarYScaIm = ZERO_VECTORS3D_Real2(EzTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // HxTransformFarYScaIm = ZERO_VECTORS3D_Real2(HxTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // HyTransformFarYScaIm = ZERO_VECTORS3D_Real2(HyTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // HzTransformFarYScaIm = ZERO_VECTORS3D_Real2(HzTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        //
        // ExTransformFarXScaIm = ZERO_VECTORS3D_Real2(ExTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // EyTransformFarXScaIm = ZERO_VECTORS3D_Real2(EyTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // EzTransformFarXScaIm = ZERO_VECTORS3D_Real2(EzTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // HxTransformFarXScaIm = ZERO_VECTORS3D_Real2(HxTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // HyTransformFarXScaIm = ZERO_VECTORS3D_Real2(HyTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        // HzTransformFarXScaIm = ZERO_VECTORS3D_Real2(HzTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
        //
        //
        // ExTransformFarZAbsRe = ZERO_VECTORS3D_Real2(ExTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // EyTransformFarZAbsRe = ZERO_VECTORS3D_Real2(EyTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // EzTransformFarZAbsRe = ZERO_VECTORS3D_Real2(EzTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // HxTransformFarZAbsRe = ZERO_VECTORS3D_Real2(HxTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // HyTransformFarZAbsRe = ZERO_VECTORS3D_Real2(HyTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        // HzTransformFarZAbsRe = ZERO_VECTORS3D_Real2(HzTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        //
        // ExTransformFarYAbsRe = ZERO_VECTORS3D_Real2(ExTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // EyTransformFarYAbsRe = ZERO_VECTORS3D_Real2(EyTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // EzTransformFarYAbsRe = ZERO_VECTORS3D_Real2(EzTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HxTransformFarYAbsRe = ZERO_VECTORS3D_Real2(HxTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HyTransformFarYAbsRe = ZERO_VECTORS3D_Real2(HyTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HzTransformFarYAbsRe = ZERO_VECTORS3D_Real2(HzTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //
        // ExTransformFarXAbsRe = ZERO_VECTORS3D_Real2(ExTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // EyTransformFarXAbsRe = ZERO_VECTORS3D_Real2(EyTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // EzTransformFarXAbsRe = ZERO_VECTORS3D_Real2(EzTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HxTransformFarXAbsRe = ZERO_VECTORS3D_Real2(HxTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HyTransformFarXAbsRe = ZERO_VECTORS3D_Real2(HyTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        // HzTransformFarXAbsRe = ZERO_VECTORS3D_Real2(HzTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //
        //
        //
        //     ExTransformFarZAbsIm = ZERO_VECTORS3D_Real2(ExTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        //     EyTransformFarZAbsIm = ZERO_VECTORS3D_Real2(EyTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        //     EzTransformFarZAbsIm = ZERO_VECTORS3D_Real2(EzTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        //     HxTransformFarZAbsIm = ZERO_VECTORS3D_Real2(HxTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        //     HyTransformFarZAbsIm = ZERO_VECTORS3D_Real2(HyTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        //     HzTransformFarZAbsIm = ZERO_VECTORS3D_Real2(HzTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
        //
        //     ExTransformFarYAbsIm = ZERO_VECTORS3D_Real2(ExTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //     EyTransformFarYAbsIm = ZERO_VECTORS3D_Real2(EyTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //     EzTransformFarYAbsIm = ZERO_VECTORS3D_Real2(EzTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //     HxTransformFarYAbsIm = ZERO_VECTORS3D_Real2(HxTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //     HyTransformFarYAbsIm = ZERO_VECTORS3D_Real2(HyTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //     HzTransformFarYAbsIm = ZERO_VECTORS3D_Real2(HzTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //
        //     ExTransformFarXAbsIm = ZERO_VECTORS3D_Real2(ExTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //     EyTransformFarXAbsIm = ZERO_VECTORS3D_Real2(EyTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //     EzTransformFarXAbsIm = ZERO_VECTORS3D_Real2(EzTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //     HxTransformFarXAbsIm = ZERO_VECTORS3D_Real2(HxTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //     HyTransformFarXAbsIm = ZERO_VECTORS3D_Real2(HyTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
        //     HzTransformFarXAbsIm = ZERO_VECTORS3D_Real2(HzTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);



                ExTransformFarZScaRe = ZERO_VECTORS3D_Real2(ExTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
                EyTransformFarZScaRe = ZERO_VECTORS3D_Real2(EyTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
                EzTransformFarZScaRe = ZERO_VECTORS3D_Real2(EzTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
                HxTransformFarZScaRe = ZERO_VECTORS3D_Real2(HxTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
                HyTransformFarZScaRe = ZERO_VECTORS3D_Real2(HyTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);
                HzTransformFarZScaRe = ZERO_VECTORS3D_Real2(HzTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca +1,YENDSca - YSTARTSca+1);


                hipMemcpy(ExTransformFarZScaRedev,ExTransformFarZScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);
                hipMemcpy(EyTransformFarZScaRedev,EyTransformFarZScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);
                hipMemcpy(HxTransformFarZScaRedev,HxTransformFarZScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);
                hipMemcpy(HyTransformFarZScaRedev,HyTransformFarZScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);

                ExTransformFarYScaRe = ZERO_VECTORS3D_Real2(ExTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
                EyTransformFarYScaRe = ZERO_VECTORS3D_Real2(EyTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
                EzTransformFarYScaRe = ZERO_VECTORS3D_Real2(EzTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
                HxTransformFarYScaRe = ZERO_VECTORS3D_Real2(HxTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
                HyTransformFarYScaRe = ZERO_VECTORS3D_Real2(HyTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);
                HzTransformFarYScaRe = ZERO_VECTORS3D_Real2(HzTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca +1,ZENDSca - ZSTARTSca+1);

                hipMemcpy(ExTransformFarYScaRedev,ExTransformFarYScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
                hipMemcpy(EzTransformFarYScaRedev,EzTransformFarYScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
                hipMemcpy(HxTransformFarYScaRedev,HxTransformFarYScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
                hipMemcpy(HzTransformFarYScaRedev,HzTransformFarYScaRe,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);

                ExTransformFarXScaRe = ZERO_VECTORS3D_Real2(ExTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                EyTransformFarXScaRe = ZERO_VECTORS3D_Real2(EyTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                EzTransformFarXScaRe = ZERO_VECTORS3D_Real2(EzTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                HxTransformFarXScaRe = ZERO_VECTORS3D_Real2(HxTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                HyTransformFarXScaRe = ZERO_VECTORS3D_Real2(HyTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                HzTransformFarXScaRe = ZERO_VECTORS3D_Real2(HzTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);

                hipMemcpy(EyTransformFarXScaRedev,EyTransformFarXScaRe,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
                hipMemcpy(EzTransformFarXScaRedev,EzTransformFarXScaRe,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
                hipMemcpy(HyTransformFarXScaRedev,HyTransformFarXScaRe,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
                hipMemcpy(HzTransformFarXScaRedev,HzTransformFarXScaRe,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);

                    ExTransformFarZScaIm = ZERO_VECTORS3D_Real2(ExTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
                    EyTransformFarZScaIm = ZERO_VECTORS3D_Real2(EyTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
                    EzTransformFarZScaIm = ZERO_VECTORS3D_Real2(EzTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
                    HxTransformFarZScaIm = ZERO_VECTORS3D_Real2(HxTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
                    HyTransformFarZScaIm = ZERO_VECTORS3D_Real2(HyTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);
                    HzTransformFarZScaIm = ZERO_VECTORS3D_Real2(HzTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,YENDSca - YSTARTSca + 1);

                    hipMemcpy(ExTransformFarZScaImdev,ExTransformFarZScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);
                    hipMemcpy(EyTransformFarZScaImdev,EyTransformFarZScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);
                    hipMemcpy(HxTransformFarZScaImdev,HxTransformFarZScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);
                    hipMemcpy(HyTransformFarZScaImdev,HyTransformFarZScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(YENDSca-YSTARTSca+1),hipMemcpyHostToDevice);

                    ExTransformFarYScaIm = ZERO_VECTORS3D_Real2(ExTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                    EyTransformFarYScaIm = ZERO_VECTORS3D_Real2(EyTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                    EzTransformFarYScaIm = ZERO_VECTORS3D_Real2(EzTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                    HxTransformFarYScaIm = ZERO_VECTORS3D_Real2(HxTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                    HyTransformFarYScaIm = ZERO_VECTORS3D_Real2(HyTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                    HzTransformFarYScaIm = ZERO_VECTORS3D_Real2(HzTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);

                    hipMemcpy(ExTransformFarYScaImdev,ExTransformFarYScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
                    hipMemcpy(EzTransformFarYScaImdev,EzTransformFarYScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
                    hipMemcpy(HxTransformFarYScaImdev,HxTransformFarYScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
                    hipMemcpy(HzTransformFarYScaImdev,HzTransformFarYScaIm,sizeof(real2)*NUM_freq*(XENDSca-XSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);

                    ExTransformFarXScaIm = ZERO_VECTORS3D_Real2(ExTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                    EyTransformFarXScaIm = ZERO_VECTORS3D_Real2(EyTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                    EzTransformFarXScaIm = ZERO_VECTORS3D_Real2(EzTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                    HxTransformFarXScaIm = ZERO_VECTORS3D_Real2(HxTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                    HyTransformFarXScaIm = ZERO_VECTORS3D_Real2(HyTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);
                    HzTransformFarXScaIm = ZERO_VECTORS3D_Real2(HzTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca + 1 ,ZENDSca - ZSTARTSca + 1);

                    hipMemcpy(EyTransformFarXScaImdev,EyTransformFarXScaIm,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
                    hipMemcpy(EzTransformFarXScaImdev,EzTransformFarXScaIm,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
                    hipMemcpy(HyTransformFarXScaImdev,HyTransformFarXScaIm,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);
                    hipMemcpy(HzTransformFarXScaImdev,HzTransformFarXScaIm,sizeof(real2)*NUM_freq*(YENDSca-YSTARTSca+1)*(ZENDSca-ZSTARTSca+1),hipMemcpyHostToDevice);



                            ExTransformFarZAbsRe = ZERO_VECTORS3D_Real2(ExTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,YENDAbs - YSTARTAbs+1);
                            EyTransformFarZAbsRe = ZERO_VECTORS3D_Real2(EyTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,YENDAbs - YSTARTAbs+1);
                            EzTransformFarZAbsRe = ZERO_VECTORS3D_Real2(EzTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,YENDAbs - YSTARTAbs+1);
                            HxTransformFarZAbsRe = ZERO_VECTORS3D_Real2(HxTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,YENDAbs - YSTARTAbs+1);
                            HyTransformFarZAbsRe = ZERO_VECTORS3D_Real2(HyTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,YENDAbs - YSTARTAbs+1);
                            HzTransformFarZAbsRe = ZERO_VECTORS3D_Real2(HzTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,YENDAbs - YSTARTAbs+1);


                            hipMemcpy(ExTransformFarZAbsRedev,ExTransformFarZAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);
                            hipMemcpy(EyTransformFarZAbsRedev,EyTransformFarZAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);
                            hipMemcpy(HxTransformFarZAbsRedev,HxTransformFarZAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);
                            hipMemcpy(HyTransformFarZAbsRedev,HyTransformFarZAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);

                            ExTransformFarYAbsRe = ZERO_VECTORS3D_Real2(ExTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,ZENDAbs - ZSTARTAbs+1);
                            EyTransformFarYAbsRe = ZERO_VECTORS3D_Real2(EyTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,ZENDAbs - ZSTARTAbs+1);
                            EzTransformFarYAbsRe = ZERO_VECTORS3D_Real2(EzTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,ZENDAbs - ZSTARTAbs+1);
                            HxTransformFarYAbsRe = ZERO_VECTORS3D_Real2(HxTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,ZENDAbs - ZSTARTAbs+1);
                            HyTransformFarYAbsRe = ZERO_VECTORS3D_Real2(HyTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,ZENDAbs - ZSTARTAbs+1);
                            HzTransformFarYAbsRe = ZERO_VECTORS3D_Real2(HzTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs +1,ZENDAbs - ZSTARTAbs+1);

                            hipMemcpy(ExTransformFarYAbsRedev,ExTransformFarYAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
                            hipMemcpy(EzTransformFarYAbsRedev,EzTransformFarYAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
                            hipMemcpy(HxTransformFarYAbsRedev,HxTransformFarYAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
                            hipMemcpy(HzTransformFarYAbsRedev,HzTransformFarYAbsRe,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);

                            ExTransformFarXAbsRe = ZERO_VECTORS3D_Real2(ExTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                            EyTransformFarXAbsRe = ZERO_VECTORS3D_Real2(EyTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                            EzTransformFarXAbsRe = ZERO_VECTORS3D_Real2(EzTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                            HxTransformFarXAbsRe = ZERO_VECTORS3D_Real2(HxTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                            HyTransformFarXAbsRe = ZERO_VECTORS3D_Real2(HyTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                            HzTransformFarXAbsRe = ZERO_VECTORS3D_Real2(HzTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

                            hipMemcpy(EyTransformFarXAbsRedev,EyTransformFarXAbsRe,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
                            hipMemcpy(EzTransformFarXAbsRedev,EzTransformFarXAbsRe,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
                            hipMemcpy(HyTransformFarXAbsRedev,HyTransformFarXAbsRe,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
                            hipMemcpy(HzTransformFarXAbsRedev,HzTransformFarXAbsRe,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);

                                ExTransformFarZAbsIm = ZERO_VECTORS3D_Real2(ExTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
                                EyTransformFarZAbsIm = ZERO_VECTORS3D_Real2(EyTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
                                EzTransformFarZAbsIm = ZERO_VECTORS3D_Real2(EzTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
                                HxTransformFarZAbsIm = ZERO_VECTORS3D_Real2(HxTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
                                HyTransformFarZAbsIm = ZERO_VECTORS3D_Real2(HyTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);
                                HzTransformFarZAbsIm = ZERO_VECTORS3D_Real2(HzTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,YENDAbs - YSTARTAbs + 1);

                                hipMemcpy(ExTransformFarZAbsImdev,ExTransformFarZAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);
                                hipMemcpy(EyTransformFarZAbsImdev,EyTransformFarZAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);
                                hipMemcpy(HxTransformFarZAbsImdev,HxTransformFarZAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);
                                hipMemcpy(HyTransformFarZAbsImdev,HyTransformFarZAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(YENDAbs-YSTARTAbs+1),hipMemcpyHostToDevice);

                                ExTransformFarYAbsIm = ZERO_VECTORS3D_Real2(ExTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                                EyTransformFarYAbsIm = ZERO_VECTORS3D_Real2(EyTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                                EzTransformFarYAbsIm = ZERO_VECTORS3D_Real2(EzTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                                HxTransformFarYAbsIm = ZERO_VECTORS3D_Real2(HxTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                                HyTransformFarYAbsIm = ZERO_VECTORS3D_Real2(HyTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                                HzTransformFarYAbsIm = ZERO_VECTORS3D_Real2(HzTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

                                hipMemcpy(ExTransformFarYAbsImdev,ExTransformFarYAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
                                hipMemcpy(EzTransformFarYAbsImdev,EzTransformFarYAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
                                hipMemcpy(HxTransformFarYAbsImdev,HxTransformFarYAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
                                hipMemcpy(HzTransformFarYAbsImdev,HzTransformFarYAbsIm,sizeof(real2)*NUM_freq*(XENDAbs-XSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);

                                ExTransformFarXAbsIm = ZERO_VECTORS3D_Real2(ExTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                                EyTransformFarXAbsIm = ZERO_VECTORS3D_Real2(EyTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                                EzTransformFarXAbsIm = ZERO_VECTORS3D_Real2(EzTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                                HxTransformFarXAbsIm = ZERO_VECTORS3D_Real2(HxTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                                HyTransformFarXAbsIm = ZERO_VECTORS3D_Real2(HyTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);
                                HzTransformFarXAbsIm = ZERO_VECTORS3D_Real2(HzTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs + 1 ,ZENDAbs - ZSTARTAbs + 1);

                                hipMemcpy(EyTransformFarXAbsImdev,EyTransformFarXAbsIm,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
                                hipMemcpy(EzTransformFarXAbsImdev,EzTransformFarXAbsIm,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
                                hipMemcpy(HyTransformFarXAbsImdev,HyTransformFarXAbsIm,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);
                                hipMemcpy(HzTransformFarXAbsImdev,HzTransformFarXAbsIm,sizeof(real2)*NUM_freq*(YENDAbs-YSTARTAbs+1)*(ZENDAbs-ZSTARTAbs+1),hipMemcpyHostToDevice);



    Ex_Reflected = MALLOC3D_Complex2(Ex_Reflected,NUM_freq,NCELLX,NCELLY);
    Hx_Reflected = MALLOC3D_Complex2(Hx_Reflected,NUM_freq,NCELLX,NCELLY);
    Ey_Reflected = MALLOC3D_Complex2(Ey_Reflected,NUM_freq,NCELLX,NCELLY);
    Hy_Reflected = MALLOC3D_Complex2(Hy_Reflected,NUM_freq,NCELLX,NCELLY);

    Ex_Transmitted  = MALLOC3D_Complex2(Ex_Transmitted ,NUM_freq,NCELLX,NCELLY);
    Hx_Transmitted  = MALLOC3D_Complex2(Hx_Transmitted ,NUM_freq,NCELLX,NCELLY);
    Ey_Transmitted  = MALLOC3D_Complex2(Ey_Transmitted ,NUM_freq,NCELLX,NCELLY);
    Hy_Transmitted  = MALLOC3D_Complex2(Hy_Transmitted ,NUM_freq,NCELLX,NCELLY);

    E_Incident = MALLOC3D_Complex2(E_Incident,NUM_freq,NCELLX,NCELLY);
    H_Incident = MALLOC3D_Complex2(H_Incident,NUM_freq,NCELLX,NCELLY);



    Cexe=MALLOC3D(Cexe,NCELLX,NCELLY,NCELLZ);
    Cexh=MALLOC3D(Cexh,NCELLX,NCELLY,NCELLZ);
    Ceye=MALLOC3D(Ceye,NCELLX,NCELLY,NCELLZ);
    Ceyh=MALLOC3D(Ceyh,NCELLX,NCELLY,NCELLZ);
    Ceze=MALLOC3D(Ceze,NCELLX,NCELLY,NCELLZ);
    Cezh=MALLOC3D(Cezh,NCELLX,NCELLY,NCELLZ);

    Chxe=MALLOC3D(Chxe,NCELLX,NCELLY,NCELLZ);
    Chxh=MALLOC3D(Chxh,NCELLX,NCELLY,NCELLZ);
    Chye=MALLOC3D(Chye,NCELLX,NCELLY,NCELLZ);
    Chyh=MALLOC3D(Chyh,NCELLX,NCELLY,NCELLZ);
    Chze=MALLOC3D(Chze,NCELLX,NCELLY,NCELLZ);
    Chzh=MALLOC3D(Chzh,NCELLX,NCELLY,NCELLZ);

    eps=MALLOC3D(eps,NCELLX,NCELLY,NCELLZ);
    mu=MALLOC3D(mu,NCELLX,NCELLY,NCELLZ);
    sigma_e=MALLOC3D(sigma_e,NCELLX,NCELLY,NCELLZ);
    sigma_m=MALLOC3D(sigma_m,NCELLX,NCELLY,NCELLZ);
    //
    psi_Ex_y_N=MALLOC3D_Complex(psi_Ex_y_N,NCELLX,NcpmlY+1,NCELLZ);
    psi_Ez_y_N=MALLOC3D_Complex(psi_Ez_y_N,NCELLX,NcpmlY+1,NCELLZ);
    psi_Ex_y_F=MALLOC3D_Complex(psi_Ex_y_F,NCELLX,NcpmlY+1,NCELLZ);
    psi_Ez_y_F=MALLOC3D_Complex(psi_Ez_y_F,NCELLX,NcpmlY+1,NCELLZ);

    psi_Ex_z_N=MALLOC3D_Complex(psi_Ex_z_N,NCELLX,NCELLY,NcpmlZ+1);
    psi_Ey_z_N=MALLOC3D_Complex(psi_Ey_z_N,NCELLX,NCELLY,NcpmlZ+1);
    psi_Ex_z_F=MALLOC3D_Complex(psi_Ex_z_F,NCELLX,NCELLY,NcpmlZ+1);
    psi_Ey_z_F=MALLOC3D_Complex(psi_Ey_z_F,NCELLX,NCELLY,NcpmlZ+1);

    psi_Ey_x_N=MALLOC3D_Complex(psi_Ey_x_N,NcpmlX+1,NCELLY,NCELLZ);
    psi_Ez_x_N=MALLOC3D_Complex(psi_Ez_x_N,NcpmlX+1,NCELLY,NCELLZ);
    psi_Ey_x_F=MALLOC3D_Complex(psi_Ey_x_F,NcpmlX+1,NCELLY,NCELLZ);
    psi_Ez_x_F=MALLOC3D_Complex(psi_Ez_x_F,NcpmlX+1,NCELLY,NCELLZ);

    psi_Hx_y_F=MALLOC3D_Complex(psi_Hx_y_F,NCELLX,NcpmlY,NCELLZ);
    psi_Hz_y_F=MALLOC3D_Complex(psi_Hz_y_F,NCELLX,NcpmlY,NCELLZ);
    psi_Hx_y_N=MALLOC3D_Complex(psi_Hx_y_N,NCELLX,NcpmlY,NCELLZ);
    psi_Hz_y_N=MALLOC3D_Complex(psi_Hz_y_N,NCELLX,NcpmlY,NCELLZ);

    psi_Hx_z_F=MALLOC3D_Complex(psi_Hx_z_F,NCELLX,NCELLY,NcpmlZ);
    psi_Hy_z_F=MALLOC3D_Complex(psi_Hy_z_F,NCELLX,NCELLY,NcpmlZ);
    psi_Hx_z_N=MALLOC3D_Complex(psi_Hx_z_N,NCELLX,NCELLY,NcpmlZ);
    psi_Hy_z_N=MALLOC3D_Complex(psi_Hy_z_N,NCELLX,NCELLY,NcpmlZ);

    psi_Hz_x_F=MALLOC3D_Complex(psi_Hz_x_F,NcpmlX,NCELLY,NCELLZ);
    psi_Hy_x_F=MALLOC3D_Complex(psi_Hy_x_F,NcpmlX,NCELLY,NCELLZ);
    psi_Hy_x_N=MALLOC3D_Complex(psi_Hy_x_N,NcpmlX,NCELLY,NCELLZ);
    psi_Hz_x_N=MALLOC3D_Complex(psi_Hz_x_N,NcpmlX,NCELLY,NCELLZ);

    kedx=MALLOC1D(kedx,NCELLX);
    kedy=MALLOC1D(kedy,NCELLY);
    kedz=MALLOC1D(kedz,NCELLZ);
    khdx=MALLOC1D(khdx,NCELLX);
    khdy=MALLOC1D(khdy,NCELLY);
    khdz=MALLOC1D(khdz,NCELLZ);

    be_x_N=MALLOC1D(be_x_N,NcpmlX+1);
    be_y_N=MALLOC1D(be_y_N,NcpmlY+1);
    be_z_N=MALLOC1D(be_z_N,NcpmlZ+1);
    bh_x_N=MALLOC1D(bh_x_N,NcpmlX);
    bh_y_N=MALLOC1D(bh_y_N,NcpmlY);
    bh_z_N=MALLOC1D(bh_z_N,NcpmlZ);

    ce_x_N=MALLOC1D(ce_x_N,NcpmlX+1);
    ce_y_N=MALLOC1D(ce_y_N,NcpmlY+1);
    ce_z_N=MALLOC1D(ce_z_N,NcpmlZ+1);
    ch_x_N=MALLOC1D(ch_x_N,NcpmlX);
    ch_y_N=MALLOC1D(ch_y_N,NcpmlY);
    ch_z_N=MALLOC1D(ch_z_N,NcpmlZ);

    be_x_F=MALLOC1D(be_x_F,NcpmlX+1);
    be_y_F=MALLOC1D(be_y_F,NcpmlY+1);
    be_z_F=MALLOC1D(be_z_F,NcpmlZ+1);
    bh_x_F=MALLOC1D(bh_x_F,NcpmlX);
    bh_y_F=MALLOC1D(bh_y_F,NcpmlY);
    bh_z_F=MALLOC1D(bh_z_F,NcpmlZ);

    ce_x_F=MALLOC1D(ce_x_F,NcpmlX+1);
    ce_y_F=MALLOC1D(ce_y_F,NcpmlY+1);
    ce_z_F=MALLOC1D(ce_z_F,NcpmlZ+1);
    ch_x_F=MALLOC1D(ch_x_F,NcpmlX);
    ch_y_F=MALLOC1D(ch_y_F,NcpmlY);
    ch_z_F=MALLOC1D(ch_z_F,NcpmlZ);

    sigma_e_x=MALLOC1D(sigma_e_x,NcpmlX);
    sigma_e_y=MALLOC1D(sigma_e_y,NcpmlY);
    sigma_e_z=MALLOC1D(sigma_e_z,NcpmlZ);
    sigma_h_x=MALLOC1D(sigma_h_x,NcpmlX);
    sigma_h_y=MALLOC1D(sigma_h_y,NcpmlY);
    sigma_h_z=MALLOC1D(sigma_h_z,NcpmlZ);

    t_inc = MALLOC1D_double(t_inc, NUM_freq);

    E_incident=MALLOC1D_Complex2(E_incident,NUM_freq);
    E_reflected=MALLOC1D_Complex2(E_reflected,NUM_freq);
    E_transmitted=MALLOC1D_Complex2(E_transmitted,NUM_freq);


return ;
}

void FREE_MEM(void){

    FREE1D_Complex2(E_incident);
    FREE1D_Complex2(E_reflected);

    FREE1D_Complex(e_inc);
    FREE1D_Complex(h_inc);

    FREE1D_Complex(ex_inc);
    FREE1D_Complex(ey_inc);
    FREE1D_Complex(ez_inc);
    FREE1D_Complex(hx_inc);
    FREE1D_Complex(hy_inc);
    FREE1D_Complex(hz_inc);

    FREE3D_Complex(ez,NCELLX,NCELLY);
    FREE3D_Complex(ey,NCELLX,NCELLY);
    FREE3D_Complex(ex,NCELLX,NCELLY);
    FREE3D_Complex(hx,NCELLX,NCELLY);
    FREE3D_Complex(hy,NCELLX,NCELLY);
    FREE3D_Complex(hz,NCELLX,NCELLY);
    FREE3D_Complex(hxPrev,NCELLX,NCELLY);
    FREE3D_Complex(hyPrev,NCELLX,NCELLY);
    FREE3D_Complex(hzPrev,NCELLX,NCELLY);
    // FREE3D_Complex(ExTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EyTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EzTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HxTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HyTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HzTransformNearZScaRe,NUM_freq,XENDSca - XSTARTSca);
    //
    // FREE3D_Complex(ExTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EyTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EzTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HxTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HyTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HzTransformNearYScaRe,NUM_freq,XENDSca - XSTARTSca);
    //
    // FREE3D_Complex(ExTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(EyTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(EzTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(HxTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(HyTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(HzTransformNearXScaRe,NUM_freq,YENDSca - YSTARTSca);
    //
    // FREE3D_Complex(ExTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EyTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EzTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HxTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HyTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HzTransformFarZScaRe,NUM_freq,XENDSca - XSTARTSca);
    //
    // FREE3D_Complex(ExTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EyTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EzTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HxTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HyTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HzTransformFarYScaRe,NUM_freq,XENDSca - XSTARTSca);
    //
    // FREE3D_Complex(ExTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(EyTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(EzTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(HxTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(HyTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(HzTransformFarXScaRe,NUM_freq,YENDSca - YSTARTSca);
    //
    //
    //
    //
    // FREE3D_Complex(ExTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EyTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EzTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HxTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HyTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HzTransformNearZScaIm,NUM_freq,XENDSca - XSTARTSca);
    //
    // FREE3D_Complex(ExTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EyTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EzTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HxTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HyTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HzTransformNearYScaIm,NUM_freq,XENDSca - XSTARTSca);
    //
    // FREE3D_Complex(ExTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(EyTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(EzTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(HxTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(HyTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(HzTransformNearXScaIm,NUM_freq,YENDSca - YSTARTSca);
    //
    // FREE3D_Complex(ExTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EyTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EzTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HxTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HyTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HzTransformFarZScaIm,NUM_freq,XENDSca - XSTARTSca);
    //
    // FREE3D_Complex(ExTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EyTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(EzTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HxTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HyTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca);
    // FREE3D_Complex(HzTransformFarYScaIm,NUM_freq,XENDSca - XSTARTSca);
    //
    // FREE3D_Complex(ExTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(EyTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(EzTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(HxTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(HyTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca);
    // FREE3D_Complex(HzTransformFarXScaIm,NUM_freq,YENDSca - YSTARTSca);
    //
    //
    //
    //
    //
    //
    // FREE3D_Complex(ExTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EyTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EzTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HxTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HyTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HzTransformNearZAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    //
    // FREE3D_Complex(ExTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EyTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EzTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HxTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HyTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HzTransformNearYAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    //
    // FREE3D_Complex(ExTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(EyTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(EzTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(HxTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(HyTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(HzTransformNearXAbsRe,NUM_freq,YENDAbs - YSTARTAbs);
    //
    // FREE3D_Complex(ExTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EyTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EzTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HxTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HyTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HzTransformFarZAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    //
    // FREE3D_Complex(ExTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EyTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EzTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HxTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HyTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HzTransformFarYAbsRe,NUM_freq,XENDAbs - XSTARTAbs);
    //
    // FREE3D_Complex(ExTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(EyTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(EzTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(HxTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(HyTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(HzTransformFarXAbsRe,NUM_freq,YENDAbs - YSTARTAbs);
    //
    // FREE3D_Complex(ExTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EyTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EzTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HxTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HyTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HzTransformNearZAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    //
    // FREE3D_Complex(ExTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EyTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EzTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HxTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HyTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HzTransformNearYAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    //
    // FREE3D_Complex(ExTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(EyTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(EzTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(HxTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(HyTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(HzTransformNearXAbsIm,NUM_freq,YENDAbs - YSTARTAbs);
    //
    // FREE3D_Complex(ExTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EyTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EzTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HxTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HyTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HzTransformFarZAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    //
    // FREE3D_Complex(ExTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EyTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(EzTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HxTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HyTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    // FREE3D_Complex(HzTransformFarYAbsIm,NUM_freq,XENDAbs - XSTARTAbs);
    //
    // FREE3D_Complex(ExTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(EyTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(EzTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(HxTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(HyTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs);
    // FREE3D_Complex(HzTransformFarXAbsIm,NUM_freq,YENDAbs - YSTARTAbs);

    FREE4D_Complex(Pz_cp,NCELLX,NCELLY,NCELLZ);
    FREE4D_Complex(Py_cp,NCELLX,NCELLY,NCELLZ);
    FREE4D_Complex(Px_cp,NCELLX,NCELLY,NCELLZ);

    FREE4D_Complex(Pz_cp_n,NCELLX,NCELLY,NCELLZ);
    FREE4D_Complex(Py_cp_n,NCELLX,NCELLY,NCELLZ);
    FREE4D_Complex(Px_cp_n,NCELLX,NCELLY,NCELLZ);

    FREE4D_Complex(Pz_cp_n_1,NCELLX,NCELLY,NCELLZ);
    FREE4D_Complex(Py_cp_n_1,NCELLX,NCELLY,NCELLZ);
    FREE4D_Complex(Px_cp_n_1,NCELLX,NCELLY,NCELLZ);

    FREE4D_Complex(Px_d,NCELLX,NCELLY,NCELLZ);
    FREE4D_Complex(Px_d_n,NCELLX,NCELLY,NCELLZ);
    FREE4D_Complex(Px_d_n_1,NCELLX,NCELLY,NCELLZ);

    FREE4D_Complex(Py_d,NCELLX,NCELLY,NCELLZ);
    FREE4D_Complex(Py_d_n,NCELLX,NCELLY,NCELLZ);
    FREE4D_Complex(Py_d_n_1,NCELLX,NCELLY,NCELLZ);

    FREE4D_Complex(Pz_d,NCELLX,NCELLY,NCELLZ);
    FREE4D_Complex(Pz_d_n,NCELLX,NCELLY,NCELLZ);
    FREE4D_Complex(Pz_d_n_1,NCELLX,NCELLY,NCELLZ);


    FREE3D(Cexe,NCELLX,NCELLY);
    FREE3D(Ceye,NCELLX,NCELLY);
    FREE3D(Ceze,NCELLX,NCELLY);
    FREE3D(Chxe,NCELLX,NCELLY);
    FREE3D(Chye,NCELLX,NCELLY);
    FREE3D(Chze,NCELLX,NCELLY);
    FREE3D(Cexh,NCELLX,NCELLY);
    FREE3D(Ceyh,NCELLX,NCELLY);
    FREE3D(Cezh,NCELLX,NCELLY);
    FREE3D(Chxh,NCELLX,NCELLY);
    FREE3D(Chyh,NCELLX,NCELLY);
    FREE3D(Chzh,NCELLX,NCELLY);

    FREE3D(sigma_e,NCELLX,NCELLY);
    FREE3D(sigma_m,NCELLX,NCELLY);
    FREE3D(eps,NCELLX,NCELLY);
    FREE3D(mu,NCELLX,NCELLY);

    FREE3D_Complex(psi_Ex_y_N,NCELLX,NcpmlY+1);
    FREE3D_Complex(psi_Ex_z_N,NCELLX,NCELLY);
    FREE3D_Complex(psi_Ey_x_N,NcpmlX+1,NCELLY);
    FREE3D_Complex(psi_Ey_z_N,NCELLX,NCELLY);
    FREE3D_Complex(psi_Ez_y_N,NCELLX,NcpmlY+1);
    FREE3D_Complex(psi_Ez_x_N,NcpmlX+1,NCELLY);
    FREE3D_Complex(psi_Hx_z_N,NCELLX,NCELLY);
    FREE3D_Complex(psi_Hx_y_N,NCELLX,NcpmlY);
    FREE3D_Complex(psi_Hy_x_N,NcpmlX,NCELLY);
    FREE3D_Complex(psi_Hy_z_N,NCELLX,NCELLY);
    FREE3D_Complex(psi_Hz_x_N,NcpmlX,NCELLY);
    FREE3D_Complex(psi_Hz_y_N,NCELLX,NcpmlY);

    FREE3D_Complex(psi_Ex_y_F,NCELLX,NcpmlY+1);
    FREE3D_Complex(psi_Ex_z_F,NCELLX,NCELLY);
    FREE3D_Complex(psi_Ey_x_F,NcpmlX+1,NCELLY);
    FREE3D_Complex(psi_Ey_z_F,NCELLX,NCELLY);
    FREE3D_Complex(psi_Ez_y_F,NCELLX,NcpmlY+1);
    FREE3D_Complex(psi_Ez_x_F,NcpmlX+1,NCELLY);
    FREE3D_Complex(psi_Hx_z_F,NCELLX,NCELLY);
    FREE3D_Complex(psi_Hx_y_F,NCELLX,NcpmlY);
    FREE3D_Complex(psi_Hy_x_F,NcpmlX,NCELLY);
    FREE3D_Complex(psi_Hy_z_F,NCELLX,NCELLY);
    FREE3D_Complex(psi_Hz_x_F,NcpmlX,NCELLY);
    FREE3D_Complex(psi_Hz_y_F,NCELLX,NcpmlY);

    FREE3D_Complex2(E_Incident,NUM_freq,NCELLX);
    FREE3D_Complex2(Ex_Reflected,NUM_freq,NCELLX);
    FREE3D_Complex2(H_Incident,NUM_freq,NCELLX);
    FREE3D_Complex2(Hx_Reflected,NUM_freq,NCELLX);
    FREE3D_Complex2(Ey_Reflected,NUM_freq,NCELLX);
    FREE3D_Complex2(Hy_Reflected,NUM_freq,NCELLX);
    FREE3D_Complex2(Hx_Transmitted,NUM_freq,NCELLX);
    FREE3D_Complex2(Ex_Transmitted,NUM_freq,NCELLX);
    FREE3D_Complex2(Ey_Transmitted,NUM_freq,NCELLX);
    FREE3D_Complex2(Hy_Transmitted,NUM_freq,NCELLX);




    FREE1D(be_x_N);
    FREE1D(be_y_N);
    FREE1D(be_z_N);
    FREE1D(bh_x_N);
    FREE1D(bh_y_N);
    FREE1D(bh_z_N);

    FREE1D(ce_x_N);
    FREE1D(ce_y_N);
    FREE1D(ce_z_N);
    FREE1D(ch_x_N);
    FREE1D(ch_y_N);
    FREE1D(ch_z_N);

    FREE1D(be_x_F);
    FREE1D(be_y_F);
    FREE1D(be_z_F);
    FREE1D(bh_x_F);
    FREE1D(bh_y_F);
    FREE1D(bh_z_F);

    FREE1D(ce_x_F);
    FREE1D(ce_y_F);
    FREE1D(ce_z_F);
    FREE1D(ch_x_F);
    FREE1D(ch_y_F);
    FREE1D(ch_z_F);


    FREE1D(kedx);
    FREE1D(kedy);
    FREE1D(kedz);
    FREE1D(khdx);
    FREE1D(khdy);
    FREE1D(khdz);

    FREE1D(sigma_e_x);
    FREE1D(sigma_e_y);
    FREE1D(sigma_e_z);
    FREE1D(sigma_h_x);
    FREE1D(sigma_h_y);
    FREE1D(sigma_h_z);

  //  FREE1D_double(t_inc);

    return;
}
