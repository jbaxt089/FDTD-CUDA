#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "extern_var.h"
#include <complex.h>
 #include <math.h>


 void Fourier_Transform(void){
    int freq_count,i,j,II,JJ,KK;
    int k=inc_plane - Spect_loc;
    int k2 = NCELLZ - NcpmlZ -5;
    if(PBC_CTW == 0) {
	k = NtfsfZ - 5;
	k2 = NCELLZ-NtfsfZ-5;

}

#ifdef FlOATPRECISION
  float complex TransExpE[NUM_freq];
  float complex TransExpH[NUM_freq];
#else
  double complex TransExpE[NUM_freq];
  double complex TransExpH[NUM_freq];
#endif


comp TransVecERe[NUM_freq];
comp TransVecHRe[NUM_freq];
comp TransVecEIm[NUM_freq];
comp TransVecHIm[NUM_freq];

  if(Periodic_XY){

    for(freq_count=0;freq_count<NUM_freq;freq_count++){
      TransExpE[freq_count] = cexp(-I*2*pi*t*dt*freq[freq_count]);
      //#pragma omp parallel for private(i,j) collapse(2)
            for(j=0;j<NCELLY;j++){
                for(i=0;i<NCELLX;i++){
                    if(TEz){

                      #ifdef DOUBLECOMPLEX
                      comp  ex_tot = (ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)])*cexp(I*(i+0.5)*dx*k_x)*cexp(I*j*dy*k_y);
                      comp  ey_tot = (ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)])*cexp(I*i*dx*k_x)*cexp(I*(j+0.5)*dy*k_y);

                      comp  hx_tot = (hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)])*cexp(I*(i)*dx*k_x)*cexp(I*(j+0.5)*dy*k_y);
                      comp  hy_tot = (hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)])*cexp(I*(i+0.5)*dx*k_x)*cexp(I*(j)*dy*k_y);
                        #endif

                        #ifndef DOUBLECOMPLEX
                        comp ex_tot = ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
                        comp ey_tot = ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
                        comp hx_tot = hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
                        comp hy_tot = hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)];

                        comp ex_tot2 = ex[ThreeDMap(i,j,k2,NCELLZ,NCELLY)];
                        comp ey_tot2 = ey[ThreeDMap(i,j,k2,NCELLZ,NCELLY)];
                        comp hx_tot2 = hx[ThreeDMap(i,j,k2,NCELLZ,NCELLY)];
                        comp hy_tot2 = hy[ThreeDMap(i,j,k2,NCELLZ,NCELLY)];
                        #endif



                         Ey_Reflected[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += ey_tot*TransExpE[freq_count] ;
                         Ex_Reflected[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += ex_tot*TransExpE[freq_count] ;
                         //Ez_Reflected[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += ez_tot*cexp(-I*2*pi*t*dt*freq[freq_count]);

                         Hx_Reflected[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += hx_tot*TransExpE[freq_count];
                         Hy_Reflected[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += hx_tot*TransExpE[freq_count] ;


                         Ey_Transmitted[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += ey_tot2*TransExpE[freq_count] ;
                         Ex_Transmitted[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += ex_tot2*TransExpE[freq_count] ;
                         //Ez_Reflected[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += ez_tot*cexp(-I*2*pi*t*dt*freq[freq_count]);

                         Hx_Transmitted[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += hx_tot2*TransExpE[freq_count] ;
                         Hy_Transmitted[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += hx_tot2*TransExpE[freq_count] ;

                      }

                    else if(TMz){

                      #ifdef DOUBLECOMPLEX
                      comp  ex_tot = (ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)])*cexp(I*(i+0.5)*dx*k_x)*cexp(I*j*dy*k_y);
                      comp  ey_tot = (ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)])*cexp(I*i*dx*k_x)*cexp(I*(j+0.5)*dy*k_y);

                      comp  hx_tot = (hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)])*cexp(I*(i)*dx*k_x)*cexp(I*(j+0.5)*dy*k_y);
                      comp  hy_tot = (hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)])*cexp(I*(i+0.5)*dx*k_x)*cexp(I*(j)*dy*k_y);
                        #endif

                        #ifndef DOUBLECOMPLEX
                        comp ex_tot = ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
                        comp ey_tot = ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
                        comp hx_tot = hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
                        comp hy_tot = hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)];

                        comp ex_tot2 = ex[ThreeDMap(i,j,k2,NCELLZ,NCELLY)];
                        comp ey_tot2 = ey[ThreeDMap(i,j,k2,NCELLZ,NCELLY)];
                        comp hx_tot2 = hx[ThreeDMap(i,j,k2,NCELLZ,NCELLY)];
                        comp hy_tot2 = hy[ThreeDMap(i,j,k2,NCELLZ,NCELLY)];
                        #endif


                         Ey_Reflected[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += ey_tot*cexp(-I*2*pi*t*dt*freq[freq_count]);
                         Ex_Reflected[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += ex_tot*cexp(-I*2*pi*t*dt*freq[freq_count]);
                         //Ez_Reflected[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += ez_tot*cexp(-I*2*pi*t*dt*freq[freq_count]);

                         Hx_Reflected[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += hx_tot*cexp(-I*2*pi*t*dt*freq[freq_count]);
                         Hy_Reflected[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += hx_tot*cexp(-I*2*pi*t*dt*freq[freq_count]);


                         Ey_Transmitted[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += ey_tot2*cexp(-I*2*pi*t*dt*freq[freq_count]);
                         Ex_Transmitted[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += ex_tot2*cexp(-I*2*pi*t*dt*freq[freq_count]);
                         //Ez_Reflected[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += ez_tot*cexp(-I*2*pi*t*dt*freq[freq_count]);

                         Hx_Transmitted[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += hx_tot2*cexp(-I*2*pi*t*dt*freq[freq_count]);
                         Hy_Transmitted[ThreeDMap(freq_count,i,j,NCELLY,NCELLX)] += hx_tot2*cexp(-I*2*pi*t*dt*freq[freq_count]);


                    }
                }
            }

    }
}
else{
    if(Absorption == 1){







      for(freq_count=0;freq_count<NUM_freq;freq_count++){
        TransVecERe[freq_count] = cos(2.0*pi*(t+1.0)*dt*freq[freq_count]);
        TransVecEIm[freq_count] = sin(2.0*pi*(t+1.0)*dt*freq[freq_count]);
        TransVecHRe[freq_count] = cos(2.0*pi*(t+0.5)*dt*freq[freq_count]);
        TransVecHIm[freq_count] = sin(2.0*pi*(t+0.5)*dt*freq[freq_count]);

        //#pragma omp parallel for private(i,j,k,II,JJ,KK) collapse(2)
            for(i=XSTARTAbs;i<XENDAbs;i++){
              for(j=YSTARTAbs;j<YENDAbs;j++){
              //  printf("%d\t%d\t%d\t%d\n",i,j,XENDAbs,YENDAbs);
                k = ZNEARAbs;
                II = i - XSTARTAbs;
                JJ = j - YSTARTAbs;

                ExTransformNearZAbsRe[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EyTransformNearZAbsRe[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
              //  EzTransformNearZAbsRe[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];

                ExTransformNearZAbsIm[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EyTransformNearZAbsIm[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
            //    EzTransformNearZAbsIm[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];

                HxTransformNearZAbsRe[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HyTransformNearZAbsRe[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
              //  HzTransformNearZAbsRe[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];

                HxTransformNearZAbsIm[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HyTransformNearZAbsIm[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
            //    HzTransformNearZAbsIm[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];

                k = ZFARAbs;

                ExTransformFarZAbsRe[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EyTransformFarZAbsRe[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
            //    EzTransformFarZAbsRe[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];

                ExTransformFarZAbsIm[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EyTransformFarZAbsIm[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
          //      EzTransformFarZAbsIm[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];

                HxTransformFarZAbsRe[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HyTransformFarZAbsRe[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
            //    HzTransformFarZAbsRe[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];

                HxTransformFarZAbsIm[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HyTransformFarZAbsIm[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
            //    HzTransformFarZAbsIm[ThreeDMap(freq_count,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
              }
            }

            //#pragma omp parallel for private(i,j,k,II,JJ,KK) collapse(2)
            for(i=XSTARTAbs;i<XENDAbs;i++){
              for(k=ZSTARTAbs;k<ZENDAbs;k++){
                II = i - XSTARTAbs;
                KK = k - ZSTARTAbs;

                j=YNEARAbs;

                ExTransformNearYAbsRe[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
            //    EyTransformNearYAbsRe[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EzTransformNearYAbsRe[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];

                ExTransformNearYAbsIm[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
          //      EyTransformNearYAbsIm[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EzTransformNearYAbsIm[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];

                HxTransformNearYAbsRe[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
            //    HyTransformNearYAbsRe[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HzTransformNearYAbsRe[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];

                HxTransformNearYAbsIm[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
            //    HyTransformNearYAbsIm[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HzTransformNearYAbsIm[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];

                j=YFARAbs;

                ExTransformFarYAbsRe[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
          //      EyTransformFarYAbsRe[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EzTransformFarYAbsRe[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];

                ExTransformFarYAbsIm[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
          //      EyTransformFarYAbsIm[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EzTransformFarYAbsIm[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];

                HxTransformFarYAbsRe[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
          //      HyTransformFarYAbsRe[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HzTransformFarYAbsRe[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];

                HxTransformFarYAbsIm[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
            //    HyTransformFarYAbsIm[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HzTransformFarYAbsIm[ThreeDMap(freq_count,II,KK,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
              }
            }

            //#pragma omp parallel for private(i,j,k,II,JJ,KK) collapse(2)
            for(j=YSTARTAbs;j<YENDAbs;j++){
              for(k=ZSTARTAbs;k<ZENDAbs;k++){
                KK = k - ZSTARTAbs;
                JJ = j - YSTARTAbs;

                i=XNEARAbs;

            //    ExTransformNearXAbsRe[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EyTransformNearXAbsRe[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EzTransformNearXAbsRe[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];

        //        ExTransformNearXAbsIm[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EyTransformNearXAbsIm[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EzTransformNearXAbsIm[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];

          //      HxTransformNearXAbsRe[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HyTransformNearXAbsRe[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HzTransformNearXAbsRe[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];

            //    HxTransformNearXAbsIm[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HyTransformNearXAbsIm[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HzTransformNearXAbsIm[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];

                i=XFARAbs;

            //    ExTransformFarXAbsRe[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EyTransformFarXAbsRe[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EzTransformFarXAbsRe[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];

          //      ExTransformFarXAbsIm[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EyTransformFarXAbsIm[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EzTransformFarXAbsIm[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];

            //    HxTransformFarXAbsRe[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HyTransformFarXAbsRe[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HzTransformFarXAbsRe[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];

              //  HxTransformFarXAbsIm[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HyTransformFarXAbsIm[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HzTransformFarXAbsIm[ThreeDMap(freq_count,JJ,KK,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
              }
            }
    }
  }

    if(Scattering == 1){
      for(freq_count=0;freq_count<NUM_freq;freq_count++){
        //#pragma omp parallel for private(i,j,k,II,JJ,KK) collapse(2)
            for(i=XSTARTSca;i<XENDSca;i++){
              for(j=YSTARTSca;j<YENDSca;j++){
                k = ZNEARSca;
                II = i - XSTARTSca;
                JJ = j - YSTARTSca;

                ExTransformNearZScaRe[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EyTransformNearZScaRe[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
            //    EzTransformNearZScaRe[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];

                ExTransformNearZScaIm[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EyTransformNearZScaIm[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
            //    EzTransformNearZScaIm[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];

                HxTransformNearZScaRe[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HyTransformNearZScaRe[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
            //    HzTransformNearZScaRe[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];

                HxTransformNearZScaIm[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HyTransformNearZScaIm[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
            //    HzTransformNearZScaIm[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];

                k = ZFARSca;

                ExTransformFarZScaRe[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EyTransformFarZScaRe[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
            //    EzTransformFarZScaRe[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];

                ExTransformFarZScaIm[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EyTransformFarZScaIm[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
            //    EzTransformFarZScaIm[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];

                HxTransformFarZScaRe[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HyTransformFarZScaRe[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
            //    HzTransformFarZScaRe[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];

                HxTransformFarZScaIm[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HyTransformFarZScaIm[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
            //    HzTransformFarZScaIm[ThreeDMap(freq_count,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
              }
            }

            //#pragma omp parallel for private(i,j,k,II,JJ,KK) collapse(2)
            for(i=XSTARTSca;i<XENDSca;i++){
              for(k=ZSTARTSca;k<ZENDSca;k++){

                j=YNEARSca;
                II = i - XSTARTSca;
                KK = k - ZSTARTSca;
                ExTransformNearYScaRe[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
              //  EyTransformNearYScaRe[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EzTransformNearYScaRe[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];

                ExTransformNearYScaIm[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
            //    EyTransformNearYScaIm[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EzTransformNearYScaIm[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];

                HxTransformNearYScaRe[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
              //  HyTransformNearYScaRe[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HzTransformNearYScaRe[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];

                HxTransformNearYScaIm[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
            //    HyTransformNearYScaIm[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HzTransformNearYScaIm[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];

                j=YFARSca;

                ExTransformFarYScaRe[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
            //    EyTransformFarYScaRe[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EzTransformFarYScaRe[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];

                ExTransformFarYScaIm[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
            //    EyTransformFarYScaIm[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EzTransformFarYScaIm[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];

                HxTransformFarYScaRe[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
            //    HyTransformFarYScaRe[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HzTransformFarYScaRe[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];

                HxTransformFarYScaIm[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
            //    HyTransformFarYScaIm[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HzTransformFarYScaIm[ThreeDMap(freq_count,II,KK,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
              }
            }

            //#pragma omp parallel for private(i,j,k,II,JJ,KK) collapse(2)
            for(j=YSTARTSca;j<YENDSca;j++){
              for(k=ZSTARTSca;k<ZENDSca;k++){
                KK = k - ZSTARTSca;
                JJ = j - YSTARTSca;
                i=XNEARSca;

            //    ExTransformNearXScaRe[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EyTransformNearXScaRe[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EzTransformNearXScaRe[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];

        //        ExTransformNearXScaIm[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EyTransformNearXScaIm[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EzTransformNearXScaIm[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];

        //        HxTransformNearXScaRe[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HyTransformNearXScaRe[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HzTransformNearXScaRe[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];

        //        HxTransformNearXScaIm[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HyTransformNearXScaIm[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HzTransformNearXScaIm[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];

                i=XFARSca;

          //      ExTransformFarXScaRe[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EyTransformFarXScaRe[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];
                EzTransformFarXScaRe[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecERe[freq_count];

            //    ExTransformFarXScaIm[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EyTransformFarXScaIm[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];
                EzTransformFarXScaIm[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecEIm[freq_count];

            //    HxTransformFarXScaRe[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HyTransformFarXScaRe[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];
                HzTransformFarXScaRe[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHRe[freq_count];

            //    HxTransformFarXScaIm[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HyTransformFarXScaIm[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
                HzTransformFarXScaIm[ThreeDMap(freq_count,JJ,KK,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)] += hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)]*TransVecHIm[freq_count];
              }
            }
    }
}
}
    return;
}


void CalculateAbsScatt(void){
  real  P_r,P_t,P_r_inc, Area, dsurf1, dsurf2,k_rho,k_z;
  real u,v,u2,v2;
//  #ifdef FlOATPRECISION
  float complex  zz,zz_trans;
  int i, k,j,II,JJ,KK, m;
  float complex cEx, cEy, ccHx, ccHy;
  float complex cEx_trans, cEy_trans, ccHx_trans, ccHy_trans;
  float complex cEx_inc, cEy_inc, ccHx_inc, cc_Hz_inc;
  int freq_count;
  real2 *QNEAXsca,*QFARXsca,*QNEAYsca,*QFARYsca,*QNEAZsca,*QFARZsca;
  real2 *QNEAXabs,*QFARXabs,*QNEAYabs,*QFARYabs,*QNEAZabs,*QFARZabs;
  float Amplitude;
  // #else
  // double complex  zz,zz_trans;
  // int i, k,j,II,JJ,KK, m;
  // double complex cEx, cEy, ccHx, ccHy;
  // double complex cEx_trans, cEy_trans, ccHx_trans, ccHy_trans;
  // double complex cEx_inc, cEy_inc, ccHx_inc, cc_Hz_inc;
  // int freq_count;
  // double *QNEAXsca,*QFARXsca,*QNEAYsca,*QFARYsca,*QNEAZsca,*QFARZsca;
  // double *QNEAXabs,*QFARXabs,*QNEAYabs,*QFARYabs,*QNEAZabs,*QFARZabs;
  // double Amplitude;
  // #endif
  QNEAXsca = MALLOC1D_Real2(QNEAXsca,NUM_freq);
  QNEAYsca = MALLOC1D_Real2(QNEAYsca,NUM_freq);
  QNEAZsca = MALLOC1D_Real2(QNEAZsca,NUM_freq);
  QFARXsca = MALLOC1D_Real2(QFARXsca,NUM_freq);
  QFARYsca = MALLOC1D_Real2(QFARYsca,NUM_freq);
  QFARZsca = MALLOC1D_Real2(QFARZsca,NUM_freq);
  QNEAXabs = MALLOC1D_Real2(QNEAXabs,NUM_freq);
  QNEAYabs = MALLOC1D_Real2(QNEAYabs,NUM_freq);
  QNEAZabs = MALLOC1D_Real2(QNEAZabs,NUM_freq);
  QFARXabs = MALLOC1D_Real2(QFARXabs,NUM_freq);
  QFARYabs = MALLOC1D_Real2(QFARYabs,NUM_freq);
  QFARZabs = MALLOC1D_Real2(QFARZabs,NUM_freq);




  //
  // if(Absorption == 1){
  //   for(freq_count=0;freq_count<NUM_freq;freq_count++){
  //     m = freq_count;
  //     // TransVecERe[freq_count] = cos(2.0*pi*(t+1.0)*dt*freq[freq_count]);
  //     // TransVecEIm[freq_count] = sin(2.0*pi*(t+1.0)*dt*freq[freq_count]);
  //     // TransVecHRe[freq_count] = cos(2.0*pi*(t+0.5)*dt*freq[freq_count]);
  //     // TransVecHIm[freq_count] = sin(2.0*pi*(t+0.5)*dt*freq[freq_count]);
  //
  //     //#pragma omp parallel for private(i,j) collapse(2)
  //         for(i=XSTARTAbs;i<XENDAbs;i++){
  //           for(j=YSTARTAbs;j<YENDAbs;j++){
  //           //  printf("%d\t%d\t%d\t%d\n",i,j,XENDAbs,YENDAbs);
  //             II = i - XSTARTAbs;
  //             JJ = j - YSTARTAbs;
  //
  //             ExTransformNearZAbsRe[freq_count][II][JJ] /= (Amplitude);
  //             EyTransformNearZAbsRe[freq_count][II][JJ] /= (Amplitude);
  //             EzTransformNearZAbsRe[freq_count][II][JJ] /= (Amplitude);
  //
  //             ExTransformNearZAbsIm[freq_count][II][JJ] /= (Amplitude);
  //             EyTransformNearZAbsIm[freq_count][II][JJ] /= (Amplitude);
  //             EzTransformNearZAbsIm[freq_count][II][JJ] /= (Amplitude);
  //
  //             HxTransformNearZAbsRe[freq_count][II][JJ] /= (Amplitude);
  //             HyTransformNearZAbsRe[freq_count][II][JJ] /= (Amplitude);
  //             HzTransformNearZAbsRe[freq_count][II][JJ] /= (Amplitude);
  //
  //             HxTransformNearZAbsIm[freq_count][II][JJ] /= (Amplitude);
  //             HyTransformNearZAbsIm[freq_count][II][JJ] /= (Amplitude);
  //             HzTransformNearZAbsIm[freq_count][II][JJ] /= (Amplitude);
  //
  //
  //             ExTransformFarZAbsRe[freq_count][II][JJ] /= (Amplitude);
  //             EyTransformFarZAbsRe[freq_count][II][JJ] /= (Amplitude);
  //             EzTransformFarZAbsRe[freq_count][II][JJ] /= (Amplitude);
  //
  //             ExTransformFarZAbsIm[freq_count][II][JJ] /= (Amplitude);
  //             EyTransformFarZAbsIm[freq_count][II][JJ] /= (Amplitude);
  //             EzTransformFarZAbsIm[freq_count][II][JJ] /= (Amplitude);
  //
  //             HxTransformFarZAbsRe[freq_count][II][JJ] /= (Amplitude);
  //             HyTransformFarZAbsRe[freq_count][II][JJ] /= (Amplitude);
  //             HzTransformFarZAbsRe[freq_count][II][JJ] /= (Amplitude);
  //
  //             HxTransformFarZAbsIm[freq_count][II][JJ] /= (Amplitude);
  //             HyTransformFarZAbsIm[freq_count][II][JJ] /= (Amplitude);
  //             HzTransformFarZAbsIm[freq_count][II][JJ] /= (Amplitude);
  //           }
  //         }
  //
  //         //#pragma omp parallel for private(i,k) collapse(2)
  //         for(i=XSTARTAbs;i<XENDAbs;i++){
  //           for(k=ZSTARTAbs;k<ZENDAbs;k++){
  //             II = i - XSTARTAbs;
  //             KK = k - ZSTARTAbs;
  //
  //
  //             ExTransformNearYAbsRe[freq_count][II][KK] /= (Amplitude);
  //             EyTransformNearYAbsRe[freq_count][II][KK] /= (Amplitude);
  //             EzTransformNearYAbsRe[freq_count][II][KK] /= (Amplitude);
  //
  //             ExTransformNearYAbsIm[freq_count][II][KK]/= (Amplitude);
  //             EyTransformNearYAbsIm[freq_count][II][KK]/= (Amplitude);
  //             EzTransformNearYAbsIm[freq_count][II][KK]/= (Amplitude);
  //
  //             HxTransformNearYAbsRe[freq_count][II][KK] /= (Amplitude);
  //             HyTransformNearYAbsRe[freq_count][II][KK] /= (Amplitude);
  //             HzTransformNearYAbsRe[freq_count][II][KK] /= (Amplitude);
  //
  //             HxTransformNearYAbsIm[freq_count][II][KK] /= (Amplitude);
  //             HyTransformNearYAbsIm[freq_count][II][KK] /= (Amplitude);
  //             HzTransformNearYAbsIm[freq_count][II][KK]/= (Amplitude);
  //
  //             ExTransformFarYAbsRe[freq_count][II][KK] /= (Amplitude);
  //             EyTransformFarYAbsRe[freq_count][II][KK] /= (Amplitude);
  //             EzTransformFarYAbsRe[freq_count][II][KK] /= (Amplitude);
  //
  //             ExTransformFarYAbsIm[freq_count][II][KK] /= (Amplitude);
  //             EyTransformFarYAbsIm[freq_count][II][KK] /= (Amplitude);
  //             EzTransformFarYAbsIm[freq_count][II][KK] /= (Amplitude);
  //
  //             HxTransformFarYAbsRe[freq_count][II][KK] /= (Amplitude);
  //             HyTransformFarYAbsRe[freq_count][II][KK] /= (Amplitude);
  //             HzTransformFarYAbsRe[freq_count][II][KK] /= (Amplitude);
  //
  //             HxTransformFarYAbsIm[freq_count][II][KK] /= (Amplitude);
  //             HyTransformFarYAbsIm[freq_count][II][KK] /= (Amplitude);
  //             HzTransformFarYAbsIm[freq_count][II][KK] /= (Amplitude);
  //           }
  //         }
  //
  //         //#pragma omp parallel for private(j,k) collapse(2)
  //         for(j=YSTARTAbs;j<YENDAbs;j++){
  //           for(k=ZSTARTAbs;k<ZENDAbs;k++){
  //             KK = k - ZSTARTAbs;
  //             JJ = j - YSTARTAbs;
  //
  //
  //             ExTransformNearXAbsRe[freq_count][JJ][KK] /= (Amplitude);
  //             EyTransformNearXAbsRe[freq_count][JJ][KK] /= (Amplitude);
  //             EzTransformNearXAbsRe[freq_count][JJ][KK] /= (Amplitude);
  //
  //             ExTransformNearXAbsIm[freq_count][JJ][KK] /= (Amplitude);
  //             EyTransformNearXAbsIm[freq_count][JJ][KK] /= (Amplitude);
  //             EzTransformNearXAbsIm[freq_count][JJ][KK] /= (Amplitude);
  //
  //             HxTransformNearXAbsRe[freq_count][JJ][KK]/= (Amplitude);
  //             HyTransformNearXAbsRe[freq_count][JJ][KK] /= (Amplitude);
  //             HzTransformNearXAbsRe[freq_count][JJ][KK] /= (Amplitude);
  //
  //             HxTransformNearXAbsIm[freq_count][JJ][KK] /= (Amplitude);
  //             HyTransformNearXAbsIm[freq_count][JJ][KK]/= (Amplitude);
  //             HzTransformNearXAbsIm[freq_count][JJ][KK] /= (Amplitude);
  //
  //
  //             ExTransformFarXAbsRe[freq_count][JJ][KK] /= (Amplitude);
  //             EyTransformFarXAbsRe[freq_count][JJ][KK] /= (Amplitude);
  //             EzTransformFarXAbsRe[freq_count][JJ][KK] /= (Amplitude);
  //
  //             ExTransformFarXAbsIm[freq_count][JJ][KK] /= (Amplitude);
  //             EyTransformFarXAbsIm[freq_count][JJ][KK] /= (Amplitude);
  //             EzTransformFarXAbsIm[freq_count][JJ][KK] /= (Amplitude);
  //
  //             HxTransformFarXAbsRe[freq_count][JJ][KK] /= (Amplitude);
  //             HyTransformFarXAbsRe[freq_count][JJ][KK] /= (Amplitude);
  //             HzTransformFarXAbsRe[freq_count][JJ][KK] /= (Amplitude);
  //
  //             HxTransformFarXAbsIm[freq_count][JJ][KK]/= (Amplitude);
  //             HyTransformFarXAbsIm[freq_count][JJ][KK] /= (Amplitude);
  //             HzTransformFarXAbsIm[freq_count][JJ][KK] /= (Amplitude);
  //           }
  //         }
  // }
  // }
  //
  //
  //   if(Scattering == 1){
  //     for(freq_count=0;freq_count<NUM_freq;freq_count++){
  //       // TransVecERe[freq_count] = cos(2.0*pi*(t+1.0)*dt*freq[freq_count]);
  //       // TransVecEIm[freq_count] = sin(2.0*pi*(t+1.0)*dt*freq[freq_count]);
  //       // TransVecHRe[freq_count] = cos(2.0*pi*(t+0.5)*dt*freq[freq_count]);
  //       // TransVecHIm[freq_count] = sin(2.0*pi*(t+0.5)*dt*freq[freq_count]);
  //
  //       //#pragma omp parallel for private(i,j) collapse(2)
  //           for(i=XSTARTSca;i<XENDSca;i++){
  //             for(j=YSTARTSca;j<YENDSca;j++){
  //             //  printf("%d\t%d\t%d\t%d\n",i,j,XENDSca,YENDSca);
  //               II = i - XSTARTSca;
  //               JJ = j - YSTARTSca;
  //
  //               ExTransformNearZScaRe[freq_count][II][JJ] /= (Amplitude);
  //               EyTransformNearZScaRe[freq_count][II][JJ] /= (Amplitude);
  //               EzTransformNearZScaRe[freq_count][II][JJ] /= (Amplitude);
  //
  //               ExTransformNearZScaIm[freq_count][II][JJ] /= (Amplitude);
  //               EyTransformNearZScaIm[freq_count][II][JJ] /= (Amplitude);
  //               EzTransformNearZScaIm[freq_count][II][JJ] /= (Amplitude);
  //
  //               HxTransformNearZScaRe[freq_count][II][JJ] /= (Amplitude);
  //               HyTransformNearZScaRe[freq_count][II][JJ] /= (Amplitude);
  //               HzTransformNearZScaRe[freq_count][II][JJ] /= (Amplitude);
  //
  //               HxTransformNearZScaIm[freq_count][II][JJ] /= (Amplitude);
  //               HyTransformNearZScaIm[freq_count][II][JJ] /= (Amplitude);
  //               HzTransformNearZScaIm[freq_count][II][JJ] /= (Amplitude);
  //
  //
  //               ExTransformFarZScaRe[freq_count][II][JJ] /= (Amplitude);
  //               EyTransformFarZScaRe[freq_count][II][JJ] /= (Amplitude);
  //               EzTransformFarZScaRe[freq_count][II][JJ] /= (Amplitude);
  //
  //               ExTransformFarZScaIm[freq_count][II][JJ] /= (Amplitude);
  //               EyTransformFarZScaIm[freq_count][II][JJ] /= (Amplitude);
  //               EzTransformFarZScaIm[freq_count][II][JJ] /= (Amplitude);
  //
  //               HxTransformFarZScaRe[freq_count][II][JJ] /= (Amplitude);
  //               HyTransformFarZScaRe[freq_count][II][JJ] /= (Amplitude);
  //               HzTransformFarZScaRe[freq_count][II][JJ] /= (Amplitude);
  //
  //               HxTransformFarZScaIm[freq_count][II][JJ] /= (Amplitude);
  //               HyTransformFarZScaIm[freq_count][II][JJ] /= (Amplitude);
  //               HzTransformFarZScaIm[freq_count][II][JJ] /= (Amplitude);
  //             }
  //           }
  //
  //           //#pragma omp parallel for private(i,k) collapse(2)
  //           for(i=XSTARTSca;i<XENDSca;i++){
  //             for(k=ZSTARTSca;k<ZENDSca;k++){
  //               II = i - XSTARTSca;
  //               KK = k - ZSTARTSca;
  //
  //
  //               ExTransformNearYScaRe[freq_count][II][KK] /= (Amplitude);
  //               EyTransformNearYScaRe[freq_count][II][KK] /= (Amplitude);
  //               EzTransformNearYScaRe[freq_count][II][KK] /= (Amplitude);
  //
  //               ExTransformNearYScaIm[freq_count][II][KK]/= (Amplitude);
  //               EyTransformNearYScaIm[freq_count][II][KK]/= (Amplitude);
  //               EzTransformNearYScaIm[freq_count][II][KK]/= (Amplitude);
  //
  //               HxTransformNearYScaRe[freq_count][II][KK] /= (Amplitude);
  //               HyTransformNearYScaRe[freq_count][II][KK] /= (Amplitude);
  //               HzTransformNearYScaRe[freq_count][II][KK] /= (Amplitude);
  //
  //               HxTransformNearYScaIm[freq_count][II][KK] /= (Amplitude);
  //               HyTransformNearYScaIm[freq_count][II][KK] /= (Amplitude);
  //               HzTransformNearYScaIm[freq_count][II][KK]/= (Amplitude);
  //
  //               ExTransformFarYScaRe[freq_count][II][KK] /= (Amplitude);
  //               EyTransformFarYScaRe[freq_count][II][KK] /= (Amplitude);
  //               EzTransformFarYScaRe[freq_count][II][KK] /= (Amplitude);
  //
  //               ExTransformFarYScaIm[freq_count][II][KK] /= (Amplitude);
  //               EyTransformFarYScaIm[freq_count][II][KK] /= (Amplitude);
  //               EzTransformFarYScaIm[freq_count][II][KK] /= (Amplitude);
  //
  //               HxTransformFarYScaRe[freq_count][II][KK] /= (Amplitude);
  //               HyTransformFarYScaRe[freq_count][II][KK] /= (Amplitude);
  //               HzTransformFarYScaRe[freq_count][II][KK] /= (Amplitude);
  //
  //               HxTransformFarYScaIm[freq_count][II][KK] /= (Amplitude);
  //               HyTransformFarYScaIm[freq_count][II][KK] /= (Amplitude);
  //               HzTransformFarYScaIm[freq_count][II][KK] /= (Amplitude);
  //             }
  //           }
  //
  //           //#pragma omp parallel for private(j,k) collapse(2)
  //           for(j=YSTARTSca;j<YENDSca;j++){
  //             for(k=ZSTARTSca;k<ZENDSca;k++){
  //               KK = k - ZSTARTSca;
  //               JJ = j - YSTARTSca;
  //
  //
  //               ExTransformNearXScaRe[freq_count][JJ][KK] /= (Amplitude);
  //               EyTransformNearXScaRe[freq_count][JJ][KK] /= (Amplitude);
  //               EzTransformNearXScaRe[freq_count][JJ][KK] /= (Amplitude);
  //
  //               ExTransformNearXScaIm[freq_count][JJ][KK] /= (Amplitude);
  //               EyTransformNearXScaIm[freq_count][JJ][KK] /= (Amplitude);
  //               EzTransformNearXScaIm[freq_count][JJ][KK] /= (Amplitude);
  //
  //               HxTransformNearXScaRe[freq_count][JJ][KK]/= (Amplitude);
  //               HyTransformNearXScaRe[freq_count][JJ][KK] /= (Amplitude);
  //               HzTransformNearXScaRe[freq_count][JJ][KK] /= (Amplitude);
  //
  //               HxTransformNearXScaIm[freq_count][JJ][KK] /= (Amplitude);
  //               HyTransformNearXScaIm[freq_count][JJ][KK]/= (Amplitude);
  //               HzTransformNearXScaIm[freq_count][JJ][KK] /= (Amplitude);
  //
  //
  //               ExTransformFarXScaRe[freq_count][JJ][KK] /= (Amplitude);
  //               EyTransformFarXScaRe[freq_count][JJ][KK] /= (Amplitude);
  //               EzTransformFarXScaRe[freq_count][JJ][KK] /= (Amplitude);
  //
  //               ExTransformFarXScaIm[freq_count][JJ][KK] /= (Amplitude);
  //               EyTransformFarXScaIm[freq_count][JJ][KK] /= (Amplitude);
  //               EzTransformFarXScaIm[freq_count][JJ][KK] /= (Amplitude);
  //
  //               HxTransformFarXScaRe[freq_count][JJ][KK] /= (Amplitude);
  //               HyTransformFarXScaRe[freq_count][JJ][KK] /= (Amplitude);
  //               HzTransformFarXScaRe[freq_count][JJ][KK] /= (Amplitude);
  //
  //               HxTransformFarXScaIm[freq_count][JJ][KK]/= (Amplitude);
  //               HyTransformFarXScaIm[freq_count][JJ][KK] /= (Amplitude);
  //               HzTransformFarXScaIm[freq_count][JJ][KK] /= (Amplitude);
  //             }
  //           }
  //   }
  //   }
  //



  for (m = 0; m < NUM_freq; m++) {

      Area = 0.0;
      P_r = 0.0;
      P_r_inc = 0.0;
      P_t = 0.0;
      Amplitude = (float)cabs(E_incident[m]);
      // Amplitude = 1.0;
      // if(t==0) Amplitude = 1.0;

    //  fprintf(Spectrum,"%e\t%e\t%e\t%e\t%e\n",freq[m],creal(E_Reflected[m][3][3]),cimag(E_Reflected[m][3][3]),creal(H_Reflected[m][3][3]),cimag(H_Reflected[m][3][3]));

    for(i=XSTARTSca;i<XENDSca;i++){
      dsurf1 = dx;
      for(j=YSTARTSca;j<YENDSca;j++){
        dsurf2 = dy;
        II = i - XSTARTSca;
        JJ = j - YSTARTSca;


              u = EyTransformNearZScaRe[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] / (Amplitude);
              v = EyTransformNearZScaIm[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] / (Amplitude);
              u2 = ExTransformNearZScaRe[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] / (Amplitude);
              v2 = ExTransformNearZScaIm[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] / (Amplitude);

            cEy = u + I*v;
            cEx = u2 + I*v2;

            u = EyTransformFarZScaRe[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] / (Amplitude);
            v = EyTransformFarZScaIm[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] / (Amplitude);
            u2 = ExTransformFarZScaRe[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] / (Amplitude);
            v2 = ExTransformFarZScaIm[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)] / (Amplitude);


            cEy_trans = u + I*v;
            cEx_trans = u2 + I*v2;

              u = HxTransformNearZScaRe[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
              v = HxTransformNearZScaIm[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
              u2 = HyTransformNearZScaRe[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
              v2 = HyTransformNearZScaIm[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);



            ccHx = u - I*v;
            ccHy = u2 - I*v2;


              u = HxTransformFarZScaRe[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
              v = HxTransformFarZScaIm[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
              u2 = HyTransformFarZScaRe[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
              v2 = HyTransformFarZScaIm[ThreeDMap(m,II,JJ,YENDSca-YSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);// + ImTFHz_xz_n[m][i][K];

            ccHx_trans = u - I*v;
            ccHy_trans = u2 - I*v2;

            zz = cEx*ccHy - cEy*ccHx;
            zz_trans = cEx_trans*ccHy_trans - cEy_trans*ccHx_trans;


            P_r += creal(zz)*dsurf1*dsurf2;
            P_t += creal(zz_trans)*dsurf1*dsurf2;


          }
      }
    QNEAZsca[m] = P_r;
    QFARZsca[m] = P_t;

    Area = 0.0;
    P_r = 0.0;
    P_r_inc = 0.0;
    P_t = 0.0;



  //  fprintf(Spectrum,"%e\t%e\t%e\t%e\t%e\n",freq[m],creal(E_Reflected[m][3][3]),cimag(E_Reflected[m][3][3]),creal(H_Reflected[m][3][3]),cimag(H_Reflected[m][3][3]));

  for(i=XSTARTSca;i<XENDSca;i++){
    dsurf1 = dx;
    for(j=ZSTARTSca;j<ZENDSca;j++){
      dsurf2 = dy;
      II = i - XSTARTSca;
      JJ = j - ZSTARTSca;


            u = ExTransformNearYScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
            v = ExTransformNearYScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
            u2 = EzTransformNearYScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
            v2 = EzTransformNearYScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);

          cEy = u + I*v;
          cEx = u2 + I*v2;

          u = ExTransformFarYScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
          v = ExTransformFarYScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
          u2 = EzTransformFarYScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
          v2 = EzTransformFarYScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);


          cEy_trans = u + I*v;
          cEx_trans = u2 + I*v2;

            u = HzTransformNearYScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
            v = HzTransformNearYScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
            u2 = HxTransformNearYScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
            v2 = HxTransformNearYScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);



          ccHx = u - I*v;
          ccHy = u2 - I*v2;


            u = HzTransformFarYScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
            v = HzTransformFarYScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
            u2 = HxTransformFarYScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);
            v2 = HxTransformFarYScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,XENDSca-XSTARTSca+1)]/(Amplitude);// + ImTFHz_xz_n[m][i][K];

          ccHx_trans = u - I*v;
          ccHy_trans = u2 - I*v2;

          zz = cEx*ccHy - cEy*ccHx;
          zz_trans = cEx_trans*ccHy_trans - cEy_trans*ccHx_trans;

          P_r += creal(zz)*dsurf1*dsurf2;
          P_t += creal(zz_trans)*dsurf1*dsurf2;


        }
    }
  QNEAYsca[m] = P_r;
  QFARYsca[m] = P_t;

  Area = 0.0;
  P_r = 0.0;
  P_r_inc = 0.0;
  P_t = 0.0;



//  fprintf(Spectrum,"%e\t%e\t%e\t%e\t%e\n",freq[m],creal(E_Reflected[m][3][3]),cimag(E_Reflected[m][3][3]),creal(H_Reflected[m][3][3]),cimag(H_Reflected[m][3][3]));

for(i=YSTARTSca;i<YENDSca;i++){
  dsurf1 = dx;
  for(j=ZSTARTSca;j<ZENDSca;j++){
    dsurf2 = dy;
    II = i - YSTARTSca;
    JJ = j - ZSTARTSca;


          u = EzTransformNearXScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);
          v = EzTransformNearXScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);
          u2 = EyTransformNearXScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);
          v2 = EyTransformNearXScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);

        cEy = u + I*v;
        cEx = u2 + I*v2;

        u = EzTransformFarXScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);
        v = EzTransformFarXScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);
        u2 = EyTransformFarXScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);
        v2 = EyTransformFarXScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);


        cEy_trans = u + I*v;
        cEx_trans = u2 + I*v2;

          u = HyTransformNearXScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);
          v = HyTransformNearXScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);
          u2 = HzTransformNearXScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);
          v2 = HzTransformNearXScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);



        ccHx = u - I*v;
        ccHy = u2 - I*v2;


          v = HyTransformFarXScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);
          u = HyTransformFarXScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);
          u2 = HzTransformFarXScaRe[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);
          v2 = HzTransformFarXScaIm[ThreeDMap(m,II,JJ,ZENDSca-ZSTARTSca + 1,YENDSca-YSTARTSca+1)]/(Amplitude);// + ImTFHz_xz_n[m][i][K];

        ccHx_trans = u - I*v;
        ccHy_trans = u2 - I*v2;

        zz = cEx*ccHy - cEy*ccHx;
        zz_trans = cEx_trans*ccHy_trans - cEy_trans*ccHx_trans;

        P_r += creal(zz)*dsurf1*dsurf2;
        P_t += creal(zz_trans)*dsurf1*dsurf2;


      }
  }
QNEAXsca[m] = P_r;
QFARXsca[m] = P_t;













Area = 0.0;
P_r = 0.0;
P_r_inc = 0.0;
P_t = 0.0;



//  fprintf(Spectrum,"%e\t%e\t%e\t%e\t%e\n",freq[m],creal(E_Reflected[m][3][3]),cimag(E_Reflected[m][3][3]),creal(H_Reflected[m][3][3]),cimag(H_Reflected[m][3][3]));

for(i=XSTARTAbs;i<XENDAbs;i++){
dsurf1 = dx;
for(j=YSTARTAbs;j<YENDAbs;j++){
  dsurf2 = dy;
  II = i - XSTARTAbs;
  JJ = j - YSTARTAbs;


        u = EyTransformNearZAbsRe[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
        v = EyTransformNearZAbsIm[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
        u2 = ExTransformNearZAbsRe[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
        v2 = ExTransformNearZAbsIm[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);

      cEy = u + I*v;
      cEx = u2 + I*v2;

      u = EyTransformFarZAbsRe[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
      v = EyTransformFarZAbsIm[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
      u2 = ExTransformFarZAbsRe[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
      v2 = ExTransformFarZAbsIm[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);


      cEy_trans = u + I*v;
      cEx_trans = u2 + I*v2;

        v = HxTransformNearZAbsIm[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
        u = HxTransformNearZAbsRe[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
        u2 = HyTransformNearZAbsRe[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
        v2 = HyTransformNearZAbsIm[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);



      ccHx = u - I*v;
      ccHy = u2 - I*v2;


        u = HxTransformFarZAbsRe[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
        v = HxTransformFarZAbsIm[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
        u2 = HyTransformFarZAbsRe[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
        v2 = HyTransformFarZAbsIm[ThreeDMap(m,II,JJ,YENDAbs-YSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);// + ImTFHz_xz_n[m][i][K];

      ccHx_trans = u - I*v;
      ccHy_trans = u2 - I*v2;

      zz = cEx*ccHy - cEy*ccHx;
      zz_trans = cEx_trans*ccHy_trans - cEy_trans*ccHx_trans;


      P_r += creal(zz)*dsurf1*dsurf2;
      P_t += creal(zz_trans)*dsurf1*dsurf2;


    }
}
QNEAZabs[m] = P_r;
QFARZabs[m] = P_t;

Area = 0.0;
P_r = 0.0;
P_r_inc = 0.0;
P_t = 0.0;



//  fprintf(Spectrum,"%e\t%e\t%e\t%e\t%e\n",freq[m],creal(E_Reflected[m][3][3]),cimag(E_Reflected[m][3][3]),creal(H_Reflected[m][3][3]),cimag(H_Reflected[m][3][3]));

for(i=XSTARTAbs;i<XENDAbs;i++){
dsurf1 = dx;
for(j=ZSTARTAbs;j<ZENDAbs;j++){
dsurf2 = dy;
II = i - XSTARTAbs;
JJ = j - ZSTARTAbs;


      u = ExTransformNearYAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
      v = ExTransformNearYAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
      u2 = EzTransformNearYAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
      v2 = EzTransformNearYAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);

    cEy = u + I*v;
    cEx = u2 + I*v2;

    v = ExTransformFarYAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
    u = ExTransformFarYAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
    v2 = EzTransformFarYAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
    u2 = EzTransformFarYAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);


    cEy_trans = u + I*v;
    cEx_trans = u2 + I*v2;

      u = HzTransformNearYAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
      v = HzTransformNearYAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
      u2 = HxTransformNearYAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
      v2 = HxTransformNearYAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);



    ccHx = u - I*v;
    ccHy = u2 - I*v2;


      u = HzTransformFarYAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
      v = HzTransformFarYAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
      u2 = HxTransformFarYAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);
      v2 = HxTransformFarYAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,XENDAbs-XSTARTAbs+1)]/(Amplitude);// + ImTFHz_xz_n[m][i][K];

    ccHx_trans = u - I*v;
    ccHy_trans = u2 - I*v2;

    zz = cEx*ccHy - cEy*ccHx;
    zz_trans = cEx_trans*ccHy_trans - cEy_trans*ccHx_trans;

    P_r += creal(zz)*dsurf1*dsurf2;
    P_t += creal(zz_trans)*dsurf1*dsurf2;


  }
}
QNEAYabs[m] = P_r;
QFARYabs[m] = P_t;

Area = 0.0;
P_r = 0.0;
P_r_inc = 0.0;
P_t = 0.0;



//  fprintf(Spectrum,"%e\t%e\t%e\t%e\t%e\n",freq[m],creal(E_Reflected[m][3][3]),cimag(E_Reflected[m][3][3]),creal(H_Reflected[m][3][3]),cimag(H_Reflected[m][3][3]));

for(i=YSTARTAbs;i<YENDAbs;i++){
dsurf1 = dx;
for(j=ZSTARTAbs;j<ZENDAbs;j++){
dsurf2 = dy;
II = i - YSTARTAbs;
JJ = j - ZSTARTAbs;


    u = EzTransformNearXAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);
    v = EzTransformNearXAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);
    u2 = EyTransformNearXAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);
    v2 = EyTransformNearXAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);

  cEy = u + I*v;
  cEx = u2 + I*v2;

  v = EzTransformFarXAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);
  u = EzTransformFarXAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);
  u2 = EyTransformFarXAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);
  v2 = EyTransformFarXAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);


  cEy_trans = u + I*v;
  cEx_trans = u2 + I*v2;

    u = HyTransformNearXAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);
    v = HyTransformNearXAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);
    u2 = HzTransformNearXAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);
    v2 = HzTransformNearXAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);





  ccHx = u - I*v;
  ccHy = u2 - I*v2;


    u = HyTransformFarXAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);
    v = HyTransformFarXAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);
    u2 = HzTransformFarXAbsRe[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);
    v2 = HzTransformFarXAbsIm[ThreeDMap(m,II,JJ,ZENDAbs-ZSTARTAbs + 1,YENDAbs-YSTARTAbs+1)]/(Amplitude);// + ImTFHz_xz_n[m][i][K];

  ccHx_trans = u - I*v;
  ccHy_trans = u2 - I*v2;

  zz = cEx*ccHy - cEy*ccHx;
  zz_trans = cEx_trans*ccHy_trans - cEy_trans*ccHx_trans;

  P_r += creal(zz)*dsurf1*dsurf2;
  P_t += creal(zz_trans)*dsurf1*dsurf2;


}
}
QNEAXabs[m] = P_r;
QFARXabs[m] = P_t;






  }










FILE *SCATABS;

char filename[100];
FILE *Snap;

static char name[10]={'S','c','a','t','A','b','s'};

sprintf(filename,"%s.%d.txt",name,t);
SCATABS = fopen(filename,"w");
double ScaTot,AbsTot,ExtTot;
for(m=0;m<NUM_freq;m++){

  ScaTot = -QNEAXsca[m] + QFARXsca[m] - QNEAYsca[m] + QFARYsca[m] - QNEAZsca[m] + QFARZsca[m];
  AbsTot = -QNEAXabs[m] + QFARXabs[m] - QNEAYabs[m] + QFARYabs[m] - QNEAZabs[m] + QFARZabs[m];
  if(nano_sphere == 1){
    // printf("%e\n",pi*nano_sphere_radius*nano_sphere_radius);
  ScaTot = ScaTot/(pi*nano_sphere_radius*nano_sphere_radius);
  AbsTot = -AbsTot/(pi*nano_sphere_radius*nano_sphere_radius);
  ExtTot = AbsTot + ScaTot;
}

  fprintf(SCATABS,"%e\t%e\t%e\t%e\t%e\t%e\t%e\t%e\t%e\t%e\t%e\t%e\t%e\t%e\t%e\t%e\t%e\t%e\t%e\n",c0/freq[m],ScaTot,AbsTot,ExtTot,QNEAXsca[m],QFARXsca[m] ,QNEAYsca[m] ,QFARYsca[m] ,QNEAZsca[m],QFARZsca[m],QNEAXabs[m],QFARXabs[m],QNEAYabs[m],QFARYabs[m],QNEAZabs[m],QFARZabs[m],cabs(E_incident[m]),creal(E_incident[m]),cimag(E_incident[m]));

}
fclose(SCATABS);


}
