#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "extern_var.h"
#include "hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>
// #include <>
//#include<conio.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <cudaProfiler.h>
//  __device__  int ThreeDMapD(int i,int j,int k,int SizeZ,int SizeY){
//   int num = k + SizeZ*j +SizeY*SizeZ*i;
//   return num;
// }
//
//
//  __device__  int FourDMapD(int i,int j,int k,int n,int SizeN,int SizeZ,int SizeY){
//   int num = n + SizeN*( k + SizeZ*j +SizeY*SizeZ*i);
//   return num;
// }
//
//  __device__  int TwoDMapD(int i,int j,int size){
//   int num = j + i*size;
//   return num;
// }


//update B-field
void UPDATE_B(){
	// if(TEz && polar_psi==0){
	// 	UPDATE_hx();
	// 	UPDATE_hz();
	// }
	// else if(TMz && polar_psi==0){
	// 	UPDATE_hy();
	// }
	//else{
  int Number;
	int threadsPerBlock = 256;
  Number = NCELLX * NCELLY *NCELLZ;
	int blocksPerGrid = Number/threadsPerBlock + 1;
  // hipProfilerStart() ;
        //
		    // UPDATE_hx <<<blocksPerGrid, threadsPerBlock>>> (hx,ez,ey,Chxh,Chxe,psi_Hx_z_N,psi_Hx_z_F,psi_Hx_y_N,psi_Hx_y_F,khdy,khdz,bh_z_N,bh_z_F,ch_z_N,ch_z_F,bh_y_N,bh_y_F,ch_y_N,ch_y_F,NCELLX,NCELLY,NCELLZ,Periodic_XY,dx,dy,dz,dt,cpml_N_Z,cpml_F_Z,cpml_N_Y,cpml_F_Y,cpml_z_lim,cpml_y_lim,cpml_x_lim,NcpmlZ,NcpmlY);
        // UPDATE_hy <<<blocksPerGrid, threadsPerBlock>>> (hy,ez,ex,Chyh,Chye,psi_Hy_z_N,psi_Hy_z_F,psi_Hy_x_N,psi_Hy_x_F,khdx,khdz,bh_z_N,bh_z_F,ch_z_N,ch_z_F,bh_x_N,bh_x_F,ch_x_N,ch_x_F,NCELLX,NCELLY,NCELLZ,Periodic_XY,dx,dy,dz,dt,cpml_N_Z,cpml_F_Z,cpml_N_X,cpml_F_X,cpml_z_lim,cpml_y_lim,cpml_x_lim,NcpmlZ,NcpmlX);
        // UPDATE_hz <<<blocksPerGrid, threadsPerBlock>>> (hz,ey,ex,Chzh,Chze,psi_Hz_x_N,psi_Hz_x_F,psi_Hz_y_N,psi_Hz_y_F,khdx,khdy,bh_x_N,bh_x_F,ch_x_N,ch_x_F,bh_y_N,bh_y_F,ch_y_N,ch_y_F,NCELLX,NCELLY,NCELLZ,Periodic_XY,dx,dy,dz,dt,cpml_N_X,cpml_F_X,cpml_N_Y,cpml_F_Y,cpml_z_lim,cpml_y_lim,cpml_x_lim,NcpmlY,NcpmlX);

        hipDeviceSynchronize();
    // hipProfilerStop() ;

	//			UPDATE_hz();


//	}

}
//update E-field
void UPDATE_E(){

		int i,j,k,n;
comp hold;
int Number;
int threadsPerBlock = 256;
Number = NCELLX * NCELLY *NCELLZ;
int blocksPerGrid = Number/threadsPerBlock + 1;
     //
			// // UPDATE_ex();
     //  UPDATE_ex <<<blocksPerGrid, threadsPerBlock>>> (ex,ex_n,ex_n_1,hy,hz,Cexe,Cexh,kedy,kedz,mat_matrix,mat_matrixX,first_medium_max,psi_Ex_z_N,psi_Ex_z_F,psi_Ex_y_N,psi_Ex_y_F,Px_cp,Px_cp_n,Px_cp_n_1,Px_d,Px_d_n,Px_d_n_1,
     // C_1_cp,C_2_cp,C_3_cp,C_4_cp,C_5_cp,d_1_d,d_2_d,d_3_d,d_4_d,d_5_d,d_NL,C_E,z0,N_CP_poles,N_drude_poles,ce_z_N,ce_z_F,be_z_N,be_z_F,ce_y_N,ce_y_F,be_y_N,be_y_F,dx,dy,dz,dt,NCELLX,NCELLY,NCELLZ,
     // Hydrodynamics,cpml_x_lim,cpml_y_lim,cpml_z_lim,cpml_N_Y,cpml_F_Y,cpml_N_Z,cpml_F_Z,NcpmlY,NcpmlZ, C_E_1,C_E_2,Periodic_XY);
			// //UPDATE_ey();
     //  UPDATE_ey <<<blocksPerGrid, threadsPerBlock>>> (ey,ey_n,ey_n_1,hx,hz,Ceye,Ceyh,kedx,kedz,mat_matrix,mat_matrixY,first_medium_max,psi_Ey_z_N,psi_Ey_z_F,psi_Ey_x_N,psi_Ey_x_F,Py_cp,Py_cp_n,Py_cp_n_1,Py_d,Py_d_n,Py_d_n_1,
     // C_1_cp,C_2_cp,C_3_cp,C_4_cp,C_5_cp,d_1_d,d_2_d,d_3_d,d_4_d,d_5_d,d_NL,C_E,z0,N_CP_poles,N_drude_poles,ce_z_N,ce_z_F,be_z_N,be_z_F,ce_y_N,ce_x_F,be_x_N,be_x_F,dx,dy,dz,dt,NCELLX,NCELLY,NCELLZ,
     // Hydrodynamics,cpml_x_lim,cpml_y_lim,cpml_z_lim,cpml_N_X,cpml_F_X,cpml_N_Z,cpml_F_Z,NcpmlX,NcpmlZ, C_E_1,C_E_2,Periodic_XY);
     //
			// // UPDATE_ez();
     //  UPDATE_ez <<<blocksPerGrid, threadsPerBlock>>> (ez,ez_n,ez_n_1,hx,hy,Ceze,Cezh,kedx,kedy,mat_matrix,mat_matrixZ,first_medium_max,psi_Ez_y_N,psi_Ez_y_F,psi_Ez_x_N,psi_Ez_x_F,Pz_cp,Pz_cp_n,Pz_cp_n_1,Pz_d,Pz_d_n,Pz_d_n_1,
     // C_1_cp,C_2_cp,C_3_cp,C_4_cp,C_5_cp,d_1_d,d_2_d,d_3_d,d_4_d,d_5_d,d_NL,C_E,z0,N_CP_poles,N_drude_poles,ce_y_N,ce_y_F,be_y_N,be_y_F,ce_x_N,ce_x_F,be_x_N,be_x_F,dx,dy,dz,dt,NCELLX,NCELLY,NCELLZ,
     // Hydrodynamics,cpml_x_lim,cpml_y_lim,cpml_z_lim,cpml_N_X,cpml_F_X,cpml_N_Y,cpml_F_Y,NcpmlX,NcpmlY, C_E_1,C_E_2,Periodic_XY);
     //  hipDeviceSynchronize();

// for(i=0;i<NCELLX;i++){
// 	for(j=0;j<NCELLY;j++){
// 	printf("%e\t",Px_d[i][j][dispersive_slab+20][0]);
// 	//printf("%e\t",hx[i][j][dispersive_slab+20]);
// 	}
// 	printf("\n");
// }

		////#pragma omp parallel for collapse(3) private(i,j,k,n) //// schedule(guided)
		for(i=0;i<NCELLX;i++){
			for(j=0;j<NCELLY;j++){
				for(k=0;k<NCELLZ;k++){
					if(mat_matrixX[ThreeDMap(i,j,k,NCELLZ,NCELLY)] < 6 || mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)] <6 || mat_matrixZ[ThreeDMap(i,j,k,NCELLZ,NCELLY)] <6 || mat_matrix[ThreeDMap(i,j,k,NCELLZ,NCELLY)] <6){

								for(n=0;n<N_drude_poles;n++){
									Px_d_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Px_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
									Px_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Px_d[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
									Py_d_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Py_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
									Py_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Py_d[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
									Pz_d_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Pz_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
									Pz_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)] = Pz_d[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];

								}
								for(n=0;n<N_CP_poles;n++){
									Px_cp_n_1[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Px_cp_n[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
									Px_cp_n[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Px_cp[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
									Py_cp_n_1[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Py_cp_n[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
									Py_cp_n[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Py_cp[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
									Pz_cp_n_1[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Pz_cp_n[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
									Pz_cp_n[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)] = Pz_cp[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
								}

								if(Hydrodynamics){
									hxPrev[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
									hyPrev[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
									hzPrev[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
									hold = NDx[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
									NDx[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = NDx_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
									NDx_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=hold;
									hold = NDy[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
									NDy[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = NDy_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
									NDy_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=hold;
									hold = NDz[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
									NDz[ThreeDMap(i,j,k,NCELLZ,NCELLY)] = NDz_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
									NDz_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=hold;
								}
							}
					}
			}
		}
	}

  //
  // __global__ void UPDATE_ex(real *ex,real *ex_n,real *ex_n_1,real *hy,real *hz,real *Cexe,real *Cexh,real *kedy,real *kedz,int *mat_matrix,int *mat_matrixX,int first_medium_max,real *psi_Ex_z_N,
  // real *psi_Ex_z_F,real *psi_Ex_y_N,real *psi_Ex_y_F,real *Px_cp,real *Px_cp_n,real *Px_cp_n_1,real *Px_d,real *Px_d_n,real *Px_d_n_1,real *C_1_cp,real *C_2_cp,real *C_3_cp,real *C_4_cp,real *C_5_cp,real *d_1_d,
  // real *d_2_d,real *d_3_d,real *d_4_d,real *d_5_d,real *d_NL,real C_E,real z0,int N_CP_poles,int N_drude_poles,real *ce_z_N,real *ce_z_F,real *be_z_N,real *be_z_F,real *ce_y_N,real *ce_y_F,real *be_y_N,real *be_y_F,
  // real dx,real dy,real dz,real dt,int NCELLX,int NCELLY,int NCELLZ,int Hydrodynamics,int cpml_x_lim,int cpml_y_lim,int cpml_z_lim,int cpml_N_Y,int cpml_F_Y,int cpml_N_Z,int cpml_F_Z,int NcpmlY,int NcpmlZ,real C_E_1,real C_E_2,int Periodic_XY){
  //     int i,j,k,n,k2,j2;
  //     comp Curl_H, Div_Grad=0.0,J_T,dummy_var;
  // 		comp Vx1,Vx2,Vy1,Vy2,Vz1,Vz2,Nx1,Nx2,Ny1,Ny2,Nz1,Nz2;
  // 			    comp C_P_1,C_P_2,C_P_3,C_P_4,C_P_NL;
  // 					double INV_DX = 1.0/dx;
  // 					double INV_DY = 1.0/dy;
  // 					double INV_DZ = 1.0/dz;
  //
  //           int idx = blockDim.x * blockIdx.x + threadIdx.x;
  //
  //           i = idx / (NCELLZ*NCELLY);
  //           j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
  //           k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
  //
  //     if(Periodic_XY){
  // 			////#pragma omp parallel for collapse(3) private(Curl_H,i,j,j2,k,k2,n,dummy_var,J_T,Div_Grad) // schedule(static)
  // 			// for(k=1;k<NCELLZ-1;k++){
  // 			// 	for(i=0;i<NCELLX;i++){
  // 		  //       for(j=0;j<NCELLY;j++){
  //               if(i<NCELLX && j<NCELLY && k>0 && k<(NCELLZ-1)){
  // 	                //for(k=1;k<NCELLZ-1;k++){
  // 	                	if(j==0){
  // 											#ifdef DOUBLECOMPLEX
  // 												Curl_H=(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i,NCELLY-1,k,NCELLZ,NCELLY)]*cexp(I*k_y*period_y))/kedy[j]-(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k];
  // 											#endif
  // 											#ifdef DOUBLEPRECISION
  // 												Curl_H=(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i,NCELLY-1,k,NCELLZ,NCELLY)])/kedy[j]-(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k];
  // 											#endif
  //
  // 	                	}
  // 	                	else{
  // 	                		Curl_H=(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/kedy[j]-(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k];
  // 	                	}
  //
  // 										if(mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max && mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] < 6){
  //
  //
  // 											//  Div_Grad = Calc_DIV_GRADx(i,j,k);
  //                       Div_Grad = 0.0;
  //
  //                       C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;
  //
  //                       for(n=0;n<N_drude_poles;n++){
  //                           C_P_1+=(d_1_d[n]-1)*Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  //                           C_P_3+=(d_2_d[n])*Px_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  //                          C_P_NL += d_NL[n]*Div_Grad;
  //                       }
  //                       for(n=0;n<N_CP_poles;n++){
  //                           C_P_2+=(C_1_cp[n]-1)*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  //                           C_P_4+=(C_2_cp[n])*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  //                       }
  //                       ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  //                       ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  //                       ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4 - C_P_NL);
  //
  //
  // 												//printf("%e\n",Div_Grad);
  // 												//Z-CPML
  // 												if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
  // 													//Near-Z-PML
  // 													psi_Ex_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]=be_z_N[k]*psi_Ex_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]+ce_z_N[k]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 													ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)];
  // 												}
  // 												if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
  // 													k2 = k - cpml_F_Z ;
  // 													psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 													ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=(1/C_E)*dt*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
  // 												}
  //
  // 											 	for(n=0;n<N_CP_poles;n++){
  // 											 							Px_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 											 	}
  //
  // 												for(n=0;n<N_drude_poles;n++){
  // 													Px_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Px_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_NL[n]*Div_Grad;
  //
  // 												}
  //
  // 										}
  //
  // 										else{
  // 												ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Cexe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_H;
  // 												//Z-CPML
  // 												if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
  // 													//Near-Z-PML
  // 													psi_Ex_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=be_z_N[k]*psi_Ex_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ce_z_N[k]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 													ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 												}
  // 												if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
  // 													k2 = k - cpml_F_Z ;
  // 													psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 													ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)];
  // 												}
  // 										}
  //                   }
  //
  //
  //
  // 	  //       }
  // 	  //   }
  //     // }
  // 	}
  //     //No PBCs
  //     else{
  // 			////#pragma omp target teams distribute parallel for collapse(3) schedule(static,1) private(Curl_H,i,j,j2,k,k2,n,dummy_var,J_T,Div_Grad)
  // 		//	//#pragma omp parallel for collapse(3) private(Curl_H,i,j,j2,k,k2,n,dummy_var,J_T,Div_Grad) // schedule(static)
  // 	    // for(i=0;i<NCELLX-1;i++){
  // 	    //     for(j=1;j<NCELLY-1;j++){
  // 	    //             for(k=1;k<NCELLZ-1;k++){
  //                     if(i<(NCELLX-1) && j>0 && j<(NCELLY-1) && k>0 && k<(NCELLZ-1)){
  // 	                    Curl_H=(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/kedy[j]-(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k];
  //
  // 											if(mat_matrixX[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max && mat_matrixX[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] < 6){
  //
  //
  // 													if(Hydrodynamics == 0)
  // 													{
  // 														//Div_Grad = Calc_DIV_GRADx(i,j,k);
  //                             Div_Grad = 0.0;
  // 														// CP_D_ex(i,j,k,Curl_H,Div_Grad);
  //
  //                              C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;
  //
  //                              for(n=0;n<N_drude_poles;n++){
  //                                  C_P_1+=(d_1_d[n]-1)*Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  //                                  C_P_3+=(d_2_d[n])*Px_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  //                                 C_P_NL += d_NL[n]*Div_Grad;
  //                              }
  //                              for(n=0;n<N_CP_poles;n++){
  //                                  C_P_2+=(C_1_cp[n]-1)*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  //                                  C_P_4+=(C_2_cp[n])*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  //                              }
  //                              ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  //                              ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  //                              ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4 - C_P_NL);
  //
  //
  //
  //
  // 														for(n=0;n<N_CP_poles;n++){
  // 																				Px_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 														}
  //
  // 														for(n=0;n<N_drude_poles;n++){
  // 																				Px_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Px_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Px_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_NL[n]*Div_Grad;
  //
  // 														}
  // 													}
  // 													// else{
  //                           //
  // 													// 	Vx1 = Px_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)];
  // 													// 	Vx2 = Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)];
  // 													// 	Nx1 = NDx_prev[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)];
  // 													// 	Nx2 = NDx_prev[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)];
  //                           //
  // 													// 	Vy1 = 0.5*(Py_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 													// 	Vy2 = 0.5*(Py_d_n[FourDMapD(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 													// 	Ny1 = 0.5*(NDy_prev[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)] + NDy_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
  // 													// 	Ny2 = 0.5*(NDy_prev[ThreeDMapD(i+1,j-1,k,NCELLZ,NCELLY)] + NDy_prev[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)]);
  //                           //
  // 													// 	Vz1 = 0.5*(Pz_d_n[FourDMapD(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 													// 	Vz2 = 0.5*(Pz_d_n[FourDMapD(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 													// 	Nz1 = 0.5*(NDz_prev[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)] + NDz_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
  // 													// 	Nz2 = 0.5*(NDz_prev[ThreeDMapD(i+1,j,k-1,NCELLZ,NCELLY)] + NDz_prev[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]);
  //                           //
  // 													// 	NDx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = NDx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] - 2.0*dt*INV_DX*(0.5*(Nx1*Vx1-Nx2*Vx2) + (Ny1*Vy1-Ny2*Vy2) + (Nz1*Vz1-Nz2*Vz2) + (0.5*(Vx1-Vx2) + (Vy1-Vy2) + (Vz1-Vz2))*N_EQ);
  //                           //
  //                           //
  // 												  //   C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;
  //                           //
  //                           //
  // 												  //   for(n=0;n<N_CP_poles;n++){
  // 												  //       C_P_2+=(C_1_cp[n]-1)*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  // 												  //       C_P_4+=(C_2_cp[n])*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  // 												  //   }
  // 												  //   ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 												  //   ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 												  //   ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0 + dt*Px_d[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*e0*(N_EQ + NDx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]) -C_E_2*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_2-C_P_4);
  //                           //
  // 													// 	for(n=0;n<N_CP_poles;n++){
  // 													// 							Px_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 													// 	}
  //                           //
  //                           //
  // 													// }
  //
  // 												//	printf("%e\n",Div_Grad);
  // 													//Z-CPML
  // 													// if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
  // 													// 	//Near-Z-PML
  // 													// 	psi_Ex_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=be_z_N[k]*psi_Ex_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ce_z_N[k]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 													// 	ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 													// }
  // 													// if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
  // 													// 	k2 = k - cpml_F_Z ;
  // 													// 	psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 													// 	ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=(1/C_E)*dt*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
  // 													// }
  //
  //
  //
  // 											}
  //
  // 	                    else{
  // 	                        ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Cexe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_H;
  // 	                    }
  // 											//Z-CPML
  // 											if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
  // 												//Near-Z-PML
  // 												psi_Ex_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]=be_z_N[k]*psi_Ex_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]+ce_z_N[k]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 												ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)];
  // 											}
  // 											if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){ //Far Z PML
  // 												k2 = k - cpml_F_Z;
  // 												psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 												// if(mat_matrixX[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max){
  // 												// 	ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=(1/C_E)*dt*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
  // 												// }
  // 												// else{
  // 													ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)];
  // 												//}
  // 											}
  // 											//Y PML
  // 											if(j<cpml_N_Y+1 && i<cpml_x_lim && k<cpml_z_lim){ //Near Y PML
  // 												psi_Ex_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY+1)]=be_y_N[j]*psi_Ex_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY+1)]+ce_y_N[j]*(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/dy;
  // 												ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY+1)];
  // 											}
  // 											if(j>=cpml_F_Y && i<cpml_x_lim && k<cpml_z_lim){ //Far Y PML
  // 												j2 = j - cpml_F_Y;
  // 												psi_Ex_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY+1)]=be_y_F[j2]*psi_Ex_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY+1)]+ce_y_F[j2]*(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/dy;
  // 												ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cexh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ex_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY+1)];
  // 											}
  // 									// 		if(mat_matrixX[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max){
  // 									// 				for(n=0;n<N_CP_poles;n++){
  // 									// 										Px_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Px_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Px_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 									// 				}
  // 									// 				for(n=0;n<N_drude_poles;n++){
  // 									// 										Px_d[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=d_1_d[n]*Px_d_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+d_2_d[n]*Px_d_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+d_3_d[n]*ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ex_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ex_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 									// 				}
  // 	                // }
  //                 }
  // 	//         }
  // 	//     }
  // 	// }
  // }
  //
  //   return;
  // }
  //
  //
  //
  //
  //
  //
  // __global__ void UPDATE_ey(real *ey,real *ey_n,real *ey_n_1,real *hx,real *hz,real *Ceye,real *Ceyh,real *kedx,real *kedz,int *mat_matrix,int *mat_matrixY,int first_medium_max,real *psi_Ey_z_N,
  // real *psi_Ey_z_F,real *psi_Ey_x_N,real *psi_Ey_x_F,real *Py_cp,real *Py_cp_n,real *Py_cp_n_1,real *Py_d,real *Py_d_n,real *Py_d_n_1,real *C_1_cp,real *C_2_cp,real *C_3_cp,real *C_4_cp,real *C_5_cp,real *d_1_d,
  // real *d_2_d,real *d_3_d,real *d_4_d,real *d_5_d,real *d_NL,real C_E,real z0,int N_CP_poles,int N_drude_poles,real *ce_z_N,real *ce_z_F,real *be_z_N,real *be_z_F,real *ce_x_N,real *ce_x_F,real *be_x_N,real *be_x_F,
  // real dx,real dy,real dz,real dt,int NCELLX,int NCELLY,int NCELLZ,int Hydrodynamics,int cpml_x_lim,int cpml_y_lim,int cpml_z_lim,int cpml_N_X,int cpml_F_X,int cpml_N_Z,int cpml_F_Z,int NcpmlX,int NcpmlZ,real C_E_1,real C_E_2,int Periodic_XY){
  //     int i,j,k,i2,k2,n;
  //     comp Curl_H,Div_Grad=0.0,dummy_var,J_T;
  // 		comp C_P_1,C_P_2,C_P_3,C_P_4,C_P_NL;
  // 		double INV_DX = 1.0/dx;
  // 		double INV_DY = 1.0/dy;
  // 		double INV_DZ = 1.0/dz;
  // 		comp Vx1,Vx2,Vy1,Vy2,Vz1,Vz2,Nx1,Nx2,Ny1,Ny2,Nz1,Nz2;
  //     int idx = blockDim.x * blockIdx.x + threadIdx.x;
  //
  //     i = idx / (NCELLZ*NCELLY);
  //     j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
  //     k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
  //
  //     if(Periodic_XY){
  //     	////#pragma omp parallel for collapse(3) private(Curl_H,i,j,i2,k,k2,n,dummy_var,J_T,Div_Grad) // schedule(static)
  // 			// for(k=0;k<NCELLZ-1;k++){
  // 			//   for(i=0;i<NCELLX;i++){
  // 		  //       for(j=0;j<NCELLY;j++){
  //               if(i<NCELLX && j<NCELLY && k>0 && k<(NCELLZ-1)){
  // 	               // for(k=1;k<NCELLZ-1;k++){
  // 	                	if(i==0){
  // 											#ifdef DOUBLECOMPLEX
  // 	                		Curl_H=(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k]-(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)]*cexp(I*period_x*k_x))/kedx[i];
  // 											#endif
  // 											#ifdef DOUBLEPRECISION
  // 											Curl_H=(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k]-(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)])/kedx[i];
  // 											#endif
  // 											  //printf("%d,%d,%d \t %f\t%f\t%f\n",i,j,k,creal(Curl_H),creal(hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]), creal(hz[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)]));
  //
  // 	                	}
  //
  // 	                	else{
  // 	                		Curl_H=(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k]-(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/kedx[i];
  // 	                	}
  // 										if(mat_matrixY[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max && mat_matrixY[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] <6){
  //
  // 												//Div_Grad = Calc_DIV_GRADy(i,j,k);
  //                         Div_Grad = 0.0;
  // 											// printf("%e\n",d_NL[0]*Div_Grad);
  // 												// CP_D_ey(i,j,k,Curl_H,Div_Grad);
  //                         C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;
  //                         //printf("here");
  //                         for(n=0;n<N_drude_poles;n++){
  //                             C_P_1+=(d_1_d[n]-1.0)*Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  //                             C_P_3+=(d_2_d[n])*Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  //                             C_P_NL += d_NL[n]*Div_Grad;
  //                         }
  //                         for(n=0;n<N_CP_poles;n++){
  //                             C_P_2+=(C_1_cp[n]-1.0)*Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  //                             C_P_4+=(C_2_cp[n])*Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  //                         }
  //                         ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  //                         ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  //                         ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4-C_P_NL);
  //
  //
  // 											//	printf("%e\n",ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
  // 												//Z-CPML
  // 												if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
  // 													//Here we are in the near Z-PML
  // 													psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]=be_z_N[k]*psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]+ce_z_N[k]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 													ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)];
  // 												}
  // 												if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
  // 													//Here we are in the far Z-PML
  // 														k2 = k - cpml_F_Z ;
  // 														psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 															ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=(1/C_E)*dt*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
  // 												}
  //
  // 														for(n=0;n<N_CP_poles;n++){
  // 																				Py_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 														}
  //
  // 														//if(Hydrodynamics == 0){
  // 														for(n=0;n<N_drude_poles;n++){
  // 															Py_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_NL[n]*Div_Grad;
  //
  // 														}
  // 													//}
  //
  // 										}
  //
  // 										else{
  // 												ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Ceye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_H;
  // 												//Z-CPML
  // 												if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
  // 													//Here we are in the near Z-PML
  // 													psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]=be_z_N[k]*psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]+ce_z_N[k]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 													ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)];
  // 												}
  // 												if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
  // 													//Here we are in the far Z-PML
  // 														k2 = k - cpml_F_Z ;
  // 														psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 														if(mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max){
  // 															ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=(1/C_E)*dt*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
  // 														}
  // 														else{
  // 															ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)];
  // 														}
  // 												}
  // 										}
  //
  //                   }
  //
  // 	      //       }
  // 	      //   }
  //    	    // }
  //     }
  //
  //     else{
  // 		//	//#pragma omp parallel for collapse(3) private(Curl_H,i,j,i2,k,k2,n,dummy_var,J_T,Div_Grad) // schedule(static)
  // 	    // for(i=1;i<NCELLX-1;i++){
  // 	    //     for(j=0;j<NCELLY-1;j++){
  // 	    //             for(k=1;k<NCELLZ-1;k++){
  //                     if(i>0 && i<(NCELLX-1) && j<(NCELLY-1) && k>0 && k<(NCELLZ-1)){
  // 	                    Curl_H=(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/kedz[k]-(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/kedx[i];
  // 											if(mat_matrixY[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max && mat_matrixY[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] <6){
  //
  // 												if(Hydrodynamics == 0){
  // 													//Div_Grad = Calc_DIV_GRADy(i,j,k);
  //                           Div_Grad= 0.0;
  // 												// printf("%e\n",d_NL[0]*Div_Grad);
  // 													// CP_D_ey(i,j,k,Curl_H,Div_Grad);
  //                           C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;
  //                       		//printf("here");
  //                           for(n=0;n<N_drude_poles;n++){
  //                               C_P_1+=(d_1_d[n]-1.0)*Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  //                               C_P_3+=(d_2_d[n])*Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  //                       				C_P_NL += d_NL[n]*Div_Grad;
  //                           }
  //                           for(n=0;n<N_CP_poles;n++){
  //                               C_P_2+=(C_1_cp[n]-1.0)*Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  //                               C_P_4+=(C_2_cp[n])*Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  //                           }
  //                           ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  //                           ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  //                           ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4-C_P_NL);
  //
  // 												//	printf("%e\n",ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
  // 													//Z-CPML
  // 													// if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
  // 													// 	//Here we are in the near Z-PML
  // 													// 	psi_Ey_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=be_z_N[k]*psi_Ey_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ce_z_N[k]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 													// 	ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_z_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 													// }
  // 													// if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
  // 													// 	//Here we are in the far Z-PML
  // 													// 		k2 = k - cpml_F_Z ;
  // 													// 		psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 													// 			ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=(1/C_E)*dt*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
  // 													// }
  //
  // 															for(n=0;n<N_CP_poles;n++){
  // 																					Py_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 															}
  //
  // 															for(n=0;n<N_drude_poles;n++){
  // 																Py_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Py_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Py_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_NL[n]*Div_Grad;
  //
  // 															}
  //
  // 													}
  // 											// 		else{
  //                       //
  // 											// 			Vy1 = Py_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)];
  // 											// 			Vy2 = Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)];
  // 											// 			Ny1 = NDy_prev[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)];
  // 											// 			Ny2 = NDy_prev[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)];
  //                       //
  // 											// 			Vx1 = 0.5*(Px_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 											// 			Vx2 = 0.5*(Px_d_n[FourDMapD(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 											// 			Nx1 = 0.5*(NDx_prev[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)] + NDx_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
  // 											// 			Nx2 = 0.5*(NDx_prev[ThreeDMapD(i-1,j+1,k,NCELLZ,NCELLY)] + NDx_prev[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)]);
  //                       //
  // 											// 			Vz1 = 0.5*(Pz_d_n[FourDMapD(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 											// 			Vz2 = 0.5*(Pz_d_n[FourDMapD(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 											// 			Nz1 = 0.5*(NDz_prev[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)] + NDz_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
  // 											// 			Nz2 = 0.5*(NDz_prev[ThreeDMapD(i,j+1,k-1,NCELLZ,NCELLY)] + NDz_prev[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)]);
  //                       //
  // 											// 			NDy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = NDy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] - 2.0*dt*INV_DX*((Nx1*Vx1-Nx2*Vx2) + 0.5*(Ny1*Vy1-Ny2*Vy2) + (Nz1*Vz1-Nz2*Vz2) + ((Vx1-Vx2) + 0.5*(Vy1-Vy2) + (Vz1-Vz2))*N_EQ);
  //                       //
  //                       //
  // 											//     C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;
  //                       //
  // 											//     for(n=0;n<N_CP_poles;n++){
  // 											//         C_P_2+=(C_1_cp[n]-1.0)*Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  // 											//         C_P_4+=(C_2_cp[n])*Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  // 											//     }
  // 											//     ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 											//     ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 											//     ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0 + dt*Py_d[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*e0*(N_EQ + NDy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]) + C_E_1*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_2-C_P_4);
  // 											// 		for(n=0;n<N_CP_poles;n++){
  // 											// 								Py_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Py_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Py_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ey_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ey_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 											// 		}
  //                       //
  // 											// }
  // 										}
  //
  //
  // 										else{
  // 												ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Ceye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_H;
  //
  // 										}
  //
  // 										if(k<cpml_N_Z+1 && i<cpml_x_lim && j<cpml_y_lim){
  // 											//Here we are in the near Z-PML
  // 											psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]=be_z_N[k]*psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)]+ce_z_N[k]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 											ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_z_N[ThreeDMapD(i,j,k,NcpmlZ+1,NCELLY)];
  // 										}
  // 										if(k>=cpml_F_Z && i<cpml_x_lim && j<cpml_y_lim){
  // 												//Here we are in the far Z-PML
  // 												k2 = k - cpml_F_Z;
  // 												psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]=be_z_F[k2]*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]+ce_z_F[k2]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)])/dz;
  // 												// if(mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max){
  // 												// 	ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=(1/C_E)*dt*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)]/z0;
  // 												// }
  // 												// else{
  // 													ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_z_F[ThreeDMapD(i,j,k2,NcpmlZ+1,NCELLY)];
  // 												//}
  // 										}
  // 										//X-CPML
  // 										if(i<cpml_N_X+1 && j<cpml_y_lim && k<cpml_z_lim){
  // 											//Here we are in the near-X-PML
  // 											psi_Ey_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=be_x_N[i]*psi_Ey_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ce_x_N[i]*(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/dx;
  // 											ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 										}
  // 										if(i>=cpml_F_X && j<cpml_y_lim && k<cpml_z_lim){
  // 											//Here we are in the far-X-PML
  // 											i2 = i - cpml_F_X;
  // 											psi_Ey_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]=be_x_F[i2]*psi_Ey_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]+ce_x_F[i2]*(hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hz[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/dx;
  // 											ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Ceyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ey_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)];
  // 										}
  //
  //
  //                   }
  //
  // 								}
  // 	  //       }
  //     //
  // 	  //   }
  //     // }
  //
  //   return;
  // }
  //
  //
  // //void UPDATE_ez(void){
  //   __global__ void UPDATE_ez(real *ez,real *ez_n,real *ez_n_1,real *hx,real *hy,real *Ceze,real *Cezh,real *kedx,real *kedy,int *mat_matrix,int *mat_matrixZ,int first_medium_max,real *psi_Ez_y_N,
  //   real *psi_Ez_y_F,real *psi_Ez_x_N,real *psi_Ez_x_F,real *Pz_cp,real *Pz_cp_n,real *Pz_cp_n_1,real *Pz_d,real *Pz_d_n,real *Pz_d_n_1,real *C_1_cp,real *C_2_cp,real *C_3_cp,real *C_4_cp,real *C_5_cp,real *d_1_d,
  //   real *d_2_d,real *d_3_d,real *d_4_d,real *d_5_d,real *d_NL,real C_E,real z0,int N_CP_poles,int N_drude_poles,real *ce_y_N,real *ce_y_F,real *be_y_N,real *be_y_F,real *ce_x_N,real *ce_x_F,real *be_x_N,real *be_x_F,
  //   real dx,real dy,real dz,real dt,int NCELLX,int NCELLY,int NCELLZ,int Hydrodynamics,int cpml_x_lim,int cpml_y_lim,int cpml_z_lim,int cpml_N_X,int cpml_F_X,int cpml_N_Y,int cpml_F_Y,int NcpmlX,int NcpmlY,real C_E_1,real C_E_2,int Periodic_XY){
  //
  //     int i,j,k,i2,j2,n;
  //     comp Curl_H,Div_Grad=0.0,dummy_var,J_T;
  // 		comp C_P_1,C_P_2,C_P_3,C_P_4,C_P_NL;
  // 		comp Vx1,Vx2,Vy1,Vy2,Vz1,Vz2,Nx1,Nx2,Ny1,Ny2,Nz1,Nz2;
  //
  // 		double INV_DX = 1.0/dx;
  // 		double INV_DY = 1.0/dy;
  // 		double INV_DZ = 1.0/dz;
  //     int idx = blockDim.x * blockIdx.x + threadIdx.x;
  //
  //     i = idx / (NCELLZ*NCELLY);
  //     j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
  //     k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
  //
  //     if(Periodic_XY){
  // 	//	//#pragma omp parallel for collapse(3) private(Curl_H,i,j,k,i2,j2,n,dummy_var,J_T,Div_Grad) // schedule(static)
  // 		// for(k=0;k<NCELLZ-1;k++){
  // 		// for(i=0;i<NCELLX;i++){
  // 		//         for(j=0;j<NCELLY;j++){
  //               if(i<NCELLX,j<NCELLY,k<NCELLZ){
  // 		           //     for(k=0;k<NCELLZ-1;k++){
  // 											//printf("Thread %d, ready to work\n",omp_get_thread_num());
  //
  // 		                	if(i==0 || j==0){
  // 		                		if(i==0 && j==0){
  // 													#ifdef DOUBLECOMPLEX
  // 				        						Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)]*cexp(I*k_x*period_x))/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,NCELLY-1,k,NCELLZ,NCELLY)]*cexp(I*k_y*period_y))/kedy[j];
  // 														#endif
  // 														#ifdef DOUBLEPRECISION
  // 														Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)])/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,NCELLY-1,k,NCELLZ,NCELLY)])/kedy[j];
  // 														#endif
  //
  //
  // 			               	    }
  // 			               	    else if(i==0){
  // 													#ifdef DOUBLECOMPLEX
  // 				                 	Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)]*cexp(I*k_x*period_x))/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/kedy[j];
  // 													#endif
  // 													#ifdef DOUBLEPRECISION
  // 													Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(NCELLX-1,j,k,NCELLZ,NCELLY)])/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/kedy[j];
  // 													#endif
  //
  //
  // 				                }
  // 				                else{
  // 													#ifdef DOUBLECOMPLEX
  // 				                	Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,NCELLY-1,k,NCELLZ,NCELLY)]*cexp(I*k_y*period_y))/kedy[j];
  // 													#endif
  // 													#ifdef DOUBLEPRECISION
  // 													Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,NCELLY-1,k,NCELLZ,NCELLY)])/kedy[j];
  // 													#endif
  //
  //
  // 				                }
  // 		        			}
  //
  // 		        			else{
  // 		        				Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/kedy[j];
  //
  // 		        			}
  // 									if(mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max && mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]<6){
  //
  // 											// Div_Grad  = Calc_DIV_GRADz(i,j,k);
  //                       Div_Grad = 0.0;
  //
  // 											// CP_D_ez(i,j,k,Curl_H,Div_Grad);
  //                       C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;
  //
  //                       for(n=0;n<N_drude_poles;n++){
  //                           C_P_1+=(d_1_d[n]-1)*Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  //                           C_P_3+=(d_2_d[n])*Pz_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  //                           C_P_NL += d_NL[n]*Div_Grad;
  //                       }
  //                       for(n=0;n<N_CP_poles;n++){
  //                           C_P_2+=(C_1_cp[n]-1)*Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  //                           C_P_4+=(C_2_cp[n])*Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  //                       }
  //                       ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  //                       ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  //                       ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4-C_P_NL);
  //
  //
  // 												for(n=0;n<N_CP_poles;n++){
  // 																		Pz_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 												}
  //
  // 												for(n=0;n<N_drude_poles;n++){
  // 													Pz_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Pz_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_NL[n]*Div_Grad;
  // 												}
  // 									}
  //
  // 									else{
  // 											ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Ceze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_H;
  // 									}
  //
  //                 }
  // 		      //     }
  // 		      //   }
  // 					// }
  //     }
  //
  //     else{
  // 	//		//#pragma omp parallel for collapse(3) private(Curl_H,i,j,k,i2,j2,n,dummy_var,J_T,Div_Grad) // schedule(static)
  // 	    // for(i=1;i<NCELLX-1;i++){
  // 	    //     for(j=1;j<NCELLY-1;j++){
  // 	    //             for(k=0;k<NCELLZ-1;k++){
  //                     if(i>0 && i<(NCELLX-1) && j>0 && j<(NCELLY-1) && k<(NCELLZ-1)){
  // 	                    Curl_H=(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/kedx[i]-(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/kedy[j];
  // 											if(mat_matrixZ[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max && mat_matrixZ[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]<6){
  //
  // 												if(Hydrodynamics == 0){
  // 			 									 //Div_Grad  = Calc_DIV_GRADz(i,j,k);
  //                          Div_Grad = 0.0;
  // 			 									// CP_D_ez(i,j,k,Curl_H,Div_Grad);
  //                         C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;
  //
  //                         for(n=0;n<N_drude_poles;n++){
  //                             C_P_1+=(d_1_d[n]-1)*Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  //                             C_P_3+=(d_2_d[n])*Pz_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  //                             C_P_NL += d_NL[n]*Div_Grad;
  //                         }
  //                         for(n=0;n<N_CP_poles;n++){
  //                             C_P_2+=(C_1_cp[n]-1)*Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  //                             C_P_4+=(C_2_cp[n])*Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  //                         }
  //                         ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  //                         ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  //                         ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4-C_P_NL);
  //
  //
  // 			 										 for(n=0;n<N_CP_poles;n++){
  // 			 																 Pz_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 			 										 }
  // 			 										 for(n=0;n<N_drude_poles;n++){
  // 			 											 Pz_d[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Pz_d_n[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Pz_d_n_1[FourDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + d_NL[n]*Div_Grad;
  // 			 										 }
  //
  // 											 }
  // 											 // else{
  //                        //
  // 												//  C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;
  //                        //
  // 												//  Vz1 = Pz_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)];
  // 												//  Vz2 = Pz_d_n[FourDMapD(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)];
  // 												//  Nz1 = NDz_prev[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)];
  // 												//  Nz2 = NDz_prev[ThreeDMapD(i,j,k-1,NCELLZ,NCELLY)];
  //                        //
  // 												//  Vx1 = 0.5*(Px_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 												//  Vx2 = 0.5*(Px_d_n[FourDMapD(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMapD(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 												//  Nx1 = 0.5*(NDx_prev[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)] + NDx_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
  // 												//  Nx2 = 0.5*(NDx_prev[ThreeDMapD(i-1,j,k+1,NCELLZ,NCELLY)] + NDx_prev[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)]);
  //                        //
  // 												//  Vy1 = 0.5*(Py_d_n[FourDMapD(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 												//  Vy2 = 0.5*(Py_d_n[FourDMapD(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMapD(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 												//  Ny1 = 0.5*(NDy_prev[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)] + NDy_prev[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]);
  // 												//  Ny2 = 0.5*(NDy_prev[ThreeDMapD(i,j-1,k+1,NCELLZ,NCELLY)] + NDy_prev[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)]);
  //                        //
  // 												//  NDz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] = NDz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] - 2.0*dt*INV_DX*((Nx1*Vx1-Nx2*Vx2) + (Ny1*Vy1-Ny2*Vy2) + 0.5*(Vz1-Vz2) + ((Vx1-Vx2) + (Vy1-Vy2) + 0.5*(Vz1-Vz2))*N_EQ);
  //                        //
  //                        //
  // 												//  for(n=0;n<N_CP_poles;n++){
  // 												// 		 C_P_2+=(C_1_cp[n]-1)*Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  // 												// 		 C_P_4+=(C_2_cp[n])*Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
  // 												//  }
  // 												//  ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 												//  ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 												//  ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0 + dt*Pz_d[FourDMapD(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*e0*(NDz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] + N_EQ)+C_E_1*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_E_2*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-C_P_2-C_P_4);
  //                        //
  // 												//  for(n=0;n<N_CP_poles;n++){
  // 												// 						 Pz_cp[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Pz_cp_n[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Pz_cp_n_1[FourDMapD(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 												//  }
  // 											 // }
  // 			 							 }
  //
  //
  // 	                    else{
  // 	                        ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Ceze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_H;
  // 	                    }
  // 											//Y CPML
  // 											if(j<cpml_N_Y && i<cpml_x_lim && k<cpml_z_lim){ //Near Y PML
  // 												psi_Ez_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY+1)]=be_y_N[j]*psi_Ez_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY+1)]+ce_y_N[j]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/dy;
  // 												ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ez_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY+1)];
  // 											}
  // 											if(j>=cpml_F_Y && i<cpml_x_lim &&  k<cpml_z_lim){
  // 												j2 = j - cpml_F_Y;
  // 												psi_Ez_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY+1)]=be_y_F[j2]*psi_Ez_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY+1)]+ce_y_F[j2]*(hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hx[ThreeDMapD(i,j-1,k,NCELLZ,NCELLY)])/dy;
  // 												ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ez_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY+1)];
  // 											}
  // 											//X PML
  // 											if(i<cpml_N_X+1 && j<cpml_y_lim && k<cpml_z_lim){//Near X-PML
  // 												psi_Ez_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=be_x_N[i]*psi_Ez_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ce_x_N[i]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/dx;
  // 												ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ez_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 											}
  // 											if(i>=cpml_F_X && j<cpml_y_lim && k<cpml_z_lim){//far X-PML
  // 												i2 = i - cpml_F_X;
  // 												psi_Ez_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]=be_x_F[i2]*psi_Ez_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]+ce_x_F[i2]*(hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-hy[ThreeDMapD(i-1,j,k,NCELLZ,NCELLY)])/dx;
  // 												ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Cezh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Ez_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)];
  // 											}
  // 											// if(mat_matrix[ThreeDMapD(i,j,k,NCELLZ,NCELLY)] > first_medium_max){
  // 											// 		 for(n=0;n<N_CP_poles;n++){
  // 											// 				Pz_cp[ThreeDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=C_1_cp[n]*Pz_cp_n[ThreeDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+C_2_cp[n]*Pz_cp_n_1[ThreeDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+C_3_cp[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_4_cp[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+C_5_cp[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 											// 				}
  // 											// 		for(n=0;n<N_drude_poles;n++){
  // 											// 				Pz_d[ThreeDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]=d_1_d[n]*Pz_d_n[ThreeDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_2_d[n]*Pz_d_n_1[ThreeDMapD(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)]+d_3_d[n]*ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_4_d[n]*ez_n[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+d_5_d[n]*ez_n_1[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
  // 											// 				}
  // 											// }
  //
  // 	                }
  //       //           }
  // 	    //     }
  //       //
  // 	    // }
  //     }
  //
  //   return;
  // }
  //





void CP_D_ex(int i,int j, int k, comp Curl_H,comp Div_Grad){
      int n;
      comp C_P_1,C_P_2,C_P_3,C_P_4,C_P_NL;
      C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;

      for(n=0;n<N_drude_poles;n++){
          C_P_1+=(d_1_d[n]-1)*Px_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
          C_P_3+=(d_2_d[n])*Px_d_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  				C_P_NL += d_NL[n]*Div_Grad;
      }
      for(n=0;n<N_CP_poles;n++){
          C_P_2+=(C_1_cp[n]-1)*Px_cp_n[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
          C_P_4+=(C_2_cp[n])*Px_cp_n_1[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
      }
      ex_n_1[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=ex_n[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
      ex_n[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
      ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ex_n[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-C_E_2*ex_n_1[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4 - C_P_NL);
  }

  void CP_D_ey(int i,int j, int k, comp Curl_H,comp Div_Grad ){
      int n;
      comp C_P_1,C_P_2,C_P_3,C_P_4,C_P_NL;
      C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;
  		//printf("here");
      for(n=0;n<N_drude_poles;n++){
          C_P_1+=(d_1_d[n]-1.0)*Py_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
          C_P_3+=(d_2_d[n])*Py_d_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  				C_P_NL += d_NL[n]*Div_Grad;
      }
      for(n=0;n<N_CP_poles;n++){
          C_P_2+=(C_1_cp[n]-1.0)*Py_cp_n[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
          C_P_4+=(C_2_cp[n])*Py_cp_n_1[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
      }
      ey_n_1[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=ey_n[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
      ey_n[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
      ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ey_n[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-C_E_2*ey_n_1[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4-C_P_NL);

  }

  void CP_D_ez(int i,int j, int k, comp Curl_H, comp Div_Grad){
      int n;
      comp C_P_1,C_P_2,C_P_3,C_P_4,C_P_NL;
      C_P_1=C_P_2=C_P_3=C_P_4=C_P_NL=0.0;

      for(n=0;n<N_drude_poles;n++){
          C_P_1+=(d_1_d[n]-1)*Pz_d_n[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
          C_P_3+=(d_2_d[n])*Pz_d_n_1[FourDMap(i,j,k,n,N_drude_poles,NCELLZ,NCELLY)];
  				C_P_NL += d_NL[n]*Div_Grad;
      }
      for(n=0;n<N_CP_poles;n++){
          C_P_2+=(C_1_cp[n]-1)*Pz_cp_n[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
          C_P_4+=(C_2_cp[n])*Pz_cp_n_1[FourDMap(i,j,k,n,N_CP_poles,NCELLZ,NCELLY)];
      }
      ez_n_1[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=ez_n[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
      ez_n[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
      ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)]=(1/C_E)*(dt*Curl_H/z0+C_E_1*ez_n[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-C_E_2*ez_n_1[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-C_P_1-C_P_2-C_P_3-C_P_4-C_P_NL);

  }

//
//
//   __global__ void UPDATE_hx(real *hx,real *ez,real *ey,real *Chxh,real *Chxe,real *psi_Hx_z_N,real *psi_Hx_z_F,real *psi_Hx_y_N,real *psi_Hx_y_F,real *khdy,real
//     *khdz,real *bh_z_N,real *bh_z_F,real *ch_z_N,real *ch_z_F,real *bh_y_N,real *bh_y_F,real *ch_y_N,real *ch_y_F,int NCELLX,int NCELLY,int NCELLZ,int Periodic_XY,real dx,real dy,real dz,real dt,int cpml_N_Z,int cpml_F_Z,int cpml_N_Y,int cpml_F_Y,int cpml_z_lim,int cpml_y_lim,int cpml_x_lim,int NcpmlZ,int NcpmlY){
// //void UPDATE_hx(void){
// // hipProfilerStart();
//
//       int i,j,k,j2,k2;
//       comp Curl_E;
//       int idx = blockDim.x * blockIdx.x + threadIdx.x;
//
//       i = idx / (NCELLZ*NCELLY);
//       j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
//       k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
//
//       if(Periodic_XY){
//   	////#pragma omp parallel for collapse(3) private(i,j,k,Curl_E,j2,k2) // schedule(static)
//   	// for(k=0;k<NCELLZ;k++){
//   	// 	for(i=0;i<NCELLX;i++){
//   	//         for(j=0;j<NCELLY;j++){
//
//               if(i<NCELLX && j<NCELLY && k<NCELLZ){
//   	           //     for(k=0;k<NCELLZ-1;k++){
//   									 //if(i==1) printf("%d %d %d\n",i,j,k);
//   		                	if(j==NCELLY-1){
//   												#ifdef DOUBLECOMPLEX
//   		                				Curl_E=(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k]-(ez[ThreeDMapD(i,0,k,NCELLZ,NCELLY)]*cexp(-I*k_y*period_y)-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j];
//   												#endif
//   												#ifdef DOUBLEPRECISION
//   		                				Curl_E=(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k]-(ez[ThreeDMapD(i,0,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j];
//   												#endif
//   		                   	    hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chxh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
//
//   		                	}
//   		                	 else{
//   														Curl_E=(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k]-(ez[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j];
//   		                   	    hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chxh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
//   		                	}
//   										 //Z-CPML
//   										 if(k<cpml_N_Z && i<cpml_x_lim && j<cpml_y_lim){
//   											 	//Near Z-PML
//   											 		psi_Hx_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]=bh_z_N[k]*psi_Hx_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]+ch_z_N[k]*(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
//   													hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hx_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)];
//   										 }
//   										 if(k>=cpml_F_Z && j<cpml_y_lim && i<cpml_x_lim){
//   											 //Far Z-PML
//   											 		k2 = k - cpml_F_Z;
//   													psi_Hx_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]=bh_z_F[k2]*psi_Hx_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]+ch_z_F[k2]*(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
//   													hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hx_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)];
//   										 }
//                      }
//   	    //             }
//   	    //     }
//   	    // }
//       }
//
//       else{
//   			//  //#pragma omp target device(0) MapD(Chxe[:NCELLX-1][:NCELLY-1][:NCELLZ-1],Chxh[:NCELLX-1][:NCELLY-1][:NCELLZ-1],ez[:NCELLX-1][:NCELLY-1][:NCELLZ-1],ey[:NCELLX-1][:NCELLY-1][:NCELLZ-1],khdy[:NCELLY-1],khdz[:NCELLZ-1],bh_z_N[:NcpmlZ-1],bh_z_F[:NcpmlZ-1],ch_z_N[:NcpmlZ-1],ch_z_F[:NcpmlZ-1],bh_y_N[:NcpmlY-1],bh_y_F[:NcpmlY-1],ch_y_N[:NcpmlY-1],ch_y_F[:NcpmlY-1]) 		MapD(tofrom:hx[:NCELLX-1][:NCELLY-1][:NCELLZ-1],psi_Hx_z_N[:NCELLX-1][:NCELLY-1][:cpml_N_Z-1],psi_Hx_z_F[:NCELLX-1][:NCELLY-1][:cpml_N_Z-1],psi_Hx_y_N[:NCELLX-1][:cpml_N_Y-1][:NCELLZ-1],psi_Hx_y_F[:NCELLX-1][:cpml_N_Y-1][:NCELLZ-1])
//   			//  {
//   			// //#pragma omp parallel for collapse(3) private(i,j,k,Curl_E,j2,k2) // schedule(static)
//   	    // for(i=0;i<NCELLX;i++){
//   	    //     for(j=0;j<NCELLY-1;j++){
//   	    //             for(k=0;k<NCELLZ-1;k++){
//                       if(i<NCELLX && j<(NCELLY-1) && k<(NCELLZ-1)){
//
//   	                    Curl_E=(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k]-(ez[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j];
//   	                    hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chxh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
//   											//Z-CPML
//   											if(k<cpml_N_Z && i<cpml_x_lim && j<cpml_y_lim){
//   												 //Near Z-PML
//   													 psi_Hx_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]=bh_z_N[k]*psi_Hx_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]+ch_z_N[k]*(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
//   													 hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hx_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)];
//   											}
//   											if(k>=cpml_F_Z && j<cpml_y_lim && i<cpml_x_lim){
//   												//Far Z-PML
//   													 k2 = k - cpml_F_Z;
//   													 psi_Hx_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]=bh_z_F[k2]*psi_Hx_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]+ch_z_F[k2]*(ey[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
//   													 hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hx_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)];
//   											}
//   										   //Y- PML
//   											if(j<cpml_N_Y && i<cpml_x_lim && j<cpml_y_lim){
//   													 psi_Hx_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY)]=bh_y_N[j]*psi_Hx_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY)]+ch_y_N[j]*(ez[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dy;
//   													 hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hx_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY)];
//   											}
//   											if(j>=cpml_F_Y && i<cpml_x_lim && k<cpml_z_lim){
//   													j2 = j - cpml_F_Y;
//   													psi_Hx_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY)]=bh_y_F[j2]*psi_Hx_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY)]+ch_y_F[j2]*(ez[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dy;
//   													hx[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chxe[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hx_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY)];
//   											}
//                       }
//   	  //               }
//   	  //       }
//   	  //  // }
//   		// }
//     }
//     // hipProfilerStop();
//
//     return;
//   }
//
//   __global__   void UPDATE_hy(real *hy,real *ez,real *ex,real *Chyh,real *Chye,real *psi_Hy_z_N,real *psi_Hy_z_F,real *psi_Hy_x_N,real *psi_Hy_x_F,real *khdx,real
//     *khdz,real *bh_z_N,real *bh_z_F,real *ch_z_N,real *ch_z_F,real *bh_x_N,real *bh_x_F,real *ch_x_N,real *ch_x_F,int NCELLX,int NCELLY,int NCELLZ,int Periodic_XY,real dx,real dy,real dz,real dt,int cpml_N_Z,int cpml_F_Z,int cpml_N_X,int cpml_F_X,int cpml_z_lim,int cpml_y_lim,int cpml_x_lim,int NcpmlZ,int NcpmlX){
//
//       int i,j,k,n,i2,k2;
//       comp Curl_E;
//       int idx = blockDim.x * blockIdx.x + threadIdx.x;
//
//       i = idx / (NCELLZ*NCELLY);
//       j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
//       k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
//       if(Periodic_XY){
//   		////#pragma omp parallel for collapse(3) private(Curl_E,i,i2,j,k,k2) // schedule(static)
//   		// for(k=0;k<NCELLZ;k++){
//   		// for(i=0;i<NCELLX;i++){
//   	  //       for(j=0;j<NCELLY;j++){
//               if(i<NCELLX && j<NCELLY && k<NCELLZ){
//   	             //   for(k=0;k<NCELLZ-1;k++){
//   	                	if(i==NCELLX-1){
//   											#ifdef DOUBLECOMPLEX
//   	                		 Curl_E=(ez[ThreeDMapD(0,j,k,NCELLZ,NCELLY)]*cexp(-I*k_x*period_x)-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i]-(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k];
//   											 #endif
//   											 #ifdef DOUBLEPRECISION
//   											 Curl_E=(ez[ThreeDMapD(0,j,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i]-(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k];
//   											 #endif
//   	                    	 hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
//   	                	}
//   	                	else{
//   	                		 Curl_E=(ez[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i]-(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k];
//   	                    	 hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
//   	                }
//   										//Z-PML
//   										if(k<cpml_N_Z && j<cpml_y_lim && k<cpml_z_lim){
//   											psi_Hy_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]=bh_z_N[k]*psi_Hy_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]+ch_z_N[k]*(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
//   											hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hy_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)];
//   										}
//   										if(k>=cpml_F_Z && j<cpml_y_lim && k<cpml_z_lim){
//   											k2 = k - cpml_F_Z;
//   											psi_Hy_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]=bh_z_F[k2]*psi_Hy_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]+ch_z_F[k2]*(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
//   											hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hy_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)];
//   										}
//                     }
//   	    //             }
//   	    //     }
//   	    // }
//       }
//
//       else{
//   			////#pragma omp parallel for collapse(3) private(Curl_E,i,i2,j,k,k2) // schedule(static)
//   	    // for(i=0;i<NCELLX-1;i++){
//   	    //     for(j=0;j<NCELLY;j++){
//   	    //             for(k=0;k<NCELLZ-1;k++){
//                       if(i<(NCELLX-1) && j<NCELLY && k<(NCELLZ-1)){
//
//   	                    Curl_E=(ez[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i]-(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdz[k];
//   	                    hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chyh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
//
//   											//Z-PML
//   											if(k<cpml_N_Z && j<cpml_y_lim && k<cpml_z_lim){
//   												psi_Hy_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]=bh_z_N[k]*psi_Hy_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)]+ch_z_N[k]*(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
//   												hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hy_z_N[ThreeDMapD(i,j,k,NcpmlZ,NCELLY)];
//   											}
//   											if(k>=cpml_F_Z && j<cpml_y_lim && k<cpml_z_lim){
//   												k2 = k - cpml_F_Z;
//   												psi_Hy_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]=bh_z_F[k2]*psi_Hy_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)]+ch_z_F[k2]*(ex[ThreeDMapD(i,j,k+1,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dz;
//   												hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hy_z_F[ThreeDMapD(i,j,k2,NcpmlZ,NCELLY)];
//   											}
//   											//X-PML
//   											if(i<cpml_N_X && j<cpml_y_lim && k<cpml_z_lim){
//   												psi_Hy_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=bh_x_N[i]*psi_Hy_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ch_x_N[i]*(ez[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dx;
//   												hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hy_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
//   											}
//   											if(i>=cpml_F_X && j<cpml_y_lim && k<cpml_z_lim){
//   												i2 = i - cpml_F_X;
//   												psi_Hy_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]=bh_x_F[i2]*psi_Hy_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]+ch_x_F[i2]*(ez[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ez[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dx;
//   												hy[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chye[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hy_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)];
//   											}
//                       }
//   	    //             }
//   	    //     }
//   	    // }
//       }
//     return;
//   }
//
//   //void UPDATE_hz(void){
//   __global__   void UPDATE_hz(real *hz,real *ey,real *ex,real *Chzh,real *Chze,real *psi_Hz_x_N,real *psi_Hz_x_F,real *psi_Hz_y_N,real *psi_Hz_y_F,real *khdx,real
//     *khdy,real *bh_x_N,real *bh_x_F,real *ch_x_N,real *ch_x_F,real *bh_y_N,real *bh_y_F,real *ch_y_N,real *ch_y_F,int NCELLX,int NCELLY,int NCELLZ,int Periodic_XY,real dx,real dy,real dz,real dt,int cpml_N_X,int cpml_F_X,int cpml_N_Y,int cpml_F_Y,int cpml_z_lim,int cpml_y_lim,int cpml_x_lim,int NcpmlY,int NcpmlX){
//
//       int i,j,k,j2,i2;
//       comp Curl_E;
//       int idx = blockDim.x * blockIdx.x + threadIdx.x;
//
//       i = idx / (NCELLZ*NCELLY);
//       j = (idx - i*NCELLZ*NCELLY) / NCELLZ;
//       k = idx - i*NCELLZ*NCELLY - j*NCELLZ;
//       if(Periodic_XY){
//     //  //#pragma omp parallel for collapse(3) private(Curl_E,i,j,k,j2,i2) // schedule(static)
//   		// for(k=0;k<NCELLZ;k++){
//   	  //   for(i=0;i<NCELLX;i++){
//   	  //       for(j=0;j<NCELLY;j++){
//           if(i<NCELLX && j<NCELLY && k<NCELLZ){
//   	          //      for(k=0;k<NCELLZ;k++){
//   	                	if(i==NCELLX-1 || j== NCELLY-1){
//   	                		if(i==NCELLX-1 && j==NCELLY-1){
//   	                			//printf("%d,%d,%d\n",i,j,k);
//   												#ifdef DOUBLECOMPLEX
//   	                			 Curl_E=(ex[ThreeDMapD(i,0,k,NCELLZ,NCELLY)]*cexp(-I*k_y*period_y)-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(0,j,k,NCELLZ,NCELLY)]*cexp(-I*k_x*period_x)-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
//   												 #endif
//   												 #ifdef DOUBLEPRECISION
//   													Curl_E=(ex[ThreeDMapD(i,0,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(0,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
//   													#endif
//   	                   			 hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chzh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
//   	                		}
//   	                		else if(i==NCELLX-1){
//   												#ifdef DOUBLECOMPLEX
//   	                			 Curl_E=(ex[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(0,j,k,NCELLZ,NCELLY)]*cexp(-I*k_x*period_x)-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
//   												 #endif
//   												 #ifdef DOUBLEPRECISION
//   												 Curl_E=(ex[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(0,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
//   												 #endif
//   	                    		 hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chzh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
//   	                		}
//   	                		else{
//   												#ifdef DOUBLECOMPLEX
//   	                			 Curl_E=(ex[ThreeDMapD(i,0,k,NCELLZ,NCELLY)]*cexp(-I*k_y*period_y)-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
//   												 #endif
//   												 #ifdef DOUBLEPRECISION
//   												 Curl_E=(ex[ThreeDMapD(i,0,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
//   												 #endif
//   	                    		 hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chzh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
//   	                		}
//   	                	}
//   	                	else{
//   							 Curl_E=(ex[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
//   	                    	 hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chzh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
//   	               	}
//                   }
//   	    //             }
//   	    //     }
//   	    // }
//       }
//
//       else{
//   		//	//#pragma omp parallel for collapse(3) private(Curl_E,i,j,k,j2,i2) // schedule(static)
//   	    // for(i=0;i<NCELLX-1;i++){
//   	    //     for(j=0;j<NCELLY-1;j++){
//   	    //             for(k=0;k<NCELLZ;k++){
//                       if(i<(NCELLX-1) && j<(NCELLY-1) && k<NCELLZ){
//
//   	                    Curl_E=(ex[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdy[j]-(ey[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/khdx[i];
//   	                    hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=Chzh[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*Curl_E;
//   											//X-PML
//   											if(i<cpml_N_X && j<cpml_y_lim && k<cpml_z_lim){
//   												psi_Hz_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]=bh_x_N[i]*psi_Hz_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+ch_x_N[i]*(ey[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dx;
//   												hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hz_x_N[ThreeDMapD(i,j,k,NCELLZ,NCELLY)];
//   											}
//   											if(i>=cpml_F_X && j<cpml_y_lim && k<cpml_z_lim){
//   												i2 = i - cpml_F_X;
//   												psi_Hz_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]=bh_x_F[i2]*psi_Hz_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)]+ch_x_F[i2]*(ey[ThreeDMapD(i+1,j,k,NCELLZ,NCELLY)]-ey[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dx;
//   												hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]-=Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hz_x_F[ThreeDMapD(i2,j,k,NCELLZ,NCELLY)];
//   											}
//   											//Y-PML
//   											if(j<cpml_N_Y && i<cpml_x_lim && k<cpml_z_lim){
//   												psi_Hz_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY)]=bh_y_N[j]*psi_Hz_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY)]+ch_y_N[j]*(ex[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dy;
//   												hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hz_y_N[ThreeDMapD(i,j,k,NCELLZ,NcpmlY)];
//   											}
//   											if(j>=cpml_F_Y && i<cpml_x_lim && k<cpml_z_lim){
//   												j2 = j - cpml_F_Y;
//   												psi_Hz_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY)]=bh_y_F[j2]*psi_Hz_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY)]+ch_y_F[j2]*(ex[ThreeDMapD(i,j+1,k,NCELLZ,NCELLY)]-ex[ThreeDMapD(i,j,k,NCELLZ,NCELLY)])/dy;
//   												hz[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]+=Chze[ThreeDMapD(i,j,k,NCELLZ,NCELLY)]*psi_Hz_y_F[ThreeDMapD(i,j2,k,NCELLZ,NcpmlY)];
//   											}
//                       }
//   	    //             }
//   	    //     }
//   	    // }
//       }
//     return;
//   }























  // void UpdateHydroPx(void){
  // 	int i,j,k;
  // 	real Vx1,Vx2,Vx3,Vy1,Vy2,Vy3,Vz1,Vz2,Vz3,Hx1,Hz1,Hy1,Hx2,Hz2,Hy2,Ex1,Ey1,Ez1,VdotGrad,VdotGrad2,DivV,VcrossH,VcrossH2,Pressure,ND1,ND2,ND3,Grad_Div,Grad_Div2;
  // 	real INV_DX,INV_DY,INV_DZ;
  // 	INV_DX = 1.0/dx;
  // 	INV_DY = 1.0/dy;
  // 	INV_DZ = 1.0/dz;
  // 	Grad_Div = 0.0;
  // 	Grad_Div2 =0.0;
  // 	////#pragma omp parallel for collapse(3)  // schedule(static)
  // 	for(i=0;i<NCELLX-1;i++){
  // 			for(j=1;j<NCELLY-1;j++){
  // 							for(k=1;k<NCELLZ-1;k++){
  // 								if(mat_matrixX[ThreeDMap(i,j,k,NCELLZ,NCELLY)]> first_medium && mat_matrixX[ThreeDMap(i,j,k,NCELLZ,NCELLY)] < 6){
  // 									ND1 = N_EQ + NDx_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
  //
  // 									Grad_Div = Calc_DIV_GRADx(i,j,k);
  // 									Grad_Div2 = Calc_DIV_GRADx2(i,j,k);
  //
  //
  // 									Vx1 = 0.5 * (Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 									Vx2 = 0.5 * (Px_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  //
  // 									Vy1 = 0.25 * (Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 									Vy2 = 0.25 * (Py_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMap(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 									//Vy1 = 0.5 * (Vy1 + Vy2);
  // 									Vz1 = 0.25 * (Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 									Vz2 = 0.25 * (Pz_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMap(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 									//Vz1 = 0.5 * (Vz1 + Vz2);
  // 									Hy2 = 0.5 * (hyPrev[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + hyPrev[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]);
  // 									Hz2 = 0.5 * (hzPrev[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + hzPrev[ThreeDMap(i,j-1,k,NCELLZ,NCELLY)]);
  // 									Hy1 = 0.5 * (hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + hy[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]);
  // 									Hz1 = 0.5 * (hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + hz[ThreeDMap(i,j-1,k,NCELLZ,NCELLY)]);
  //
  // 									if(WithConvection) {
  // 										VdotGrad = 0.5*(Vx1*(Px_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy1*(Px_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz1*(Px_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
  // 										VdotGrad2 = 0.5*(Vx2*(Px_d_n_1[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n_1[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy2*(Px_d_n_1[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n_1[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz2*(Px_d_n_1[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Px_d_n_1[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
  // 										VdotGrad = (VdotGrad - VdotGrad2)/dt;
  // 									}
  // 									else VdotGrad = 0.0;
  //
  // 									if(WithMagField){
  // 										VcrossH = Vy1*Hz1 - Vz1*Hy1;
  // 										VcrossH2 = Vy2*Hz2 - Vz2*Hy2;
  // 										VcrossH = (VcrossH - VcrossH2)/dt;
  // 									}
  // 									else VcrossH = 0.0;
  //
  // 										 Px_d[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] = d_1_d[0]*Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + d_2_d[0]*Px_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] +d_3_d[0]*ex[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + d_4_d[0]*ex_n_1[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + d_NL[0]*(Grad_Div + Grad_Div2/N_EQ)/pow(ND1,1.0/3.0) + d_5_d[0]*(VdotGrad + VcrossH);
  //
  // 								}
  //
  // 							}
  // 						}
  // 					}
  // }
  //
  // void UpdateHydroPy(void){
  // 	int i,j,k;
  // 	real Vx1,Vx2,Vx3,Vy1,Vy2,Vy3,Vz1,Vz2,Vz3,Hx1,Hz1,Hy1,Hx2,Hz2,Hy2,Ex1,Ey1,Ez1,VdotGrad,VdotGrad2,DivV,VcrossH,VcrossH2,Pressure,ND1,ND2,ND3,Grad_Div,Grad_Div2;
  // 	real INV_DX,INV_DY,INV_DZ;
  // 	INV_DX = 1.0/dx;
  // 	INV_DY = 1.0/dy;
  // 	INV_DZ = 1.0/dz;
  // 	Grad_Div = 0.0;
  // 	Grad_Div2 =0.0;
  // 	////#pragma omp parallel for collapse(3) // schedule(static)
  // 	for(i=1;i<NCELLX-1;i++){
  // 			for(j=0;j<NCELLY-1;j++){
  // 							for(k=1;k<NCELLZ-1;k++){
  // 								if(mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)]> first_medium && mat_matrixY[ThreeDMap(i,j,k,NCELLZ,NCELLY)] < 6){
  //
  // 									Grad_Div = Calc_DIV_GRADy(i,j,k);
  // 									Grad_Div2 = Calc_DIV_GRADy2(i,j,k);
  //
  // 									ND1 = N_EQ + NDy_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
  //
  // 									Vy1 = 0.5 * (Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 									Vy2 = 0.5 * (Py_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  //
  // 									Vx1 = 0.25 * (Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 									Vx2 = 0.25 * (Px_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMap(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 								 //	Vx1 = 0.5 * (Vx1 + Vx2);
  // 									Vz1 = 0.25 * (Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 									Vz2 = 0.25 * (Pz_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMap(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 									//Vz1 = 0.5 * (Vz1 + Vz2);
  // 									Hx1 = 0.5 * (hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + hx[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]);
  // 									Hz1 = 0.5 * (hz[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + hz[ThreeDMap(i-1,j,k,NCELLZ,NCELLY)]);
  // 									Hx2 = 0.5 * (hxPrev[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + hxPrev[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]);
  // 									Hz2 = 0.5 * (hzPrev[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + hzPrev[ThreeDMap(i-1,j,k,NCELLZ,NCELLY)]);
  //
  //
  // 									 if(WithMagField){
  // 										 VcrossH = Vz1*Hx1 - Vx1*Hz1;
  // 										 VcrossH2 = Vz2*Hx2 - Vx2*Hz2;
  // 										 VcrossH = (VcrossH - VcrossH2)/dt;
  // 									 }
  // 									 else VcrossH = 0.0;
  // 									 if(WithConvection) {
  // 										 VdotGrad = 0.5*(Vx1*(Py_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy1*(Py_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz1*(Py_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
  // 										 VdotGrad2 = 0.5*(Vx2*(Py_d_n_1[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n_1[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy2*(Py_d_n_1[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n_1[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz2*(Py_d_n_1[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Py_d_n_1[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
  // 										 VdotGrad = (VdotGrad - VdotGrad2)/dt;
  // 									 }
  // 									 else VdotGrad = 0.0;
  //
  // 								 Py_d[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] = d_1_d[0]*Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + d_2_d[0]*Py_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + d_3_d[0]*ey[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + d_4_d[0]*ey_n_1[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + d_NL[0]*(Grad_Div + Grad_Div2/N_EQ)/pow(ND1,1.0/3.0) + d_5_d[0]*(VdotGrad + VcrossH);
  //
  // 								}
  //
  //
  // 							}
  // 						}
  // 					}
  // }
  //
  // void UpdateHydroPz(void){
  // 	int i,j,k;
  // 	real Vx1,Vx2,Vx3,Vy1,Vy2,Vy3,Vz1,Vz2,Vz3,Hx1,Hz1,Hy1,Hx2,Hz2,Hy2,Ex1,Ey1,Ez1,VdotGrad,VdotGrad2,DivV,VcrossH,VcrossH2,Pressure,ND1,ND2,ND3,Grad_Div,Grad_Div2;
  // 	real INV_DX,INV_DY,INV_DZ;
  // 	INV_DX = 1.0/dx;
  // 	INV_DY = 1.0/dy;
  // 	INV_DZ = 1.0/dz;
  // 	Grad_Div = 0.0;
  // 	Grad_Div2 =0.0;
  // 	////#pragma omp parallel for collapse(3) // schedule(static)
  // 	for(i=1;i<NCELLX-1;i++){
  // 			for(j=1;j<NCELLY-1;j++){
  // 							for(k=0;k<NCELLZ-1;k++){
  // 								if(mat_matrixZ[ThreeDMap(i,j,k,NCELLZ,NCELLY)]> first_medium && mat_matrixZ[ThreeDMap(i,j,k,NCELLZ,NCELLY)] < 6){
  //
  // 									Grad_Div = Calc_DIV_GRADz(i,j,k);
  // 									Grad_Div2 = Calc_DIV_GRADz2(i,j,k);
  //
  // 									ND1 = N_EQ + NDz_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)];
  //
  // 								 Vz1 = 0.5 * (Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 								 Vz2 = 0.5 * (Pz_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  //
  // 								 Vx1 = 0.25 * (Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 								 Vx2 = 0.25 * (Px_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMap(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n_1[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 								 //Vx1 = 0.5 * (Vx1 + Vx2);
  // 								 Vy1 = 0.25 * (Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 								 Vy2 = 0.25 * (Py_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMap(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n_1[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  // 								 //	Vy1 = 0.5 * (Vy1 + Vy2);
  // 								 Hy1 = 0.5 * (hy[ThreeDMap(i-1,j,k,NCELLZ,NCELLY)] + hy[ThreeDMap(i,j,k,NCELLZ,NCELLY)]);
  // 								 Hx1 = 0.5 * (hx[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + hx[ThreeDMap(i,j-1,k,NCELLZ,NCELLY)]);
  // 								 Hy2 = 0.5 * (hyPrev[ThreeDMap(i-1,j,k,NCELLZ,NCELLY)] + hyPrev[ThreeDMap(i,j,k,NCELLZ,NCELLY)]);
  // 								 Hx2 = 0.5 * (hxPrev[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + hxPrev[ThreeDMap(i,j-1,k,NCELLZ,NCELLY)]);
  //
  // 								 if(WithMagField){
  // 								 	VcrossH = Vx1*Hy1 - Vy1*Hx1;
  // 								 	VcrossH2 = Vx2*Hy2 - Vy2*Hx2;
  // 								 	VcrossH = (VcrossH - VcrossH2)/dt;
  // 								 }
  // 								 else VcrossH = 0.0;
  //
  // 								 if(WithConvection){
  // 								 	VdotGrad = 0.5*(Vx1*(Pz_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy1*(Pz_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz1*(Pz_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
  // 								 	VdotGrad2 = 0.5*(Vx2*(Pz_d_n_1[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n_1[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DX + Vy2*(Pz_d_n_1[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n_1[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DY  + Vz2*(Pz_d_n_1[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - Pz_d_n_1[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])*INV_DZ);
  // 								 	VdotGrad = (VdotGrad - VdotGrad2)/dt;
  // 								 }
  // 								 else VdotGrad = 0.0;
  //
  // 								 Pz_d[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] = d_1_d[0]*Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + d_2_d[0]*Pz_d_n_1[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + d_3_d[0]*ez[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + d_4_d[0]*ez_n_1[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + d_NL[0]*(Grad_Div + Grad_Div2/N_EQ)/pow(ND1,1.0/3.0) + d_5_d[0]*(VdotGrad + VcrossH);
  //
  // 								}
  //
  //
  // 							}
  // 						}
  // 					}
  // }
  //
  //
  //













  comp Calc_DIV_GRADx2(int i,int j, int k){
  	comp Div_Grad;
  	real INV_DX = 1.0/dx;
  	real INV_DY = 1.0/dy;
  	real INV_DZ = 1.0/dz;
  	int n;

  	Div_Grad = INV_DX*INV_DX*(Px_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMap(i+1,j,k,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMap(i-1,j,k,NCELLZ,NCELLY)])
  						 + INV_DX*INV_DY*(Py_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMap(i+1,j,k,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMap(i+1,j-1,k,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMap(i,j-1,k,NCELLZ,NCELLY)])
  						 + INV_DX*INV_DZ*(Pz_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMap(i+1,j,k,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMap(i+1,j,k-1,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]);
  }

  comp Calc_DIV_GRADx(int i,int j,int k){
  	comp Div_Grad;
  	real INV_DX = 1.0/dx;
  	real INV_DY = 1.0/dy;
  	real INV_DZ = 1.0/dz;
  	int n;


  	for(n=0;n<N_drude_poles;n++){
  	if(Diverge_Gradient){

  		if(i==0 && j==0){
  				Div_Grad = INV_DX*INV_DX*(Px_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
  				 					 + INV_DX*INV_DY*(Py_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i+1,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  									 + INV_DX*INV_DZ*(Pz_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}

  		else if(i==NCELLX-1 && j==0){
  				Div_Grad = INV_DX*INV_DX*(Px_d_n[FourDMap(0,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
  				 					 + INV_DX*INV_DY*(Py_d_n[FourDMap(0,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(0,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  									 + INV_DX*INV_DZ*(Pz_d_n[FourDMap(0,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(0,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}
  		else if(i==NCELLX-1){
  				Div_Grad = INV_DX*INV_DX*(Px_d_n[FourDMap(0,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
  									 + INV_DX*INV_DY*(Py_d_n[FourDMap(0,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(0,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  									 + INV_DX*INV_DZ*(Pz_d_n[FourDMap(0,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(0,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}
  		else if(i==0){
  				Div_Grad = INV_DX*INV_DX*(Px_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
  									 + INV_DX*INV_DY*(Py_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  									 + INV_DX*INV_DZ*(Pz_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}
  		else if(j==0){
  				Div_Grad = INV_DX*INV_DX*(Px_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
  				 					 + INV_DX*INV_DY*(Py_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i+1,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  									 + INV_DX*INV_DZ*(Pz_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}
  		else{
  				Div_Grad = INV_DX*INV_DX*(Px_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
  				 					 + INV_DX*INV_DY*(Py_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i+1,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  									 + INV_DX*INV_DZ*(Pz_d_n[FourDMap(i+1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}

  	}

   }
  	return Div_Grad;
  }


  comp Calc_DIV_GRADy2(int i,int j,int k){
  	comp Div_Grad;
  	real INV_DX = 1.0/dx;
  	real INV_DY = 1.0/dy;
  	real INV_DZ = 1.0/dz;
  	int n;

  	Div_Grad = INV_DY*INV_DY*(Py_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMap(i,j+1,k,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMap(i,j-1,k,NCELLZ,NCELLY)])
  						 + INV_DX*INV_DY*(Px_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMap(i,j+1,k,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMap(i-1,j+1,k,NCELLZ,NCELLY)]+Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMap(i-1,j,k,NCELLZ,NCELLY)])
  						 + INV_DY*INV_DZ*(Pz_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMap(i,j+1,k,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMap(i,j+1,k-1,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)]);

  	return Div_Grad;
  }


  comp Calc_DIV_GRADy(int i,int j,int k){
  	comp Div_Grad;
  	real INV_DX = 1.0/dx;
  	real INV_DY = 1.0/dy;
  	real INV_DZ = 1.0/dz;
  	int n;
  	for(n=0;n<N_drude_poles;n++){
  	if(Diverge_Gradient){
  		if(i==0 && j==0){
  			Div_Grad = INV_DY*INV_DY*(Py_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DX*INV_DY*(Px_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(NCELLX-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMap(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DY*INV_DZ*(Pz_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}

  		else if(i==0 && j==NCELLY-1){
  			Div_Grad = INV_DY*INV_DY*(Py_d_n[FourDMap(i,0,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DX*INV_DY*(Px_d_n[FourDMap(i,0,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(NCELLX-1,0,k,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMap(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DY*INV_DZ*(Pz_d_n[FourDMap(i,0,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,0,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  			}
  			else if(j==NCELLY-1){
  				Div_Grad = INV_DY*INV_DY*(Py_d_n[FourDMap(i,0,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  									 + INV_DX*INV_DY*(Px_d_n[FourDMap(i,0,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i-1,0,k,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
  									 + INV_DY*INV_DZ*(Pz_d_n[FourDMap(i,0,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,0,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  			}
  		else if(j==0){
  			Div_Grad = INV_DY*INV_DY*(Py_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DX*INV_DY*(Px_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DY*INV_DZ*(Pz_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}
  		else if(i==0){
  			Div_Grad = INV_DY*INV_DY*(Py_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DX*INV_DY*(Px_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(NCELLX-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMap(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DY*INV_DZ*(Pz_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}
  		else{
  			Div_Grad = INV_DY*INV_DY*(Py_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DX*INV_DY*(Px_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i-1,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DY*INV_DZ*(Pz_d_n[FourDMap(i,j+1,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Pz_d_n[FourDMap(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}

  	}


  }
  	//printf("%e\n",Div_Grad);
  	return Div_Grad;
  }


  comp Calc_DIV_GRADz2(int i,int j,int k){
  	comp Div_Grad;
  	real INV_DX = 1.0/dx;
  	real INV_DY = 1.0/dy;
  	real INV_DZ = 1.0/dz;
  	int n;

  	Div_Grad = INV_DZ*INV_DZ*(Pz_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMap(i,j,k+1,NCELLZ,NCELLY)] - 2.0*Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]*NDz_prev[ThreeDMap(i,j,k-1,NCELLZ,NCELLY)])
  						 + INV_DZ*INV_DY*(Py_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMap(i,j,k+1,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMap(i,j-1,k+1,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDy_prev[ThreeDMap(i,j-1,k,NCELLZ,NCELLY)])
  						 + INV_DX*INV_DZ*(Px_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMap(i,j,k+1,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMap(i,j,k,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMap(i-1,j,k+1,NCELLZ,NCELLY)]+Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]*NDx_prev[ThreeDMap(i-1,j,k,NCELLZ,NCELLY)]);
  return Div_Grad;
  }

  comp Calc_DIV_GRADz(int i,int j,int k){
  	comp Div_Grad;
  	real INV_DX = 1.0/dx;
  	real INV_DY = 1.0/dy;
  	real INV_DZ = 1.0/dz;
  	int n;
  	for(n=0;n<N_drude_poles;n++){
  	if(Diverge_Gradient){

  		if(i==0 && j==0){
  			Div_Grad = INV_DZ*INV_DZ*(Pz_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])
  			 					 + INV_DZ*INV_DY*(Py_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,NCELLY-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DX*INV_DZ*(Px_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(NCELLX-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMap(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}
  		else if(i==0){
  			Div_Grad = INV_DZ*INV_DZ*(Pz_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])
  			 					 + INV_DZ*INV_DY*(Py_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DX*INV_DZ*(Px_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(NCELLX-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMap(NCELLX-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}
  		else if(j==0){
  			Div_Grad = INV_DZ*INV_DZ*(Pz_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DZ*INV_DY*(Py_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,NCELLY-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,NCELLY-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DX*INV_DZ*(Px_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);
  		}
  		else{
  			Div_Grad = INV_DZ*INV_DZ*(Pz_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)])
  			 					 + INV_DZ*INV_DY*(Py_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,j-1,k,0,N_drude_poles,NCELLZ,NCELLY)])
  								 + INV_DX*INV_DZ*(Px_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[FourDMap(i-1,j,k,0,N_drude_poles,NCELLZ,NCELLY)]);

  								//  if(t==173 && k==102){
  								// 	 printf("%e\t%e\t%e\n",Pz_d_n[FourDMap(i,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)] - 2.0*Pz_d_n[FourDMap(i,j,k,0,N_drude_poles,NCELLZ,NCELLY)] + Pz_d_n[FourDMap(i,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)],Py_d_n[i][j+1][k+1][0]-Py_d_n[FourDMap(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]-Py_d_n[FourDMap(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[i][j-1][k-1][0],Px_d_n[i+1][j][k+1][0]-Px_d_n[FourDMap(i+1,j,k-1,0,N_drude_poles,NCELLZ,NCELLY)]-Px_d_n[FourDMap(i-1,j,k+1,0,N_drude_poles,NCELLZ,NCELLY)]+Px_d_n[i-1][j][k-1][0]);
  								// 	 printf("%e\n",Py_d_n[i][j+1][k+1][0]-(Py_d_n[FourDMap(i,j+1,k-1,0,N_drude_poles,NCELLZ,NCELLY)]+Py_d_n[FourDMap(i,j-1,k+1,0,N_drude_poles,NCELLZ,NCELLY)])+Py_d_n[i][j-1][k-1][0]);
  								 //
  								//  }
  								 //1.186946e-66
  		}
  	//	printf("%e\n",Div_Grad);
  	}

  }
  return Div_Grad;
  }
