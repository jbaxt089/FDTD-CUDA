#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "extern_var.h"
#include <math.h>

//pre-name of snapshot files
static char snapshot_name[10]={'s','i','m'};
static char aux_snapshot_name[10]={'a','u','x','_','s','i','m'};

void SETUP_SNAPSHOT(void){
    //how many spacial steps do you want to skip in each direction?
    // x_skip = 1;
    // y_skip = 1;
    // z_skip = 1;
    // if(NCELLX > 30) x_skip=2;
    // if(NCELLY > 30) y_skip=1;
    // if(NCELLZ >100) z_skip=1;
    //how many time steps in between snapshots?
    //t_skip=10;
    //counter
    snapshot_count=0;

}

void SNAPSHOT_2D(){
    int i,j,k;
    char filename[100];
    real ETOT;
    //Is it the appropriate time step?
    FILE *Snap;

    //fprintf(Snap,"%d\t%d\n",NCELLX/x_skip,NCELLY/y_skip);
    int XZ = 0;
    int XY = 0;
    int YZ = 1;
    if(XZ == 1){
      sprintf(filename,"videofiles/XZ_%s.%d.txt",snapshot_name,snapshot_count);
      Snap=fopen(filename,"w");
    for(i=0;i<NCELLX;i+=x_skip){
       for(k=0;k<NCELLZ;k+=z_skip){
           // for(k=0;k<NCELLZ;k+=z_skip){

                    #ifdef DOUBLECOMPLEX
                    fprintf(Snap,"%f,",creal(ez[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)]));
                    #endif
                    #ifndef DOUBLECOMPLEX
		                 ETOT = ex[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] * ex[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] + ey[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)]*ey[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] + ez[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)]*ez[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)];
                    // ETOT = hx[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] * hx[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)];
                  //  ETOT = hx[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] * hx[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] + hy[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)]*hy[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] + hz[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)]*hz[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)];
              //    ETOT = ez[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)]*ez[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)];
                    ETOT = pow(ETOT,0.5);
		    fprintf(Snap,"%e ",ETOT);
                    #endif
            }
            fprintf(Snap,"\n");

          //  }
      }
    }
    if(YZ == 1){
      sprintf(filename,"videofiles/YZ_%s.%d.txt",snapshot_name,snapshot_count);
      Snap=fopen(filename,"w");
      for(j=0;j<NCELLY;j+=y_skip){
         for(k=0;k<NCELLZ;k+=z_skip){
           ETOT = ex[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)] * ex[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)]+  ey[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)]*ey[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)] + ez[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)]*ez[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)];
           ETOT = pow(ETOT,0.5);
           fprintf(Snap,"%e ",ETOT);

         }
         fprintf(Snap,"\n");

       }
    }
        fclose(Snap);
        snapshot_count++;

    return;
}



void SNAPSHOT_2D_N(){
    int i,j,k;
    char filename[100];
    real ETOT;
    //Is it the appropriate time step?
    FILE *Snap;

    //fprintf(Snap,"%d\t%d\n",NCELLX/x_skip,NCELLY/y_skip);
    int XZ = 0;
    int XY = 0;
    int YZ = 1;

    if(XZ == 1){
      sprintf(filename,"videofiles/XZ_%s.%d_N.txt",snapshot_name,snapshot_count);
      Snap=fopen(filename,"w");
    for(i=0;i<NCELLX;i+=x_skip){
       for(k=0;k<NCELLZ;k+=z_skip){
           // for(k=0;k<NCELLZ;k+=z_skip){

                    #ifdef DOUBLECOMPLEX
                    fprintf(Snap,"%f,",creal(ez[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)]));
                    #endif
                    #ifndef DOUBLECOMPLEX
		                 // ETOT = ex[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] * ex[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] + ey[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)]*ey[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] + ez[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)]*ez[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)];
                    // ETOT = hx[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] * hx[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)];
                  //  ETOT = hx[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] * hx[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] + hy[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)]*hy[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] + hz[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)]*hz[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)];
              //    ETOT = ez[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)]*ez[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)];
                    // ETOT = pow(ETOT,0.5);
                    ETOT = NDx[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] + NDy[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)] + NDz[ThreeDMap(i,(int)(NCELLY/2),k,NCELLZ,NCELLY)];
		    fprintf(Snap,"%e ",ETOT/3.0);
                    #endif
            }
            fprintf(Snap,"\n");

          //  }
      }
    }
    if(YZ == 1){
      sprintf(filename,"videofiles/YZ_%s.%d_N.txt",snapshot_name,snapshot_count);
      Snap=fopen(filename,"w");
      for(j=0;j<NCELLY;j+=y_skip){
         for(k=0;k<NCELLZ;k+=z_skip){
           // ETOT = ex[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)] * ex[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)]+  ey[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)]*ey[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)] + ez[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)]*ez[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)];
           // ETOT = pow(ETOT,0.5);
           // printf("%e\t%d\t%d\n", NDx[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)] ,j,k);
           ETOT = NDx[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)] + NDy[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)] + NDz[ThreeDMap((int)(NCELLX/2),j,k,NCELLZ,NCELLY)];
           fprintf(Snap,"%e ",ETOT/3.0);

         }
         fprintf(Snap,"\n");

       }
    }
        fclose(Snap);
        snapshot_count++;

    return;
}


void SNAPSHOT_1D(){
    int i;
    char filename[100];
    FILE *Snap;
    sprintf(filename,"%s.%d.txt",aux_snapshot_name,snapshot_count);
    Snap=fopen(filename,"w");
    for(i=0;i<NCELLZ;i+=z_skip){
      #ifdef DOUBLECOMPLEX
        fprintf(Snap,"%e\n",creal(hx[ThreeDMap(1,1,i,NCELLZ,NCELLY)]));
      #endif

      #ifndef DOUBLECOMPLEX
      //fprintf(Snap,"%e\n",Py_d[2][2][i][0]);
    fprintf(Snap,"%e\n",ey[ThreeDMap(2,2,i,NCELLZ,NCELLY)]);
      #endif
    }
    fclose(Snap);




    sprintf(filename,"%s.%d_2.txt",aux_snapshot_name,snapshot_count);
    Snap=fopen(filename,"w");
    for(i=0;i<inc_Length;i+=z_skip){
      #ifdef DOUBLECOMPLEX
        fprintf(Snap,"%e\n",creal(hx[ThreeDMap(1,1,i,NCELLZ,NCELLY)]));
      #endif

      #ifndef DOUBLECOMPLEX
      fprintf(Snap,"%e\n",e_inc[i]);
    // fprintf(Snap,"%e\n",ey[2][2][i]);
      #endif
    }
    fclose(Snap);
    snapshot_count++;


    return;
}
